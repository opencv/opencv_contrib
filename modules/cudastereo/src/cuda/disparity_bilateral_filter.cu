#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/limits.hpp"

#include "disparity_bilateral_filter.hpp"

namespace cv { namespace cuda { namespace device
{
    namespace disp_bilateral_filter
    {
        template <int channels>
        struct DistRgbMax
        {
            static __device__ __forceinline__ uchar calc(const uchar* a, const uchar* b)
            {
                uchar x = ::abs(a[0] - b[0]);
                uchar y = ::abs(a[1] - b[1]);
                uchar z = ::abs(a[2] - b[2]);
                return (::max(::max(x, y), z));
            }
        };

        template <>
        struct DistRgbMax<1>
        {
            static __device__ __forceinline__ uchar calc(const uchar* a, const uchar* b)
            {
                return ::abs(a[0] - b[0]);
            }
        };

        template <int channels, typename T>
        __global__ void disp_bilateral_filter(int t, T* disp, size_t disp_step,
            const uchar* img, size_t img_step, int h, int w,
            const float* ctable_color, const float * ctable_space, size_t ctable_space_step,
            int cradius,
            short cedge_disc, short cmax_disc)
        {
            const int y = blockIdx.y * blockDim.y + threadIdx.y;
            const int x = ((blockIdx.x * blockDim.x + threadIdx.x) << 1) + ((y + t) & 1);

            T dp[5];

            if (y > 0 && y < h - 1 && x > 0 && x < w - 1)
            {
                dp[0] = *(disp + (y  ) * disp_step + x + 0);
                dp[1] = *(disp + (y-1) * disp_step + x + 0);
                dp[2] = *(disp + (y  ) * disp_step + x - 1);
                dp[3] = *(disp + (y+1) * disp_step + x + 0);
                dp[4] = *(disp + (y  ) * disp_step + x + 1);

                if(::abs(dp[1] - dp[0]) >= cedge_disc || ::abs(dp[2] - dp[0]) >= cedge_disc || ::abs(dp[3] - dp[0]) >= cedge_disc || ::abs(dp[4] - dp[0]) >= cedge_disc)
                {
                    const int ymin = ::max(0, y - cradius);
                    const int xmin = ::max(0, x - cradius);
                    const int ymax = ::min(h - 1, y + cradius);
                    const int xmax = ::min(w - 1, x + cradius);

                    float cost[] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f};

                    const uchar* ic = img + y * img_step + channels * x;

                    for(int yi = ymin; yi <= ymax; yi++)
                    {
                        const T* disp_y = disp + yi * disp_step;

                        for(int xi = xmin; xi <= xmax; xi++)
                        {
                            const uchar* in = img + yi * img_step + channels * xi;

                            uchar dist_rgb = DistRgbMax<channels>::calc(in, ic);

                            const float weight = ctable_color[dist_rgb] * (ctable_space + ::abs(y-yi)* ctable_space_step)[::abs(x-xi)];

                            const T disp_reg = disp_y[xi];

                            cost[0] += ::min((float)cmax_disc, (float)::abs(disp_reg - dp[0])) * weight;
                            cost[1] += ::min((float)cmax_disc, (float)::abs(disp_reg - dp[1])) * weight;
                            cost[2] += ::min((float)cmax_disc, (float)::abs(disp_reg - dp[2])) * weight;
                            cost[3] += ::min((float)cmax_disc, (float)::abs(disp_reg - dp[3])) * weight;
                            cost[4] += ::min((float)cmax_disc, (float)::abs(disp_reg - dp[4])) * weight;
                        }
                    }

                    float minimum = numeric_limits<float>::max();
                    int id = 0;

                    if (cost[0] < minimum)
                    {
                        minimum = cost[0];
                        id = 0;
                    }
                    if (cost[1] < minimum)
                    {
                        minimum = cost[1];
                        id = 1;
                    }
                    if (cost[2] < minimum)
                    {
                        minimum = cost[2];
                        id = 2;
                    }
                    if (cost[3] < minimum)
                    {
                        minimum = cost[3];
                        id = 3;
                    }
                    if (cost[4] < minimum)
                    {
                        minimum = cost[4];
                        id = 4;
                    }

                    *(disp + y * disp_step + x) = dp[id];
                }
            }
        }

        template <typename T>
        void disp_bilateral_filter(PtrStepSz<T> disp, PtrStepSzb img, int channels, int iters, const float *table_color, const float* table_space, size_t table_step, int radius, short edge_disc, short max_disc, hipStream_t stream)
        {
            dim3 threads(32, 8, 1);
            dim3 grid(1, 1, 1);
            grid.x = divUp(disp.cols, threads.x << 1);
            grid.y = divUp(disp.rows, threads.y);

            switch (channels)
            {
            case 1:
                for (int i = 0; i < iters; ++i)
                {
                    disp_bilateral_filter<1><<<grid, threads, 0, stream>>>(0, disp.data, disp.step/sizeof(T), img.data, img.step, disp.rows, disp.cols, table_color, table_space, table_step, radius, edge_disc, max_disc);
                    cudaSafeCall( hipGetLastError() );

                    disp_bilateral_filter<1><<<grid, threads, 0, stream>>>(1, disp.data, disp.step/sizeof(T), img.data, img.step, disp.rows, disp.cols, table_color, table_space, table_step, radius, edge_disc, max_disc);
                    cudaSafeCall( hipGetLastError() );
                }
                break;
            case 3:
                for (int i = 0; i < iters; ++i)
                {
                    disp_bilateral_filter<3><<<grid, threads, 0, stream>>>(0, disp.data, disp.step/sizeof(T), img.data, img.step, disp.rows, disp.cols, table_color, table_space, table_step, radius, edge_disc, max_disc);
                    cudaSafeCall( hipGetLastError() );

                    disp_bilateral_filter<3><<<grid, threads, 0, stream>>>(1, disp.data, disp.step/sizeof(T), img.data, img.step, disp.rows, disp.cols, table_color, table_space, table_step, radius, edge_disc, max_disc);
                    cudaSafeCall( hipGetLastError() );
                }
                break;
            default:
                CV_Error(cv::Error::BadNumChannels, "Unsupported channels count");
            }

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template void disp_bilateral_filter<uchar>(PtrStepSz<uchar> disp, PtrStepSzb img, int channels, int iters, const float *table_color, const float *table_space, size_t table_step, int radius, short, short, hipStream_t stream);
        template void disp_bilateral_filter<short>(PtrStepSz<short> disp, PtrStepSzb img, int channels, int iters, const float *table_color, const float *table_space, size_t table_step, int radius, short, short, hipStream_t stream);
        template void disp_bilateral_filter<float>(PtrStepSz<float> disp, PtrStepSzb img, int channels, int iters, const float *table_color, const float *table_space, size_t table_step, int radius, short, short, hipStream_t stream);
    } // namespace bilateral_filter
}}} // namespace cv { namespace cuda { namespace cudev

#endif /* CUDA_DISABLER */
