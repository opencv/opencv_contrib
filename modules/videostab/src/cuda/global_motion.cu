#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include <thrust/device_ptr.h>
#include <thrust/remove.h>
#include <thrust/functional.h>
#include "opencv2/core/cuda/common.hpp"

namespace cv { namespace cuda { namespace device { namespace globmotion {

__constant__ float cml[9];
__constant__ float cmr[9];

struct is_zero
{
    __host__ __device__ bool operator()(uchar x) const { return x == 0; }
};

int compactPoints(int N, float *points0, float *points1, const uchar *mask)
{
    thrust::device_ptr<float2> dpoints0((float2*)points0);
    thrust::device_ptr<float2> dpoints1((float2*)points1);
    thrust::device_ptr<const uchar> dmask(mask);

    return (int)(thrust::remove_if(thrust::make_zip_iterator(thrust::make_tuple(dpoints0, dpoints1)),
                             thrust::make_zip_iterator(thrust::make_tuple(dpoints0 + N, dpoints1 + N)),
                             dmask, is_zero())
           - thrust::make_zip_iterator(make_tuple(dpoints0, dpoints1)));
}


__global__ void calcWobbleSuppressionMapsKernel(
        const int left, const int idx, const int right, const int width, const int height,
        PtrStepf mapx, PtrStepf mapy)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x < width && y < height)
    {
        float xl = cml[0]*x + cml[1]*y + cml[2];
        float yl = cml[3]*x + cml[4]*y + cml[5];
        float izl = 1.f / (cml[6]*x + cml[7]*y + cml[8]);
        xl *= izl;
        yl *= izl;

        float xr = cmr[0]*x + cmr[1]*y + cmr[2];
        float yr = cmr[3]*x + cmr[4]*y + cmr[5];
        float izr = 1.f / (cmr[6]*x + cmr[7]*y + cmr[8]);
        xr *= izr;
        yr *= izr;

        float wl = idx - left;
        float wr = right - idx;
        mapx(y,x) = (wr * xl + wl * xr) / (wl + wr);
        mapy(y,x) = (wr * yl + wl * yr) / (wl + wr);
    }
}


void calcWobbleSuppressionMaps(
        int left, int idx, int right, int width, int height,
        const float *ml, const float *mr, PtrStepSzf mapx, PtrStepSzf mapy)
{
    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(cml), ml, 9*sizeof(float)));
    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(cmr), mr, 9*sizeof(float)));

    dim3 threads(32, 8);
    dim3 grid(divUp(width, threads.x), divUp(height, threads.y));

    calcWobbleSuppressionMapsKernel<<<grid, threads>>>(
            left, idx, right, width, height, mapx, mapy);

    cudaSafeCall(hipGetLastError());
    cudaSafeCall(hipDeviceSynchronize());
}

}}}}


#endif /* CUDA_DISABLER */
