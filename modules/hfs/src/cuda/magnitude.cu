#include "hip/hip_runtime.h"
// This file is part of OpenCV project.
// It is subject to the license terms in the LICENSE file found in the top-level directory
// of this distribution and at http://opencv.org/license.html.


// #if defined _HFS_CUDA_ON_

#include "../precomp.hpp"
#include "../magnitude/magnitude.hpp"

namespace cv { namespace hfs {

__global__ void derrivativeXYDevice(const uchar *gray_img,
    int *delta_x, int *delta_y, int *mag, Vector2i img_size)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x > img_size.x - 1 || y > img_size.y - 1)
        return;

    int idx = y*img_size.x + x;

    if (x == 0)
        delta_x[idx] = gray_img[idx + 1] - gray_img[idx];
    else if (x == img_size.x - 1)
        delta_x[idx] = gray_img[idx] - gray_img[idx - 1];
    else
        delta_x[idx] = gray_img[idx + 1] - gray_img[idx - 1];

    if (y == 0)
        delta_y[idx] = gray_img[idx + img_size.x] - gray_img[idx];
    else if (y == img_size.y - 1)
        delta_y[idx] = gray_img[idx] - gray_img[idx - img_size.x];
    else
        delta_y[idx] = gray_img[idx + img_size.x] - gray_img[idx - img_size.x];

    mag[idx] = (int)(0.5 +
        sqrt((double)(delta_x[idx] * delta_x[idx] + delta_y[idx] * delta_y[idx])));
}

__device__ __forceinline__ int dmin(int a, int b)
{
    return a < b ? a : b;
}

__device__ __forceinline__ int dmax(int a, int b)
{
    return a > b ? a : b;
}

__global__ void nonMaxSuppDevice(uchar *nms_mag,
    int *delta_x, int *delta_y, int *mag, Vector2i img_size)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x > img_size.x - 1 || y > img_size.y - 1) return;

    int idx = y*img_size.x + x;

    if (x == 0 || x == img_size.x - 1 || y == 0 || y == img_size.y - 1)
    {
        nms_mag[idx] = 0;
        return;
    }

    int m00, gx, gy, z1, z2;
    double mag1, mag2, xprep, yprep;

    m00 = mag[idx];
    if (m00 == 0)
    {
        nms_mag[idx] = 0;
        return;
    }
    else
    {
        xprep = -(gx = delta_x[idx]) / ((double)m00);
        yprep = (gy = delta_y[idx]) / ((double)m00);
    }

    if (gx >= 0)
    {
        if (gy >= 0)
        {
            if (gx >= gy)
            {
                z1 = mag[idx - 1];
                z2 = mag[idx - img_size.x - 1];
                mag1 = (m00 - z1)*xprep + (z2 - z1)*yprep;

                z1 = mag[idx + 1];
                z2 = mag[idx + img_size.x + 1];
                mag2 = (m00 - z1)*xprep + (z2 - z1)*yprep;
            }
            else
            {
                z1 = mag[idx - img_size.x];
                z2 = mag[idx - img_size.x - 1];
                mag1 = (z1 - z2)*xprep + (z1 - m00)*yprep;

                z1 = mag[idx + img_size.x];
                z2 = mag[idx + img_size.x + 1];
                mag2 = (z1 - z2)*xprep + (z1 - m00)*yprep;
            }
        }
        else
        {
            if (gx >= -gy)
            {
                z1 = mag[idx - 1];
                z2 = mag[idx + img_size.x - 1];
                mag1 = (m00 - z1)*xprep + (z1 - z2)*yprep;

                z1 = mag[idx + 1];
                z2 = mag[idx - img_size.x + 1];
                mag2 = (m00 - z1)*xprep + (z1 - z2)*yprep;
            }
            else
            {
                z1 = mag[idx + img_size.x];
                z2 = mag[idx + img_size.x - 1];
                mag1 = (z1 - z2)*xprep + (m00 - z1)*yprep;

                z1 = mag[idx - img_size.x];
                z2 = mag[idx - img_size.x + 1];
                mag2 = (z1 - z2)*xprep + (m00 - z1)*yprep;
            }
        }
    }
    else
    {
        if (gy >= 0)
        {
            if (-gx >= gy)
            {
                z1 = mag[idx + 1];
                z2 = mag[idx - img_size.x + 1];
                mag1 = (z1 - m00)*xprep + (z2 - z1)*yprep;

                z1 = mag[idx - 1];
                z2 = mag[idx + img_size.x - 1];
                mag2 = (z1 - m00)*xprep + (z2 - z1)*yprep;
            }
            else
            {
                z1 = mag[idx - img_size.x];
                z2 = mag[idx - img_size.x + 1];
                mag1 = (z2 - z1)*xprep + (z1 - m00)*yprep;

                z1 = mag[idx + img_size.x];
                z2 = mag[idx + img_size.x - 1];
                mag2 = (z2 - z1)*xprep + (z1 - m00)*yprep;
            }
        }
        else
        {
            if (-gx > -gy)
            {
                z1 = mag[idx + 1];
                z2 = mag[idx + img_size.x + 1];
                mag1 = (z1 - m00)*xprep + (z1 - z2)*yprep;

                z1 = mag[idx - 1];
                z2 = mag[idx - img_size.x - 1];
                mag2 = (z1 - m00)*xprep + (z1 - z2)*yprep;
            }
            else
            {
                z1 = mag[idx + img_size.x];
                z2 = mag[idx + img_size.x + 1];
                mag1 = (z2 - z1)*xprep + (m00 - z1)*yprep;

                z1 = mag[idx - img_size.x];
                z2 = mag[idx - img_size.x - 1];
                mag2 = (z2 - z1)*xprep + (m00 - z1)*yprep;
            }
        }
    }

    if (mag1 > 0 || mag2 >= 0)
        nms_mag[idx] = 0;
    else
        nms_mag[idx] = (uchar)dmin(dmax(m00, 0), 255);
}

void Magnitude::derrivativeXYGpu()
{
    uchar *gray_ptr = gray_img->getGpuData();
    int *dx_ptr = delta_x->getGpuData();
    int *dy_ptr = delta_y->getGpuData();
    int *mag_ptr = mag->getGpuData();

    dim3 blockSize(HFS_BLOCK_DIM, HFS_BLOCK_DIM);
    dim3 gridSize((int)ceil((float)img_size.x / (float)blockSize.x),
        (int)ceil((float)img_size.y / (float)blockSize.y));

    derrivativeXYDevice <<<gridSize, blockSize >>>
        (gray_ptr, dx_ptr, dy_ptr, mag_ptr, img_size);
}

void Magnitude::nonMaxSuppGpu()
{
    int *dx_ptr = delta_x->getGpuData();
    int *dy_ptr = delta_y->getGpuData();
    int *mag_ptr = mag->getGpuData();
    uchar *nms_ptr = nms_mag->getGpuData();

    dim3 blockSize(HFS_BLOCK_DIM, HFS_BLOCK_DIM);
    dim3 gridSize((int)ceil((float)img_size.x / (float)blockSize.x),
        (int)ceil((float)img_size.y / (float)blockSize.y));

    nonMaxSuppDevice <<<gridSize, blockSize >>>
        (nms_ptr, dx_ptr, dy_ptr, mag_ptr, img_size);
}

void Magnitude::processImgGpu(const Mat& bgr3u, Mat& mag1u)
{
    Mat gray, blur1u;
    cvtColor(bgr3u, gray, COLOR_BGR2GRAY);
    GaussianBlur(gray, blur1u, Size(7, 7), 1, 1);

    img_size.x = bgr3u.cols;
    img_size.y = bgr3u.rows;

    loadImage(blur1u, gray_img);
    gray_img->updateDeviceFromHost();
    derrivativeXYGpu();
    nonMaxSuppGpu();
    mag1u.create(bgr3u.rows, bgr3u.cols, CV_8UC1);
    nms_mag->updateHostFromDevice();
    loadImage(nms_mag, mag1u);
}

}}

// #endif
