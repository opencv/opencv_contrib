#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"

namespace cv { namespace cuda { namespace device
{
    namespace optical_flow
    {
        #define NEEDLE_MAP_SCALE 16
        #define NUM_VERTS_PER_ARROW 6

        __global__ void NeedleMapAverageKernel(const PtrStepSzf u, const PtrStepf v, PtrStepf u_avg, PtrStepf v_avg)
        {
            __shared__ float smem[2 * NEEDLE_MAP_SCALE];

            volatile float* u_col_sum = smem;
            volatile float* v_col_sum = u_col_sum + NEEDLE_MAP_SCALE;

            const int x = blockIdx.x * NEEDLE_MAP_SCALE + threadIdx.x;
            const int y = blockIdx.y * NEEDLE_MAP_SCALE;

            u_col_sum[threadIdx.x] = 0;
            v_col_sum[threadIdx.x] = 0;

            #pragma unroll
            for(int i = 0; i < NEEDLE_MAP_SCALE; ++i)
            {
                u_col_sum[threadIdx.x] += u(::min(y + i, u.rows - 1), x);
                v_col_sum[threadIdx.x] += v(::min(y + i, u.rows - 1), x);
            }

            if (threadIdx.x < 8)
            {
                // now add the column sums
                const uint X = threadIdx.x;

                if (X | (0xfe == 0xfe))  // bit 0 is 0
                {
                    u_col_sum[threadIdx.x] += u_col_sum[threadIdx.x + 1];
                    v_col_sum[threadIdx.x] += v_col_sum[threadIdx.x + 1];
                }

                if (X | (0xfe == 0xfc)) // bits 0 & 1 == 0
                {
                    u_col_sum[threadIdx.x] += u_col_sum[threadIdx.x + 2];
                    v_col_sum[threadIdx.x] += v_col_sum[threadIdx.x + 2];
                }

                if (X | (0xf8 == 0xf8))
                {
                    u_col_sum[threadIdx.x] += u_col_sum[threadIdx.x + 4];
                    v_col_sum[threadIdx.x] += v_col_sum[threadIdx.x + 4];
                }

                if (X == 0)
                {
                    u_col_sum[threadIdx.x] += u_col_sum[threadIdx.x + 8];
                    v_col_sum[threadIdx.x] += v_col_sum[threadIdx.x + 8];
                }
            }

            if (threadIdx.x == 0)
            {
                const float coeff = 1.0f / (NEEDLE_MAP_SCALE * NEEDLE_MAP_SCALE);

                u_col_sum[0] *= coeff;
                v_col_sum[0] *= coeff;

                u_avg(blockIdx.y, blockIdx.x) = u_col_sum[0];
                v_avg(blockIdx.y, blockIdx.x) = v_col_sum[0];
            }
        }

        void NeedleMapAverage_gpu(PtrStepSzf u, PtrStepSzf v, PtrStepSzf u_avg, PtrStepSzf v_avg)
        {
            const dim3 block(NEEDLE_MAP_SCALE);
            const dim3 grid(u_avg.cols, u_avg.rows);

            NeedleMapAverageKernel<<<grid, block>>>(u, v, u_avg, v_avg);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }

        __global__ void NeedleMapVertexKernel(const PtrStepSzf u_avg, const PtrStepf v_avg, float* vertex_data, float* color_data, float max_flow, float xscale, float yscale)
        {
            // test - just draw a triangle at each pixel
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            const float arrow_x = x * NEEDLE_MAP_SCALE + NEEDLE_MAP_SCALE / 2.0f;
            const float arrow_y = y * NEEDLE_MAP_SCALE + NEEDLE_MAP_SCALE / 2.0f;

            float3 v[NUM_VERTS_PER_ARROW];

            if (x < u_avg.cols && y < u_avg.rows)
            {
                const float u_avg_val = u_avg(y, x);
                const float v_avg_val = v_avg(y, x);

                const float theta = ::atan2f(v_avg_val, u_avg_val);

                float r = ::sqrtf(v_avg_val * v_avg_val + u_avg_val * u_avg_val);
                r = fmin(14.0f * (r / max_flow), 14.0f);

                v[0].z = 1.0f;
                v[1].z = 0.7f;
                v[2].z = 0.7f;
                v[3].z = 0.7f;
                v[4].z = 0.7f;
                v[5].z = 1.0f;

                v[0].x = arrow_x;
                v[0].y = arrow_y;
                v[5].x = arrow_x;
                v[5].y = arrow_y;

                v[2].x = arrow_x + r * ::cosf(theta);
                v[2].y = arrow_y + r * ::sinf(theta);
                v[3].x = v[2].x;
                v[3].y = v[2].y;

                r = ::fmin(r, 2.5f);

                v[1].x = arrow_x + r * ::cosf(theta - CV_PI_F / 2.0f);
                v[1].y = arrow_y + r * ::sinf(theta - CV_PI_F / 2.0f);

                v[4].x = arrow_x + r * ::cosf(theta + CV_PI_F / 2.0f);
                v[4].y = arrow_y + r * ::sinf(theta + CV_PI_F / 2.0f);

                int indx = (y * u_avg.cols + x) * NUM_VERTS_PER_ARROW * 3;

                color_data[indx] = (theta - CV_PI_F) / CV_PI_F * 180.0f;
                vertex_data[indx++] = v[0].x * xscale;
                vertex_data[indx++] = v[0].y * yscale;
                vertex_data[indx++] = v[0].z;

                color_data[indx] = (theta - CV_PI_F) / CV_PI_F * 180.0f;
                vertex_data[indx++] = v[1].x * xscale;
                vertex_data[indx++] = v[1].y * yscale;
                vertex_data[indx++] = v[1].z;

                color_data[indx] = (theta - CV_PI_F) / CV_PI_F * 180.0f;
                vertex_data[indx++] = v[2].x * xscale;
                vertex_data[indx++] = v[2].y * yscale;
                vertex_data[indx++] = v[2].z;

                color_data[indx] = (theta - CV_PI_F) / CV_PI_F * 180.0f;
                vertex_data[indx++] = v[3].x * xscale;
                vertex_data[indx++] = v[3].y * yscale;
                vertex_data[indx++] = v[3].z;

                color_data[indx] = (theta - CV_PI_F) / CV_PI_F * 180.0f;
                vertex_data[indx++] = v[4].x * xscale;
                vertex_data[indx++] = v[4].y * yscale;
                vertex_data[indx++] = v[4].z;

                color_data[indx] = (theta - CV_PI_F) / CV_PI_F * 180.0f;
                vertex_data[indx++] = v[5].x * xscale;
                vertex_data[indx++] = v[5].y * yscale;
                vertex_data[indx++] = v[5].z;
            }
        }

        void CreateOpticalFlowNeedleMap_gpu(PtrStepSzf u_avg, PtrStepSzf v_avg, float* vertex_buffer, float* color_data, float max_flow, float xscale, float yscale)
        {
            const dim3 block(16);
            const dim3 grid(divUp(u_avg.cols, block.x), divUp(u_avg.rows, block.y));

            NeedleMapVertexKernel<<<grid, block>>>(u_avg, v_avg, vertex_buffer, color_data, max_flow, xscale, yscale);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }
    }
}}}

#endif /* CUDA_DISABLER */
