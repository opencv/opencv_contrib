#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

////////////////////////////////////////////////////////////////////////////////
//
// NVIDIA CUDA implementation of Brox et al Optical Flow algorithm
//
// Algorithm is explained in the original paper:
//      T. Brox, A. Bruhn, N. Papenberg, J. Weickert:
//      High accuracy optical flow estimation based on a theory for warping.
//      ECCV 2004.
//
// Implementation by Mikhail Smirnov
// email: msmirnov@nvidia.com, devsupport@nvidia.com
//
// Credits for help with the code to:
// Alexey Mendelenko, Anton Obukhov, and Alexander Kharlamov.
//
////////////////////////////////////////////////////////////////////////////////

#include <iostream>
#include <vector>
#include <memory>

#include "opencv2/core/cuda/utility.hpp"

#include "opencv2/cudalegacy/NPP_staging.hpp"
#include "opencv2/cudalegacy/NCVBroxOpticalFlow.hpp"
#include <opencv2/cudev/ptr2d/texture.hpp>


typedef NCVVectorAlloc<Ncv32f> FloatVector;
typedef cv::cudev::TexturePtr<float> Ptr2D;
typedef cv::cudev::Texture<float> Texture;

/////////////////////////////////////////////////////////////////////////////////////////
// Implementation specific constants
/////////////////////////////////////////////////////////////////////////////////////////
__device__ const float eps2 = 1e-6f;

/////////////////////////////////////////////////////////////////////////////////////////
// Additional defines
/////////////////////////////////////////////////////////////////////////////////////////

// rounded up division
inline int iDivUp(int a, int b)
{
    return (a + b - 1)/b;
}

/////////////////////////////////////////////////////////////////////////////////////////
// SUPPLEMENTARY FUNCTIONS
/////////////////////////////////////////////////////////////////////////////////////////

///////////////////////////////////////////////////////////////////////////////
/// \brief performs pointwise summation of two vectors stored in device memory
/// \param d_res    - pointer to resulting vector (device memory)
/// \param d_op1    - term #1 (device memory)
/// \param d_op2    - term #2 (device memory)
/// \param len    - vector size
///////////////////////////////////////////////////////////////////////////////
__global__ void pointwise_add(float *d_res, const float *d_op1, const float *d_op2, const int len)
{
    const int pos = blockIdx.x*blockDim.x + threadIdx.x;

    if(pos >= len) return;

    d_res[pos] = d_op1[pos] + d_op2[pos];
}

///////////////////////////////////////////////////////////////////////////////
/// \brief wrapper for summation kernel.
///  Computes \b op1 + \b op2 and stores result to \b res
/// \param res   array, containing op1 + op2 (device memory)
/// \param op1   term #1 (device memory)
/// \param op2   term #2 (device memory)
/// \param count vector size
///////////////////////////////////////////////////////////////////////////////
static void add(float *res, const float *op1, const float *op2, const int count, hipStream_t stream)
{
    dim3 threads(256);
    dim3 blocks(iDivUp(count, threads.x));

    pointwise_add<<<blocks, threads, 0, stream>>>(res, op1, op2, count);
}

///////////////////////////////////////////////////////////////////////////////
/// \brief wrapper for summation kernel.
/// Increments \b res by \b rhs
/// \param res   initial vector, will be replaced with result (device memory)
/// \param rhs   increment (device memory)
/// \param count vector size
///////////////////////////////////////////////////////////////////////////////
static void add(float *res, const float *rhs, const int count, hipStream_t stream)
{
    add(res, res, rhs, count, stream);
}

///////////////////////////////////////////////////////////////////////////////
/// \brief kernel for scaling vector by scalar
/// \param d_res  scaled vector (device memory)
/// \param d_src  source vector (device memory)
/// \param scale  scalar to scale by
/// \param len    vector size (number of elements)
///////////////////////////////////////////////////////////////////////////////
__global__ void scaleVector(float *d_res, const float *d_src, float scale, const int len)
{
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;

    if (pos >= len) return;

    d_res[pos] = d_src[pos] * scale;
}

///////////////////////////////////////////////////////////////////////////////
/// \brief scale vector by scalar
///
/// kernel wrapper
/// \param d_res  scaled vector (device memory)
/// \param d_src  source vector (device memory)
/// \param scale  scalar to scale by
/// \param len    vector size (number of elements)
/// \param stream CUDA stream
///////////////////////////////////////////////////////////////////////////////
static void ScaleVector(float *d_res, const float *d_src, float scale, const int len, hipStream_t stream)
{
    dim3 threads(256);
    dim3 blocks(iDivUp(len, threads.x));

    scaleVector<<<blocks, threads, 0, stream>>>(d_res, d_src, scale, len);
}

const int SOR_TILE_WIDTH = 32;
const int SOR_TILE_HEIGHT = 6;
const int PSOR_TILE_WIDTH = 32;
const int PSOR_TILE_HEIGHT = 6;
const int PSOR_PITCH = PSOR_TILE_WIDTH + 4;
const int PSOR_HEIGHT = PSOR_TILE_HEIGHT + 4;

///////////////////////////////////////////////////////////////////////////////
///\brief Utility function. Compute smooth term diffusivity along x axis
///\param s (out) pointer to memory location for result (diffusivity)
///\param pos (in) position within shared memory array containing \b u
///\param u (in) shared memory array containing \b u
///\param v (in) shared memory array containing \b v
///\param du (in) shared memory array containing \b du
///\param dv (in) shared memory array containing \b dv
///////////////////////////////////////////////////////////////////////////////
__forceinline__ __device__ void diffusivity_along_x(float *s, int pos, const float *u, const float *v, const float *du, const float *dv)
{
    //x derivative between pixels (i,j) and (i-1,j)
    const int left = pos-1;
    float u_x = u[pos] + du[pos] - u[left] - du[left];
    float v_x = v[pos] + dv[pos] - v[left] - dv[left];
    const int up        = pos + PSOR_PITCH;
    const int down      = pos - PSOR_PITCH;
    const int up_left   = up - 1;
    const int down_left = down-1;
    //y derivative between pixels (i,j) and (i-1,j)
    float u_y = 0.25f*(u[up] + du[up] + u[up_left] + du[up_left] - u[down] - du[down] - u[down_left] - du[down_left]);
    float v_y = 0.25f*(v[up] + dv[up] + v[up_left] + dv[up_left] - v[down] - dv[down] - v[down_left] - dv[down_left]);
    *s = 0.5f / sqrtf(u_x*u_x + v_x*v_x + u_y*u_y + v_y*v_y + eps2);
}

///////////////////////////////////////////////////////////////////////////////
///\brief Utility function. Compute smooth term diffusivity along y axis
///\param s (out) pointer to memory location for result (diffusivity)
///\param pos (in) position within shared memory array containing \b u
///\param u (in) shared memory array containing \b u
///\param v (in) shared memory array containing \b v
///\param du (in) shared memory array containing \b du
///\param dv (in) shared memory array containing \b dv
///////////////////////////////////////////////////////////////////////////////
__forceinline__ __device__ void diffusivity_along_y(float *s, int pos, const float *u, const float *v, const float *du, const float *dv)
{
    //y derivative between pixels (i,j) and (i,j-1)
    const int down = pos-PSOR_PITCH;
    float u_y = u[pos] + du[pos] - u[down] - du[down];
    float v_y = v[pos] + dv[pos] - v[down] - dv[down];
    const int right      = pos + 1;
    const int left       = pos - 1;
    const int down_right = down + 1;
    const int down_left  = down - 1;
    //x derivative between pixels (i,j) and (i,j-1);
    float u_x = 0.25f*(u[right] + u[down_right] + du[right] + du[down_right] - u[left] - u[down_left] - du[left] - du[down_left]);
    float v_x = 0.25f*(v[right] + v[down_right] + dv[right] + dv[down_right] - v[left] - v[down_left] - dv[left] - dv[down_left]);
    *s = 0.5f/sqrtf(u_x*u_x + v_x*v_x + u_y*u_y + v_y*v_y + eps2);
}

///////////////////////////////////////////////////////////////////////////////
///\brief Utility function. Load element of 2D global memory to shared memory
///\param smem pointer to shared memory array
///\param is shared memory array column
///\param js shared memory array row
///\param w number of columns in global memory array
///\param h number of rows in global memory array
///\param p global memory array pitch in floats
///////////////////////////////////////////////////////////////////////////////
__forceinline__ __device__ void load_array_element(Ptr2D texSrc, float *smem, int is, int js, int i, int j, int w, int h, int p)
{
    //position within shared memory array
    const int ijs = js * PSOR_PITCH + is;
    //mirror reflection across borders
    i = max(i, -i-1);
    i = min(i, w-i+w-1);
    j = max(j, -j-1);
    j = min(j, h-j+h-1);
    const int pos = j * p + i;
    smem[ijs] = texSrc(pos);
}

///////////////////////////////////////////////////////////////////////////////
///\brief Utility function. Load part (tile) of 2D global memory to shared memory
///\param smem pointer to target shared memory array
///\param ig column number within source
///\param jg row number within source
///\param w number of columns in global memory array
///\param h number of rows in global memory array
///\param p global memory array pitch in floats
///////////////////////////////////////////////////////////////////////////////
__forceinline__ __device__ void load_array(Ptr2D texSrc, float *smem, int ig, int jg, int w, int h, int p)
{
    const int i = threadIdx.x + 2;
    const int j = threadIdx.y + 2;
    load_array_element(texSrc, smem, i, j, ig, jg, w, h, p);//load current pixel
    __syncthreads();
    if(threadIdx.y < 2)
    {
        //load bottom shadow elements
        load_array_element(texSrc, smem, i, j-2, ig, jg-2, w, h, p);
        if(threadIdx.x < 2)
        {
            //load bottom right shadow elements
            load_array_element(texSrc, smem, i+PSOR_TILE_WIDTH, j-2, ig+PSOR_TILE_WIDTH, jg-2, w, h, p);
            //load middle right shadow elements
            load_array_element(texSrc, smem, i+PSOR_TILE_WIDTH, j, ig+PSOR_TILE_WIDTH, jg, w, h, p);
        }
        else if(threadIdx.x >= PSOR_TILE_WIDTH-2)
        {
            //load bottom left shadow elements
            load_array_element(texSrc, smem, i-PSOR_TILE_WIDTH, j-2, ig-PSOR_TILE_WIDTH, jg-2, w, h, p);
            //load middle left shadow elements
            load_array_element(texSrc, smem, i-PSOR_TILE_WIDTH, j, ig-PSOR_TILE_WIDTH, jg, w, h, p);
        }
    }
    else if(threadIdx.y >= PSOR_TILE_HEIGHT-2)
    {
        //load upper shadow elements
        load_array_element(texSrc, smem, i, j+2, ig, jg+2, w, h, p);
        if(threadIdx.x < 2)
        {
            //load upper right shadow elements
            load_array_element(texSrc, smem, i+PSOR_TILE_WIDTH, j+2, ig+PSOR_TILE_WIDTH, jg+2, w, h, p);
            //load middle right shadow elements
            load_array_element(texSrc, smem, i+PSOR_TILE_WIDTH, j, ig+PSOR_TILE_WIDTH, jg, w, h, p);
        }
        else if(threadIdx.x >= PSOR_TILE_WIDTH-2)
        {
            //load upper left shadow elements
            load_array_element(texSrc, smem, i-PSOR_TILE_WIDTH, j+2, ig-PSOR_TILE_WIDTH, jg+2, w, h, p);
            //load middle left shadow elements
            load_array_element(texSrc, smem, i-PSOR_TILE_WIDTH, j, ig-PSOR_TILE_WIDTH, jg, w, h, p);
        }
    }
    else
    {
        //load middle shadow elements
        if(threadIdx.x < 2)
        {
            //load middle right shadow elements
            load_array_element(texSrc, smem, i+PSOR_TILE_WIDTH, j, ig+PSOR_TILE_WIDTH, jg, w, h, p);
        }
        else if(threadIdx.x >= PSOR_TILE_WIDTH-2)
        {
            //load middle left shadow elements
            load_array_element(texSrc, smem, i-PSOR_TILE_WIDTH, j, ig-PSOR_TILE_WIDTH, jg, w, h, p);
        }
    }
    __syncthreads();
}

///////////////////////////////////////////////////////////////////////////////
/// \brief computes matrix of linearised system for \c du, \c dv
/// Computed values reside in GPU memory. \n
/// Matrix computation is divided into two steps. This kernel performs first step\n
/// - compute smoothness term diffusivity between pixels - psi dash smooth
/// - compute robustness factor in the data term - psi dash data
/// \param diffusivity_x (in/out) diffusivity between pixels along x axis in smoothness term
/// \param diffusivity_y (in/out) diffusivity between pixels along y axis in smoothness term
/// \param denominator_u (in/out) precomputed part of expression for new du value in SOR iteration
/// \param denominator_v (in/out) precomputed part of expression for new dv value in SOR iteration
/// \param numerator_dudv (in/out) precomputed part of expression for new du and dv value in SOR iteration
/// \param numerator_u (in/out) precomputed part of expression for new du value in SOR iteration
/// \param numerator_v (in/out) precomputed part of expression for new dv value in SOR iteration
/// \param w (in) frame width
/// \param h (in) frame height
/// \param pitch (in) pitch in floats
/// \param alpha (in) alpha in Brox model (flow smoothness)
/// \param gamma (in) gamma in Brox model (edge importance)
///////////////////////////////////////////////////////////////////////////////
__global__ void prepare_sor_stage_1_tex(Ptr2D texU, Ptr2D texV, Ptr2D texDu, Ptr2D texDv, Ptr2D texI0, Ptr2D texI1, Ptr2D texIx, Ptr2D texIxx, Ptr2D texIx0, Ptr2D texIy, Ptr2D texIyy,
    Ptr2D texIy0, Ptr2D texIxy, float *diffusivity_x, float *diffusivity_y, float *denominator_u, float *denominator_v, float *numerator_dudv, float *numerator_u, float *numerator_v,
    int w, int h, int s, float alpha, float gamma)
{
    __shared__ float u[PSOR_PITCH * PSOR_HEIGHT];
    __shared__ float v[PSOR_PITCH * PSOR_HEIGHT];
    __shared__ float du[PSOR_PITCH * PSOR_HEIGHT];
    __shared__ float dv[PSOR_PITCH * PSOR_HEIGHT];

    //position within tile
    const int i = threadIdx.x;
    const int j = threadIdx.y;
    //position within smem arrays
    const int ijs = (j+2) * PSOR_PITCH + i + 2;
    //position within global memory
    const int ig  = blockIdx.x * blockDim.x + threadIdx.x;
    const int jg  = blockIdx.y * blockDim.y + threadIdx.y;
    const int ijg = jg * s + ig;
    //position within texture
    float x = (float)ig + 0.5f;
    float y = (float)jg + 0.5f;
    //load u  and v to smem
    load_array(texU, u, ig, jg, w, h, s);
    load_array(texV, v, ig, jg, w, h, s);
    load_array(texDu, du, ig, jg, w, h, s);
    load_array(texDv, dv, ig, jg, w, h, s);
    //warped position
    float wx = (x + u[ijs])/(float)w;
    float wy = (y + v[ijs])/(float)h;
    x /= (float)w;
    y /= (float)h;
    //compute image derivatives
    const float Iz = texI1(wy, wx) - texI0(y,x);
    const float Ix  = texIx(wy, wx);
    const float Ixz = Ix - texIx0(y, x);
    const float Ixy = texIxy(wy, wx);
    const float Ixx = texIxx(wy, wx);
    const float Iy = texIy(wy, wx);
    const float Iyz = Iy - texIy0(y, x);
    const float Iyy = texIyy(wy, wx);
    //compute data term
    float q0, q1, q2;
    q0 = Iz  + Ix  * du[ijs] + Iy  * dv[ijs];
    q1 = Ixz + Ixx * du[ijs] + Ixy * dv[ijs];
    q2 = Iyz + Ixy * du[ijs] + Iyy * dv[ijs];
    float data_term = 0.5f * rsqrtf(q0*q0 + gamma*(q1*q1 + q2*q2) + eps2);
    //scale data term by 1/alpha
    data_term /= alpha;
    //compute smoothness term (diffusivity)
    float sx, sy;

    if(ig >= w || jg >= h) return;

    diffusivity_along_x(&sx, ijs, u, v, du, dv);
    diffusivity_along_y(&sy, ijs, u, v, du, dv);

    if(ig == 0) sx = 0.0f;
    if(jg == 0) sy = 0.0f;

    numerator_dudv[ijg] = data_term * (Ix*Iy + gamma * Ixy*(Ixx + Iyy));
    numerator_u[ijg]    = data_term * (Ix*Iz + gamma * (Ixx*Ixz + Ixy*Iyz));
    numerator_v[ijg]    = data_term * (Iy*Iz + gamma * (Iyy*Iyz + Ixy*Ixz));
    denominator_u[ijg]  = data_term * (Ix*Ix + gamma * (Ixy*Ixy + Ixx*Ixx));
    denominator_v[ijg]  = data_term * (Iy*Iy + gamma * (Ixy*Ixy + Iyy*Iyy));
    diffusivity_x[ijg]  = sx;
    diffusivity_y[ijg]  = sy;
}

///////////////////////////////////////////////////////////////////////////////
///\brief computes matrix of linearised system for \c du, \c dv
///\param inv_denominator_u
///\param inv_denominator_v
///\param w
///\param h
///\param s
///////////////////////////////////////////////////////////////////////////////
__global__ void prepare_sor_stage_2(Ptr2D texDiffX, Ptr2D texDiffY, float *inv_denominator_u, float *inv_denominator_v, int w, int h, int s)
{
    __shared__ float sx[(PSOR_TILE_WIDTH+1) * (PSOR_TILE_HEIGHT+1)];
    __shared__ float sy[(PSOR_TILE_WIDTH+1) * (PSOR_TILE_HEIGHT+1)];
    //position within tile
    const int i = threadIdx.x;
    const int j = threadIdx.y;
    //position within smem arrays
    const int ijs = j*(PSOR_TILE_WIDTH+1) + i;
    //position within global memory
    const int ig  = blockIdx.x * blockDim.x + threadIdx.x;
    const int jg  = blockIdx.y * blockDim.y + threadIdx.y;
    const int ijg = jg*s + ig;
    int inside = ig < w && jg < h;
    float denom_u;
    float denom_v;
    if(inside)
    {
        denom_u = inv_denominator_u[ijg];
        denom_v = inv_denominator_v[ijg];
    }
    if(inside)
    {
        sx[ijs] = texDiffX(ijg);
        sy[ijs] = texDiffY(ijg);
    }
    else
    {
        sx[ijs] = 0.0f;
        sy[ijs] = 0.0f;
    }
    int up = ijs+PSOR_TILE_WIDTH+1;
    if(j == PSOR_TILE_HEIGHT-1)
    {
        if(jg < h-1 && inside)
            sy[up] = texDiffY(ijg + s);
        else
            sy[up] = 0.0f;
    }
    int right = ijs + 1;
    if(threadIdx.x == PSOR_TILE_WIDTH-1)
    {
        if(ig < w-1 && inside)
            sx[right] = texDiffX(ijg + 1);
        else
            sx[right] = 0.0f;
    }
    __syncthreads();
    float diffusivity_sum;
    diffusivity_sum = sx[ijs] + sx[ijs+1] + sy[ijs] + sy[ijs+PSOR_TILE_WIDTH+1];
    if(inside)
    {
        denom_u += diffusivity_sum;
        denom_v += diffusivity_sum;
        inv_denominator_u[ijg] = 1.0f/denom_u;
        inv_denominator_v[ijg] = 1.0f/denom_v;
    }
}

/////////////////////////////////////////////////////////////////////////////////////////
// Red-Black SOR
/////////////////////////////////////////////////////////////////////////////////////////

template<int isBlack> __global__ void sor_pass(Ptr2D texU, Ptr2D texV, Ptr2D texDu, Ptr2D texDv, Ptr2D texDiffX, Ptr2D texDiffY, float *new_du, float *new_dv, const float *g_inv_denominator_u,
    const float *g_inv_denominator_v, const float *g_numerator_u, const float *g_numerator_v, const float *g_numerator_dudv, float omega, int width, int height, int stride)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if(i >= width || j >= height)
        return;

    const int pos = j * stride + i;
    const int pos_r = i < width - 1 ? pos + 1 : pos;
    const int pos_u = j < height - 1 ? pos + stride : pos;
    const int pos_d = j > 0 ? pos - stride : pos;
    const int pos_l = i > 0 ? pos - 1 : pos;

    //load smooth term
    float s_up, s_left, s_right, s_down;
    s_left = texDiffX(pos);
    s_down = texDiffY(pos);
    if(i < width-1)
        s_right = texDiffX(pos_r);
    else
        s_right = 0.0f; //Neumann BC
    if(j < height-1)
        s_up = texDiffY(pos_u);
    else
        s_up = 0.0f; //Neumann BC

    //load u, v and du, dv
    float u_up, u_left, u_right, u_down, u;
    float v_up, v_left, v_right, v_down, v;
    float du_up, du_left, du_right, du_down, du;
    float dv_up, dv_left, dv_right, dv_down, dv;

    u_left = texU(pos_l);
    u_right = texU(pos_r);
    u_down = texU(pos_d);
    u_up = texU(pos_u);
    u = texU(pos);

    v_left = texV(pos_l);
    v_right = texV(pos_r);
    v_down = texV(pos_d);
    v = texV(pos);
    v_up = texV(pos_u);

    du =  texDu(pos);
    du_left = texDu(pos_l);
    du_right = texDu(pos_r);
    du_down = texDu(pos_d);
    du_up = texDu(pos_u);

    dv = texDv(pos);
    dv_left = texDv(pos_l);
    dv_right = texDv(pos_r);
    dv_down = texDv(pos_d);
    dv_up = texDv(pos_u);
    float numerator_dudv    = g_numerator_dudv[pos];

    if((i+j)%2 == isBlack)
    {
        // update du
        float numerator_u = (s_left*(u_left + du_left) + s_up*(u_up + du_up) + s_right*(u_right + du_right) + s_down*(u_down + du_down) -
                             u * (s_left + s_right + s_up + s_down) - g_numerator_u[pos] - numerator_dudv*dv);

        du = (1.0f - omega) * du + omega * g_inv_denominator_u[pos] * numerator_u;

        // update dv
        float numerator_v = (s_left*(v_left + dv_left) + s_up*(v_up + dv_up) + s_right*(v_right + dv_right) + s_down*(v_down + dv_down) -
                             v * (s_left + s_right + s_up + s_down) - g_numerator_v[pos] - numerator_dudv*du);

        dv = (1.0f - omega) * dv + omega * g_inv_denominator_v[pos] * numerator_v;
    }
    new_du[pos] = du;
    new_dv[pos] = dv;
}

///////////////////////////////////////////////////////////////////////////////
// utility functions
///////////////////////////////////////////////////////////////////////////////
namespace
{
    struct ImagePyramid
    {
        std::vector<FloatVector*> img0;
        std::vector<FloatVector*> img1;

        std::vector<Ncv32u> w;
        std::vector<Ncv32u> h;

        explicit ImagePyramid(int outer_iterations)
        {
            img0.reserve(outer_iterations);
            img1.reserve(outer_iterations);

            w.reserve(outer_iterations);
            h.reserve(outer_iterations);
        }

        ~ImagePyramid()
        {
            w.clear();
            h.clear();

            for (int i = static_cast<int>(img0.size()) - 1; i >= 0; --i)
            {
                delete img1[i];
                delete img0[i];
            }

            img0.clear();
            img1.clear();
        }
    };
}

/////////////////////////////////////////////////////////////////////////////////////////
// MAIN FUNCTION
/////////////////////////////////////////////////////////////////////////////////////////
NCVStatus NCVBroxOpticalFlow(const NCVBroxOpticalFlowDescriptor desc,
                             INCVMemAllocator &gpu_mem_allocator,
                             const NCVMatrix<Ncv32f> &frame0,
                             const NCVMatrix<Ncv32f> &frame1,
                             NCVMatrix<Ncv32f> &uOut,
                             NCVMatrix<Ncv32f> &vOut,
                             hipStream_t stream)
{
    ncvAssertPrintReturn(desc.alpha > 0.0f                   , "Invalid alpha"                      , NCV_INCONSISTENT_INPUT);
    ncvAssertPrintReturn(desc.gamma >= 0.0f                  , "Invalid gamma"                      , NCV_INCONSISTENT_INPUT);
    ncvAssertPrintReturn(desc.number_of_inner_iterations > 0 , "Invalid number of inner iterations" , NCV_INCONSISTENT_INPUT);
    ncvAssertPrintReturn(desc.number_of_outer_iterations > 0 , "Invalid number of outer iterations" , NCV_INCONSISTENT_INPUT);
    ncvAssertPrintReturn(desc.number_of_solver_iterations > 0, "Invalid number of solver iterations", NCV_INCONSISTENT_INPUT);

    const Ncv32u kSourceWidth  = frame0.width();
    const Ncv32u kSourceHeight = frame0.height();

    ncvAssertPrintReturn(frame1.width() == kSourceWidth && frame1.height() == kSourceHeight, "Frame dims do not match", NCV_INCONSISTENT_INPUT);
    ncvAssertReturn(uOut.width() == kSourceWidth && vOut.width() == kSourceWidth &&
        uOut.height() == kSourceHeight && vOut.height() == kSourceHeight, NCV_INCONSISTENT_INPUT);

    ncvAssertReturn(gpu_mem_allocator.isInitialized(), NCV_ALLOCATOR_NOT_INITIALIZED);

    bool kSkipProcessing = gpu_mem_allocator.isCounting();

    int cuda_device;
    ncvAssertCUDAReturn(hipGetDevice(&cuda_device), NCV_CUDA_ERROR);

    hipDeviceProp_t device_props;
    ncvAssertCUDAReturn(hipGetDeviceProperties(&device_props, cuda_device), NCV_CUDA_ERROR);

    Ncv32u alignmentValue = gpu_mem_allocator.alignment ();

    const Ncv32u kStrideAlignmentFloat = alignmentValue / sizeof(float);
    const Ncv32u kSourcePitch = alignUp(kSourceWidth, kStrideAlignmentFloat) * sizeof(float);

    const Ncv32f scale_factor = desc.scale_factor;
    const Ncv32f alpha = desc.alpha;
    const Ncv32f gamma = desc.gamma;

    const Ncv32u kSizeInPixelsAligned = alignUp(kSourceWidth, kStrideAlignmentFloat)*kSourceHeight;

#if defined SAFE_VECTOR_DECL
#undef SAFE_VECTOR_DECL
#endif
#define SAFE_VECTOR_DECL(name, allocator, size) \
    FloatVector name((allocator), (size)); \
    ncvAssertReturn(name.isMemAllocated(), NCV_ALLOCATOR_BAD_ALLOC);

    // matrix elements
    SAFE_VECTOR_DECL(diffusivity_x,  gpu_mem_allocator, kSizeInPixelsAligned);
    SAFE_VECTOR_DECL(diffusivity_y,  gpu_mem_allocator, kSizeInPixelsAligned);
    SAFE_VECTOR_DECL(denom_u,  gpu_mem_allocator, kSizeInPixelsAligned);
    SAFE_VECTOR_DECL(denom_v,  gpu_mem_allocator, kSizeInPixelsAligned);
    SAFE_VECTOR_DECL(num_dudv, gpu_mem_allocator, kSizeInPixelsAligned);
    SAFE_VECTOR_DECL(num_u,    gpu_mem_allocator, kSizeInPixelsAligned);
    SAFE_VECTOR_DECL(num_v,    gpu_mem_allocator, kSizeInPixelsAligned);

    // flow components
    SAFE_VECTOR_DECL(u, gpu_mem_allocator, kSizeInPixelsAligned);
    SAFE_VECTOR_DECL(v, gpu_mem_allocator, kSizeInPixelsAligned);

    SAFE_VECTOR_DECL(u_new, gpu_mem_allocator, kSizeInPixelsAligned);
    SAFE_VECTOR_DECL(v_new, gpu_mem_allocator, kSizeInPixelsAligned);

    // flow increments
    SAFE_VECTOR_DECL(du, gpu_mem_allocator, kSizeInPixelsAligned);
    SAFE_VECTOR_DECL(dv, gpu_mem_allocator, kSizeInPixelsAligned);

    SAFE_VECTOR_DECL(du_new, gpu_mem_allocator, kSizeInPixelsAligned);
    SAFE_VECTOR_DECL(dv_new, gpu_mem_allocator, kSizeInPixelsAligned);

    // temporary storage
    SAFE_VECTOR_DECL(device_buffer, gpu_mem_allocator,
        alignUp(kSourceWidth, kStrideAlignmentFloat) * alignUp(kSourceHeight, kStrideAlignmentFloat));

    // image derivatives
    SAFE_VECTOR_DECL(Ix,  gpu_mem_allocator, kSizeInPixelsAligned);
    SAFE_VECTOR_DECL(Ixx, gpu_mem_allocator, kSizeInPixelsAligned);
    SAFE_VECTOR_DECL(Ix0, gpu_mem_allocator, kSizeInPixelsAligned);
    SAFE_VECTOR_DECL(Iy,  gpu_mem_allocator, kSizeInPixelsAligned);
    SAFE_VECTOR_DECL(Iyy, gpu_mem_allocator, kSizeInPixelsAligned);
    SAFE_VECTOR_DECL(Iy0, gpu_mem_allocator, kSizeInPixelsAligned);
    SAFE_VECTOR_DECL(Ixy, gpu_mem_allocator, kSizeInPixelsAligned);

    // spatial derivative filter size
    const int kDFilterSize = 5;
    SAFE_VECTOR_DECL(derivativeFilter, gpu_mem_allocator, kDFilterSize);

    if (!kSkipProcessing)
    {
        const float derivativeFilterHost[kDFilterSize] = {1.0f, -8.0f, 0.0f, 8.0f, -1.0f};

        ncvAssertCUDAReturn(hipMemcpy(derivativeFilter.ptr(), derivativeFilterHost, sizeof(float) * kDFilterSize,
            hipMemcpyHostToDevice), NCV_CUDA_ERROR);
    }

    //prepare image pyramid
    ImagePyramid pyr(desc.number_of_outer_iterations);

    float scale = 1.0f;

    //cuda arrays for frames
    std::unique_ptr<FloatVector> pI0(new FloatVector(gpu_mem_allocator, kSizeInPixelsAligned));
    ncvAssertReturn(pI0->isMemAllocated(), NCV_ALLOCATOR_BAD_ALLOC);

    std::unique_ptr<FloatVector> pI1(new FloatVector(gpu_mem_allocator, kSizeInPixelsAligned));
    ncvAssertReturn(pI1->isMemAllocated(), NCV_ALLOCATOR_BAD_ALLOC);

    if (!kSkipProcessing)
    {
        //copy frame data to device
        size_t dst_width_in_bytes = alignUp(kSourceWidth, kStrideAlignmentFloat) * sizeof(float);
        size_t src_width_in_bytes = kSourceWidth * sizeof(float);
        size_t src_pitch_in_bytes = frame0.pitch();

        ncvAssertCUDAReturn( hipMemcpy2DAsync(pI0->ptr(), dst_width_in_bytes, frame0.ptr(),
            src_pitch_in_bytes, src_width_in_bytes, kSourceHeight, hipMemcpyDeviceToDevice, stream), NCV_CUDA_ERROR );

        ncvAssertCUDAReturn( hipMemcpy2DAsync(pI1->ptr(), dst_width_in_bytes, frame1.ptr(),
            src_pitch_in_bytes, src_width_in_bytes, kSourceHeight, hipMemcpyDeviceToDevice, stream), NCV_CUDA_ERROR );
    }

    FloatVector* I0 = pI0.release();
    FloatVector* I1 = pI1.release();

        //prepare pyramid
    pyr.img0.push_back(I0);
    pyr.img1.push_back(I1);

    pyr.w.push_back(kSourceWidth);
    pyr.h.push_back(kSourceHeight);

    scale *= scale_factor;

    Ncv32u prev_level_width  = kSourceWidth;
    Ncv32u prev_level_height = kSourceHeight;
    while((prev_level_width > 15) && (prev_level_height > 15) && (static_cast<Ncv32u>(pyr.img0.size()) < desc.number_of_outer_iterations))
    {
        //current resolution
        Ncv32u level_width  = static_cast<Ncv32u>(ceilf(kSourceWidth  * scale));
        Ncv32u level_height = static_cast<Ncv32u>(ceilf(kSourceHeight * scale));

        Ncv32u level_width_aligned  = alignUp(level_width,  kStrideAlignmentFloat);

        Ncv32u buffer_size = alignUp(level_width, kStrideAlignmentFloat) * level_height; // buffer size in floats

        Ncv32u prev_level_pitch = alignUp(prev_level_width, kStrideAlignmentFloat) * sizeof(float);

        std::unique_ptr<FloatVector> level_frame0(new FloatVector(gpu_mem_allocator, buffer_size));
        ncvAssertReturn(level_frame0->isMemAllocated(), NCV_ALLOCATOR_BAD_ALLOC);

        std::unique_ptr<FloatVector> level_frame1(new FloatVector(gpu_mem_allocator, buffer_size));
        ncvAssertReturn(level_frame1->isMemAllocated(), NCV_ALLOCATOR_BAD_ALLOC);

        if (!kSkipProcessing)
        {
            ncvAssertCUDAReturn(hipStreamSynchronize(stream), NCV_CUDA_ERROR);

            NcvSize32u srcSize (prev_level_width, prev_level_height);
            NcvSize32u dstSize (level_width, level_height);
            NcvRect32u srcROI (0, 0, prev_level_width, prev_level_height);
            NcvRect32u dstROI (0, 0, level_width, level_height);

            // frame 0
            ncvAssertReturnNcvStat( nppiStResize_32f_C1R (I0->ptr(), srcSize, prev_level_pitch, srcROI,
                level_frame0->ptr(), dstSize, level_width_aligned * sizeof (float), dstROI, scale_factor, scale_factor, nppStSupersample) );

            // frame 1
            ncvAssertReturnNcvStat( nppiStResize_32f_C1R (I1->ptr(), srcSize, prev_level_pitch, srcROI,
                level_frame1->ptr(), dstSize, level_width_aligned * sizeof (float), dstROI, scale_factor, scale_factor, nppStSupersample) );
        }

        I0 = level_frame0.release();
        I1 = level_frame1.release();

        //store pointers
        pyr.img0.push_back(I0);
        pyr.img1.push_back(I1);

        pyr.w.push_back(level_width);
        pyr.h.push_back(level_height);

        scale *= scale_factor;

        prev_level_width  = level_width;
        prev_level_height = level_height;
    }

    if (!kSkipProcessing)
    {
        //initial values for flow is 0
        ncvAssertCUDAReturn(hipMemsetAsync(u.ptr(), 0, kSizeInPixelsAligned * sizeof(float), stream), NCV_CUDA_ERROR);
        ncvAssertCUDAReturn(hipMemsetAsync(v.ptr(), 0, kSizeInPixelsAligned * sizeof(float), stream), NCV_CUDA_ERROR);

        //select images with lowest resolution
        ncvAssertCUDAReturn(hipStreamSynchronize(stream), NCV_CUDA_ERROR);

        FloatVector* ptrU = &u;
        FloatVector* ptrV = &v;
        FloatVector* ptrUNew = &u_new;
        FloatVector* ptrVNew = &v_new;

        std::vector<FloatVector*>::const_reverse_iterator img0Iter = pyr.img0.rbegin();
        std::vector<FloatVector*>::const_reverse_iterator img1Iter = pyr.img1.rbegin();

        //outer loop
        //warping fixed point iteration
        while(!pyr.w.empty())
        {
            //current grid dimensions
            const Ncv32u kLevelWidth  = pyr.w.back();
            const Ncv32u kLevelHeight = pyr.h.back();
            const Ncv32u kLevelStride = alignUp(kLevelWidth, kStrideAlignmentFloat);

            //size of current image in bytes
            const int kLevelSizeInBytes = kLevelStride * kLevelHeight * sizeof(float);

            //number of points at current resolution
            const int kLevelSizeInPixels = kLevelStride * kLevelHeight;

            //initial guess for du and dv
            ncvAssertCUDAReturn(hipMemsetAsync(du.ptr(), 0, kLevelSizeInBytes, stream), NCV_CUDA_ERROR);
            ncvAssertCUDAReturn(hipMemsetAsync(dv.ptr(), 0, kLevelSizeInBytes, stream), NCV_CUDA_ERROR);

            I0 = *img0Iter;
            I1 = *img1Iter;

            ++img0Iter;
            ++img1Iter;

            Texture texI0(kLevelHeight, kLevelWidth, I0->ptr(), kLevelStride * sizeof(float), true, hipFilterModeLinear, hipAddressModeMirror);
            Texture texI1(kLevelHeight, kLevelWidth, I1->ptr(), kLevelStride * sizeof(float), true, hipFilterModeLinear, hipAddressModeMirror);

            //compute derivatives
            dim3 dBlocks(iDivUp(kLevelWidth, 32), iDivUp(kLevelHeight, 6));
            //dim3 dThreads(32, 6);

            const int kPitchTex = kLevelStride * sizeof(float);

            NcvSize32u srcSize(kLevelWidth, kLevelHeight);
            Ncv32u nSrcStep = kLevelStride * sizeof(float);
            NcvRect32u oROI(0, 0, kLevelWidth, kLevelHeight);

            // Ix0
            ncvAssertReturnNcvStat( nppiStFilterRowBorder_32f_C1R (I0->ptr(), srcSize, nSrcStep, Ix0.ptr(), srcSize, nSrcStep, oROI,
                nppStBorderMirror, derivativeFilter.ptr(), kDFilterSize, kDFilterSize/2, 1.0f/12.0f) );

            // Iy0
            ncvAssertReturnNcvStat( nppiStFilterColumnBorder_32f_C1R (I0->ptr(), srcSize, nSrcStep, Iy0.ptr(), srcSize, nSrcStep, oROI,
                nppStBorderMirror, derivativeFilter.ptr(), kDFilterSize, kDFilterSize/2, 1.0f/12.0f) );

            // Ix
            ncvAssertReturnNcvStat( nppiStFilterRowBorder_32f_C1R (I1->ptr(), srcSize, nSrcStep, Ix.ptr(), srcSize, nSrcStep, oROI,
                nppStBorderMirror, derivativeFilter.ptr(), kDFilterSize, kDFilterSize/2, 1.0f/12.0f) );

            // Iy
            ncvAssertReturnNcvStat( nppiStFilterColumnBorder_32f_C1R (I1->ptr(), srcSize, nSrcStep, Iy.ptr(), srcSize, nSrcStep, oROI,
                nppStBorderMirror, derivativeFilter.ptr(), kDFilterSize, kDFilterSize/2, 1.0f/12.0f) );

            // Ixx
            ncvAssertReturnNcvStat( nppiStFilterRowBorder_32f_C1R (Ix.ptr(), srcSize, nSrcStep, Ixx.ptr(), srcSize, nSrcStep, oROI,
                nppStBorderMirror, derivativeFilter.ptr(), kDFilterSize, kDFilterSize/2, 1.0f/12.0f) );

            // Iyy
            ncvAssertReturnNcvStat( nppiStFilterColumnBorder_32f_C1R (Iy.ptr(), srcSize, nSrcStep, Iyy.ptr(), srcSize, nSrcStep, oROI,
                nppStBorderMirror, derivativeFilter.ptr(), kDFilterSize, kDFilterSize/2, 1.0f/12.0f) );

            // Ixy
            ncvAssertReturnNcvStat( nppiStFilterRowBorder_32f_C1R (Iy.ptr(), srcSize, nSrcStep, Ixy.ptr(), srcSize, nSrcStep, oROI,
                nppStBorderMirror, derivativeFilter.ptr(), kDFilterSize, kDFilterSize/2, 1.0f/12.0f) );

            Texture texIx(kLevelHeight, kLevelWidth, Ix.ptr(), kPitchTex, true, hipFilterModeLinear, hipAddressModeMirror);
            Texture texIxx(kLevelHeight, kLevelWidth, Ixx.ptr(), kPitchTex, true, hipFilterModeLinear, hipAddressModeMirror);
            Texture texIx0(kLevelHeight, kLevelWidth, Ix0.ptr(), kPitchTex, true, hipFilterModeLinear, hipAddressModeMirror);
            Texture texIy(kLevelHeight, kLevelWidth, Iy.ptr(), kPitchTex, true, hipFilterModeLinear, hipAddressModeMirror);
            Texture texIyy(kLevelHeight, kLevelWidth, Iyy.ptr(), kPitchTex, true, hipFilterModeLinear, hipAddressModeMirror);
            Texture texIy0(kLevelHeight, kLevelWidth, Iy0.ptr(), kPitchTex, true, hipFilterModeLinear, hipAddressModeMirror);
            Texture texIxy(kLevelHeight, kLevelWidth, Ixy.ptr(), kPitchTex, true, hipFilterModeLinear, hipAddressModeMirror);
            Texture texDiffX(kLevelSizeInBytes, diffusivity_x.ptr());
            Texture texDiffY(kLevelSizeInBytes, diffusivity_y.ptr());

            //    flow
            Texture texU(kLevelSizeInBytes, ptrU->ptr());
            Texture texV(kLevelSizeInBytes, ptrV->ptr());
            //    flow increments
            Texture texDu(kLevelSizeInBytes, du.ptr());
            Texture texDv(kLevelSizeInBytes, dv.ptr());
            Texture texDuNew(kLevelSizeInBytes, du_new.ptr());
            Texture texDvNew(kLevelSizeInBytes, dv_new.ptr());

            dim3 psor_blocks(iDivUp(kLevelWidth, PSOR_TILE_WIDTH), iDivUp(kLevelHeight, PSOR_TILE_HEIGHT));
            dim3 psor_threads(PSOR_TILE_WIDTH, PSOR_TILE_HEIGHT);

            dim3 sor_blocks(iDivUp(kLevelWidth, SOR_TILE_WIDTH), iDivUp(kLevelHeight, SOR_TILE_HEIGHT));
            dim3 sor_threads(SOR_TILE_WIDTH, SOR_TILE_HEIGHT);

            // inner loop
            // lagged nonlinearity fixed point iteration
            ncvAssertCUDAReturn(hipStreamSynchronize(stream), NCV_CUDA_ERROR);
            for (Ncv32u current_inner_iteration = 0; current_inner_iteration < desc.number_of_inner_iterations; ++current_inner_iteration)
            {
                //compute coefficients
                prepare_sor_stage_1_tex<<<psor_blocks, psor_threads, 0, stream>>> (texU, texV, texDu, texDv, texI0, texI1, texIx, texIxx, texIx0, texIy, texIyy, texIy0, texIxy,
                    diffusivity_x.ptr(), diffusivity_y.ptr(), denom_u.ptr(), denom_v.ptr(), num_dudv.ptr(), num_u.ptr(), num_v.ptr(), kLevelWidth, kLevelHeight, kLevelStride, alpha, gamma);

                ncvAssertCUDALastErrorReturn(NCV_CUDA_ERROR);

                prepare_sor_stage_2<<<psor_blocks, psor_threads, 0, stream>>>(texDiffX, texDiffY, denom_u.ptr(), denom_v.ptr(), kLevelWidth, kLevelHeight, kLevelStride);

                ncvAssertCUDALastErrorReturn(NCV_CUDA_ERROR);


                //solve linear system
                for (Ncv32u solver_iteration = 0; solver_iteration < desc.number_of_solver_iterations; ++solver_iteration)
                {
                    float omega = 1.99f;
                    sor_pass<0><<<sor_blocks, sor_threads, 0, stream>>>(texU, texV, texDu, texDv, texDiffX, texDiffY, du_new.ptr(), dv_new.ptr(), denom_u.ptr(), denom_v.ptr(),
                        num_u.ptr(), num_v.ptr(), num_dudv.ptr(), omega, kLevelWidth, kLevelHeight, kLevelStride);

                    ncvAssertCUDALastErrorReturn(NCV_CUDA_ERROR);


                    sor_pass<1><<<sor_blocks, sor_threads, 0, stream>>>(texU, texV, texDuNew, texDvNew, texDiffX, texDiffY, du.ptr(), dv.ptr(), denom_u.ptr(), denom_v.ptr(), num_u.ptr(),
                        num_v.ptr(),num_dudv.ptr(), omega, kLevelWidth, kLevelHeight, kLevelStride);

                    ncvAssertCUDALastErrorReturn(NCV_CUDA_ERROR);
                }//end of solver loop
            }// end of inner loop

            //update u and v
            add(ptrU->ptr(), du.ptr(), kLevelSizeInPixels, stream);
            ncvAssertCUDALastErrorReturn(NCV_CUDA_ERROR);
            add(ptrV->ptr(), dv.ptr(), kLevelSizeInPixels, stream);
            ncvAssertCUDALastErrorReturn(NCV_CUDA_ERROR);

            //prolongate using texture
            pyr.w.pop_back();
            pyr.h.pop_back();
            if (!pyr.w.empty())
            {
                //compute new image size
                Ncv32u nw = pyr.w.back();
                Ncv32u nh = pyr.h.back();
                Ncv32u ns = alignUp(nw, kStrideAlignmentFloat);

                dim3 p_blocks(iDivUp(nw, 32), iDivUp(nh, 8));
                //dim3 p_threads(32, 8);

                NcvSize32u inner_srcSize (kLevelWidth, kLevelHeight);
                NcvSize32u dstSize (nw, nh);
                NcvRect32u srcROI (0, 0, kLevelWidth, kLevelHeight);
                NcvRect32u dstROI (0, 0, nw, nh);

                ncvAssertReturnNcvStat( nppiStResize_32f_C1R (ptrU->ptr(), inner_srcSize, kLevelStride * sizeof (float), srcROI,
                    ptrUNew->ptr(), dstSize, ns * sizeof (float), dstROI, 1.0f/scale_factor, 1.0f/scale_factor, nppStBicubic) );

                ScaleVector(ptrUNew->ptr(), ptrUNew->ptr(), 1.0f/scale_factor, ns * nh, stream);
                ncvAssertCUDALastErrorReturn(NCV_CUDA_ERROR);

                ncvAssertReturnNcvStat( nppiStResize_32f_C1R (ptrV->ptr(), inner_srcSize, kLevelStride * sizeof (float), srcROI,
                    ptrVNew->ptr(), dstSize, ns * sizeof (float), dstROI, 1.0f/scale_factor, 1.0f/scale_factor, nppStBicubic) );

                ScaleVector(ptrVNew->ptr(), ptrVNew->ptr(), 1.0f/scale_factor, ns * nh, stream);
                ncvAssertCUDALastErrorReturn((int)NCV_CUDA_ERROR);

                cv::cuda::device::swap<FloatVector*>(ptrU, ptrUNew);
                cv::cuda::device::swap<FloatVector*>(ptrV, ptrVNew);
            }
            scale /= scale_factor;
        }

        // end of warping iterations
        ncvAssertCUDAReturn(hipStreamSynchronize(stream), (int)NCV_CUDA_ERROR);

        ncvAssertCUDAReturn( hipMemcpy2DAsync
            (uOut.ptr(), uOut.pitch(), ptrU->ptr(),
            kSourcePitch, kSourceWidth*sizeof(float), kSourceHeight, hipMemcpyDeviceToDevice, stream), (int)NCV_CUDA_ERROR );

        ncvAssertCUDAReturn( hipMemcpy2DAsync
            (vOut.ptr(), vOut.pitch(), ptrV->ptr(),
            kSourcePitch, kSourceWidth*sizeof(float), kSourceHeight, hipMemcpyDeviceToDevice, stream), (int)NCV_CUDA_ERROR );

        ncvAssertCUDAReturn(hipStreamSynchronize(stream), (int)NCV_CUDA_ERROR);
    }

    return NCV_SUCCESS;
}
