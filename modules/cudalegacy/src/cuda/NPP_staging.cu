#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include <vector>
#include <hip/hip_runtime.h>

#include "opencv2/cudev.hpp"

#include "opencv2/cudalegacy/NPP_staging.hpp"
#include <opencv2/cudev/ptr2d/texture.hpp>

//==============================================================================
//
// CUDA streams handling
//
//==============================================================================


static hipStream_t nppStream = 0;


hipStream_t nppStGetActiveCUDAstream(void)
{
    return nppStream;
}


hipStream_t nppStSetActiveCUDAstream(hipStream_t cudaStream)
{
    hipStream_t tmp = nppStream;
    nppStream = cudaStream;
    return tmp;
}


//==============================================================================
//
// IntegralImage.cu
//
//==============================================================================


const Ncv32u NUM_SCAN_THREADS = 256;
const Ncv32u LOG2_NUM_SCAN_THREADS = 8;


template<class T_in, class T_out, bool tbDoSqr>
struct _scanElemOp
{
    static __host__ __device__ T_out scanElemOp(T_in elem);
};

template<class T_in, class T_out>
struct _scanElemOp<T_in, T_out, false>
{
    static inline __host__ __device__ T_out scanElemOp(T_in elem) {
        return (T_out)(elem);
    }
};

template<class T_in, class T_out>
struct _scanElemOp<T_in, T_out, true>
{
    static inline __host__ __device__ T_out scanElemOp(T_in elem) {
        return (T_out)(elem*elem);
    }
};


template<class T>
inline __device__ T readElem(cv::cudev::TexturePtr<Ncv8u> tex8u, T *d_src, Ncv32u texOffs, Ncv32u srcStride, Ncv32u curElemOffs);


template<>
inline __device__ Ncv8u readElem<Ncv8u>(cv::cudev::TexturePtr<Ncv8u> tex8u, Ncv8u* d_src, Ncv32u texOffs, Ncv32u srcStride, Ncv32u curElemOffs)
{
    return tex8u(texOffs + srcStride * blockIdx.x + curElemOffs);
}


template<>
inline __device__ Ncv32u readElem<Ncv32u>(cv::cudev::TexturePtr<Ncv8u> tex8u, Ncv32u *d_src, Ncv32u texOffs, Ncv32u srcStride, Ncv32u curElemOffs)
{
    return d_src[curElemOffs];
}


template<>
inline __device__ Ncv32f readElem<Ncv32f>(cv::cudev::TexturePtr<Ncv8u> tex8u, Ncv32f *d_src, Ncv32u texOffs, Ncv32u srcStride, Ncv32u curElemOffs)
{
    return d_src[curElemOffs];
}


/**
* \brief Segmented scan kernel
*
* Calculates per-row prefix scans of the input image.
* Out-of-bounds safe: reads 'size' elements, writes 'size+1' elements
*
* \tparam T_in      Type of input image elements
* \tparam T_out     Type of output image elements
* \tparam T_op      Defines an operation to be performed on the input image pixels
*
* \param d_src      [IN] Source image pointer
* \param srcWidth   [IN] Source image width
* \param srcStride  [IN] Source image stride
* \param d_II       [OUT] Output image pointer
* \param IIstride   [IN] Output image stride
*
* \return None
*/
template <class T_in, class T_out, bool tbDoSqr>
__global__ void scanRows(cv::cudev::TexturePtr<Ncv8u> tex8u, T_in *d_src, Ncv32u texOffs, Ncv32u srcWidth, Ncv32u srcStride, T_out *d_II, Ncv32u IIstride)
{
    //advance pointers to the current line
    if (sizeof(T_in) != 1)
    {
        d_src += srcStride * blockIdx.x;
    }
    //for initial image 8bit source we use texref tex8u
    d_II += IIstride * blockIdx.x;

    Ncv32u numBuckets = (srcWidth + NUM_SCAN_THREADS - 1) >> LOG2_NUM_SCAN_THREADS;
    Ncv32u offsetX = 0;

    __shared__ T_out shmem[NUM_SCAN_THREADS];
    __shared__ T_out carryElem;
    carryElem = 0;
    __syncthreads();

    while (numBuckets--)
    {
        Ncv32u curElemOffs = offsetX + threadIdx.x;
        T_out curScanElem;

        T_in curElem = 0;
        T_out curElemMod;

        if (curElemOffs < srcWidth)
        {
            //load elements
            curElem = readElem<T_in>(tex8u, d_src, texOffs, srcStride, curElemOffs);
        }
        curElemMod = _scanElemOp<T_in, T_out, tbDoSqr>::scanElemOp(curElem);

        //inclusive scan
        curScanElem = cv::cudev::blockScanInclusive<NUM_SCAN_THREADS>(curElemMod, shmem, threadIdx.x);

        if (curElemOffs <= srcWidth)
        {
            //make scan exclusive and write the bucket to the output buffer
            d_II[curElemOffs] = carryElem + curScanElem - curElemMod;
            offsetX += NUM_SCAN_THREADS;
        }

        //remember last element for subsequent buckets adjustment
        __syncthreads();
        if (threadIdx.x == NUM_SCAN_THREADS-1)
        {
            carryElem += curScanElem;
        }
        __syncthreads();
    }

    if (offsetX == srcWidth && !threadIdx.x)
    {
        d_II[offsetX] = carryElem;
    }
}


template <bool tbDoSqr, class T_in, class T_out>
NCVStatus scanRowsWrapperDevice(T_in *d_src, Ncv32u srcStride,
                                T_out *d_dst, Ncv32u dstStride, NcvSize32u roi)
{
    cv::cudev::Texture<Ncv8u> tex8u(static_cast<size_t>(roi.height * srcStride), (Ncv8u*)d_src);
    scanRows <T_in, T_out, tbDoSqr> <<<roi.height, NUM_SCAN_THREADS, 0, nppStGetActiveCUDAstream()>>> (tex8u, d_src, 0, roi.width, srcStride, d_dst, dstStride);
    ncvAssertCUDALastErrorReturn(NPPST_CUDA_KERNEL_EXECUTION_ERROR);
    return NPPST_SUCCESS;
}


static Ncv32u getPaddedDimension(Ncv32u dim, Ncv32u elemTypeSize, Ncv32u allocatorAlignment)
{
    Ncv32u alignMask = allocatorAlignment-1;
    Ncv32u inverseAlignMask = ~alignMask;
    Ncv32u dimBytes = dim * elemTypeSize;
    Ncv32u pitch = (dimBytes + alignMask) & inverseAlignMask;
    Ncv32u PaddedDim = pitch / elemTypeSize;
    return PaddedDim;
}


template <class T_in, class T_out>
NCVStatus ncvIntegralImage_device(T_in *d_src, Ncv32u srcStep,
                                  T_out *d_dst, Ncv32u dstStep, NcvSize32u roi,
                                  INCVMemAllocator &gpuAllocator)
{
    ncvAssertReturn(sizeof(T_out) == sizeof(Ncv32u), NPPST_MEM_INTERNAL_ERROR);
    ncvAssertReturn(gpuAllocator.memType() == NCVMemoryTypeDevice ||
                      gpuAllocator.memType() == NCVMemoryTypeNone, NPPST_MEM_RESIDENCE_ERROR);
    ncvAssertReturn(gpuAllocator.isInitialized(), NPPST_MEM_INTERNAL_ERROR);
    ncvAssertReturn((d_src != NULL && d_dst != NULL) || gpuAllocator.isCounting(), NPPST_NULL_POINTER_ERROR);
    ncvAssertReturn(roi.width > 0 && roi.height > 0, NPPST_INVALID_ROI);
    ncvAssertReturn(srcStep >= roi.width * sizeof(T_in) &&
                      dstStep >= (roi.width + 1) * sizeof(T_out) &&
                      srcStep % sizeof(T_in) == 0 &&
                      dstStep % sizeof(T_out) == 0, NPPST_INVALID_STEP);
    srcStep /= sizeof(T_in);
    dstStep /= sizeof(T_out);

    Ncv32u WidthII = roi.width + 1;
    Ncv32u HeightII = roi.height + 1;
    Ncv32u PaddedWidthII32 = getPaddedDimension(WidthII, sizeof(Ncv32u), gpuAllocator.alignment());
    Ncv32u PaddedHeightII32 = getPaddedDimension(HeightII, sizeof(Ncv32u), gpuAllocator.alignment());

    NCVMatrixAlloc<T_out> Tmp32_1(gpuAllocator, PaddedWidthII32, PaddedHeightII32);
    ncvAssertReturn(gpuAllocator.isCounting() || Tmp32_1.isMemAllocated(), NPPST_MEM_INTERNAL_ERROR);
    NCVMatrixAlloc<T_out> Tmp32_2(gpuAllocator, PaddedHeightII32, PaddedWidthII32);
    ncvAssertReturn(gpuAllocator.isCounting() || Tmp32_2.isMemAllocated(), NPPST_MEM_INTERNAL_ERROR);
    ncvAssertReturn(Tmp32_1.pitch() * Tmp32_1.height() == Tmp32_2.pitch() * Tmp32_2.height(), NPPST_MEM_INTERNAL_ERROR);

    NCVStatus ncvStat;
    NCV_SET_SKIP_COND(gpuAllocator.isCounting());

    NCV_SKIP_COND_BEGIN

    ncvStat = scanRowsWrapperDevice
        <false>
        (d_src, srcStep, Tmp32_1.ptr(), PaddedWidthII32, roi);
    ncvAssertReturnNcvStat(ncvStat);

    ncvStat = nppiStTranspose_32u_C1R((Ncv32u *)Tmp32_1.ptr(), PaddedWidthII32*sizeof(Ncv32u),
                                      (Ncv32u *)Tmp32_2.ptr(), PaddedHeightII32*sizeof(Ncv32u), NcvSize32u(WidthII, roi.height));
    ncvAssertReturnNcvStat(ncvStat);

    ncvStat = scanRowsWrapperDevice
        <false>
        (Tmp32_2.ptr(), PaddedHeightII32, Tmp32_1.ptr(), PaddedHeightII32, NcvSize32u(roi.height, WidthII));
    ncvAssertReturnNcvStat(ncvStat);

    ncvStat = nppiStTranspose_32u_C1R((Ncv32u *)Tmp32_1.ptr(), PaddedHeightII32*sizeof(Ncv32u),
                                      (Ncv32u *)d_dst, dstStep*sizeof(Ncv32u), NcvSize32u(HeightII, WidthII));
    ncvAssertReturnNcvStat(ncvStat);

    NCV_SKIP_COND_END

    return NPPST_SUCCESS;
}


NCVStatus ncvSquaredIntegralImage_device(Ncv8u *d_src, Ncv32u srcStep,
                                         Ncv64u *d_dst, Ncv32u dstStep, NcvSize32u roi,
                                         INCVMemAllocator &gpuAllocator)
{
    ncvAssertReturn(gpuAllocator.isInitialized(), NPPST_MEM_INTERNAL_ERROR);
    ncvAssertReturn(gpuAllocator.memType() == NCVMemoryTypeDevice ||
                      gpuAllocator.memType() == NCVMemoryTypeNone, NPPST_MEM_RESIDENCE_ERROR);
    ncvAssertReturn((d_src != NULL && d_dst != NULL) || gpuAllocator.isCounting(), NPPST_NULL_POINTER_ERROR);
    ncvAssertReturn(roi.width > 0 && roi.height > 0, NPPST_INVALID_ROI);
    ncvAssertReturn(srcStep >= roi.width &&
                      dstStep >= (roi.width + 1) * sizeof(Ncv64u) &&
                      dstStep % sizeof(Ncv64u) == 0, NPPST_INVALID_STEP);
    dstStep /= sizeof(Ncv64u);

    Ncv32u WidthII = roi.width + 1;
    Ncv32u HeightII = roi.height + 1;
    Ncv32u PaddedWidthII32 = getPaddedDimension(WidthII, sizeof(Ncv32u), gpuAllocator.alignment());
    Ncv32u PaddedHeightII32 = getPaddedDimension(HeightII, sizeof(Ncv32u), gpuAllocator.alignment());
    Ncv32u PaddedWidthII64 = getPaddedDimension(WidthII, sizeof(Ncv64u), gpuAllocator.alignment());
    Ncv32u PaddedHeightII64 = getPaddedDimension(HeightII, sizeof(Ncv64u), gpuAllocator.alignment());
    Ncv32u PaddedWidthMax = PaddedWidthII32 > PaddedWidthII64 ? PaddedWidthII32 : PaddedWidthII64;
    Ncv32u PaddedHeightMax = PaddedHeightII32 > PaddedHeightII64 ? PaddedHeightII32 : PaddedHeightII64;

    NCVMatrixAlloc<Ncv32u> Tmp32_1(gpuAllocator, PaddedWidthII32, PaddedHeightII32);
    ncvAssertReturn(Tmp32_1.isMemAllocated(), NPPST_MEM_INTERNAL_ERROR);
    NCVMatrixAlloc<Ncv64u> Tmp64(gpuAllocator, PaddedWidthMax, PaddedHeightMax);
    ncvAssertReturn(Tmp64.isMemAllocated(), NPPST_MEM_INTERNAL_ERROR);

    NCVMatrixReuse<Ncv32u> Tmp32_2(Tmp64.getSegment(), gpuAllocator.alignment(), PaddedWidthII32, PaddedHeightII32);
    ncvAssertReturn(Tmp32_2.isMemReused(), NPPST_MEM_INTERNAL_ERROR);
    NCVMatrixReuse<Ncv64u> Tmp64_2(Tmp64.getSegment(), gpuAllocator.alignment(), PaddedWidthII64, PaddedHeightII64);
    ncvAssertReturn(Tmp64_2.isMemReused(), NPPST_MEM_INTERNAL_ERROR);

    NCVStatus ncvStat;
    NCV_SET_SKIP_COND(gpuAllocator.isCounting());

    NCV_SKIP_COND_BEGIN

    ncvStat = scanRowsWrapperDevice
        <true, Ncv8u, Ncv32u>
        (d_src, srcStep, Tmp32_2.ptr(), PaddedWidthII32, roi);
    ncvAssertReturnNcvStat(ncvStat);

    ncvStat = nppiStTranspose_32u_C1R(Tmp32_2.ptr(), PaddedWidthII32*sizeof(Ncv32u),
                                      Tmp32_1.ptr(), PaddedHeightII32*sizeof(Ncv32u), NcvSize32u(WidthII, roi.height));
    ncvAssertReturnNcvStat(ncvStat);

    ncvStat = scanRowsWrapperDevice
        <false, Ncv32u, Ncv64u>
        (Tmp32_1.ptr(), PaddedHeightII32, Tmp64_2.ptr(), PaddedHeightII64, NcvSize32u(roi.height, WidthII));
    ncvAssertReturnNcvStat(ncvStat);

    ncvStat = nppiStTranspose_64u_C1R(Tmp64_2.ptr(), PaddedHeightII64*sizeof(Ncv64u),
                                      d_dst, dstStep*sizeof(Ncv64u), NcvSize32u(HeightII, WidthII));
    ncvAssertReturnNcvStat(ncvStat);

    NCV_SKIP_COND_END

    return NPPST_SUCCESS;
}


NCVStatus nppiStIntegralGetSize_8u32u(NcvSize32u roiSize, Ncv32u *pBufsize, hipDeviceProp_t &devProp)
{
    ncvAssertReturn(pBufsize != NULL, NPPST_NULL_POINTER_ERROR);
    ncvAssertReturn(roiSize.width > 0 && roiSize.height > 0, NPPST_INVALID_ROI);

    NCVMemStackAllocator gpuCounter(static_cast<Ncv32u>(devProp.textureAlignment));
    ncvAssertReturn(gpuCounter.isInitialized(), NPPST_MEM_INTERNAL_ERROR);

    NCVStatus ncvStat = ncvIntegralImage_device((Ncv8u*)NULL, roiSize.width,
                                                  (Ncv32u*)NULL, (roiSize.width+1) * sizeof(Ncv32u),
                                                  roiSize, gpuCounter);
    ncvAssertReturnNcvStat(ncvStat);

    *pBufsize = (Ncv32u)gpuCounter.maxSize();
    return NPPST_SUCCESS;
}


NCVStatus nppiStIntegralGetSize_32f32f(NcvSize32u roiSize, Ncv32u *pBufsize, hipDeviceProp_t &devProp)
{
    ncvAssertReturn(pBufsize != NULL, NPPST_NULL_POINTER_ERROR);
    ncvAssertReturn(roiSize.width > 0 && roiSize.height > 0, NPPST_INVALID_ROI);

    NCVMemStackAllocator gpuCounter(static_cast<Ncv32u>(devProp.textureAlignment));
    ncvAssertReturn(gpuCounter.isInitialized(), NPPST_MEM_INTERNAL_ERROR);

    NCVStatus ncvStat = ncvIntegralImage_device((Ncv32f*)NULL, roiSize.width * sizeof(Ncv32f),
                                                  (Ncv32f*)NULL, (roiSize.width+1) * sizeof(Ncv32f),
                                                  roiSize, gpuCounter);
    ncvAssertReturnNcvStat(ncvStat);

    *pBufsize = (Ncv32u)gpuCounter.maxSize();
    return NPPST_SUCCESS;
}


NCVStatus nppiStSqrIntegralGetSize_8u64u(NcvSize32u roiSize, Ncv32u *pBufsize, hipDeviceProp_t &devProp)
{
    ncvAssertReturn(pBufsize != NULL, NPPST_NULL_POINTER_ERROR);
    ncvAssertReturn(roiSize.width > 0 && roiSize.height > 0, NPPST_INVALID_ROI);

    NCVMemStackAllocator gpuCounter(static_cast<Ncv32u>(devProp.textureAlignment));
    ncvAssertReturn(gpuCounter.isInitialized(), NPPST_MEM_INTERNAL_ERROR);

    NCVStatus ncvStat = ncvSquaredIntegralImage_device(NULL, roiSize.width,
                                                         NULL, (roiSize.width+1) * sizeof(Ncv64u),
                                                         roiSize, gpuCounter);
    ncvAssertReturnNcvStat(ncvStat);

    *pBufsize = (Ncv32u)gpuCounter.maxSize();
    return NPPST_SUCCESS;
}


NCVStatus nppiStIntegral_8u32u_C1R(Ncv8u *d_src, Ncv32u srcStep,
                                   Ncv32u *d_dst, Ncv32u dstStep,
                                   NcvSize32u roiSize, Ncv8u *pBuffer,
                                   Ncv32u bufSize, hipDeviceProp_t &devProp)
{
    NCVMemStackAllocator gpuAllocator(NCVMemoryTypeDevice, bufSize, static_cast<Ncv32u>(devProp.textureAlignment), pBuffer);
    ncvAssertReturn(gpuAllocator.isInitialized(), NPPST_MEM_INTERNAL_ERROR);

    NCVStatus ncvStat = ncvIntegralImage_device(d_src, srcStep, d_dst, dstStep, roiSize, gpuAllocator);
    ncvAssertReturnNcvStat(ncvStat);

    return NPPST_SUCCESS;
}


NCVStatus nppiStIntegral_32f32f_C1R(Ncv32f *d_src, Ncv32u srcStep,
                                    Ncv32f *d_dst, Ncv32u dstStep,
                                    NcvSize32u roiSize, Ncv8u *pBuffer,
                                    Ncv32u bufSize, hipDeviceProp_t &devProp)
{
    NCVMemStackAllocator gpuAllocator(NCVMemoryTypeDevice, bufSize, static_cast<Ncv32u>(devProp.textureAlignment), pBuffer);
    ncvAssertReturn(gpuAllocator.isInitialized(), NPPST_MEM_INTERNAL_ERROR);

    NCVStatus ncvStat = ncvIntegralImage_device(d_src, srcStep, d_dst, dstStep, roiSize, gpuAllocator);
    ncvAssertReturnNcvStat(ncvStat);

    return NPPST_SUCCESS;
}


NCVStatus nppiStSqrIntegral_8u64u_C1R(Ncv8u *d_src, Ncv32u srcStep,
                                      Ncv64u *d_dst, Ncv32u dstStep,
                                      NcvSize32u roiSize, Ncv8u *pBuffer,
                                      Ncv32u bufSize, hipDeviceProp_t &devProp)
{
    NCVMemStackAllocator gpuAllocator(NCVMemoryTypeDevice, bufSize, static_cast<Ncv32u>(devProp.textureAlignment), pBuffer);
    ncvAssertReturn(gpuAllocator.isInitialized(), NPPST_MEM_INTERNAL_ERROR);

    NCVStatus ncvStat = ncvSquaredIntegralImage_device(d_src, srcStep, d_dst, dstStep, roiSize, gpuAllocator);
    ncvAssertReturnNcvStat(ncvStat);

    return NPPST_SUCCESS;
}


NCVStatus nppiStIntegral_8u32u_C1R_host(Ncv8u *h_src, Ncv32u srcStep,
                                        Ncv32u *h_dst, Ncv32u dstStep,
                                        NcvSize32u roiSize)
{
    ncvAssertReturn(h_src != NULL && h_dst != NULL, NPPST_NULL_POINTER_ERROR);
    ncvAssertReturn(roiSize.width > 0 && roiSize.height > 0, NPPST_INVALID_ROI);
    ncvAssertReturn(srcStep >= roiSize.width &&
                      dstStep >= (roiSize.width + 1) * sizeof(Ncv32u) &&
                      dstStep % sizeof(Ncv32u) == 0, NPPST_INVALID_STEP);
    dstStep /= sizeof(Ncv32u);

    Ncv32u WidthII = roiSize.width + 1;
    Ncv32u HeightII = roiSize.height + 1;

    memset(h_dst, 0, WidthII * sizeof(Ncv32u));
    for (Ncv32u i=1; i<HeightII; i++)
    {
        h_dst[i * dstStep] = 0;
        for (Ncv32u j=1; j<WidthII; j++)
        {
            Ncv32u top = h_dst[(i-1) * dstStep + j];
            Ncv32u left = h_dst[i * dstStep + (j - 1)];
            Ncv32u topleft = h_dst[(i - 1) * dstStep + (j - 1)];
            Ncv32u elem = h_src[(i - 1) * srcStep + (j - 1)];
            h_dst[i * dstStep + j] = elem + left - topleft + top;
        }
    }

    return NPPST_SUCCESS;
}


NCVStatus nppiStIntegral_32f32f_C1R_host(Ncv32f *h_src, Ncv32u srcStep,
                                         Ncv32f *h_dst, Ncv32u dstStep,
                                         NcvSize32u roiSize)
{
    ncvAssertReturn(h_src != NULL && h_dst != NULL, NPPST_NULL_POINTER_ERROR);
    ncvAssertReturn(roiSize.width > 0 && roiSize.height > 0, NPPST_INVALID_ROI);
    ncvAssertReturn(srcStep >= roiSize.width * sizeof(Ncv32f) &&
                      dstStep >= (roiSize.width + 1) * sizeof(Ncv32f) &&
                      srcStep % sizeof(Ncv32f) == 0 &&
                      dstStep % sizeof(Ncv32f) == 0, NPPST_INVALID_STEP);
    srcStep /= sizeof(Ncv32f);
    dstStep /= sizeof(Ncv32f);

    Ncv32u WidthII = roiSize.width + 1;
    Ncv32u HeightII = roiSize.height + 1;

    memset(h_dst, 0, WidthII * sizeof(Ncv32u));
    for (Ncv32u i=1; i<HeightII; i++)
    {
        h_dst[i * dstStep] = 0.0f;
        for (Ncv32u j=1; j<WidthII; j++)
        {
            Ncv32f top = h_dst[(i-1) * dstStep + j];
            Ncv32f left = h_dst[i * dstStep + (j - 1)];
            Ncv32f topleft = h_dst[(i - 1) * dstStep + (j - 1)];
            Ncv32f elem = h_src[(i - 1) * srcStep + (j - 1)];
            h_dst[i * dstStep + j] = elem + left - topleft + top;
        }
    }

    return NPPST_SUCCESS;
}


NCVStatus nppiStSqrIntegral_8u64u_C1R_host(Ncv8u *h_src, Ncv32u srcStep,
                                           Ncv64u *h_dst, Ncv32u dstStep,
                                           NcvSize32u roiSize)
{
    ncvAssertReturn(h_src != NULL && h_dst != NULL, NPPST_NULL_POINTER_ERROR);
    ncvAssertReturn(roiSize.width > 0 && roiSize.height > 0, NPPST_INVALID_ROI);
    ncvAssertReturn(srcStep >= roiSize.width &&
                      dstStep >= (roiSize.width + 1) * sizeof(Ncv64u) &&
                      dstStep % sizeof(Ncv64u) == 0, NPPST_INVALID_STEP);
    dstStep /= sizeof(Ncv64u);

    Ncv32u WidthII = roiSize.width + 1;
    Ncv32u HeightII = roiSize.height + 1;

    memset(h_dst, 0, WidthII * sizeof(Ncv64u));
    for (Ncv32u i=1; i<HeightII; i++)
    {
        h_dst[i * dstStep] = 0;
        for (Ncv32u j=1; j<WidthII; j++)
        {
            Ncv64u top = h_dst[(i-1) * dstStep + j];
            Ncv64u left = h_dst[i * dstStep + (j - 1)];
            Ncv64u topleft = h_dst[(i - 1) * dstStep + (j - 1)];
            Ncv64u elem = h_src[(i - 1) * srcStep + (j - 1)];
            h_dst[i * dstStep + j] = elem*elem + left - topleft + top;
        }
    }

    return NPPST_SUCCESS;
}


//==============================================================================
//
// Decimate.cu
//
//==============================================================================


const Ncv32u NUM_DOWNSAMPLE_NEAREST_THREADS_X = 32;
const Ncv32u NUM_DOWNSAMPLE_NEAREST_THREADS_Y = 8;

template <class T>
__global__ void decimate_C1R(T* d_src, Ncv32u srcStep, T* d_dst, Ncv32u dstStep, NcvSize32u dstRoi, Ncv32u scale)
{
    int curX = blockIdx.x * blockDim.x + threadIdx.x;
    int curY = blockIdx.y * blockDim.y + threadIdx.y;
    if (curX >= dstRoi.width || curY >= dstRoi.height) return;
    d_dst[curY * dstStep + curX] = d_src[(curY * srcStep + curX) * scale];
}

template <class T>
__global__ void decimate_C1R(cv::cudev::TexturePtr<T> texSrc, Ncv32u srcStep, T* d_dst, Ncv32u dstStep,
    NcvSize32u dstRoi, Ncv32u scale)
{
    int curX = blockIdx.x * blockDim.x + threadIdx.x;
    int curY = blockIdx.y * blockDim.y + threadIdx.y;
    if (curX >= dstRoi.width || curY >= dstRoi.height) return;
    d_dst[curY * dstStep + curX] = texSrc((curY * srcStep + curX) * scale);
}

template <class T>
static NCVStatus decimateWrapperDevice(T *d_src, Ncv32u srcStep,
                                                T *d_dst, Ncv32u dstStep,
                                                NcvSize32u srcRoi, Ncv32u scale,
                                                NcvBool readThruTexture)
{
    ncvAssertReturn(d_src != NULL && d_dst != NULL, NPPST_NULL_POINTER_ERROR);
    ncvAssertReturn(srcRoi.width > 0 && srcRoi.height > 0, NPPST_INVALID_ROI);
    ncvAssertReturn(scale != 0, NPPST_INVALID_SCALE);
    ncvAssertReturn(srcStep >= (Ncv32u)(srcRoi.width) * sizeof(T) &&
                      dstStep >= (Ncv32u)(srcRoi.width * sizeof(T) / scale), NPPST_INVALID_STEP);
    srcStep /= sizeof(T);
    dstStep /= sizeof(T);

    NcvSize32u dstRoi;
    dstRoi.width = srcRoi.width / scale;
    dstRoi.height = srcRoi.height / scale;

    dim3 grid((dstRoi.width + NUM_DOWNSAMPLE_NEAREST_THREADS_X - 1) / NUM_DOWNSAMPLE_NEAREST_THREADS_X,
              (dstRoi.height + NUM_DOWNSAMPLE_NEAREST_THREADS_Y - 1) / NUM_DOWNSAMPLE_NEAREST_THREADS_Y);
    dim3 block(NUM_DOWNSAMPLE_NEAREST_THREADS_X, NUM_DOWNSAMPLE_NEAREST_THREADS_Y);
    if (!readThruTexture) {
        decimate_C1R<T><<<grid, block, 0, nppStGetActiveCUDAstream()>>>(d_src, srcStep, d_dst, dstStep, dstRoi, scale);
    }
    else {
        cv::cudev::Texture<T> texSrc(srcRoi.height * srcStep * sizeof(T), d_src);
        decimate_C1R<T><<<grid, block, 0, nppStGetActiveCUDAstream()>>>(texSrc, srcStep, d_dst, dstStep, dstRoi, scale);
    }

    ncvAssertCUDALastErrorReturn(NPPST_CUDA_KERNEL_EXECUTION_ERROR);

    return NPPST_SUCCESS;
}


template <class T>
static NCVStatus decimateWrapperHost(T *h_src, Ncv32u srcStep,
                                              T *h_dst, Ncv32u dstStep,
                                              NcvSize32u srcRoi, Ncv32u scale)
{
    ncvAssertReturn(h_src != NULL && h_dst != NULL, NPPST_NULL_POINTER_ERROR);
    ncvAssertReturn(srcRoi.width != 0 && srcRoi.height != 0, NPPST_INVALID_ROI);
    ncvAssertReturn(scale != 0, NPPST_INVALID_SCALE);
    ncvAssertReturn(srcStep >= (Ncv32u)(srcRoi.width) * sizeof(T) &&
                      dstStep >= (Ncv32u)(srcRoi.width * sizeof(T) / scale) &&
                      srcStep % sizeof(T) == 0 && dstStep % sizeof(T) == 0, NPPST_INVALID_STEP);
    srcStep /= sizeof(T);
    dstStep /= sizeof(T);

    NcvSize32u dstRoi;
    dstRoi.width = srcRoi.width / scale;
    dstRoi.height = srcRoi.height / scale;

    for (Ncv32u i=0; i<dstRoi.height; i++)
    {
        for (Ncv32u j=0; j<dstRoi.width; j++)
        {
            h_dst[i*dstStep+j] = h_src[i*scale*srcStep + j*scale];
        }
    }

    return NPPST_SUCCESS;
}


#define implementNppDecimate(bit, typ) \
    NCVStatus nppiStDecimate_##bit##typ##_C1R(Ncv##bit##typ *d_src, Ncv32u srcStep, \
                                                     Ncv##bit##typ *d_dst, Ncv32u dstStep, \
                                                     NcvSize32u srcRoi, Ncv32u scale, NcvBool readThruTexture) \
    { \
        return decimateWrapperDevice<Ncv##bit##u>((Ncv##bit##u *)d_src, srcStep, \
                                                           (Ncv##bit##u *)d_dst, dstStep, \
                                                           srcRoi, scale, readThruTexture); \
    }


#define implementNppDecimateHost(bit, typ) \
    NCVStatus nppiStDecimate_##bit##typ##_C1R_host(Ncv##bit##typ *h_src, Ncv32u srcStep, \
                                                          Ncv##bit##typ *h_dst, Ncv32u dstStep, \
                                                          NcvSize32u srcRoi, Ncv32u scale) \
    { \
        return decimateWrapperHost<Ncv##bit##u>((Ncv##bit##u *)h_src, srcStep, \
                                                         (Ncv##bit##u *)h_dst, dstStep, \
                                                         srcRoi, scale); \
    }


implementNppDecimate(32, u)
implementNppDecimate(64, u)
implementNppDecimateHost(32, u)
implementNppDecimateHost(32, s)
implementNppDecimateHost(32, f)
implementNppDecimateHost(64, u)
implementNppDecimateHost(64, s)
implementNppDecimateHost(64, f)


//==============================================================================
//
// RectStdDev.cu
//
//==============================================================================


const Ncv32u NUM_RECTSTDDEV_THREADS = 128;


template <NcvBool tbCacheTexture, class Ptr2D>
__device__ Ncv32u getElemSum(Ptr2D tex, Ncv32u x, Ncv32u *d_sum)
{
    if (tbCacheTexture)
        return tex(x);
    else
        return d_sum[x];
}


template <NcvBool tbCacheTexture, class Ptr2D>
__device__ Ncv64u getElemSqSum(Ptr2D tex, Ncv32u x, Ncv64u *d_sqsum)
{
    if (tbCacheTexture)
        return tex(x);
    else
        return d_sqsum[x];
}


template <NcvBool tbCacheTexture>
__global__ void rectStdDev_32f_C1R(cv::cudev::TexturePtr<Ncv32u> texSum, cv::cudev::TexturePtr<Ncv64u> texSumSq, Ncv32u *d_sum, Ncv32u sumStep, Ncv64u *d_sqsum, Ncv32u sqsumStep,
    Ncv32f *d_norm, Ncv32u normStep, NcvSize32u roi, NcvRect32u rect, Ncv32f invRectArea)
{
    Ncv32u x_offs = blockIdx.x * NUM_RECTSTDDEV_THREADS + threadIdx.x;
    if (x_offs >= roi.width)
    {
        return;
    }

    Ncv32u sum_offset = blockIdx.y * sumStep + x_offs;
    Ncv32u sqsum_offset = blockIdx.y * sqsumStep + x_offs;

    //OPT: try swapping order (could change cache hit/miss ratio)
    Ncv32u sum_tl = getElemSum<tbCacheTexture>(texSum, sum_offset + rect.y * sumStep + rect.x, d_sum);
    Ncv32u sum_bl = getElemSum<tbCacheTexture>(texSum, sum_offset + (rect.y + rect.height) * sumStep + rect.x, d_sum);
    Ncv32u sum_tr = getElemSum<tbCacheTexture>(texSum, sum_offset + rect.y * sumStep + rect.x + rect.width, d_sum);
    Ncv32u sum_br = getElemSum<tbCacheTexture>(texSum, sum_offset + (rect.y + rect.height) * sumStep + rect.x + rect.width, d_sum);
    Ncv32u sum_val = sum_br + sum_tl - sum_tr - sum_bl;

    Ncv64u sqsum_tl, sqsum_bl, sqsum_tr, sqsum_br;
    sqsum_tl = getElemSqSum<tbCacheTexture>(texSumSq, sqsum_offset + rect.y * sqsumStep + rect.x, d_sqsum);
    sqsum_bl = getElemSqSum<tbCacheTexture>(texSumSq, sqsum_offset + (rect.y + rect.height) * sqsumStep + rect.x, d_sqsum);
    sqsum_tr = getElemSqSum<tbCacheTexture>(texSumSq, sqsum_offset + rect.y * sqsumStep + rect.x + rect.width, d_sqsum);
    sqsum_br = getElemSqSum<tbCacheTexture>(texSumSq, sqsum_offset + (rect.y + rect.height) * sqsumStep + rect.x + rect.width, d_sqsum);
    Ncv64u sqsum_val = sqsum_br + sqsum_tl - sqsum_tr - sqsum_bl;

    Ncv32f mean = sum_val * invRectArea;

    //////////////////////////////////////////////////////////////////////////
    // sqsum_val_res = sqsum_val / rectArea
    //////////////////////////////////////////////////////////////////////////

    Ncv32f sqsum_val_1 = __ull2float_rz(sqsum_val);
    Ncv64u sqsum_val_2 = __float2ull_rz(sqsum_val_1);
    Ncv64u sqsum_val_3 = sqsum_val - sqsum_val_2;
    Ncv32f sqsum_val_4 = __ull2float_rn(sqsum_val_3);
    sqsum_val_1 *= invRectArea;
    sqsum_val_4 *= invRectArea;
    Ncv32f sqsum_val_res = sqsum_val_1 + sqsum_val_4;

    //////////////////////////////////////////////////////////////////////////
    // variance = sqsum_val_res - mean * mean
    //////////////////////////////////////////////////////////////////////////

#if defined DISABLE_MAD_SELECTIVELY
    Ncv32f variance = sqsum_val_2 - __fmul_rn(mean, mean);
#else
    Ncv32f variance = sqsum_val_res - mean * mean;
#endif

    //////////////////////////////////////////////////////////////////////////
    // stddev = sqrtf(variance)
    //////////////////////////////////////////////////////////////////////////

    //Ncv32f stddev = sqrtf(variance);
    Ncv32f stddev = __fsqrt_rn(variance);

    d_norm[blockIdx.y * normStep + x_offs] = stddev;
}


NCVStatus nppiStRectStdDev_32f_C1R(Ncv32u *d_sum, Ncv32u sumStep,
                                   Ncv64u *d_sqsum, Ncv32u sqsumStep,
                                   Ncv32f *d_norm, Ncv32u normStep,
                                   NcvSize32u roi, NcvRect32u rect,
                                   Ncv32f scaleArea, NcvBool readThruTexture)
{
    ncvAssertReturn(d_sum != NULL && d_sqsum != NULL && d_norm != NULL, NPPST_NULL_POINTER_ERROR);
    ncvAssertReturn(roi.width > 0 && roi.height > 0, NPPST_INVALID_ROI);
    ncvAssertReturn(sumStep >= (Ncv32u)(roi.width + rect.x + rect.width - 1) * sizeof(Ncv32u) &&
                      sqsumStep >= (Ncv32u)(roi.width + rect.x + rect.width - 1) * sizeof(Ncv64u) &&
                      normStep >= (Ncv32u)roi.width * sizeof(Ncv32f) &&
                      sumStep % sizeof(Ncv32u) == 0 &&
                      sqsumStep % sizeof(Ncv64u) == 0 &&
                      normStep % sizeof(Ncv32f) == 0, NPPST_INVALID_STEP);
    ncvAssertReturn(scaleArea >= 1.0f, NPPST_INVALID_SCALE);
    sumStep /= sizeof(Ncv32u);
    sqsumStep /= sizeof(Ncv64u);
    normStep /= sizeof(Ncv32f);

    Ncv32f rectArea = rect.width * rect.height * scaleArea;
    Ncv32f invRectArea = 1.0f / rectArea;

    dim3 grid(((roi.width + NUM_RECTSTDDEV_THREADS - 1) / NUM_RECTSTDDEV_THREADS), roi.height);
    dim3 block(NUM_RECTSTDDEV_THREADS);

    cv::cudev::Texture<Ncv32u> texSum((roi.height + rect.y + rect.height) * sumStep * sizeof(Ncv32u), d_sum);
    cv::cudev::Texture<Ncv64u> texSumSq((roi.height + rect.y + rect.height) * sqsumStep * sizeof(Ncv64u), d_sqsum);
    if (!readThruTexture)
        rectStdDev_32f_C1R<false><<<grid, block, 0, nppStGetActiveCUDAstream()>>>(texSum, texSumSq, d_sum, sumStep, d_sqsum, sqsumStep, d_norm, normStep, roi, rect, invRectArea);
    else
        rectStdDev_32f_C1R<true><<<grid, block, 0, nppStGetActiveCUDAstream()>>>(texSum, texSumSq, NULL, sumStep, NULL, sqsumStep, d_norm, normStep, roi, rect, invRectArea);

    ncvAssertCUDALastErrorReturn(NPPST_CUDA_KERNEL_EXECUTION_ERROR);

    return NPPST_SUCCESS;
}


NCVStatus nppiStRectStdDev_32f_C1R_host(Ncv32u *h_sum, Ncv32u sumStep,
                                        Ncv64u *h_sqsum, Ncv32u sqsumStep,
                                        Ncv32f *h_norm, Ncv32u normStep,
                                        NcvSize32u roi, NcvRect32u rect,
                                        Ncv32f scaleArea)
{
    ncvAssertReturn(h_sum != NULL && h_sqsum != NULL && h_norm != NULL, NPPST_NULL_POINTER_ERROR);
    ncvAssertReturn(roi.width > 0 && roi.height > 0, NPPST_INVALID_ROI);
    ncvAssertReturn(sumStep >= (Ncv32u)(roi.width + rect.x + rect.width - 1) * sizeof(Ncv32u) &&
                      sqsumStep >= (Ncv32u)(roi.width + rect.x + rect.width - 1) * sizeof(Ncv64u) &&
                      normStep >= (Ncv32u)roi.width * sizeof(Ncv32f) &&
                      sumStep % sizeof(Ncv32u) == 0 &&
                      sqsumStep % sizeof(Ncv64u) == 0 &&
                      normStep % sizeof(Ncv32f) == 0, NPPST_INVALID_STEP);
    ncvAssertReturn(scaleArea >= 1.0f, NPPST_INVALID_SCALE);
    sumStep /= sizeof(Ncv32u);
    sqsumStep /= sizeof(Ncv64u);
    normStep /= sizeof(Ncv32f);

    Ncv32f rectArea = rect.width * rect.height * scaleArea;
    Ncv32f invRectArea = 1.0f / rectArea;

    for (Ncv32u i=0; i<roi.height; i++)
    {
        for (Ncv32u j=0; j<roi.width; j++)
        {
            Ncv32u sum_offset = i * sumStep + j;
            Ncv32u sqsum_offset = i * sqsumStep + j;

            Ncv32u sum_tl = h_sum[sum_offset + rect.y * sumStep + rect.x];
            Ncv32u sum_bl = h_sum[sum_offset + (rect.y + rect.height) * sumStep + rect.x];
            Ncv32u sum_tr = h_sum[sum_offset + rect.y * sumStep + rect.x + rect.width];
            Ncv32u sum_br = h_sum[sum_offset + (rect.y + rect.height) * sumStep + rect.x + rect.width];
            Ncv64f sum_val = sum_br + sum_tl - sum_tr - sum_bl;

            Ncv64u sqsum_tl = h_sqsum[sqsum_offset + rect.y * sqsumStep + rect.x];
            Ncv64u sqsum_bl = h_sqsum[sqsum_offset + (rect.y + rect.height) * sqsumStep + rect.x];
            Ncv64u sqsum_tr = h_sqsum[sqsum_offset + rect.y * sqsumStep + rect.x + rect.width];
            Ncv64u sqsum_br = h_sqsum[sqsum_offset + (rect.y + rect.height) * sqsumStep + rect.x + rect.width];
            Ncv64f sqsum_val = (Ncv64f)(sqsum_br + sqsum_tl - sqsum_tr - sqsum_bl);

            Ncv64f mean = sum_val * invRectArea;
            Ncv64f sqsum_val_2 = sqsum_val / rectArea;
            Ncv64f variance = sqsum_val_2 - mean * mean;

            h_norm[i * normStep + j] = (Ncv32f)sqrt(variance);
        }
    }

    return NPPST_SUCCESS;
}


//==============================================================================
//
// Transpose.cu
//
//==============================================================================


const Ncv32u TRANSPOSE_TILE_DIM   = 16;
const Ncv32u TRANSPOSE_BLOCK_ROWS = 16;


/**
* \brief Matrix transpose kernel
*
* Calculates transpose of the input image
* \see TRANSPOSE_TILE_DIM
*
* \tparam T_in      Type of input image elements
* \tparam T_out     Type of output image elements
*
* \param d_src      [IN] Source image pointer
* \param srcStride  [IN] Source image stride
* \param d_dst      [OUT] Output image pointer
* \param dstStride  [IN] Output image stride
*
* \return None
*/
template <class T>
__global__ void transpose(T *d_src, Ncv32u srcStride,
                          T *d_dst, Ncv32u dstStride, NcvSize32u srcRoi)
{
    __shared__ T tile[TRANSPOSE_TILE_DIM][TRANSPOSE_TILE_DIM+1];

    Ncv32u blockIdx_x, blockIdx_y;

    // do diagonal reordering
    if (gridDim.x == gridDim.y)
    {
        blockIdx_y = blockIdx.x;
        blockIdx_x = (blockIdx.x + blockIdx.y) % gridDim.x;
    }
    else
    {
        Ncv32u bid = blockIdx.x + gridDim.x * blockIdx.y;
        blockIdx_y = bid % gridDim.y;
        blockIdx_x = ((bid / gridDim.y) + blockIdx_y) % gridDim.x;
    }

    Ncv32u xIndex = blockIdx_x * TRANSPOSE_TILE_DIM + threadIdx.x;
    Ncv32u yIndex = blockIdx_y * TRANSPOSE_TILE_DIM + threadIdx.y;
    Ncv32u index_gmem = xIndex + yIndex * srcStride;

    if (xIndex < srcRoi.width)
    {
        for (Ncv32u i=0; i<TRANSPOSE_TILE_DIM; i+=TRANSPOSE_BLOCK_ROWS)
        {
            if (yIndex + i < srcRoi.height)
            {
                tile[threadIdx.y+i][threadIdx.x] = d_src[index_gmem+i*srcStride];
            }
        }
    }

    __syncthreads();

    xIndex = blockIdx_y * TRANSPOSE_TILE_DIM + threadIdx.x;
    yIndex = blockIdx_x * TRANSPOSE_TILE_DIM + threadIdx.y;
    index_gmem = xIndex + yIndex * dstStride;

    if (xIndex < srcRoi.height)
    {
        for (Ncv32u i=0; i<TRANSPOSE_TILE_DIM; i+=TRANSPOSE_BLOCK_ROWS)
        {
            if (yIndex + i < srcRoi.width)
            {
                d_dst[index_gmem+i*dstStride] = tile[threadIdx.x][threadIdx.y+i];
            }
        }
    }
}


template <class T>
NCVStatus transposeWrapperDevice(T *d_src, Ncv32u srcStride,
                                   T *d_dst, Ncv32u dstStride, NcvSize32u srcRoi)
{
    ncvAssertReturn(d_src != NULL && d_dst != NULL, NPPST_NULL_POINTER_ERROR);
    ncvAssertReturn(srcRoi.width > 0 && srcRoi.height > 0, NPPST_INVALID_ROI);
    ncvAssertReturn(srcStride >= srcRoi.width * sizeof(T) &&
                      dstStride >= srcRoi.height * sizeof(T) &&
                      srcStride % sizeof(T) == 0 && dstStride % sizeof(T) == 0, NPPST_INVALID_STEP);
    srcStride /= sizeof(T);
    dstStride /= sizeof(T);

    dim3 grid((srcRoi.width + TRANSPOSE_TILE_DIM - 1) / TRANSPOSE_TILE_DIM,
              (srcRoi.height + TRANSPOSE_TILE_DIM - 1) / TRANSPOSE_TILE_DIM);
    dim3 block(TRANSPOSE_TILE_DIM, TRANSPOSE_TILE_DIM);
    transpose
        <T>
        <<<grid, block, 0, nppStGetActiveCUDAstream()>>>
        (d_src, srcStride, d_dst, dstStride, srcRoi);
    ncvAssertCUDALastErrorReturn(NPPST_CUDA_KERNEL_EXECUTION_ERROR);

    return NPPST_SUCCESS;
}


template <class T>
static NCVStatus transposeWrapperHost(T *h_src, Ncv32u srcStride,
                                        T *h_dst, Ncv32u dstStride, NcvSize32u srcRoi)
{
    ncvAssertReturn(h_src != NULL && h_dst != NULL, NPPST_NULL_POINTER_ERROR);
    ncvAssertReturn(srcRoi.width > 0 && srcRoi.height > 0, NPPST_INVALID_ROI);
    ncvAssertReturn(srcStride >= srcRoi.width * sizeof(T) &&
                      dstStride >= srcRoi.height * sizeof(T) &&
                      srcStride % sizeof(T) == 0 && dstStride % sizeof(T) == 0, NPPST_INVALID_STEP);
    srcStride /= sizeof(T);
    dstStride /= sizeof(T);

    for (Ncv32u i=0; i<srcRoi.height; i++)
    {
        for (Ncv32u j=0; j<srcRoi.width; j++)
        {
            h_dst[j*dstStride+i] = h_src[i*srcStride + j];
        }
    }

    return NPPST_SUCCESS;
}


#define implementNppTranspose(bit, typ) \
    NCVStatus nppiStTranspose_##bit##typ##_C1R(Ncv##bit##typ *d_src, Ncv32u srcStep, \
                                             Ncv##bit##typ *d_dst, Ncv32u dstStep, NcvSize32u srcRoi) \
    { \
        return transposeWrapperDevice<Ncv##bit##u>((Ncv##bit##u *)d_src, srcStep, \
                                                   (Ncv##bit##u *)d_dst, dstStep, srcRoi); \
    }


#define implementNppTransposeHost(bit, typ) \
    NCVStatus nppiStTranspose_##bit##typ##_C1R_host(Ncv##bit##typ *h_src, Ncv32u srcStep, \
                                                  Ncv##bit##typ *h_dst, Ncv32u dstStep, \
                                                  NcvSize32u srcRoi) \
    { \
        return transposeWrapperHost<Ncv##bit##u>((Ncv##bit##u *)h_src, srcStep, \
                                                 (Ncv##bit##u *)h_dst, dstStep, srcRoi); \
    }


implementNppTranspose(32,u)
implementNppTranspose(32,s)
implementNppTranspose(32,f)
implementNppTranspose(64,u)
implementNppTranspose(64,s)
implementNppTranspose(64,f)

implementNppTransposeHost(32,u)
implementNppTransposeHost(32,s)
implementNppTransposeHost(32,f)
implementNppTransposeHost(64,u)
implementNppTransposeHost(64,s)
implementNppTransposeHost(64,f)


NCVStatus nppiStTranspose_128_C1R(void *d_src, Ncv32u srcStep,
                                  void *d_dst, Ncv32u dstStep, NcvSize32u srcRoi)
{
    return transposeWrapperDevice<uint4>((uint4 *)d_src, srcStep, (uint4 *)d_dst, dstStep, srcRoi);
}


NCVStatus nppiStTranspose_128_C1R_host(void *d_src, Ncv32u srcStep,
                                       void *d_dst, Ncv32u dstStep, NcvSize32u srcRoi)
{
    return transposeWrapperHost<uint4>((uint4 *)d_src, srcStep, (uint4 *)d_dst, dstStep, srcRoi);
}


//==============================================================================
//
// Compact.cu
//
//==============================================================================


const Ncv32u NUM_REMOVE_THREADS = 256;


template <bool bRemove, bool bWritePartial>
__global__ void removePass1Scan(Ncv32u *d_src, Ncv32u srcLen,
                                Ncv32u *d_offsets, Ncv32u *d_blockSums,
                                Ncv32u elemRemove)
{
    Ncv32u blockId = blockIdx.y * 65535 + blockIdx.x;
    Ncv32u elemAddrIn = blockId * NUM_REMOVE_THREADS + threadIdx.x;

    if (elemAddrIn > srcLen + blockDim.x)
    {
        return;
    }

    __shared__ Ncv32u shmem[NUM_REMOVE_THREADS];

    Ncv32u scanElem = 0;
    if (elemAddrIn < srcLen)
    {
        if (bRemove)
        {
            scanElem = (d_src[elemAddrIn] != elemRemove) ? 1 : 0;
        }
        else
        {
            scanElem = d_src[elemAddrIn];
        }
    }

    Ncv32u localScanInc = cv::cudev::blockScanInclusive<NUM_REMOVE_THREADS>(scanElem, shmem, threadIdx.x);
    __syncthreads();

    if (elemAddrIn < srcLen)
    {
        if (threadIdx.x == NUM_REMOVE_THREADS-1 && bWritePartial)
        {
            d_blockSums[blockId] = localScanInc;
        }

        if (bRemove)
        {
            d_offsets[elemAddrIn] = localScanInc - scanElem;
        }
        else
        {
            d_src[elemAddrIn] = localScanInc - scanElem;
        }
    }
}


__global__ void removePass2Adjust(Ncv32u *d_offsets, Ncv32u srcLen, Ncv32u *d_blockSums)
{
    Ncv32u blockId = blockIdx.y * 65535 + blockIdx.x;
    Ncv32u elemAddrIn = blockId * NUM_REMOVE_THREADS + threadIdx.x;
    if (elemAddrIn >= srcLen)
    {
        return;
    }

    __shared__ Ncv32u valOffs;
    valOffs = d_blockSums[blockId];
    __syncthreads();

    d_offsets[elemAddrIn] += valOffs;
}


__global__ void removePass3Compact(Ncv32u *d_src, Ncv32u srcLen,
                                   Ncv32u *d_offsets, Ncv32u *d_dst,
                                   Ncv32u elemRemove, Ncv32u *dstLenValue)
{
    Ncv32u blockId = blockIdx.y * 65535 + blockIdx.x;
    Ncv32u elemAddrIn = blockId * NUM_REMOVE_THREADS + threadIdx.x;
    if (elemAddrIn >= srcLen)
    {
        return;
    }

    Ncv32u elem = d_src[elemAddrIn];
    Ncv32u elemAddrOut = d_offsets[elemAddrIn];
    if (elem != elemRemove)
    {
        d_dst[elemAddrOut] = elem;
    }

    if (elemAddrIn == srcLen-1)
    {
        if (elem != elemRemove)
        {
            *dstLenValue = elemAddrOut + 1;
        }
        else
        {
            *dstLenValue = elemAddrOut;
        }
    }
}


NCVStatus compactVector_32u_device(Ncv32u *d_src, Ncv32u srcLen,
                                   Ncv32u *d_dst, Ncv32u *dstLenPinned,
                                   Ncv32u elemRemove,
                                   INCVMemAllocator &gpuAllocator)
{
    ncvAssertReturn(gpuAllocator.isInitialized(), NPPST_MEM_INTERNAL_ERROR);
    ncvAssertReturn((d_src != NULL && d_dst != NULL) || gpuAllocator.isCounting(), NPPST_NULL_POINTER_ERROR);

    if (srcLen == 0)
    {
        if (dstLenPinned != NULL)
        {
            *dstLenPinned = 0;
        }
        return NPPST_SUCCESS;
    }

    std::vector<Ncv32u> partSumNums;
    std::vector<Ncv32u> partSumOffsets;
    Ncv32u partSumLastNum = srcLen;
    Ncv32u partSumLastOffs = 0;
    do
    {
        partSumNums.push_back(partSumLastNum);
        partSumOffsets.push_back(partSumLastOffs);

        Ncv32u curPartSumAlignedLength = alignUp(partSumLastNum * sizeof(Ncv32u),
                                                 gpuAllocator.alignment()) / sizeof(Ncv32u);
        partSumLastOffs += curPartSumAlignedLength;

        partSumLastNum = (partSumLastNum + NUM_REMOVE_THREADS - 1) / NUM_REMOVE_THREADS;
    }
    while (partSumLastNum>1);
    partSumNums.push_back(partSumLastNum);
    partSumOffsets.push_back(partSumLastOffs);

    NCVVectorAlloc<Ncv32u> d_hierSums(gpuAllocator, partSumLastOffs+1);
    ncvAssertReturn(gpuAllocator.isCounting() || d_hierSums.isMemAllocated(), NPPST_MEM_INTERNAL_ERROR);
    NCVVectorAlloc<Ncv32u> d_numDstElements(gpuAllocator, 1);
    ncvAssertReturn(gpuAllocator.isCounting() || d_numDstElements.isMemAllocated(), NPPST_MEM_INTERNAL_ERROR);

    NCV_SET_SKIP_COND(gpuAllocator.isCounting());
    NCV_SKIP_COND_BEGIN

    dim3 block(NUM_REMOVE_THREADS);

    //calculate zero-level partial sums for indices calculation
    if (partSumNums.size() > 2)
    {
        dim3 grid(partSumNums[1]);

        if (grid.x > 65535)
        {
            grid.y = (grid.x + 65534) / 65535;
            grid.x = 65535;
        }
        removePass1Scan
            <true, true>
            <<<grid, block, 0, nppStGetActiveCUDAstream()>>>
            (d_src, srcLen,
             d_hierSums.ptr(),
             d_hierSums.ptr() + partSumOffsets[1],
             elemRemove);

        ncvAssertCUDALastErrorReturn(NPPST_CUDA_KERNEL_EXECUTION_ERROR);

        //calculate hierarchical partial sums
        for (Ncv32u i=1; i<partSumNums.size()-1; i++)
        {
            dim3 grid_partial(partSumNums[i+1]);
            if (grid_partial.x > 65535)
            {
                grid_partial.y = (grid_partial.x + 65534) / 65535;
                grid_partial.x = 65535;
            }
            if (grid_partial.x != 1)
            {
                removePass1Scan
                    <false, true>
                    <<<grid_partial, block, 0, nppStGetActiveCUDAstream()>>>
                    (d_hierSums.ptr() + partSumOffsets[i],
                     partSumNums[i], NULL,
                     d_hierSums.ptr() + partSumOffsets[i+1],
                     0);
            }
            else
            {
                removePass1Scan
                    <false, false>
                    <<<grid_partial, block, 0, nppStGetActiveCUDAstream()>>>
                    (d_hierSums.ptr() + partSumOffsets[i],
                     partSumNums[i], NULL,
                     NULL,
                     0);
            }

            ncvAssertCUDALastErrorReturn(NPPST_CUDA_KERNEL_EXECUTION_ERROR);
        }

        //adjust hierarchical partial sums
        for (Ncv32s i=(Ncv32s)partSumNums.size()-3; i>=0; i--)
        {
            dim3 grid_local(partSumNums[i+1]);
            if (grid_local.x > 65535)
            {
                grid_local.y = (grid_local.x + 65534) / 65535;
                grid_local.x = 65535;
            }
            removePass2Adjust
                <<<grid_local, block, 0, nppStGetActiveCUDAstream()>>>
                (d_hierSums.ptr() + partSumOffsets[i], partSumNums[i],
                 d_hierSums.ptr() + partSumOffsets[i+1]);

            ncvAssertCUDALastErrorReturn(NPPST_CUDA_KERNEL_EXECUTION_ERROR);
        }
    }
    else
    {
        dim3 grid_local(partSumNums[1]);
        removePass1Scan
            <true, false>
            <<<grid_local, block, 0, nppStGetActiveCUDAstream()>>>
            (d_src, srcLen,
             d_hierSums.ptr(),
             NULL, elemRemove);

        ncvAssertCUDALastErrorReturn(NPPST_CUDA_KERNEL_EXECUTION_ERROR);
    }

    //compact source vector using indices
    dim3 grid(partSumNums[1]);
    if (grid.x > 65535)
    {
        grid.y = (grid.x + 65534) / 65535;
        grid.x = 65535;
    }
    removePass3Compact
        <<<grid, block, 0, nppStGetActiveCUDAstream()>>>
        (d_src, srcLen, d_hierSums.ptr(), d_dst,
         elemRemove, d_numDstElements.ptr());

    ncvAssertCUDALastErrorReturn(NPPST_CUDA_KERNEL_EXECUTION_ERROR);

    //get number of dst elements
    if (dstLenPinned != NULL)
    {
        ncvAssertCUDAReturn(hipMemcpyAsync(dstLenPinned, d_numDstElements.ptr(), sizeof(Ncv32u),
                                              hipMemcpyDeviceToHost, nppStGetActiveCUDAstream()), NPPST_MEM_RESIDENCE_ERROR);
        ncvAssertCUDAReturn(hipStreamSynchronize(nppStGetActiveCUDAstream()), NPPST_MEM_RESIDENCE_ERROR);
    }

    NCV_SKIP_COND_END

    return NPPST_SUCCESS;
}


NCVStatus nppsStCompactGetSize_32u(Ncv32u srcLen, Ncv32u *pBufsize, hipDeviceProp_t &devProp)
{
    ncvAssertReturn(pBufsize != NULL, NPPST_NULL_POINTER_ERROR);

    if (srcLen == 0)
    {
        *pBufsize = 0;
        return NPPST_SUCCESS;
    }

    NCVMemStackAllocator gpuCounter(static_cast<Ncv32u>(devProp.textureAlignment));
    ncvAssertReturn(gpuCounter.isInitialized(), NPPST_MEM_INTERNAL_ERROR);

    NCVStatus ncvStat = compactVector_32u_device(NULL, srcLen, NULL, NULL, 0xC001C0DE,
                                                 gpuCounter);
    ncvAssertReturnNcvStat(ncvStat);

    *pBufsize = (Ncv32u)gpuCounter.maxSize();
    return NPPST_SUCCESS;
}


NCVStatus nppsStCompactGetSize_32s(Ncv32u srcLen, Ncv32u *pBufsize, hipDeviceProp_t &devProp)
{
    return nppsStCompactGetSize_32u(srcLen, pBufsize, devProp);
}


NCVStatus nppsStCompactGetSize_32f(Ncv32u srcLen, Ncv32u *pBufsize, hipDeviceProp_t &devProp)
{
    return nppsStCompactGetSize_32u(srcLen, pBufsize, devProp);
}


NCVStatus nppsStCompact_32u(Ncv32u *d_src, Ncv32u srcLen,
                            Ncv32u *d_dst, Ncv32u *p_dstLen,
                            Ncv32u elemRemove, Ncv8u *pBuffer,
                            Ncv32u bufSize, hipDeviceProp_t &devProp)
{
    NCVMemStackAllocator gpuAllocator(NCVMemoryTypeDevice, bufSize, static_cast<Ncv32u>(devProp.textureAlignment), pBuffer);
    ncvAssertReturn(gpuAllocator.isInitialized(), NPPST_MEM_INTERNAL_ERROR);

    NCVStatus ncvStat = compactVector_32u_device(d_src, srcLen, d_dst, p_dstLen, elemRemove,
                                                 gpuAllocator);
    ncvAssertReturnNcvStat(ncvStat);

    return NPPST_SUCCESS;
}


NCVStatus nppsStCompact_32s(Ncv32s *d_src, Ncv32u srcLen,
                            Ncv32s *d_dst, Ncv32u *p_dstLen,
                            Ncv32s elemRemove, Ncv8u *pBuffer,
                            Ncv32u bufSize, hipDeviceProp_t &devProp)
{
    return nppsStCompact_32u((Ncv32u *)d_src, srcLen, (Ncv32u *)d_dst, p_dstLen,
                             *(Ncv32u *)&elemRemove, pBuffer, bufSize, devProp);
}


#if defined __GNUC__ && (__GNUC__*100 + __GNUC_MINOR__ > 204)
typedef Ncv32u __attribute__((__may_alias__)) Ncv32u_a;
#else
typedef Ncv32u Ncv32u_a;
#endif

NCVStatus nppsStCompact_32f(Ncv32f *d_src, Ncv32u srcLen,
                            Ncv32f *d_dst, Ncv32u *p_dstLen,
                            Ncv32f elemRemove, Ncv8u *pBuffer,
                            Ncv32u bufSize, hipDeviceProp_t &devProp)
{
    return nppsStCompact_32u((Ncv32u *)d_src, srcLen, (Ncv32u *)d_dst, p_dstLen,
                             *(Ncv32u_a *)&elemRemove, pBuffer, bufSize, devProp);
}

NCVStatus nppsStCompact_32u_host(Ncv32u *h_src, Ncv32u srcLen,
                                 Ncv32u *h_dst, Ncv32u *dstLen, Ncv32u elemRemove)
{
    ncvAssertReturn(h_src != NULL && h_dst != NULL, NPPST_NULL_POINTER_ERROR);

    if (srcLen == 0)
    {
        if (dstLen != NULL)
        {
            *dstLen = 0;
        }
        return NPPST_SUCCESS;
    }

    Ncv32u dstIndex = 0;
    for (Ncv32u srcIndex=0; srcIndex<srcLen; srcIndex++)
    {
        if (h_src[srcIndex] != elemRemove)
        {
            h_dst[dstIndex++] = h_src[srcIndex];
        }
    }

    if (dstLen != NULL)
    {
        *dstLen = dstIndex;
    }

    return NPPST_SUCCESS;
}


NCVStatus nppsStCompact_32s_host(Ncv32s *h_src, Ncv32u srcLen,
                                 Ncv32s *h_dst, Ncv32u *dstLen, Ncv32s elemRemove)
{
    return nppsStCompact_32u_host((Ncv32u *)h_src, srcLen, (Ncv32u *)h_dst, dstLen, *(Ncv32u_a *)&elemRemove);
}


NCVStatus nppsStCompact_32f_host(Ncv32f *h_src, Ncv32u srcLen,
                                 Ncv32f *h_dst, Ncv32u *dstLen, Ncv32f elemRemove)
{
    return nppsStCompact_32u_host((Ncv32u *)h_src, srcLen, (Ncv32u *)h_dst, dstLen, *(Ncv32u_a *)&elemRemove);
}

//==============================================================================
//
// Filter.cu
//
//==============================================================================

__forceinline__ __device__ float getValueMirrorRow(cv::cudev::TexturePtr< Ncv32f> tex, const int rowOffset, int i, int w)
{
    if (i < 0) i = 1 - i;
    if (i >= w) i = w + w - i - 1;
    return tex(rowOffset + i);
}


__forceinline__ __device__ float getValueMirrorColumn(cv::cudev::TexturePtr< Ncv32f> tex, const int offset, const int rowStep, int j, int h)
{
    if (j < 0) j = 1 - j;
    if (j >= h) j = h + h - j - 1;
    return tex(offset + j * rowStep);
}


__global__ void FilterRowBorderMirror_32f_C1R(cv::cudev::TexturePtr<Ncv32f> texSrc, cv::cudev::TexturePtr<Ncv32f> texKernel1, Ncv32u srcStep, Ncv32f *pDst, NcvSize32u dstSize, Ncv32u dstStep,
    NcvRect32u roi, Ncv32s nKernelSize, Ncv32s nAnchor, Ncv32f multiplier)
{
    // position within ROI
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if (ix >= roi.width || iy >= roi.height)
    {
        return;
    }

    const int p = nKernelSize - nAnchor - 1;

    const int j = roi.y + iy;

    const int rowOffset = j * srcStep + roi.x;

    float sum = 0.0f;
    for (int m = 0; m < nKernelSize; ++m)
    {
        sum += getValueMirrorRow(texSrc, rowOffset, ix + m - p, roi.width)
            * texKernel1(m);
    }

    pDst[iy * dstStep + ix] = sum * multiplier;
}


__global__ void FilterColumnBorderMirror_32f_C1R(cv::cudev::TexturePtr<Ncv32f> texSrc, cv::cudev::TexturePtr<Ncv32f> texKernel, Ncv32u srcStep, Ncv32f *pDst, NcvSize32u dstSize, Ncv32u dstStep,
    NcvRect32u roi, Ncv32s nKernelSize, Ncv32s nAnchor, Ncv32f multiplier)
{
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if (ix >= roi.width || iy >= roi.height)
    {
        return;
    }

    const int p = nKernelSize - nAnchor - 1;
    const int i = roi.x + ix;
    const int offset = i + roi.y * srcStep;

    float sum = 0.0f;
    for (int m = 0; m < nKernelSize; ++m)
    {
        sum += getValueMirrorColumn(texSrc, offset, srcStep, iy + m - p, roi.height)
            * texKernel(m);
    }

    pDst[ix + iy * dstStep] = sum * multiplier;
}


NCVStatus nppiStFilterRowBorder_32f_C1R(Ncv32f *pSrc,
                                        NcvSize32u srcSize,
                                        Ncv32u nSrcStep,
                                        Ncv32f *pDst,
                                        NcvSize32u dstSize,
                                        Ncv32u nDstStep,
                                        NcvRect32u oROI,
                                        NppStBorderType borderType,
                                        Ncv32f *pKernel,
                                        Ncv32s nKernelSize,
                                        Ncv32s nAnchor,
                                        Ncv32f multiplier)
{
    ncvAssertReturn (pSrc != NULL &&
        pDst != NULL &&
        pKernel != NULL, NCV_NULL_PTR);

    ncvAssertReturn (oROI.width > 0 && oROI.height > 0, NPPST_INVALID_ROI);

    ncvAssertReturn (srcSize.width * sizeof (Ncv32f) <= nSrcStep &&
        dstSize.width * sizeof (Ncv32f) <= nDstStep &&
        oROI.width * sizeof (Ncv32f) <= nSrcStep &&
        oROI.width * sizeof (Ncv32f) <= nDstStep &&
        nSrcStep % sizeof (Ncv32f) == 0 &&
        nDstStep % sizeof (Ncv32f) == 0, NPPST_INVALID_STEP);

    Ncv32u srcStep = nSrcStep / sizeof (Ncv32f);
    Ncv32u dstStep = nDstStep / sizeof (Ncv32f);

    // adjust ROI size to be within source image
    if (oROI.x + oROI.width > srcSize.width)
    {
        oROI.width = srcSize.width - oROI.x;
    }

    if (oROI.y + oROI.height > srcSize.height)
    {
        oROI.height = srcSize.height - oROI.y;
    }

    cv::cudev::Texture<Ncv32f> texSrc(srcSize.height * nSrcStep, pSrc);
    cv::cudev::Texture<Ncv32f> texKernel(nKernelSize * sizeof(Ncv32f), pKernel);

    dim3 ctaSize (32, 6);
    dim3 gridSize ((oROI.width + ctaSize.x - 1) / ctaSize.x,
        (oROI.height + ctaSize.y - 1) / ctaSize.y);

    switch (borderType)
    {
    case nppStBorderNone:
        return NPPST_ERROR;
    case nppStBorderClamp:
        return NPPST_ERROR;
    case nppStBorderWrap:
        return NPPST_ERROR;
    case nppStBorderMirror:
        FilterRowBorderMirror_32f_C1R <<<gridSize, ctaSize, 0, nppStGetActiveCUDAstream ()>>>(texSrc, texKernel, srcStep, pDst, dstSize, dstStep, oROI, nKernelSize, nAnchor, multiplier);
        ncvAssertCUDALastErrorReturn(NPPST_CUDA_KERNEL_EXECUTION_ERROR);
        break;
    default:
        return NPPST_ERROR;
    }

    return NPPST_SUCCESS;
}


NCVStatus nppiStFilterColumnBorder_32f_C1R(Ncv32f *pSrc,
                                           NcvSize32u srcSize,
                                           Ncv32u nSrcStep,
                                           Ncv32f *pDst,
                                           NcvSize32u dstSize,
                                           Ncv32u nDstStep,
                                           NcvRect32u oROI,
                                           NppStBorderType borderType,
                                           Ncv32f *pKernel,
                                           Ncv32s nKernelSize,
                                           Ncv32s nAnchor,
                                           Ncv32f multiplier)
{
    ncvAssertReturn (pSrc != NULL &&
        pDst != NULL &&
        pKernel != NULL, NCV_NULL_PTR);

    ncvAssertReturn (oROI.width > 0 && oROI.height > 0, NPPST_INVALID_ROI);

    ncvAssertReturn (srcSize.width * sizeof (Ncv32f) <= nSrcStep &&
        dstSize.width * sizeof (Ncv32f) <= nDstStep &&
        oROI.width * sizeof (Ncv32f) <= nSrcStep &&
        oROI.width * sizeof (Ncv32f) <= nDstStep &&
        nSrcStep % sizeof (Ncv32f) == 0 &&
        nDstStep % sizeof (Ncv32f) == 0, NPPST_INVALID_STEP);

    Ncv32u srcStep = nSrcStep / sizeof (Ncv32f);
    Ncv32u dstStep = nDstStep / sizeof (Ncv32f);

    // adjust ROI size to be within source image
    if (oROI.x + oROI.width > srcSize.width)
    {
        oROI.width = srcSize.width - oROI.x;
    }

    if (oROI.y + oROI.height > srcSize.height)
    {
        oROI.height = srcSize.height - oROI.y;
    }

    cv::cudev::Texture<Ncv32f> texSrc(srcSize.height * nSrcStep, pSrc);
    cv::cudev::Texture<Ncv32f> texKernel(nKernelSize * sizeof(Ncv32f), pKernel);

    dim3 ctaSize (32, 6);
    dim3 gridSize ((oROI.width + ctaSize.x - 1) / ctaSize.x,
        (oROI.height + ctaSize.y - 1) / ctaSize.y);

    switch (borderType)
    {
    case nppStBorderClamp:
        return NPPST_ERROR;
    case nppStBorderWrap:
        return NPPST_ERROR;
    case nppStBorderMirror:
        FilterColumnBorderMirror_32f_C1R <<<gridSize, ctaSize, 0, nppStGetActiveCUDAstream ()>>>(texSrc, texKernel, srcStep, pDst, dstSize, dstStep, oROI, nKernelSize, nAnchor, multiplier);
        ncvAssertCUDALastErrorReturn(NPPST_CUDA_KERNEL_EXECUTION_ERROR);
        break;
    default:
        return NPPST_ERROR;
    }

    return NPPST_SUCCESS;
}


//==============================================================================
//
// FrameInterpolate.cu
//
//==============================================================================


inline Ncv32u iDivUp(Ncv32u num, Ncv32u denom)
{
    return (num + denom - 1)/denom;
}

__global__ void BlendFramesKernel(cv::cudev::TexturePtr<Ncv32f> texSrc0, cv::cudev::TexturePtr<Ncv32f> texSrc1,
    const float *u, const float *v,   // forward flow
    const float *ur, const float *vr, // backward flow
    const float *o0, const float *o1, // coverage masks
    int w, int h, int s, float theta, float *out)
{
    const int ix = threadIdx.x + blockDim.x * blockIdx.x;
    const int iy = threadIdx.y + blockDim.y * blockIdx.y;

    const int pos = ix + s * iy;

    if (ix >= w || iy >= h) return;

    float _u = u[pos];
    float _v = v[pos];

    float _ur = ur[pos];
    float _vr = vr[pos];

    float x = (float)ix + 0.5f;
    float y = (float)iy + 0.5f;
    bool b0 = o0[pos] > 1e-4f;
    bool b1 = o1[pos] > 1e-4f;

    if (b0 && b1) // pixel is visible on both frames
        out[pos] = texSrc0(y - _v * theta, x - _u * theta)* (1.0f - theta) + texSrc0(y + _v * (1.0f - theta), x + _u * (1.0f - theta)) * theta;
    else if (b0) // visible on the first frame only
        out[pos] = texSrc0(y - _v * theta, x - _u * theta);
    else // visible on the second frame only
        out[pos] = texSrc1(y - _vr * (1.0f - theta), x - _ur * (1.0f - theta));
}


NCVStatus BlendFrames(Ncv32f *src0,
                      Ncv32f *src1,
                      const Ncv32f *ufi,
                      const Ncv32f *vfi,
                      const Ncv32f *ubi,
                      const Ncv32f *vbi,
                      const Ncv32f *o1,
                      const Ncv32f *o2,
                      Ncv32u width,
                      Ncv32u height,
                      Ncv32u stride,
                      Ncv32f theta,
                      Ncv32f *out)
{
    const Ncv32u pitch = stride * sizeof (float);
    cv::cudev::Texture<Ncv32f> texSrc0(height, width, src0, pitch, false, hipFilterModeLinear);
    cv::cudev::Texture<Ncv32f> texSrc1(height, width, src1, pitch, false, hipFilterModeLinear);
    dim3 threads (32, 4);
    dim3 blocks (iDivUp (width, threads.x), iDivUp (height, threads.y));
    BlendFramesKernel<<<blocks, threads, 0, nppStGetActiveCUDAstream ()>>>(texSrc0, texSrc1, ufi, vfi, ubi, vbi, o1, o2, width, height, stride, theta, out);
    ncvAssertCUDALastErrorReturn(NPPST_CUDA_KERNEL_EXECUTION_ERROR);
    return NPPST_SUCCESS;
}


NCVStatus nppiStGetInterpolationBufferSize(NcvSize32u srcSize,
                                           Ncv32u nStep,
                                           Ncv32u *hpSize)
{
    NCVStatus status = NPPST_ERROR;
    status = nppiStVectorWarpGetBufferSize(srcSize, nStep, hpSize);
    return status;
}


NCVStatus nppiStInterpolateFrames(const NppStInterpolationState *pState)
{
    // check state validity
    ncvAssertReturn (pState->pSrcFrame0 != 0 &&
        pState->pSrcFrame1 != 0 &&
        pState->pFU != 0 &&
        pState->pFV != 0 &&
        pState->pBU != 0 &&
        pState->pBV != 0 &&
        pState->pNewFrame != 0 &&
        pState->ppBuffers[0] != 0 &&
        pState->ppBuffers[1] != 0 &&
        pState->ppBuffers[2] != 0 &&
        pState->ppBuffers[3] != 0 &&
        pState->ppBuffers[4] != 0 &&
        pState->ppBuffers[5] != 0, NPPST_NULL_POINTER_ERROR);

    ncvAssertReturn (pState->size.width  > 0 &&
        pState->size.height > 0, NPPST_ERROR);

    ncvAssertReturn (pState->nStep >= pState->size.width * sizeof (Ncv32f) &&
        pState->nStep > 0 &&
        pState->nStep % sizeof (Ncv32f) == 0,
        NPPST_INVALID_STEP);

    // change notation
    Ncv32f *cov0 = pState->ppBuffers[0];
    Ncv32f *cov1 = pState->ppBuffers[1];
    Ncv32f *fwdU = pState->ppBuffers[2]; // forward u
    Ncv32f *fwdV = pState->ppBuffers[3]; // forward v
    Ncv32f *bwdU = pState->ppBuffers[4]; // backward u
    Ncv32f *bwdV = pState->ppBuffers[5]; // backward v
    // warp flow
    ncvAssertReturnNcvStat (
        nppiStVectorWarp_PSF2x2_32f_C1 (pState->pFU,
        pState->size,
        pState->nStep,
        pState->pFU,
        pState->pFV,
        pState->nStep,
        cov0,
        pState->pos,
        fwdU) );
    ncvAssertReturnNcvStat (
        nppiStVectorWarp_PSF2x2_32f_C1 (pState->pFV,
        pState->size,
        pState->nStep,
        pState->pFU,
        pState->pFV,
        pState->nStep,
        cov0,
        pState->pos,
        fwdV) );
    // warp backward flow
    ncvAssertReturnNcvStat (
        nppiStVectorWarp_PSF2x2_32f_C1 (pState->pBU,
        pState->size,
        pState->nStep,
        pState->pBU,
        pState->pBV,
        pState->nStep,
        cov1,
        1.0f - pState->pos,
        bwdU) );
    ncvAssertReturnNcvStat (
        nppiStVectorWarp_PSF2x2_32f_C1 (pState->pBV,
        pState->size,
        pState->nStep,
        pState->pBU,
        pState->pBV,
        pState->nStep,
        cov1,
        1.0f - pState->pos,
        bwdU) );
    // interpolate frame
    ncvAssertReturnNcvStat (
        BlendFrames (pState->pSrcFrame0,
        pState->pSrcFrame1,
        fwdU,
        fwdV,
        bwdU,
        bwdV,
        cov0,
        cov1,
        pState->size.width,
        pState->size.height,
        pState->nStep / sizeof (Ncv32f),
        pState->pos,
        pState->pNewFrame) );

    return NPPST_SUCCESS;
}


//==============================================================================
//
// VectorWarpFrame.cu
//
//==============================================================================


#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 200)

// FP32 atomic add
static __forceinline__ __device__ float _atomicAdd(float *addr, float val)
{
    float old = *addr, assumed;

    do {
        assumed = old;
        old = int_as_float(__iAtomicCAS((int*)addr,
              float_as_int(assumed),
              float_as_int(val+assumed)));
    } while( assumed!=old );

    return old;
}
#else
#define _atomicAdd atomicAdd
#endif


__global__ void ForwardWarpKernel_PSF2x2(const float *u,
                                         const float *v,
                                         const float *src,
                                         const int w,
                                         const int h,
                                         const int flow_stride,
                                         const int image_stride,
                                         const float time_scale,
                                         float *normalization_factor,
                                         float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    //bottom left corner of a target pixel
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;
    // pixel containing bottom left corner
    float px;
    float py;
    float dx = modff (cx, &px);
    float dy = modff (cy, &py);
    // target pixel integer coords
    int tx;
    int ty;
    tx = (int) px;
    ty = (int) py;
    float value = src[image_row_offset + j];
    float weight;
    // fill pixel containing bottom right corner
    if (!((tx >= w) || (tx < 0) || (ty >= h) || (ty < 0)))
    {
        weight = dx * dy;
        _atomicAdd (dst + ty * image_stride + tx, value * weight);
        _atomicAdd (normalization_factor + ty * image_stride + tx, weight);
    }

    // fill pixel containing bottom left corner
    tx -= 1;
    if (!((tx >= w) || (tx < 0) || (ty >= h) || (ty < 0)))
    {
        weight = (1.0f - dx) * dy;
        _atomicAdd (dst + ty * image_stride + tx, value * weight);
        _atomicAdd (normalization_factor + ty * image_stride + tx, weight);
    }

    // fill pixel containing upper left corner
    ty -= 1;
    if (!((tx >= w) || (tx < 0) || (ty >= h) || (ty < 0)))
    {
        weight = (1.0f - dx) * (1.0f - dy);
        _atomicAdd (dst + ty * image_stride + tx, value * weight);
        _atomicAdd (normalization_factor + ty * image_stride + tx, weight);
    }

    // fill pixel containing upper right corner
    tx += 1;
    if (!((tx >= w) || (tx < 0) || (ty >= h) || (ty < 0)))
    {
        weight = dx * (1.0f - dy);
        _atomicAdd (dst + ty * image_stride + tx, value * weight);
        _atomicAdd (normalization_factor + ty * image_stride + tx, weight);
    }
}


__global__ void ForwardWarpKernel_PSF1x1(const float *u,
                                         const float *v,
                                         const float *src,
                                         const int w,
                                         const int h,
                                         const int flow_stride,
                                         const int image_stride,
                                         const float time_scale,
                                         float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset = i * flow_stride;
    int image_row_offset = i * image_stride;

    float u_ = u[flow_row_offset + j];
    float v_ = v[flow_row_offset + j];

    //bottom left corner of target pixel
    float cx = u_ * time_scale + (float)j + 1.0f;
    float cy = v_ * time_scale + (float)i + 1.0f;
    // pixel containing bottom left corner
    int tx = __float2int_rn (cx);
    int ty = __float2int_rn (cy);

    float value = src[image_row_offset + j];
    // fill pixel
    if (!((tx >= w) || (tx < 0) || (ty >= h) || (ty < 0)))
    {
        _atomicAdd (dst + ty * image_stride + tx, value);
    }
}


__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    int i = threadIdx.y + blockDim.y * blockIdx.y;
    int j = threadIdx.x + blockDim.x * blockIdx.x;

    if (i >= h || j >= w) return;

    const int pos = i * s + j;

    float scale = normalization_factor[pos];

    float invScale = (scale == 0.0f) ? 1.0f : (1.0f / scale);

    image[pos] *= invScale;
}


__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    int i = threadIdx.y + blockDim.y * blockIdx.y;
    int j = threadIdx.x + blockDim.x * blockIdx.x;

    if (i >= h || j >= w) return;

    const int pos = i * w + j;

    image[pos] = value;
}


NCVStatus nppiStVectorWarpGetBufferSize (NcvSize32u srcSize, Ncv32u nSrcStep, Ncv32u *hpSize)
{
    ncvAssertReturn (hpSize != NULL, NPPST_NULL_POINTER_ERROR);
    ncvAssertReturn (srcSize.width * sizeof (Ncv32f) <= nSrcStep,
        NPPST_INVALID_STEP);

    *hpSize = nSrcStep * srcSize.height;

    return NPPST_SUCCESS;
}


// does not require normalization
NCVStatus nppiStVectorWarp_PSF1x1_32f_C1(const Ncv32f *pSrc,
                                         NcvSize32u srcSize,
                                         Ncv32u nSrcStep,
                                         const Ncv32f *pU,
                                         const Ncv32f *pV,
                                         Ncv32u nVFStep,
                                         Ncv32f timeScale,
                                         Ncv32f *pDst)
{
    ncvAssertReturn (pSrc != NULL &&
        pU   != NULL &&
        pV   != NULL &&
        pDst != NULL, NPPST_NULL_POINTER_ERROR);

    ncvAssertReturn (srcSize.width * sizeof (Ncv32f) <= nSrcStep &&
        srcSize.width * sizeof (Ncv32f) <= nVFStep,
        NPPST_INVALID_STEP);

    Ncv32u srcStep = nSrcStep / sizeof (Ncv32f);
    Ncv32u vfStep  = nVFStep / sizeof (Ncv32f);

    dim3 ctaSize (32, 6);
    dim3 gridSize (iDivUp (srcSize.width, ctaSize.x), iDivUp (srcSize.height, ctaSize.y));

    ForwardWarpKernel_PSF1x1 <<<gridSize, ctaSize, 0, nppStGetActiveCUDAstream()>>>
        (pU, pV, pSrc, srcSize.width, srcSize.height, vfStep, srcStep, timeScale, pDst);

    ncvAssertCUDALastErrorReturn(NPPST_CUDA_KERNEL_EXECUTION_ERROR);

    return NPPST_SUCCESS;
}


NCVStatus nppiStVectorWarp_PSF2x2_32f_C1(const Ncv32f *pSrc,
                                         NcvSize32u srcSize,
                                         Ncv32u nSrcStep,
                                         const Ncv32f *pU,
                                         const Ncv32f *pV,
                                         Ncv32u nVFStep,
                                         Ncv32f *pBuffer,
                                         Ncv32f timeScale,
                                         Ncv32f *pDst)
{
    ncvAssertReturn (pSrc != NULL &&
        pU   != NULL &&
        pV   != NULL &&
        pDst != NULL &&
        pBuffer != NULL, NPPST_NULL_POINTER_ERROR);

    ncvAssertReturn (srcSize.width * sizeof (Ncv32f) <= nSrcStep &&
        srcSize.width * sizeof (Ncv32f) <= nVFStep, NPPST_INVALID_STEP);

    Ncv32u srcStep = nSrcStep / sizeof (Ncv32f);
    Ncv32u vfStep = nVFStep / sizeof(Ncv32f);

    dim3 ctaSize(32, 6);
    dim3 gridSize (iDivUp (srcSize.width, ctaSize.x), iDivUp (srcSize.height, ctaSize.y));

    MemsetKernel <<<gridSize, ctaSize, 0, nppStGetActiveCUDAstream()>>>
        (0, srcSize.width, srcSize.height, pBuffer);

    ncvAssertCUDALastErrorReturn(NPPST_CUDA_KERNEL_EXECUTION_ERROR);

    ForwardWarpKernel_PSF2x2 <<<gridSize, ctaSize, 0, nppStGetActiveCUDAstream()>>>
        (pU, pV, pSrc, srcSize.width, srcSize.height, vfStep, srcStep, timeScale, pBuffer, pDst);

    ncvAssertCUDALastErrorReturn(NPPST_CUDA_KERNEL_EXECUTION_ERROR);

    NormalizeKernel <<<gridSize, ctaSize, 0, nppStGetActiveCUDAstream()>>>
        (pBuffer, srcSize.width, srcSize.height, srcStep, pDst);

    ncvAssertCUDALastErrorReturn(NPPST_CUDA_KERNEL_EXECUTION_ERROR);

    return NPPST_SUCCESS;
}


//==============================================================================
//
// Resize.cu
//
//==============================================================================

__forceinline__
__device__ float processLine(cv::cudev::TexturePtr<Ncv32f> tex, int spos, float xmin, float xmax, int ixmin, int ixmax, float fxmin, float cxmax)
{
    // first element
    float wsum = 1.0f - xmin + fxmin;
    float sum = tex( spos) * (1.0f - xmin + fxmin);
    spos++;
    for (int ix = ixmin + 1; ix < ixmax; ++ix)
    {
        sum += tex(spos);
        spos++;
        wsum += 1.0f;
    }
    sum += tex(spos) * (cxmax - xmax);
    wsum += cxmax - xmax;
    return sum / wsum;
}


__global__ void resizeSuperSample_32f(cv::cudev::TexturePtr<Ncv32f> texSrc, NcvSize32u srcSize, Ncv32u srcStep, NcvRect32u srcROI, Ncv32f *dst, NcvSize32u dstSize, Ncv32u dstStep,
    NcvRect32u dstROI, Ncv32f scaleX, Ncv32f scaleY)
{
    // position within dst ROI
    const int ix = blockIdx.x * blockDim.x + threadIdx.x;
    const int iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix >= dstROI.width || iy >= dstROI.height)
    {
        return;
    }

    float rw = (float) srcROI.width;
    float rh = (float) srcROI.height;

    // source position
    float x = scaleX * (float) ix;
    float y = scaleY * (float) iy;

    // x sampling range
    float xBegin = fmax (x - scaleX, 0.0f);
    float xEnd   = fmin (x + scaleX, rw - 1.0f);
    // y sampling range
    float yBegin = fmax (y - scaleY, 0.0f);
    float yEnd   = fmin (y + scaleY, rh - 1.0f);
    // x range of source samples
    float floorXBegin = floorf (xBegin);
    float ceilXEnd    = ceilf (xEnd);
    int iXBegin = srcROI.x + (int) floorXBegin;
    int iXEnd   = srcROI.x + (int) ceilXEnd;
    // y range of source samples
    float floorYBegin = floorf (yBegin);
    float ceilYEnd    = ceilf (yEnd);
    int iYBegin = srcROI.y + (int) floorYBegin;
    int iYEnd   = srcROI.y + (int) ceilYEnd;

    // first row
    int pos = iYBegin * srcStep + iXBegin;

    float wsum = 1.0f - yBegin + floorYBegin;

    float sum = processLine (texSrc, pos, xBegin, xEnd, iXBegin, iXEnd, floorXBegin,
        ceilXEnd) * (1.0f - yBegin + floorYBegin);
    pos += srcStep;
    for (int iy = iYBegin + 1; iy < iYEnd; ++iy)
    {
        sum += processLine (texSrc, pos, xBegin, xEnd, iXBegin, iXEnd, floorXBegin,
            ceilXEnd);
        pos += srcStep;
        wsum += 1.0f;
    }

    sum += processLine (texSrc, pos, xBegin, xEnd, iXBegin, iXEnd, floorXBegin,
        ceilXEnd) * (ceilYEnd - yEnd);
    wsum += ceilYEnd - yEnd;
    sum /= wsum;

    dst[(ix + dstROI.x) + (iy + dstROI.y) * dstStep] = sum;
}


// bicubic interpolation
__forceinline__
__device__ float bicubicCoeff(float x_)
{
    float x = fabsf(x_);
    if (x <= 1.0f)
    {
        return x * x * (1.5f * x - 2.5f) + 1.0f;
    }
    else if (x < 2.0f)
    {
        return x * (x * (-0.5f * x + 2.5f) - 4.0f) + 2.0f;
    }
    else
    {
        return 0.0f;
    }
}


__global__ void resizeBicubic(cv::cudev::TexturePtr<Ncv32f> texSrc, NcvSize32u srcSize, NcvRect32u srcROI, NcvSize32u dstSize, Ncv32u dstStep, Ncv32f *dst, NcvRect32u dstROI, Ncv32f scaleX, Ncv32f scaleY)
{
    const int ix = blockIdx.x * blockDim.x + threadIdx.x;
    const int iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix >= dstROI.width || iy >= dstROI.height)
    {
        return;
    }

    const float dx = 1.0f / srcROI.width;
    const float dy = 1.0f / srcROI.height;

    float rx = (float) srcROI.x;
    float ry = (float) srcROI.y;

    float rw = (float) srcROI.width;
    float rh = (float) srcROI.height;

    float x = scaleX * (float) ix;
    float y = scaleY * (float) iy;

    // sampling range
    // border mode is clamp
    float xmin = fmax (ceilf (x - 2.0f), 0.0f);
    float xmax = fmin (floorf (x + 2.0f), rw - 1.0f);

    float ymin = fmax (ceilf (y - 2.0f), 0.0f);
    float ymax = fmin (floorf (y + 2.0f), rh - 1.0f);

    // shift data window to match ROI
    rx += 0.5f;
    ry += 0.5f;

    x += rx;
    y += ry;

    xmin += rx;
    xmax += rx;
    ymin += ry;
    ymax += ry;

    float sum  = 0.0f;
    float wsum = 0.0f;

    for (float cy = ymin; cy <= ymax; cy += 1.0f)
    {
        for (float cx = xmin; cx <= xmax; cx += 1.0f)
        {
            float xDist = x - cx;
            float yDist = y - cy;
            float wx = bicubicCoeff (xDist);
            float wy = bicubicCoeff (yDist);
            wx *= wy;
            sum += wx * texSrc(cy * dy, cx * dx);
            wsum += wx;
        }
    }
    dst[(ix + dstROI.x)+ (iy + dstROI.y) * dstStep] = (!wsum)? 0 : sum / wsum;
}


NCVStatus nppiStResize_32f_C1R(Ncv32f *pSrc,
                               NcvSize32u srcSize,
                               Ncv32u nSrcStep,
                               NcvRect32u srcROI,
                               Ncv32f *pDst,
                               NcvSize32u dstSize,
                               Ncv32u nDstStep,
                               NcvRect32u dstROI,
                               Ncv32f xFactor,
                               Ncv32f yFactor,
                               NppStInterpMode interpolation)
{
    NCVStatus status = NPPST_SUCCESS;

    ncvAssertReturn (pSrc != NULL && pDst != NULL, NPPST_NULL_POINTER_ERROR);
    ncvAssertReturn (xFactor != 0.0 && yFactor != 0.0, NPPST_INVALID_SCALE);

    ncvAssertReturn (nSrcStep >= sizeof (Ncv32f) * (Ncv32u) srcSize.width &&
        nDstStep >= sizeof (Ncv32f) * (Ncv32f) dstSize.width,
        NPPST_INVALID_STEP);

    Ncv32u srcStep = nSrcStep / sizeof (Ncv32f);
    Ncv32u dstStep = nDstStep / sizeof (Ncv32f);

    // TODO: preprocess ROI to prevent out of bounds access

    if (interpolation == nppStSupersample)
    {
        cv::cudev::Texture<Ncv32f> texSrc(srcSize.height * nSrcStep, pSrc);
        dim3 ctaSize (32, 6);
        dim3 gridSize ((dstROI.width  + ctaSize.x - 1) / ctaSize.x,(dstROI.height + ctaSize.y - 1) / ctaSize.y);
        resizeSuperSample_32f <<<gridSize, ctaSize, 0, nppStGetActiveCUDAstream ()>>> (texSrc, srcSize, srcStep, srcROI, pDst, dstSize, dstStep, dstROI, 1.0f / xFactor, 1.0f / yFactor);
    }
    else if (interpolation == nppStBicubic)
    {
        cv::cudev::Texture<float> texSrc(srcSize.height, srcSize.width, pSrc, nSrcStep, true, hipFilterModePoint, hipAddressModeMirror);
        dim3 ctaSize (32, 6);
        dim3 gridSize ((dstSize.width  + ctaSize.x - 1) / ctaSize.x, (dstSize.height + ctaSize.y - 1) / ctaSize.y);
        resizeBicubic <<<gridSize, ctaSize, 0, nppStGetActiveCUDAstream ()>>> (texSrc, srcSize, srcROI, dstSize, dstStep, pDst, dstROI, 1.0f / xFactor, 1.0f / yFactor);
    }
    else
    {
        status = NPPST_ERROR;
    }

    ncvAssertCUDALastErrorReturn(NPPST_CUDA_KERNEL_EXECUTION_ERROR);

    return status;
}

#endif /* CUDA_DISABLER */
