#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

////////////////////////////////////////////////////////////////////////////////
//
// NVIDIA CUDA implementation of Viola-Jones Object Detection Framework
//
// The algorithm and code are explained in the upcoming GPU Computing Gems
// chapter in detail:
//
//   Anton Obukhov, "Haar Classifiers for Object Detection with CUDA"
//   PDF URL placeholder
//   email: aobukhov@nvidia.com, devsupport@nvidia.com
//
// Credits for help with the code to:
// Alexey Mendelenko, Cyril Crassin, and Mikhail Smirnov.
//
////////////////////////////////////////////////////////////////////////////////

#include <algorithm>
#include <cstdio>

#include "opencv2/cudev.hpp"
#include "opencv2/core/persistence.hpp"

#include "opencv2/opencv_modules.hpp"

#ifdef HAVE_OPENCV_OBJDETECT
#  include "opencv2/objdetect.hpp"
//#  include "opencv2/objdetect/objdetect_c.h"
#endif

#include "opencv2/cudalegacy/NCV.hpp"
#include "opencv2/cudalegacy/NPP_staging.hpp"
#include "opencv2/cudalegacy/NCVHaarObjectDetection.hpp"
#include <opencv2/cudev/ptr2d/texture.hpp>

#include "NCVRuntimeTemplates.hpp"
#include "NCVAlg.hpp"


//==============================================================================
//
// HaarClassifierCascade file
//
//==============================================================================


const Ncv32u MAX_GRID_DIM = 65535;


const Ncv32u NUM_THREADS_ANCHORSPARALLEL = 64;


#define NUM_THREADS_CLASSIFIERPARALLEL_LOG2     6
#define NUM_THREADS_CLASSIFIERPARALLEL          (1 << NUM_THREADS_CLASSIFIERPARALLEL_LOG2)


__device__ HaarStage64 getStage(Ncv32u iStage, HaarStage64 *d_Stages)
{
    return d_Stages[iStage];
}


template <NcvBool tbCacheTextureCascade>
__device__ HaarClassifierNode128 getClassifierNode(cv::cudev::TexturePtr<uint4> texHaarClassifierNodes, Ncv32u iNode, HaarClassifierNode128 *d_ClassifierNodes)
{
    HaarClassifierNode128 tmpNode;
    if (tbCacheTextureCascade)
        tmpNode._ui4 = texHaarClassifierNodes(iNode);
    else
        tmpNode = d_ClassifierNodes[iNode];
    return tmpNode;
}


template <NcvBool tbCacheTextureCascade>
__device__ void getFeature(cv::cudev::TexturePtr<uint2> texHaarFeatures, Ncv32u iFeature, HaarFeature64* d_Features, Ncv32f* weight, Ncv32u* rectX, Ncv32u* rectY, Ncv32u* rectWidth, Ncv32u* rectHeight)
{
    HaarFeature64 feature;
    if (tbCacheTextureCascade)
        feature._ui2 = texHaarFeatures(iFeature);
    else
        feature = d_Features[iFeature];
    feature.getRect(rectX, rectY, rectWidth, rectHeight);
    *weight = feature.getWeight();
}


template <NcvBool tbCacheTextureIImg>
__device__ Ncv32u getElemIImg(cv::cudev::TexturePtr<Ncv32u> texImg, Ncv32u x, Ncv32u *d_IImg)
{
    if (tbCacheTextureIImg)
        return texImg(x);
    else
        return d_IImg[x];
}


__device__ Ncv32u d_outMaskPosition;


__device__ void compactBlockWriteOutAnchorParallel(Ncv32u threadPassFlag, Ncv32u threadElem, Ncv32u *vectorOut)
{
#if __CUDA_ARCH__ && __CUDA_ARCH__ >= 110

    __shared__ Ncv32u shmem[NUM_THREADS_ANCHORSPARALLEL];
    __shared__ Ncv32u numPassed;
    __shared__ Ncv32u outMaskOffset;

    Ncv32u incScan = cv::cudev::blockScanInclusive<NUM_THREADS_ANCHORSPARALLEL>(threadPassFlag, shmem, threadIdx.x);
    __syncthreads();

    if (threadIdx.x == NUM_THREADS_ANCHORSPARALLEL-1)
    {
        numPassed = incScan;
        outMaskOffset = atomicAdd(&d_outMaskPosition, incScan);
    }

    if (threadPassFlag)
    {
        Ncv32u excScan = incScan - threadPassFlag;
        shmem[excScan] = threadElem;
    }

    __syncthreads();

    if (threadIdx.x < numPassed)
    {
        vectorOut[outMaskOffset + threadIdx.x] = shmem[threadIdx.x];
    }
#endif
}


template <NcvBool tbInitMaskPositively, NcvBool tbCacheTextureIImg, NcvBool tbCacheTextureCascade, NcvBool tbReadPixelIndexFromVector, NcvBool tbDoAtomicCompaction>
__global__ void applyHaarClassifierAnchorParallel(cv::cudev::TexturePtr<Ncv32u> texImg, cv::cudev::TexturePtr<uint2> texHaarFeatures, cv::cudev::TexturePtr<uint4> texHaarClassifierNodes,
    Ncv32u *d_IImg, Ncv32u IImgStride, Ncv32f *d_weights, Ncv32u weightsStride, HaarFeature64 *d_Features, HaarClassifierNode128 *d_ClassifierNodes, HaarStage64 *d_Stages, Ncv32u *d_inMask,
    Ncv32u *d_outMask, Ncv32u mask1Dlen, Ncv32u mask2Dstride,  NcvSize32u anchorsRoi, Ncv32u startStageInc, Ncv32u endStageExc, Ncv32f scaleArea)
{
    Ncv32u y_offs;
    Ncv32u x_offs;
    Ncv32u maskOffset;
    Ncv32u outMaskVal;

    NcvBool bInactiveThread = false;

    if (tbReadPixelIndexFromVector)
    {
        maskOffset = (MAX_GRID_DIM * blockIdx.y + blockIdx.x) * NUM_THREADS_ANCHORSPARALLEL + threadIdx.x;

        if (maskOffset >= mask1Dlen)
        {
            if (tbDoAtomicCompaction) bInactiveThread = true; else return;
        }

        if (!tbDoAtomicCompaction || (tbDoAtomicCompaction && !bInactiveThread))
        {
            outMaskVal = d_inMask[maskOffset];
            y_offs = outMaskVal >> 16;
            x_offs = outMaskVal & 0xFFFF;
        }
    }
    else
    {
        y_offs = blockIdx.y;
        x_offs = blockIdx.x * NUM_THREADS_ANCHORSPARALLEL + threadIdx.x;

        if (x_offs >= mask2Dstride)
        {
            if (tbDoAtomicCompaction) bInactiveThread = true; else return;
        }

        if (!tbDoAtomicCompaction || (tbDoAtomicCompaction && !bInactiveThread))
        {
            maskOffset = y_offs * mask2Dstride + x_offs;

            if ((x_offs >= anchorsRoi.width) ||
                (!tbInitMaskPositively &&
                 d_inMask != d_outMask &&
                 d_inMask[maskOffset] == OBJDET_MASK_ELEMENT_INVALID_32U))
            {
                if (tbDoAtomicCompaction)
                {
                    bInactiveThread = true;
                }
                else
                {
                    d_outMask[maskOffset] = OBJDET_MASK_ELEMENT_INVALID_32U;
                    return;
                }
            }

            outMaskVal = (y_offs << 16) | x_offs;
        }
    }

    NcvBool bPass = true;

    if (!tbDoAtomicCompaction || tbDoAtomicCompaction)
    {
        Ncv32f pixelStdDev = 0.0f;

        if (!bInactiveThread)
            pixelStdDev = d_weights[y_offs * weightsStride + x_offs];

        for (Ncv32u iStage = startStageInc; iStage < endStageExc; iStage++)
        {
            Ncv32f curStageSum = 0.0f;

            HaarStage64 curStage = getStage(iStage, d_Stages);
            Ncv32u numRootNodesInStage = curStage.getNumClassifierRootNodes();
            Ncv32u curRootNodeOffset = curStage.getStartClassifierRootNodeOffset();
            Ncv32f stageThreshold = curStage.getStageThreshold();

            while (numRootNodesInStage--)
            {
                NcvBool bMoreNodesToTraverse = true;
                Ncv32u iNode = curRootNodeOffset;

                if (bPass && !bInactiveThread)
                {
                    while (bMoreNodesToTraverse)
                    {
                        HaarClassifierNode128 curNode = getClassifierNode<tbCacheTextureCascade>(texHaarClassifierNodes, iNode, d_ClassifierNodes);
                        HaarFeatureDescriptor32 featuresDesc = curNode.getFeatureDesc();
                        Ncv32u curNodeFeaturesNum = featuresDesc.getNumFeatures();
                        Ncv32u iFeature = featuresDesc.getFeaturesOffset();

                        Ncv32f curNodeVal = 0.0f;

                        for (Ncv32u iRect=0; iRect<curNodeFeaturesNum; iRect++)
                        {
                            Ncv32f rectWeight;
                            Ncv32u rectX, rectY, rectWidth, rectHeight;
                            getFeature<tbCacheTextureCascade> (texHaarFeatures, iFeature + iRect, d_Features, &rectWeight, &rectX, &rectY, &rectWidth, &rectHeight);

                            Ncv32u iioffsTL = (y_offs + rectY) * IImgStride + (x_offs + rectX);
                            Ncv32u iioffsTR = iioffsTL + rectWidth;
                            Ncv32u iioffsBL = iioffsTL + rectHeight * IImgStride;
                            Ncv32u iioffsBR = iioffsBL + rectWidth;

                            Ncv32u rectSum = getElemIImg<tbCacheTextureIImg>(texImg, iioffsBR, d_IImg) -
                                             getElemIImg<tbCacheTextureIImg>(texImg, iioffsBL, d_IImg) +
                                             getElemIImg<tbCacheTextureIImg>(texImg, iioffsTL, d_IImg) -
                                             getElemIImg<tbCacheTextureIImg>(texImg, iioffsTR, d_IImg);

    #if defined CPU_FP_COMPLIANCE || defined DISABLE_MAD_SELECTIVELY
                        curNodeVal += __fmul_rn((Ncv32f)rectSum, rectWeight);
    #else
                        curNodeVal += (Ncv32f)rectSum * rectWeight;
    #endif
                        }

                        HaarClassifierNodeDescriptor32 nodeLeft = curNode.getLeftNodeDesc();
                        HaarClassifierNodeDescriptor32 nodeRight = curNode.getRightNodeDesc();
                        Ncv32f nodeThreshold = curNode.getThreshold();

                        HaarClassifierNodeDescriptor32 nextNodeDescriptor;
                        NcvBool nextNodeIsLeaf;

                        if (curNodeVal < scaleArea * pixelStdDev * nodeThreshold)
                        {
                            nextNodeDescriptor = nodeLeft;
                            nextNodeIsLeaf = featuresDesc.isLeftNodeLeaf();
                        }
                        else
                        {
                            nextNodeDescriptor = nodeRight;
                            nextNodeIsLeaf = featuresDesc.isRightNodeLeaf();
                        }

                        if (nextNodeIsLeaf)
                        {
                            Ncv32f tmpLeafValue = nextNodeDescriptor.getLeafValue();
                            curStageSum += tmpLeafValue;
                            bMoreNodesToTraverse = false;
                        }
                        else
                        {
                            iNode = nextNodeDescriptor.getNextNodeOffset();
                        }
                    }
                }

                __syncthreads();
                curRootNodeOffset++;
            }

            if (curStageSum < stageThreshold)
            {
                bPass = false;
                outMaskVal = OBJDET_MASK_ELEMENT_INVALID_32U;
            }
        }
    }

    __syncthreads();

    if (!tbDoAtomicCompaction)
    {
        if (!tbReadPixelIndexFromVector ||
            (tbReadPixelIndexFromVector && (!bPass || d_inMask != d_outMask)))
        {
            d_outMask[maskOffset] = outMaskVal;
        }
    }
    else
    {
        compactBlockWriteOutAnchorParallel(bPass && !bInactiveThread,
                                           outMaskVal,
                                           d_outMask);
    }
}


template <NcvBool tbCacheTextureIImg, NcvBool tbCacheTextureCascade, NcvBool tbDoAtomicCompaction>
__global__ void applyHaarClassifierClassifierParallel(cv::cudev::TexturePtr<Ncv32u> texImg, cv::cudev::TexturePtr<uint2> texHaarFeatures, cv::cudev::TexturePtr<uint4> texHaarClassifierNodes, Ncv32u *d_IImg,
    Ncv32u IImgStride, Ncv32f *d_weights, Ncv32u weightsStride, HaarFeature64 *d_Features, HaarClassifierNode128 *d_ClassifierNodes, HaarStage64 *d_Stages, Ncv32u *d_inMask, Ncv32u *d_outMask,
    Ncv32u mask1Dlen, Ncv32u mask2Dstride, NcvSize32u anchorsRoi, Ncv32u startStageInc, Ncv32u endStageExc, Ncv32f scaleArea)
{
    Ncv32u maskOffset = MAX_GRID_DIM * blockIdx.y + blockIdx.x;

    if (maskOffset >= mask1Dlen)
    {
        return;
    }

    Ncv32u outMaskVal = d_inMask[maskOffset];
    Ncv32u y_offs = outMaskVal >> 16;
    Ncv32u x_offs = outMaskVal & 0xFFFF;

    Ncv32f pixelStdDev = d_weights[y_offs * weightsStride + x_offs];
    NcvBool bPass = true;

    for (Ncv32u iStage = startStageInc; iStage<endStageExc; iStage++)
    {
        //this variable is subject to reduction
        Ncv32f curStageSum = 0.0f;

        HaarStage64 curStage = getStage(iStage, d_Stages);
        Ncv32s numRootNodesInStage = curStage.getNumClassifierRootNodes();
        Ncv32u curRootNodeOffset = curStage.getStartClassifierRootNodeOffset() + threadIdx.x;
        Ncv32f stageThreshold = curStage.getStageThreshold();

        Ncv32u numRootChunks = (numRootNodesInStage + NUM_THREADS_CLASSIFIERPARALLEL - 1) >> NUM_THREADS_CLASSIFIERPARALLEL_LOG2;

        for (Ncv32u chunkId=0; chunkId<numRootChunks; chunkId++)
        {
            NcvBool bMoreNodesToTraverse = true;

            if (chunkId * NUM_THREADS_CLASSIFIERPARALLEL + threadIdx.x < numRootNodesInStage)
            {
                Ncv32u iNode = curRootNodeOffset;

                while (bMoreNodesToTraverse)
                {
                    HaarClassifierNode128 curNode = getClassifierNode<tbCacheTextureCascade>(texHaarClassifierNodes, iNode, d_ClassifierNodes);
                    HaarFeatureDescriptor32 featuresDesc = curNode.getFeatureDesc();
                    Ncv32u curNodeFeaturesNum = featuresDesc.getNumFeatures();
                    Ncv32u iFeature = featuresDesc.getFeaturesOffset();

                    Ncv32f curNodeVal = 0.0f;
                    //TODO: fetch into shmem if size suffices. Shmem can be shared with reduce
                    for (Ncv32u iRect=0; iRect<curNodeFeaturesNum; iRect++)
                    {
                        Ncv32f rectWeight;
                        Ncv32u rectX, rectY, rectWidth, rectHeight;
                        getFeature<tbCacheTextureCascade> (texHaarFeatures, iFeature + iRect, d_Features, &rectWeight, &rectX, &rectY, &rectWidth, &rectHeight);

                        Ncv32u iioffsTL = (y_offs + rectY) * IImgStride + (x_offs + rectX);
                        Ncv32u iioffsTR = iioffsTL + rectWidth;
                        Ncv32u iioffsBL = iioffsTL + rectHeight * IImgStride;
                        Ncv32u iioffsBR = iioffsBL + rectWidth;

                        Ncv32u rectSum = getElemIImg<tbCacheTextureIImg>(texImg, iioffsBR, d_IImg) -
                                         getElemIImg<tbCacheTextureIImg>(texImg, iioffsBL, d_IImg) +
                                         getElemIImg<tbCacheTextureIImg>(texImg, iioffsTL, d_IImg) -
                                         getElemIImg<tbCacheTextureIImg>(texImg, iioffsTR, d_IImg);

#if defined CPU_FP_COMPLIANCE || defined DISABLE_MAD_SELECTIVELY
                        curNodeVal += __fmul_rn((Ncv32f)rectSum, rectWeight);
#else
                        curNodeVal += (Ncv32f)rectSum * rectWeight;
#endif
                    }

                    HaarClassifierNodeDescriptor32 nodeLeft = curNode.getLeftNodeDesc();
                    HaarClassifierNodeDescriptor32 nodeRight = curNode.getRightNodeDesc();
                    Ncv32f nodeThreshold = curNode.getThreshold();

                    HaarClassifierNodeDescriptor32 nextNodeDescriptor;
                    NcvBool nextNodeIsLeaf;

                    if (curNodeVal < scaleArea * pixelStdDev * nodeThreshold)
                    {
                        nextNodeDescriptor = nodeLeft;
                        nextNodeIsLeaf = featuresDesc.isLeftNodeLeaf();
                    }
                    else
                    {
                        nextNodeDescriptor = nodeRight;
                        nextNodeIsLeaf = featuresDesc.isRightNodeLeaf();
                    }

                    if (nextNodeIsLeaf)
                    {
                        Ncv32f tmpLeafValue = nextNodeDescriptor.getLeafValue();
                        curStageSum += tmpLeafValue;
                        bMoreNodesToTraverse = false;
                    }
                    else
                    {
                        iNode = nextNodeDescriptor.getNextNodeOffset();
                    }
                }
            }
            __syncthreads();

            curRootNodeOffset += NUM_THREADS_CLASSIFIERPARALLEL;
        }

        Ncv32f finalStageSum = subReduce<Ncv32f, functorAddValues<Ncv32f>, NUM_THREADS_CLASSIFIERPARALLEL>(curStageSum);

        if (finalStageSum < stageThreshold)
        {
            bPass = false;
            outMaskVal = OBJDET_MASK_ELEMENT_INVALID_32U;
            break;
        }
    }

    if (!tbDoAtomicCompaction)
    {
        if (!bPass || d_inMask != d_outMask)
        {
            if (!threadIdx.x)
            {
                d_outMask[maskOffset] = outMaskVal;
            }
        }
    }
    else
    {
#if __CUDA_ARCH__ && __CUDA_ARCH__ >= 110
        if (bPass && !threadIdx.x)
        {
            Ncv32u outMaskOffset = atomicAdd(&d_outMaskPosition, 1);
            d_outMask[outMaskOffset] = outMaskVal;
        }
#endif
    }
}


template <NcvBool tbMaskByInmask,
          NcvBool tbDoAtomicCompaction>
__global__ void initializeMaskVector(Ncv32u *d_inMask, Ncv32u *d_outMask,
                                     Ncv32u mask1Dlen, Ncv32u mask2Dstride,
                                     NcvSize32u anchorsRoi, Ncv32u step)
{
    Ncv32u y_offs = blockIdx.y;
    Ncv32u x_offs = blockIdx.x * NUM_THREADS_ANCHORSPARALLEL + threadIdx.x;
    Ncv32u outMaskOffset = y_offs * gridDim.x * blockDim.x + x_offs;

    Ncv32u y_offs_upsc = step * y_offs;
    Ncv32u x_offs_upsc = step * x_offs;
    Ncv32u inMaskOffset = y_offs_upsc * mask2Dstride + x_offs_upsc;

    Ncv32u outElem = OBJDET_MASK_ELEMENT_INVALID_32U;

    if (x_offs_upsc < anchorsRoi.width &&
        (!tbMaskByInmask || d_inMask[inMaskOffset] != OBJDET_MASK_ELEMENT_INVALID_32U))
    {
        outElem = (y_offs_upsc << 16) | x_offs_upsc;
    }

    if (!tbDoAtomicCompaction)
    {
        d_outMask[outMaskOffset] = outElem;
    }
    else
    {
        compactBlockWriteOutAnchorParallel(outElem != OBJDET_MASK_ELEMENT_INVALID_32U,
                                           outElem,
                                           d_outMask);
    }
}


struct applyHaarClassifierAnchorParallelFunctor
{
    dim3 gridConf, blockConf;
    hipStream_t cuStream;
    cv::cudev::TexturePtr<Ncv32u> texImg;
    cv::cudev::TexturePtr<uint2> texHaarFeatures;
    cv::cudev::TexturePtr<uint4> texHaarClassifierNodes;
    Ncv32u *d_IImg;
    Ncv32u IImgStride;
    Ncv32f *d_weights;
    Ncv32u weightsStride;
    HaarFeature64 *d_Features;
    HaarClassifierNode128 *d_ClassifierNodes;
    HaarStage64 *d_Stages;
    Ncv32u *d_inMask;
    Ncv32u *d_outMask;
    Ncv32u mask1Dlen;
    Ncv32u mask2Dstride;
    NcvSize32u anchorsRoi;
    Ncv32u startStageInc;
    Ncv32u endStageExc;
    Ncv32f scaleArea;

    //Arguments are passed through the constructor
    applyHaarClassifierAnchorParallelFunctor(cv::cudev::TexturePtr<Ncv32u> texImg_, cv::cudev::TexturePtr<uint2> texHaarFeatures_, cv::cudev::TexturePtr<uint4> texHaarClassifierNodes_, dim3 _gridConf,
        dim3 _blockConf, hipStream_t _cuStream, Ncv32u *_d_IImg, Ncv32u _IImgStride, Ncv32f *_d_weights, Ncv32u _weightsStride, HaarFeature64 *_d_Features, HaarClassifierNode128 *_d_ClassifierNodes,
        HaarStage64 *_d_Stages, Ncv32u *_d_inMask, Ncv32u *_d_outMask, Ncv32u _mask1Dlen, Ncv32u _mask2Dstride, NcvSize32u _anchorsRoi, Ncv32u _startStageInc, Ncv32u _endStageExc, Ncv32f _scaleArea) :
        gridConf(_gridConf), blockConf(_blockConf), cuStream(_cuStream), texImg(texImg_), texHaarFeatures(texHaarFeatures_), texHaarClassifierNodes(texHaarClassifierNodes_), d_IImg(_d_IImg), IImgStride(_IImgStride),
        d_weights(_d_weights), weightsStride(_weightsStride), d_Features(_d_Features), d_ClassifierNodes(_d_ClassifierNodes), d_Stages(_d_Stages), d_inMask(_d_inMask), d_outMask(_d_outMask), mask1Dlen(_mask1Dlen),
        mask2Dstride(_mask2Dstride), anchorsRoi(_anchorsRoi), startStageInc(_startStageInc), endStageExc(_endStageExc), scaleArea(_scaleArea)
    {}

    template<class TList>
    void call(TList tl)
    {
        CV_UNUSED(tl);
        applyHaarClassifierAnchorParallel <
            Loki::TL::TypeAt<TList, 0>::Result::value,
            Loki::TL::TypeAt<TList, 1>::Result::value,
            Loki::TL::TypeAt<TList, 2>::Result::value,
            Loki::TL::TypeAt<TList, 3>::Result::value,
            Loki::TL::TypeAt<TList, 4>::Result::value >
            <<<gridConf, blockConf, 0, cuStream>>> (texImg, texHaarFeatures, texHaarClassifierNodes, d_IImg, IImgStride, d_weights, weightsStride, d_Features, d_ClassifierNodes, d_Stages, d_inMask,
                d_outMask, mask1Dlen, mask2Dstride, anchorsRoi, startStageInc, endStageExc, scaleArea);
    }
};


void applyHaarClassifierAnchorParallelDynTemplate(NcvBool tbInitMaskPositively, NcvBool tbCacheTextureIImg, NcvBool tbCacheTextureCascade, NcvBool tbReadPixelIndexFromVector, NcvBool tbDoAtomicCompaction,
    dim3 gridConf, dim3 blockConf, hipStream_t cuStream, cv::cudev::TexturePtr<Ncv32u> texImg, cv::cudev::TexturePtr<uint2> texHaarFeatures, cv::cudev::TexturePtr<uint4> texHaarClassifierNodes, Ncv32u *d_IImg,
    Ncv32u IImgStride, Ncv32f *d_weights, Ncv32u weightsStride, HaarFeature64 *d_Features, HaarClassifierNode128 *d_ClassifierNodes, HaarStage64 *d_Stages, Ncv32u *d_inMask, Ncv32u *d_outMask,
    Ncv32u mask1Dlen, Ncv32u mask2Dstride, NcvSize32u anchorsRoi, Ncv32u startStageInc, Ncv32u endStageExc, Ncv32f scaleArea)
{
    applyHaarClassifierAnchorParallelFunctor functor(texImg, texHaarFeatures, texHaarClassifierNodes, gridConf, blockConf, cuStream, d_IImg, IImgStride, d_weights, weightsStride, d_Features, d_ClassifierNodes, d_Stages,
                                                     d_inMask, d_outMask, mask1Dlen, mask2Dstride, anchorsRoi, startStageInc, endStageExc, scaleArea);

    //Second parameter is the number of "dynamic" template parameters
    NCVRuntimeTemplateBool::KernelCaller<Loki::NullType, 5, applyHaarClassifierAnchorParallelFunctor>
        ::call( &functor,
                tbInitMaskPositively,
                tbCacheTextureIImg,
                tbCacheTextureCascade,
                tbReadPixelIndexFromVector,
                tbDoAtomicCompaction);
}


struct applyHaarClassifierClassifierParallelFunctor
{
    dim3 gridConf, blockConf;
    hipStream_t cuStream;
    cv::cudev::TexturePtr<Ncv32u> texImg;
    cv::cudev::TexturePtr<uint2> texHaarFeatures;
    cv::cudev::TexturePtr<uint4> texHaarClassifierNodes;
    Ncv32u *d_IImg;
    Ncv32u IImgStride;
    Ncv32f *d_weights;
    Ncv32u weightsStride;
    HaarFeature64 *d_Features;
    HaarClassifierNode128 *d_ClassifierNodes;
    HaarStage64 *d_Stages;
    Ncv32u *d_inMask;
    Ncv32u *d_outMask;
    Ncv32u mask1Dlen;
    Ncv32u mask2Dstride;
    NcvSize32u anchorsRoi;
    Ncv32u startStageInc;
    Ncv32u endStageExc;
    Ncv32f scaleArea;

    //Arguments are passed through the constructor
    applyHaarClassifierClassifierParallelFunctor(dim3 _gridConf, dim3 _blockConf, hipStream_t _cuStream, cv::cudev::TexturePtr<Ncv32u> texImg_, cv::cudev::TexturePtr<uint2> texHaarFeatures_,
        cv::cudev::TexturePtr<uint4> texHaarClassifierNodes_, Ncv32u *_d_IImg, Ncv32u _IImgStride, Ncv32f *_d_weights, Ncv32u _weightsStride, HaarFeature64 *_d_Features,
        HaarClassifierNode128 *_d_ClassifierNodes, HaarStage64 *_d_Stages, Ncv32u *_d_inMask, Ncv32u *_d_outMask, Ncv32u _mask1Dlen, Ncv32u _mask2Dstride, NcvSize32u _anchorsRoi,
        Ncv32u _startStageInc, Ncv32u _endStageExc, Ncv32f _scaleArea) : gridConf(_gridConf), blockConf(_blockConf), cuStream(_cuStream), texImg(texImg_), texHaarFeatures(texHaarFeatures_),
        texHaarClassifierNodes(texHaarClassifierNodes_), d_IImg(_d_IImg), IImgStride(_IImgStride), d_weights(_d_weights), weightsStride(_weightsStride), d_Features(_d_Features),
        d_ClassifierNodes(_d_ClassifierNodes), d_Stages(_d_Stages), d_inMask(_d_inMask), d_outMask(_d_outMask), mask1Dlen(_mask1Dlen), mask2Dstride(_mask2Dstride), anchorsRoi(_anchorsRoi),
        startStageInc(_startStageInc), endStageExc(_endStageExc), scaleArea(_scaleArea)
    {}

    template<class TList>
    void call(TList tl)
    {
        CV_UNUSED(tl);
        applyHaarClassifierClassifierParallel <
            Loki::TL::TypeAt<TList, 0>::Result::value,
            Loki::TL::TypeAt<TList, 1>::Result::value,
            Loki::TL::TypeAt<TList, 2>::Result::value >
            <<<gridConf, blockConf, 0, cuStream>>> (texImg, texHaarFeatures, texHaarClassifierNodes, d_IImg, IImgStride, d_weights, weightsStride, d_Features, d_ClassifierNodes, d_Stages, d_inMask,
                d_outMask, mask1Dlen, mask2Dstride, anchorsRoi, startStageInc, endStageExc, scaleArea);
    }
};


void applyHaarClassifierClassifierParallelDynTemplate(NcvBool tbCacheTextureIImg, NcvBool tbCacheTextureCascade, NcvBool tbDoAtomicCompaction, dim3 gridConf, dim3 blockConf, hipStream_t cuStream,
    cv::cudev::TexturePtr<Ncv32u> texImg, cv::cudev::TexturePtr<uint2> texHaarFeatures, cv::cudev::TexturePtr<uint4> texHaarClassifierNodes, Ncv32u *d_IImg, Ncv32u IImgStride, Ncv32f *d_weights,
    Ncv32u weightsStride, HaarFeature64 *d_Features, HaarClassifierNode128 *d_ClassifierNodes, HaarStage64 *d_Stages, Ncv32u *d_inMask, Ncv32u *d_outMask, Ncv32u mask1Dlen, Ncv32u mask2Dstride,
    NcvSize32u anchorsRoi, Ncv32u startStageInc, Ncv32u endStageExc, Ncv32f scaleArea)
{
    applyHaarClassifierClassifierParallelFunctor functor(gridConf, blockConf, cuStream, texImg, texHaarFeatures, texHaarClassifierNodes, d_IImg, IImgStride, d_weights, weightsStride, d_Features,
        d_ClassifierNodes, d_Stages, d_inMask, d_outMask, mask1Dlen, mask2Dstride, anchorsRoi, startStageInc, endStageExc, scaleArea);

    //Second parameter is the number of "dynamic" template parameters
    NCVRuntimeTemplateBool::KernelCaller<Loki::NullType, 3, applyHaarClassifierClassifierParallelFunctor>
        ::call( &functor,
                tbCacheTextureIImg,
                tbCacheTextureCascade,
                tbDoAtomicCompaction);
}


struct initializeMaskVectorFunctor
{
    dim3 gridConf, blockConf;
    hipStream_t cuStream;

    //Kernel arguments are stored as members;
    Ncv32u *d_inMask;
    Ncv32u *d_outMask;
    Ncv32u mask1Dlen;
    Ncv32u mask2Dstride;
    NcvSize32u anchorsRoi;
    Ncv32u step;

    //Arguments are passed through the constructor
    initializeMaskVectorFunctor(dim3 _gridConf, dim3 _blockConf, hipStream_t _cuStream,
                                Ncv32u *_d_inMask, Ncv32u *_d_outMask,
                                Ncv32u _mask1Dlen, Ncv32u _mask2Dstride,
                                NcvSize32u _anchorsRoi, Ncv32u _step) :
    gridConf(_gridConf),
    blockConf(_blockConf),
    cuStream(_cuStream),
    d_inMask(_d_inMask),
    d_outMask(_d_outMask),
    mask1Dlen(_mask1Dlen),
    mask2Dstride(_mask2Dstride),
    anchorsRoi(_anchorsRoi),
    step(_step)
    {}

    template<class TList>
    void call(TList tl)
    {
        CV_UNUSED(tl);
        initializeMaskVector <
            Loki::TL::TypeAt<TList, 0>::Result::value,
            Loki::TL::TypeAt<TList, 1>::Result::value >
            <<<gridConf, blockConf, 0, cuStream>>>
            (d_inMask, d_outMask,
             mask1Dlen, mask2Dstride,
             anchorsRoi, step);
    }
};


void initializeMaskVectorDynTemplate(NcvBool tbMaskByInmask,
                                     NcvBool tbDoAtomicCompaction,

                                     dim3 gridConf, dim3 blockConf, hipStream_t cuStream,

                                     Ncv32u *d_inMask, Ncv32u *d_outMask,
                                     Ncv32u mask1Dlen, Ncv32u mask2Dstride,
                                     NcvSize32u anchorsRoi, Ncv32u step)
{
    initializeMaskVectorFunctor functor(gridConf, blockConf, cuStream,
                                        d_inMask, d_outMask,
                                        mask1Dlen, mask2Dstride,
                                        anchorsRoi, step);

    //Second parameter is the number of "dynamic" template parameters
    NCVRuntimeTemplateBool::KernelCaller<Loki::NullType, 2, initializeMaskVectorFunctor>
        ::call( &functor,
                tbMaskByInmask,
                tbDoAtomicCompaction);
}


Ncv32u getStageNumWithNotLessThanNclassifiers(Ncv32u N, HaarClassifierCascadeDescriptor &haar,
                                              NCVVector<HaarStage64> &h_HaarStages)
{
    Ncv32u i = 0;
    for (; i<haar.NumStages; i++)
    {
        if (h_HaarStages.ptr()[i].getNumClassifierRootNodes() >= N)
        {
            break;
        }
    }
    return i;
}


NCVStatus ncvApplyHaarClassifierCascade_device(NCVMatrix<Ncv32u> &integral,
                                               NCVMatrix<Ncv32f> &d_weights,
                                               NCVMatrixAlloc<Ncv32u> &d_pixelMask,
                                               Ncv32u &numDetections,
                                               HaarClassifierCascadeDescriptor &haar,
                                               NCVVector<HaarStage64> &h_HaarStages,
                                               NCVVector<HaarStage64> &d_HaarStages,
                                               NCVVector<HaarClassifierNode128> &d_HaarNodes,
                                               NCVVector<HaarFeature64> &d_HaarFeatures,
                                               NcvBool bMaskElements,
                                               NcvSize32u anchorsRoi,
                                               Ncv32u pixelStep,
                                               Ncv32f scaleArea,
                                               INCVMemAllocator &gpuAllocator,
                                               INCVMemAllocator &cpuAllocator,
                                               hipDeviceProp_t &devProp,
                                               hipStream_t cuStream)
{
    ncvAssertReturn(integral.memType() == d_weights.memType()&&
                    integral.memType() == d_pixelMask.memType() &&
                    integral.memType() == gpuAllocator.memType() &&
                   (integral.memType() == NCVMemoryTypeDevice ||
                    integral.memType() == NCVMemoryTypeNone), NCV_MEM_RESIDENCE_ERROR);

    ncvAssertReturn(d_HaarStages.memType() == d_HaarNodes.memType() &&
                    d_HaarStages.memType() == d_HaarFeatures.memType() &&
                     (d_HaarStages.memType() == NCVMemoryTypeDevice ||
                      d_HaarStages.memType() == NCVMemoryTypeNone), NCV_MEM_RESIDENCE_ERROR);

    ncvAssertReturn(h_HaarStages.memType() != NCVMemoryTypeDevice, NCV_MEM_RESIDENCE_ERROR);

    ncvAssertReturn(gpuAllocator.isInitialized() && cpuAllocator.isInitialized(), NCV_ALLOCATOR_NOT_INITIALIZED);

    ncvAssertReturn((integral.ptr() != NULL && d_weights.ptr() != NULL && d_pixelMask.ptr() != NULL &&
                     h_HaarStages.ptr() != NULL && d_HaarStages.ptr() != NULL && d_HaarNodes.ptr() != NULL &&
                     d_HaarFeatures.ptr() != NULL) || gpuAllocator.isCounting(), NCV_NULL_PTR);

    ncvAssertReturn(anchorsRoi.width > 0 && anchorsRoi.height > 0 &&
                    d_pixelMask.width() >= anchorsRoi.width && d_pixelMask.height() >= anchorsRoi.height &&
                    d_weights.width() >= anchorsRoi.width && d_weights.height() >= anchorsRoi.height &&
                    integral.width() >= anchorsRoi.width + haar.ClassifierSize.width &&
                    integral.height() >= anchorsRoi.height + haar.ClassifierSize.height, NCV_DIMENSIONS_INVALID);

    ncvAssertReturn(scaleArea > 0, NCV_INVALID_SCALE);

    ncvAssertReturn(d_HaarStages.length() >= haar.NumStages &&
                    d_HaarNodes.length() >= haar.NumClassifierTotalNodes &&
                    d_HaarFeatures.length() >= haar.NumFeatures &&
                    d_HaarStages.length() == h_HaarStages.length() &&
                    haar.NumClassifierRootNodes <= haar.NumClassifierTotalNodes, NCV_DIMENSIONS_INVALID);

    ncvAssertReturn(haar.bNeedsTiltedII == false || gpuAllocator.isCounting(), NCV_NOIMPL_HAAR_TILTED_FEATURES);

    ncvAssertReturn(pixelStep == 1 || pixelStep == 2, NCV_HAAR_INVALID_PIXEL_STEP);

    NCV_SET_SKIP_COND(gpuAllocator.isCounting());

#if defined _SELF_TEST_

    NCVStatus ncvStat;

    NCVMatrixAlloc<Ncv32u> h_integralImage(cpuAllocator, integral.width, integral.height, integral.pitch);
    ncvAssertReturn(h_integralImage.isMemAllocated(), NCV_ALLOCATOR_BAD_ALLOC);
    NCVMatrixAlloc<Ncv32f> h_weights(cpuAllocator, d_weights.width, d_weights.height, d_weights.pitch);
    ncvAssertReturn(h_weights.isMemAllocated(), NCV_ALLOCATOR_BAD_ALLOC);
    NCVMatrixAlloc<Ncv32u> h_pixelMask(cpuAllocator, d_pixelMask.width, d_pixelMask.height, d_pixelMask.pitch);
    ncvAssertReturn(h_pixelMask.isMemAllocated(), NCV_ALLOCATOR_BAD_ALLOC);
    NCVVectorAlloc<HaarClassifierNode128> h_HaarNodes(cpuAllocator, d_HaarNodes.length);
    ncvAssertReturn(h_HaarNodes.isMemAllocated(), NCV_ALLOCATOR_BAD_ALLOC);
    NCVVectorAlloc<HaarFeature64> h_HaarFeatures(cpuAllocator, d_HaarFeatures.length);
    ncvAssertReturn(h_HaarFeatures.isMemAllocated(), NCV_ALLOCATOR_BAD_ALLOC);

    NCVMatrixAlloc<Ncv32u> h_pixelMask_d(cpuAllocator, d_pixelMask.width, d_pixelMask.height, d_pixelMask.pitch);
    ncvAssertReturn(h_pixelMask_d.isMemAllocated(), NCV_ALLOCATOR_BAD_ALLOC);

    NCV_SKIP_COND_BEGIN

    ncvStat = d_pixelMask.copySolid(h_pixelMask, 0);
    ncvAssertReturnNcvStat(ncvStat);
    ncvStat = integral.copySolid(h_integralImage, 0);
    ncvAssertReturnNcvStat(ncvStat);
    ncvStat = d_weights.copySolid(h_weights, 0);
    ncvAssertReturnNcvStat(ncvStat);
    ncvStat = d_HaarNodes.copySolid(h_HaarNodes, 0);
    ncvAssertReturnNcvStat(ncvStat);
    ncvStat = d_HaarFeatures.copySolid(h_HaarFeatures, 0);
    ncvAssertReturnNcvStat(ncvStat);
    ncvAssertCUDAReturn(hipStreamSynchronize(0), NCV_CUDA_ERROR);

    for (Ncv32u i=0; i<(Ncv32u)anchorsRoi.height; i++)
    {
        for (Ncv32u j=0; j<d_pixelMask.stride(); j++)
        {
            if ((i%pixelStep==0) && (j%pixelStep==0) && (j<(Ncv32u)anchorsRoi.width))
            {
                if (!bMaskElements || h_pixelMask.ptr[i*d_pixelMask.stride()+j] != OBJDET_MASK_ELEMENT_INVALID_32U)
                {
                    h_pixelMask.ptr[i*d_pixelMask.stride()+j] = (i << 16) | j;
                }
            }
            else
            {
                h_pixelMask.ptr[i*d_pixelMask.stride()+j] = OBJDET_MASK_ELEMENT_INVALID_32U;
            }
        }
    }

    NCV_SKIP_COND_END

#endif

    NCVVectorReuse<Ncv32u> d_vecPixelMask(d_pixelMask.getSegment(), anchorsRoi.height * d_pixelMask.stride());
    ncvAssertReturn(d_vecPixelMask.isMemReused(), NCV_ALLOCATOR_BAD_REUSE);

    NCVVectorAlloc<Ncv32u> d_vecPixelMaskTmp(gpuAllocator, static_cast<Ncv32u>(d_vecPixelMask.length()));
    ncvAssertReturn(d_vecPixelMaskTmp.isMemAllocated(), NCV_ALLOCATOR_BAD_ALLOC);

    NCVVectorAlloc<Ncv32u> hp_pool32u(cpuAllocator, 2);
    ncvAssertReturn(hp_pool32u.isMemAllocated(), NCV_ALLOCATOR_BAD_ALLOC);
    Ncv32u *hp_zero = &hp_pool32u.ptr()[0];
    Ncv32u *hp_numDet = &hp_pool32u.ptr()[1];

    NCV_SKIP_COND_BEGIN
    *hp_zero = 0;
    *hp_numDet = 0;
    NCV_SKIP_COND_END

    Ncv32f scaleAreaPixels = scaleArea * ((haar.ClassifierSize.width - 2*HAAR_STDDEV_BORDER) *
                                          (haar.ClassifierSize.height - 2*HAAR_STDDEV_BORDER));

    NcvBool bTexCacheCascade = devProp.major < 2;
    NcvBool bTexCacheIImg = true; //this works better even on Fermi so far
    NcvBool bDoAtomicCompaction = devProp.major >= 2 || (devProp.major == 1 && devProp.minor >= 3);

    NCVVector<Ncv32u> *d_ptrNowData = &d_vecPixelMask;
    NCVVector<Ncv32u> *d_ptrNowTmp = &d_vecPixelMaskTmp;

    Ncv32u szNppCompactTmpBuf;
    nppsStCompactGetSize_32u(static_cast<Ncv32u>(d_vecPixelMask.length()), &szNppCompactTmpBuf, devProp);
    if (bDoAtomicCompaction)
    {
        szNppCompactTmpBuf = 0;
    }
    NCVVectorAlloc<Ncv8u> d_tmpBufCompact(gpuAllocator, szNppCompactTmpBuf);

    NCV_SKIP_COND_BEGIN

    cv::cudev::Texture<Ncv32u> texImg;
    if (bTexCacheIImg)
        texImg = cv::cudev::Texture<Ncv32u>((anchorsRoi.height + haar.ClassifierSize.height) * integral.pitch(), integral.ptr());

    cv::cudev::Texture<uint2> texHaarFeatures;
    cv::cudev::Texture<uint4> texHaarClassifierNodes;
    if (bTexCacheCascade) {
        texHaarFeatures = cv::cudev::Texture<uint2>(sizeof(HaarFeature64) * haar.NumFeatures, reinterpret_cast<uint2*>(d_HaarFeatures.ptr()));
        texHaarClassifierNodes = cv::cudev::Texture<uint4>(sizeof(HaarClassifierNode128) * haar.NumClassifierTotalNodes, reinterpret_cast<uint4*>(d_HaarNodes.ptr()));
    }

    Ncv32u stageStartAnchorParallel = 0;
    Ncv32u stageMiddleSwitch = getStageNumWithNotLessThanNclassifiers(NUM_THREADS_CLASSIFIERPARALLEL,
        haar, h_HaarStages);
    Ncv32u stageEndClassifierParallel = haar.NumStages;
    if (stageMiddleSwitch == 0)
    {
        stageMiddleSwitch = 1;
    }

    //create stages subdivision for pixel-parallel processing
    const Ncv32u compactEveryNstage = bDoAtomicCompaction ? 7 : 1;
    Ncv32u curStop = stageStartAnchorParallel;
    std::vector<Ncv32u> pixParallelStageStops;
    while (curStop < stageMiddleSwitch)
    {
        pixParallelStageStops.push_back(curStop);
        curStop += compactEveryNstage;
    }
    if (curStop > compactEveryNstage && curStop - stageMiddleSwitch > compactEveryNstage / 2)
    {
        pixParallelStageStops[pixParallelStageStops.size()-1] =
            (stageMiddleSwitch - (curStop - 2 * compactEveryNstage)) / 2;
    }
    pixParallelStageStops.push_back(stageMiddleSwitch);
    Ncv32u pixParallelStageStopsIndex = 0;

    if (pixelStep != 1 || bMaskElements)
    {
        if (bDoAtomicCompaction)
        {
            ncvAssertCUDAReturn(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_outMaskPosition), hp_zero, sizeof(Ncv32u),
                                                        0, hipMemcpyHostToDevice, cuStream), NCV_CUDA_ERROR);
            ncvAssertCUDAReturn(hipStreamSynchronize(cuStream), NCV_CUDA_ERROR);
        }

        dim3 gridInit((((anchorsRoi.width + pixelStep - 1) / pixelStep + NUM_THREADS_ANCHORSPARALLEL - 1) / NUM_THREADS_ANCHORSPARALLEL),
                        (anchorsRoi.height + pixelStep - 1) / pixelStep);
        dim3 blockInit(NUM_THREADS_ANCHORSPARALLEL);

        if (gridInit.x == 0 || gridInit.y == 0)
        {
            numDetections = 0;
            return NCV_SUCCESS;
        }

        initializeMaskVectorDynTemplate(bMaskElements,
                                        bDoAtomicCompaction,
                                        gridInit, blockInit, cuStream,
                                        d_ptrNowData->ptr(),
                                        d_ptrNowTmp->ptr(),
                                        static_cast<Ncv32u>(d_vecPixelMask.length()), d_pixelMask.stride(),
                                        anchorsRoi, pixelStep);
        ncvAssertCUDAReturn(hipGetLastError(), NCV_CUDA_ERROR);

        if (bDoAtomicCompaction)
        {
            ncvAssertCUDAReturn(hipStreamSynchronize(cuStream), NCV_CUDA_ERROR);
            ncvAssertCUDAReturn(hipMemcpyFromSymbolAsync(hp_numDet, HIP_SYMBOL(d_outMaskPosition), sizeof(Ncv32u),
                                                          0, hipMemcpyDeviceToHost, cuStream), NCV_CUDA_ERROR);
            ncvAssertCUDAReturn(hipStreamSynchronize(cuStream), NCV_CUDA_ERROR);
            swap(d_ptrNowData, d_ptrNowTmp);
        }
        else
        {
            NCVStatus nppSt;
            nppSt = nppsStCompact_32u(d_ptrNowTmp->ptr(), static_cast<Ncv32u>(d_vecPixelMask.length()),
                                      d_ptrNowData->ptr(), hp_numDet, OBJDET_MASK_ELEMENT_INVALID_32U,
                                      d_tmpBufCompact.ptr(), szNppCompactTmpBuf, devProp);
            ncvAssertReturn(nppSt == NPPST_SUCCESS, NCV_NPP_ERROR);
        }
        numDetections = *hp_numDet;
    }
    else
    {
        //
        // 1. Run the first pixel-input pixel-parallel classifier for few stages
        //

        if (bDoAtomicCompaction)
        {
            ncvAssertCUDAReturn(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_outMaskPosition), hp_zero, sizeof(Ncv32u),
                                                        0, hipMemcpyHostToDevice, cuStream), NCV_CUDA_ERROR);
            ncvAssertCUDAReturn(hipStreamSynchronize(cuStream), NCV_CUDA_ERROR);
        }

        dim3 grid1(((d_pixelMask.stride() + NUM_THREADS_ANCHORSPARALLEL - 1) / NUM_THREADS_ANCHORSPARALLEL),
                   anchorsRoi.height);
        dim3 block1(NUM_THREADS_ANCHORSPARALLEL);
        applyHaarClassifierAnchorParallelDynTemplate( true, bTexCacheIImg, bTexCacheCascade,  pixParallelStageStops[pixParallelStageStopsIndex] != 0, bDoAtomicCompaction, grid1, block1, cuStream,
            texImg, texHaarFeatures, texHaarClassifierNodes, integral.ptr(), integral.stride(), d_weights.ptr(), d_weights.stride(), d_HaarFeatures.ptr(), d_HaarNodes.ptr(), d_HaarStages.ptr(),
            d_ptrNowData->ptr(), bDoAtomicCompaction ? d_ptrNowTmp->ptr() : d_ptrNowData->ptr(), 0, d_pixelMask.stride(), anchorsRoi, pixParallelStageStops[pixParallelStageStopsIndex],
            pixParallelStageStops[pixParallelStageStopsIndex+1], scaleAreaPixels);
        ncvAssertCUDAReturn(hipGetLastError(), NCV_CUDA_ERROR);

        if (bDoAtomicCompaction)
        {
            ncvAssertCUDAReturn(hipStreamSynchronize(cuStream), NCV_CUDA_ERROR);
            ncvAssertCUDAReturn(hipMemcpyFromSymbolAsync(hp_numDet, HIP_SYMBOL(d_outMaskPosition), sizeof(Ncv32u),
                                                          0, hipMemcpyDeviceToHost, cuStream), NCV_CUDA_ERROR);
            ncvAssertCUDAReturn(hipStreamSynchronize(cuStream), NCV_CUDA_ERROR);
        }
        else
        {
            NCVStatus nppSt;
            nppSt = nppsStCompact_32u(d_ptrNowData->ptr(), static_cast<Ncv32u>(d_vecPixelMask.length()),
                                      d_ptrNowTmp->ptr(), hp_numDet, OBJDET_MASK_ELEMENT_INVALID_32U,
                                      d_tmpBufCompact.ptr(), szNppCompactTmpBuf, devProp);
            ncvAssertReturnNcvStat(nppSt);
        }

        swap(d_ptrNowData, d_ptrNowTmp);
        numDetections = *hp_numDet;

        pixParallelStageStopsIndex++;
    }

    //
    // 2. Run pixel-parallel stages
    //

    for (; pixParallelStageStopsIndex < pixParallelStageStops.size()-1; pixParallelStageStopsIndex++)
    {
        if (numDetections == 0)
        {
            break;
        }

        if (bDoAtomicCompaction)
        {
            ncvAssertCUDAReturn(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_outMaskPosition), hp_zero, sizeof(Ncv32u),
                                                        0, hipMemcpyHostToDevice, cuStream), NCV_CUDA_ERROR);
            ncvAssertCUDAReturn(hipStreamSynchronize(cuStream), NCV_CUDA_ERROR);
        }

        dim3 grid2((numDetections + NUM_THREADS_ANCHORSPARALLEL - 1) / NUM_THREADS_ANCHORSPARALLEL);
        if (numDetections > MAX_GRID_DIM)
        {
            grid2.x = MAX_GRID_DIM;
            grid2.y = (numDetections + MAX_GRID_DIM - 1) / MAX_GRID_DIM;
        }
        dim3 block2(NUM_THREADS_ANCHORSPARALLEL);

        applyHaarClassifierAnchorParallelDynTemplate( false, bTexCacheIImg, bTexCacheCascade, pixParallelStageStops[pixParallelStageStopsIndex] != 0 || pixelStep != 1 || bMaskElements, bDoAtomicCompaction,
            grid2, block2, cuStream, texImg, texHaarFeatures, texHaarClassifierNodes, integral.ptr(), integral.stride(), d_weights.ptr(), d_weights.stride(), d_HaarFeatures.ptr(), d_HaarNodes.ptr(),
            d_HaarStages.ptr(), d_ptrNowData->ptr(), bDoAtomicCompaction ? d_ptrNowTmp->ptr() : d_ptrNowData->ptr(), numDetections, d_pixelMask.stride(), anchorsRoi,
            pixParallelStageStops[pixParallelStageStopsIndex], pixParallelStageStops[pixParallelStageStopsIndex+1], scaleAreaPixels);
        ncvAssertCUDAReturn(hipGetLastError(), NCV_CUDA_ERROR);

        if (bDoAtomicCompaction)
        {
            ncvAssertCUDAReturn(hipStreamSynchronize(cuStream), NCV_CUDA_ERROR);
            ncvAssertCUDAReturn(hipMemcpyFromSymbolAsync(hp_numDet, HIP_SYMBOL(d_outMaskPosition), sizeof(Ncv32u),
                                                          0, hipMemcpyDeviceToHost, cuStream), NCV_CUDA_ERROR);
            ncvAssertCUDAReturn(hipStreamSynchronize(cuStream), NCV_CUDA_ERROR);
        }
        else
        {
            NCVStatus nppSt;
            nppSt = nppsStCompact_32u(d_ptrNowData->ptr(), numDetections,
                                      d_ptrNowTmp->ptr(), hp_numDet, OBJDET_MASK_ELEMENT_INVALID_32U,
                                      d_tmpBufCompact.ptr(), szNppCompactTmpBuf, devProp);
            ncvAssertReturnNcvStat(nppSt);
        }

        swap(d_ptrNowData, d_ptrNowTmp);
        numDetections = *hp_numDet;
    }

    //
    // 3. Run all left stages in one stage-parallel kernel
    //

    if (numDetections > 0 && stageMiddleSwitch < stageEndClassifierParallel)
    {
        if (bDoAtomicCompaction)
        {
            ncvAssertCUDAReturn(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_outMaskPosition), hp_zero, sizeof(Ncv32u),
                                                        0, hipMemcpyHostToDevice, cuStream), NCV_CUDA_ERROR);
            ncvAssertCUDAReturn(hipStreamSynchronize(cuStream), NCV_CUDA_ERROR);
        }

        dim3 grid3(numDetections);
        if (numDetections > MAX_GRID_DIM)
        {
            grid3.x = MAX_GRID_DIM;
            grid3.y = (numDetections + MAX_GRID_DIM - 1) / MAX_GRID_DIM;
        }
        dim3 block3(NUM_THREADS_CLASSIFIERPARALLEL);

        applyHaarClassifierClassifierParallelDynTemplate(bTexCacheIImg, bTexCacheCascade, bDoAtomicCompaction, grid3, block3, cuStream, texImg, texHaarFeatures, texHaarClassifierNodes, integral.ptr(), integral.stride(),
            d_weights.ptr(), d_weights.stride(), d_HaarFeatures.ptr(), d_HaarNodes.ptr(), d_HaarStages.ptr(), d_ptrNowData->ptr(), bDoAtomicCompaction ? d_ptrNowTmp->ptr() : d_ptrNowData->ptr(), numDetections,
            d_pixelMask.stride(), anchorsRoi, stageMiddleSwitch, stageEndClassifierParallel, scaleAreaPixels);
        ncvAssertCUDAReturn(hipGetLastError(), NCV_CUDA_ERROR);

        if (bDoAtomicCompaction)
        {
            ncvAssertCUDAReturn(hipStreamSynchronize(cuStream), NCV_CUDA_ERROR);
            ncvAssertCUDAReturn(hipMemcpyFromSymbolAsync(hp_numDet, HIP_SYMBOL(d_outMaskPosition), sizeof(Ncv32u),
                                                          0, hipMemcpyDeviceToHost, cuStream), NCV_CUDA_ERROR);
            ncvAssertCUDAReturn(hipStreamSynchronize(cuStream), NCV_CUDA_ERROR);
        }
        else
        {
            NCVStatus nppSt;
            nppSt = nppsStCompact_32u(d_ptrNowData->ptr(), numDetections,
                                      d_ptrNowTmp->ptr(), hp_numDet, OBJDET_MASK_ELEMENT_INVALID_32U,
                                      d_tmpBufCompact.ptr(), szNppCompactTmpBuf, devProp);
            ncvAssertReturnNcvStat(nppSt);
        }

        swap(d_ptrNowData, d_ptrNowTmp);
        numDetections = *hp_numDet;
    }

    if (d_ptrNowData != &d_vecPixelMask)
    {
        d_vecPixelMaskTmp.copySolid(d_vecPixelMask, cuStream);
        ncvAssertCUDAReturn(hipStreamSynchronize(cuStream), NCV_CUDA_ERROR);
    }

#if defined _SELF_TEST_

    ncvStat = d_pixelMask.copySolid(h_pixelMask_d, 0);
    ncvAssertReturnNcvStat(ncvStat);
    ncvAssertCUDAReturn(hipStreamSynchronize(cuStream), NCV_CUDA_ERROR);

    if (bDoAtomicCompaction)
    {
        std::sort(h_pixelMask_d.ptr, h_pixelMask_d.ptr + numDetections);
    }

    Ncv32u fpu_oldcw, fpu_cw;
    _controlfp_s(&fpu_cw, 0, 0);
    fpu_oldcw = fpu_cw;
    _controlfp_s(&fpu_cw, _PC_24, _MCW_PC);
    Ncv32u numDetGold;
    ncvStat = ncvApplyHaarClassifierCascade_host(h_integralImage, h_weights, h_pixelMask, numDetGold, haar,
                                                 h_HaarStages, h_HaarNodes, h_HaarFeatures,
                                                 bMaskElements, anchorsRoi, pixelStep, scaleArea);
    ncvAssertReturnNcvStat(ncvStat);
    _controlfp_s(&fpu_cw, fpu_oldcw, _MCW_PC);

    bool bPass = true;

    if (numDetGold != numDetections)
    {
        printf("NCVHaarClassifierCascade::applyHaarClassifierCascade numdetections don't match: cpu=%d, gpu=%d\n", numDetGold, numDetections);
        bPass = false;
    }
    else
    {
        for (Ncv32u i=0; i<std::max(numDetGold, numDetections) && bPass; i++)
        {
            if (h_pixelMask.ptr[i] != h_pixelMask_d.ptr[i])
            {
                printf("NCVHaarClassifierCascade::applyHaarClassifierCascade self test failed: i=%d, cpu=%d, gpu=%d\n", i, h_pixelMask.ptr[i], h_pixelMask_d.ptr[i]);
                bPass = false;
            }
        }
    }

    printf("NCVHaarClassifierCascade::applyHaarClassifierCascade %s\n", bPass?"PASSED":"FAILED");
#endif

    NCV_SKIP_COND_END

    return NCV_SUCCESS;
}


//==============================================================================
//
// HypothesesOperations file
//
//==============================================================================


const Ncv32u NUM_GROW_THREADS = 128;


__device__ __host__ NcvRect32u pixelToRect(Ncv32u pixel, Ncv32u width, Ncv32u height, Ncv32f scale)
{
    NcvRect32u res;
    res.x = (Ncv32u)(scale * (pixel & 0xFFFF));
    res.y = (Ncv32u)(scale * (pixel >> 16));
    res.width = (Ncv32u)(scale * width);
    res.height = (Ncv32u)(scale * height);
    return res;
}


__global__ void growDetectionsKernel(Ncv32u *pixelMask, Ncv32u numElements,
                                     NcvRect32u *hypotheses,
                                     Ncv32u rectWidth, Ncv32u rectHeight, Ncv32f curScale)
{
    Ncv32u blockId = blockIdx.y * 65535 + blockIdx.x;
    Ncv32u elemAddr = blockId * NUM_GROW_THREADS + threadIdx.x;
    if (elemAddr >= numElements)
    {
        return;
    }
    hypotheses[elemAddr] = pixelToRect(pixelMask[elemAddr], rectWidth, rectHeight, curScale);
}


NCVStatus ncvGrowDetectionsVector_device(NCVVector<Ncv32u> &pixelMask,
                                         Ncv32u numPixelMaskDetections,
                                         NCVVector<NcvRect32u> &hypotheses,
                                         Ncv32u &totalDetections,
                                         Ncv32u totalMaxDetections,
                                         Ncv32u rectWidth,
                                         Ncv32u rectHeight,
                                         Ncv32f curScale,
                                         hipStream_t cuStream)
{
    ncvAssertReturn(pixelMask.ptr() != NULL && hypotheses.ptr() != NULL, NCV_NULL_PTR);

    ncvAssertReturn(pixelMask.memType() == hypotheses.memType() &&
                    pixelMask.memType() == NCVMemoryTypeDevice, NCV_MEM_RESIDENCE_ERROR);

    ncvAssertReturn(rectWidth > 0 && rectHeight > 0 && curScale > 0, NCV_INVALID_ROI);

    ncvAssertReturn(curScale > 0, NCV_INVALID_SCALE);

    ncvAssertReturn(totalMaxDetections <= hypotheses.length() &&
                    numPixelMaskDetections <= pixelMask.length() &&
                    totalDetections <= totalMaxDetections, NCV_INCONSISTENT_INPUT);

    NCVStatus ncvStat = NCV_SUCCESS;
    Ncv32u numDetsToCopy = numPixelMaskDetections;

    if (numDetsToCopy == 0)
    {
        return ncvStat;
    }

    if (totalDetections + numPixelMaskDetections > totalMaxDetections)
    {
        ncvStat = NCV_WARNING_HAAR_DETECTIONS_VECTOR_OVERFLOW;
        numDetsToCopy = totalMaxDetections - totalDetections;
    }

    dim3 block(NUM_GROW_THREADS);
    dim3 grid((numDetsToCopy + NUM_GROW_THREADS - 1) / NUM_GROW_THREADS);
    if (grid.x > 65535)
    {
        grid.y = (grid.x + 65534) / 65535;
        grid.x = 65535;
    }
    growDetectionsKernel<<<grid, block, 0, cuStream>>>(pixelMask.ptr(), numDetsToCopy,
                                                       hypotheses.ptr() + totalDetections,
                                                       rectWidth, rectHeight, curScale);
    ncvAssertCUDAReturn(hipGetLastError(), NCV_CUDA_ERROR);

    totalDetections += numDetsToCopy;
    return ncvStat;
}


//==============================================================================
//
// Pipeline file
//
//==============================================================================


NCVStatus ncvDetectObjectsMultiScale_device(NCVMatrix<Ncv8u> &d_srcImg,
                                            NcvSize32u srcRoi,
                                            NCVVector<NcvRect32u> &d_dstRects,
                                            Ncv32u &dstNumRects,

                                            HaarClassifierCascadeDescriptor &haar,
                                            NCVVector<HaarStage64> &h_HaarStages,
                                            NCVVector<HaarStage64> &d_HaarStages,
                                            NCVVector<HaarClassifierNode128> &d_HaarNodes,
                                            NCVVector<HaarFeature64> &d_HaarFeatures,

                                            NcvSize32u minObjSize,
                                            Ncv32u minNeighbors,      //default 4
                                            Ncv32f scaleStep,         //default 1.2f
                                            Ncv32u pixelStep,         //default 1
                                            Ncv32u flags,             //default NCVPipeObjDet_Default

                                            INCVMemAllocator &gpuAllocator,
                                            INCVMemAllocator &cpuAllocator,
                                            hipDeviceProp_t &devProp,
                                            hipStream_t cuStream)
{
    ncvAssertReturn(d_srcImg.memType() == d_dstRects.memType() &&
                    d_srcImg.memType() == gpuAllocator.memType() &&
                     (d_srcImg.memType() == NCVMemoryTypeDevice ||
                      d_srcImg.memType() == NCVMemoryTypeNone), NCV_MEM_RESIDENCE_ERROR);

    ncvAssertReturn(d_HaarStages.memType() == d_HaarNodes.memType() &&
                    d_HaarStages.memType() == d_HaarFeatures.memType() &&
                     (d_HaarStages.memType() == NCVMemoryTypeDevice ||
                      d_HaarStages.memType() == NCVMemoryTypeNone), NCV_MEM_RESIDENCE_ERROR);

    ncvAssertReturn(h_HaarStages.memType() != NCVMemoryTypeDevice, NCV_MEM_RESIDENCE_ERROR);

    ncvAssertReturn(gpuAllocator.isInitialized() && cpuAllocator.isInitialized(), NCV_ALLOCATOR_NOT_INITIALIZED);

    ncvAssertReturn((d_srcImg.ptr() != NULL && d_dstRects.ptr() != NULL &&
                     h_HaarStages.ptr() != NULL && d_HaarStages.ptr() != NULL && d_HaarNodes.ptr() != NULL &&
                     d_HaarFeatures.ptr() != NULL) || gpuAllocator.isCounting(), NCV_NULL_PTR);
    ncvAssertReturn(srcRoi.width > 0 && srcRoi.height > 0 &&
                    d_srcImg.width() >= srcRoi.width && d_srcImg.height() >= srcRoi.height &&
                    srcRoi.width >= minObjSize.width && srcRoi.height >= minObjSize.height &&
                    d_dstRects.length() >= 1, NCV_DIMENSIONS_INVALID);

    ncvAssertReturn(scaleStep > 1.0f, NCV_INVALID_SCALE);

    ncvAssertReturn(d_HaarStages.length() >= haar.NumStages &&
                    d_HaarNodes.length() >= haar.NumClassifierTotalNodes &&
                    d_HaarFeatures.length() >= haar.NumFeatures &&
                    d_HaarStages.length() == h_HaarStages.length() &&
                    haar.NumClassifierRootNodes <= haar.NumClassifierTotalNodes, NCV_DIMENSIONS_INVALID);

    ncvAssertReturn(haar.bNeedsTiltedII == false, NCV_NOIMPL_HAAR_TILTED_FEATURES);

    ncvAssertReturn(pixelStep == 1 || pixelStep == 2, NCV_HAAR_INVALID_PIXEL_STEP);

    //TODO: set NPP active stream to cuStream

    NCVStatus ncvStat;
    NCV_SET_SKIP_COND(gpuAllocator.isCounting());

    Ncv32u integralWidth = d_srcImg.width() + 1;
    Ncv32u integralHeight = d_srcImg.height() + 1;

    NCVMatrixAlloc<Ncv32u> integral(gpuAllocator, integralWidth, integralHeight);
    ncvAssertReturn(integral.isMemAllocated(), NCV_ALLOCATOR_BAD_ALLOC);
    NCVMatrixAlloc<Ncv64u> d_sqIntegralImage(gpuAllocator, integralWidth, integralHeight);
    ncvAssertReturn(d_sqIntegralImage.isMemAllocated(), NCV_ALLOCATOR_BAD_ALLOC);

    NCVMatrixAlloc<Ncv32f> d_rectStdDev(gpuAllocator, d_srcImg.width(), d_srcImg.height());
    ncvAssertReturn(d_rectStdDev.isMemAllocated(), NCV_ALLOCATOR_BAD_ALLOC);
    NCVMatrixAlloc<Ncv32u> d_pixelMask(gpuAllocator, d_srcImg.width(), d_srcImg.height());
    ncvAssertReturn(d_pixelMask.isMemAllocated(), NCV_ALLOCATOR_BAD_ALLOC);

    NCVMatrixAlloc<Ncv32u> d_scaledIntegralImage(gpuAllocator, integralWidth, integralHeight);
    ncvAssertReturn(d_scaledIntegralImage.isMemAllocated(), NCV_ALLOCATOR_BAD_ALLOC);
    NCVMatrixAlloc<Ncv64u> d_scaledSqIntegralImage(gpuAllocator, integralWidth, integralHeight);
    ncvAssertReturn(d_scaledSqIntegralImage.isMemAllocated(), NCV_ALLOCATOR_BAD_ALLOC);

    NCVVectorAlloc<NcvRect32u> d_hypothesesIntermediate(gpuAllocator, d_srcImg.width() * d_srcImg.height());
    ncvAssertReturn(d_hypothesesIntermediate.isMemAllocated(), NCV_ALLOCATOR_BAD_ALLOC);
    NCVVectorAlloc<NcvRect32u> h_hypothesesIntermediate(cpuAllocator, d_srcImg.width() * d_srcImg.height());
    ncvAssertReturn(h_hypothesesIntermediate.isMemAllocated(), NCV_ALLOCATOR_BAD_ALLOC);

    NCVStatus nppStat;
    Ncv32u szTmpBufIntegral, szTmpBufSqIntegral;
    nppStat = nppiStIntegralGetSize_8u32u(NcvSize32u(d_srcImg.width(), d_srcImg.height()), &szTmpBufIntegral, devProp);
    ncvAssertReturnNcvStat(nppStat);
    nppStat = nppiStSqrIntegralGetSize_8u64u(NcvSize32u(d_srcImg.width(), d_srcImg.height()), &szTmpBufSqIntegral, devProp);
    ncvAssertReturnNcvStat(nppStat);
    NCVVectorAlloc<Ncv8u> d_tmpIIbuf(gpuAllocator, std::max(szTmpBufIntegral, szTmpBufSqIntegral));
    ncvAssertReturn(d_tmpIIbuf.isMemAllocated(), NCV_ALLOCATOR_BAD_ALLOC);

    NCV_SKIP_COND_BEGIN

    nppStat = nppiStIntegral_8u32u_C1R(d_srcImg.ptr(), d_srcImg.pitch(),
                                       integral.ptr(), integral.pitch(),
                                       NcvSize32u(d_srcImg.width(), d_srcImg.height()),
                                       d_tmpIIbuf.ptr(), szTmpBufIntegral, devProp);
    ncvAssertReturnNcvStat(nppStat);

    nppStat = nppiStSqrIntegral_8u64u_C1R(d_srcImg.ptr(), d_srcImg.pitch(),
                                          d_sqIntegralImage.ptr(), d_sqIntegralImage.pitch(),
                                          NcvSize32u(d_srcImg.width(), d_srcImg.height()),
                                          d_tmpIIbuf.ptr(), szTmpBufSqIntegral, devProp);
    ncvAssertReturnNcvStat(nppStat);

    NCV_SKIP_COND_END

    dstNumRects = 0;

    Ncv32u lastCheckedScale = 0;
    NcvBool bReverseTraverseScale = ((flags & NCVPipeObjDet_FindLargestObject) != 0);
    std::vector<Ncv32u> scalesVector;

    NcvBool bFoundLargestFace = false;

    for (Ncv32f scaleIter = 1.0f; ; scaleIter *= scaleStep)
    {
        Ncv32u scale = (Ncv32u)scaleIter;
        if (lastCheckedScale == scale)
        {
            continue;
        }
        lastCheckedScale = scale;

        if (haar.ClassifierSize.width * (Ncv32s)scale < minObjSize.width ||
            haar.ClassifierSize.height * (Ncv32s)scale < minObjSize.height)
        {
            continue;
        }

        NcvSize32s srcRoi_, srcIIRo_i, scaledIIRoi, searchRoi;

        srcRoi_.width = d_srcImg.width();
        srcRoi_.height = d_srcImg.height();

        srcIIRo_i.width = srcRoi_.width + 1;
        srcIIRo_i.height = srcRoi_.height + 1;

        scaledIIRoi.width = srcIIRo_i.width / scale;
        scaledIIRoi.height = srcIIRo_i.height / scale;

        searchRoi.width = scaledIIRoi.width - haar.ClassifierSize.width;
        searchRoi.height = scaledIIRoi.height - haar.ClassifierSize.height;

        if (searchRoi.width <= 0 || searchRoi.height <= 0)
        {
            break;
        }

        scalesVector.push_back(scale);

        if (gpuAllocator.isCounting())
        {
            break;
        }
    }

    if (bReverseTraverseScale)
    {
        std::reverse(scalesVector.begin(), scalesVector.end());
    }

    //TODO: handle _fair_scale_ flag
    for (Ncv32u i=0; i<scalesVector.size(); i++)
    {
        Ncv32u scale = scalesVector[i];

        NcvSize32u srcRoi_, scaledIIRoi, searchRoi;
        NcvSize32u srcIIRoi;
        srcRoi_.width = d_srcImg.width();
        srcRoi_.height = d_srcImg.height();
        srcIIRoi.width = srcRoi_.width + 1;
        srcIIRoi.height = srcRoi_.height + 1;
        scaledIIRoi.width = srcIIRoi.width / scale;
        scaledIIRoi.height = srcIIRoi.height / scale;
        searchRoi.width = scaledIIRoi.width - haar.ClassifierSize.width;
        searchRoi.height = scaledIIRoi.height - haar.ClassifierSize.height;

        NCV_SKIP_COND_BEGIN

        nppStat = nppiStDecimate_32u_C1R(
            integral.ptr(), integral.pitch(),
            d_scaledIntegralImage.ptr(), d_scaledIntegralImage.pitch(),
            srcIIRoi, scale, true);
        ncvAssertReturnNcvStat(nppStat);

        nppStat = nppiStDecimate_64u_C1R(
            d_sqIntegralImage.ptr(), d_sqIntegralImage.pitch(),
            d_scaledSqIntegralImage.ptr(), d_scaledSqIntegralImage.pitch(),
            srcIIRoi, scale, true);
        ncvAssertReturnNcvStat(nppStat);

        const NcvRect32u rect(
            HAAR_STDDEV_BORDER,
            HAAR_STDDEV_BORDER,
            haar.ClassifierSize.width - 2*HAAR_STDDEV_BORDER,
            haar.ClassifierSize.height - 2*HAAR_STDDEV_BORDER);
        nppStat = nppiStRectStdDev_32f_C1R(
            d_scaledIntegralImage.ptr(), d_scaledIntegralImage.pitch(),
            d_scaledSqIntegralImage.ptr(), d_scaledSqIntegralImage.pitch(),
            d_rectStdDev.ptr(), d_rectStdDev.pitch(),
            NcvSize32u(searchRoi.width, searchRoi.height), rect,
            (Ncv32f)scale*scale, true);
        ncvAssertReturnNcvStat(nppStat);

        NCV_SKIP_COND_END

        Ncv32u detectionsOnThisScale;
        ncvStat = ncvApplyHaarClassifierCascade_device(
            d_scaledIntegralImage, d_rectStdDev, d_pixelMask,
            detectionsOnThisScale,
            haar, h_HaarStages, d_HaarStages, d_HaarNodes, d_HaarFeatures, false,
            searchRoi, pixelStep, (Ncv32f)scale*scale,
            gpuAllocator, cpuAllocator, devProp, cuStream);
        ncvAssertReturnNcvStat(nppStat);

        NCV_SKIP_COND_BEGIN

        NCVVectorReuse<Ncv32u> d_vecPixelMask(d_pixelMask.getSegment());
        ncvStat = ncvGrowDetectionsVector_device(
            d_vecPixelMask,
            detectionsOnThisScale,
            d_hypothesesIntermediate,
            dstNumRects,
            static_cast<Ncv32u>(d_hypothesesIntermediate.length()),
            haar.ClassifierSize.width,
            haar.ClassifierSize.height,
            (Ncv32f)scale,
            cuStream);
        ncvAssertReturn(ncvStat == NCV_SUCCESS, ncvStat);

        if (flags & NCVPipeObjDet_FindLargestObject)
        {
            if (dstNumRects == 0)
            {
                continue;
            }

            if (dstNumRects != 0)
            {
                ncvAssertCUDAReturn(hipStreamSynchronize(cuStream), NCV_CUDA_ERROR);
                ncvStat = d_hypothesesIntermediate.copySolid(h_hypothesesIntermediate, cuStream,
                                                             dstNumRects * sizeof(NcvRect32u));
                ncvAssertReturnNcvStat(ncvStat);
                ncvAssertCUDAReturn(hipStreamSynchronize(cuStream), NCV_CUDA_ERROR);
            }

            Ncv32u numStrongHypothesesNow = dstNumRects;
            ncvStat = ncvGroupRectangles_host(
                h_hypothesesIntermediate,
                numStrongHypothesesNow,
                minNeighbors,
                RECT_SIMILARITY_PROPORTION,
                NULL);
            ncvAssertReturnNcvStat(ncvStat);

            if (numStrongHypothesesNow > 0)
            {
                NcvRect32u maxRect = h_hypothesesIntermediate.ptr()[0];
                for (Ncv32u j=1; j<numStrongHypothesesNow; j++)
                {
                    if (maxRect.width < h_hypothesesIntermediate.ptr()[j].width)
                    {
                        maxRect = h_hypothesesIntermediate.ptr()[j];
                    }
                }

                h_hypothesesIntermediate.ptr()[0] = maxRect;
                dstNumRects = 1;

                ncvStat = h_hypothesesIntermediate.copySolid(d_dstRects, cuStream, sizeof(NcvRect32u));
                ncvAssertReturnNcvStat(ncvStat);

                bFoundLargestFace = true;

                break;
            }
        }

        NCV_SKIP_COND_END

        if (gpuAllocator.isCounting())
        {
            break;
        }
    }

    NCVStatus ncvRetCode = NCV_SUCCESS;

    NCV_SKIP_COND_BEGIN

    if (flags & NCVPipeObjDet_FindLargestObject)
    {
        if (!bFoundLargestFace)
        {
            dstNumRects = 0;
        }
    }
    else
    {
        //TODO: move hypotheses filtration to GPU pipeline (the only CPU-resident element of the pipeline left)
        if (dstNumRects != 0)
        {
            ncvAssertCUDAReturn(hipStreamSynchronize(cuStream), NCV_CUDA_ERROR);
            ncvStat = d_hypothesesIntermediate.copySolid(h_hypothesesIntermediate, cuStream,
                                                         dstNumRects * sizeof(NcvRect32u));
            ncvAssertReturnNcvStat(ncvStat);
            ncvAssertCUDAReturn(hipStreamSynchronize(cuStream), NCV_CUDA_ERROR);
        }

        ncvStat = ncvGroupRectangles_host(
            h_hypothesesIntermediate,
            dstNumRects,
            minNeighbors,
            RECT_SIMILARITY_PROPORTION,
            NULL);
        ncvAssertReturnNcvStat(ncvStat);

        if (dstNumRects > d_dstRects.length())
        {
            ncvRetCode = NCV_WARNING_HAAR_DETECTIONS_VECTOR_OVERFLOW;
            dstNumRects = static_cast<Ncv32u>(d_dstRects.length());
        }

        if (dstNumRects != 0)
        {
            ncvStat = h_hypothesesIntermediate.copySolid(d_dstRects, cuStream,
                                                         dstNumRects * sizeof(NcvRect32u));
            ncvAssertReturnNcvStat(ncvStat);
        }
    }

    if (flags & NCVPipeObjDet_VisualizeInPlace)
    {
        ncvAssertCUDAReturn(hipStreamSynchronize(cuStream), NCV_CUDA_ERROR);
        ncvDrawRects_8u_device(d_srcImg.ptr(), d_srcImg.stride(),
                               d_srcImg.width(), d_srcImg.height(),
                               d_dstRects.ptr(), dstNumRects, 255, cuStream);
    }

    NCV_SKIP_COND_END

    return ncvRetCode;
}


//==============================================================================
//
// Purely Host code: classifier IO, mock-ups
//
//==============================================================================


#ifdef _SELF_TEST_
#include <float.h>
#endif


NCVStatus ncvApplyHaarClassifierCascade_host(NCVMatrix<Ncv32u> &h_integralImage,
                                             NCVMatrix<Ncv32f> &h_weights,
                                             NCVMatrixAlloc<Ncv32u> &h_pixelMask,
                                             Ncv32u &numDetections,
                                             HaarClassifierCascadeDescriptor &haar,
                                             NCVVector<HaarStage64> &h_HaarStages,
                                             NCVVector<HaarClassifierNode128> &h_HaarNodes,
                                             NCVVector<HaarFeature64> &h_HaarFeatures,
                                             NcvBool bMaskElements,
                                             NcvSize32u anchorsRoi,
                                             Ncv32u pixelStep,
                                             Ncv32f scaleArea)
{
    ncvAssertReturn(h_integralImage.memType() == h_weights.memType() &&
                    h_integralImage.memType() == h_pixelMask.memType() &&
                     (h_integralImage.memType() == NCVMemoryTypeHostPageable ||
                      h_integralImage.memType() == NCVMemoryTypeHostPinned), NCV_MEM_RESIDENCE_ERROR);
    ncvAssertReturn(h_HaarStages.memType() == h_HaarNodes.memType() &&
                    h_HaarStages.memType() == h_HaarFeatures.memType() &&
                     (h_HaarStages.memType() == NCVMemoryTypeHostPageable ||
                      h_HaarStages.memType() == NCVMemoryTypeHostPinned), NCV_MEM_RESIDENCE_ERROR);
    ncvAssertReturn(h_integralImage.ptr() != NULL && h_weights.ptr() != NULL && h_pixelMask.ptr() != NULL &&
                    h_HaarStages.ptr() != NULL && h_HaarNodes.ptr() != NULL && h_HaarFeatures.ptr() != NULL, NCV_NULL_PTR);
    ncvAssertReturn(anchorsRoi.width > 0 && anchorsRoi.height > 0 &&
                    h_pixelMask.width() >= anchorsRoi.width && h_pixelMask.height() >= anchorsRoi.height &&
                    h_weights.width() >= anchorsRoi.width && h_weights.height() >= anchorsRoi.height &&
                    h_integralImage.width() >= anchorsRoi.width + haar.ClassifierSize.width &&
                    h_integralImage.height() >= anchorsRoi.height + haar.ClassifierSize.height, NCV_DIMENSIONS_INVALID);
    ncvAssertReturn(scaleArea > 0, NCV_INVALID_SCALE);
    ncvAssertReturn(h_HaarStages.length() >= haar.NumStages &&
                    h_HaarNodes.length() >= haar.NumClassifierTotalNodes &&
                    h_HaarFeatures.length() >= haar.NumFeatures &&
                    h_HaarStages.length() == h_HaarStages.length() &&
                    haar.NumClassifierRootNodes <= haar.NumClassifierTotalNodes, NCV_DIMENSIONS_INVALID);
    ncvAssertReturn(haar.bNeedsTiltedII == false, NCV_NOIMPL_HAAR_TILTED_FEATURES);
    ncvAssertReturn(pixelStep == 1 || pixelStep == 2, NCV_HAAR_INVALID_PIXEL_STEP);

    Ncv32f scaleAreaPixels = scaleArea * ((haar.ClassifierSize.width - 2*HAAR_STDDEV_BORDER) *
                                          (haar.ClassifierSize.height - 2*HAAR_STDDEV_BORDER));

    for (Ncv32u i=0; i<anchorsRoi.height; i++)
    {
        for (Ncv32u j=0; j<h_pixelMask.stride(); j++)
        {
            if (i % pixelStep != 0 || j % pixelStep != 0 || j >= anchorsRoi.width)
            {
                h_pixelMask.ptr()[i * h_pixelMask.stride() + j] = OBJDET_MASK_ELEMENT_INVALID_32U;
            }
            else
            {
                for (Ncv32u iStage = 0; iStage < haar.NumStages; iStage++)
                {
                    Ncv32f curStageSum = 0.0f;
                    Ncv32u numRootNodesInStage = h_HaarStages.ptr()[iStage].getNumClassifierRootNodes();
                    Ncv32u curRootNodeOffset = h_HaarStages.ptr()[iStage].getStartClassifierRootNodeOffset();

                    if (iStage == 0)
                    {
                        if (bMaskElements && h_pixelMask.ptr()[i * h_pixelMask.stride() + j] == OBJDET_MASK_ELEMENT_INVALID_32U)
                        {
                            break;
                        }
                        else
                        {
                            h_pixelMask.ptr()[i * h_pixelMask.stride() + j] = ((i << 16) | j);
                        }
                    }
                    else if (h_pixelMask.ptr()[i * h_pixelMask.stride() + j] == OBJDET_MASK_ELEMENT_INVALID_32U)
                    {
                        break;
                    }

                    while (numRootNodesInStage--)
                    {
                        NcvBool bMoreNodesToTraverse = true;
                        Ncv32u curNodeOffset = curRootNodeOffset;

                        while (bMoreNodesToTraverse)
                        {
                            HaarClassifierNode128 curNode = h_HaarNodes.ptr()[curNodeOffset];
                            HaarFeatureDescriptor32 curFeatDesc = curNode.getFeatureDesc();
                            Ncv32u curNodeFeaturesNum = curFeatDesc.getNumFeatures();
                            Ncv32u curNodeFeaturesOffs = curFeatDesc.getFeaturesOffset();

                            Ncv32f curNodeVal = 0.f;
                            for (Ncv32u iRect=0; iRect<curNodeFeaturesNum; iRect++)
                            {
                                HaarFeature64 feature = h_HaarFeatures.ptr()[curNodeFeaturesOffs + iRect];
                                Ncv32u rectX, rectY, rectWidth, rectHeight;
                                feature.getRect(&rectX, &rectY, &rectWidth, &rectHeight);
                                Ncv32f rectWeight = feature.getWeight();
                                Ncv32u iioffsTL = (i + rectY) * h_integralImage.stride() + (j + rectX);
                                Ncv32u iioffsTR = iioffsTL + rectWidth;
                                Ncv32u iioffsBL = iioffsTL + rectHeight * h_integralImage.stride();
                                Ncv32u iioffsBR = iioffsBL + rectWidth;

                                Ncv32u iivalTL = h_integralImage.ptr()[iioffsTL];
                                Ncv32u iivalTR = h_integralImage.ptr()[iioffsTR];
                                Ncv32u iivalBL = h_integralImage.ptr()[iioffsBL];
                                Ncv32u iivalBR = h_integralImage.ptr()[iioffsBR];
                                Ncv32u rectSum = iivalBR - iivalBL + iivalTL - iivalTR;
                                curNodeVal += (Ncv32f)rectSum * rectWeight;
                            }

                            HaarClassifierNodeDescriptor32 nodeLeft = curNode.getLeftNodeDesc();
                            HaarClassifierNodeDescriptor32 nodeRight = curNode.getRightNodeDesc();
                            Ncv32f nodeThreshold = curNode.getThreshold();

                            HaarClassifierNodeDescriptor32 nextNodeDescriptor;
                            NcvBool nextNodeIsLeaf;

                            if (curNodeVal < scaleAreaPixels * h_weights.ptr()[i * h_weights.stride() + j] * nodeThreshold)
                            {
                                nextNodeDescriptor = nodeLeft;
                                nextNodeIsLeaf = curFeatDesc.isLeftNodeLeaf();
                            }
                            else
                            {
                                nextNodeDescriptor = nodeRight;
                                nextNodeIsLeaf = curFeatDesc.isRightNodeLeaf();
                            }

                            if (nextNodeIsLeaf)
                            {
                                Ncv32f tmpLeafValue = nextNodeDescriptor.getLeafValueHost();
                                curStageSum += tmpLeafValue;
                                bMoreNodesToTraverse = false;
                            }
                            else
                            {
                                curNodeOffset = nextNodeDescriptor.getNextNodeOffset();
                            }
                        }

                        curRootNodeOffset++;
                    }

                    Ncv32f tmpStageThreshold = h_HaarStages.ptr()[iStage].getStageThreshold();
                    if (curStageSum < tmpStageThreshold)
                    {
                        //drop
                        h_pixelMask.ptr()[i * h_pixelMask.stride() + j] = OBJDET_MASK_ELEMENT_INVALID_32U;
                        break;
                    }
                }
            }
        }
    }

    std::sort(h_pixelMask.ptr(), h_pixelMask.ptr() + anchorsRoi.height * h_pixelMask.stride());
    Ncv32u i = 0;
    for (; i<anchorsRoi.height * h_pixelMask.stride(); i++)
    {
        if (h_pixelMask.ptr()[i] == OBJDET_MASK_ELEMENT_INVALID_32U)
        {
            break;
        }
    }
    numDetections = i;

    return NCV_SUCCESS;
}


NCVStatus ncvGrowDetectionsVector_host(NCVVector<Ncv32u> &pixelMask,
                                       Ncv32u numPixelMaskDetections,
                                       NCVVector<NcvRect32u> &hypotheses,
                                       Ncv32u &totalDetections,
                                       Ncv32u totalMaxDetections,
                                       Ncv32u rectWidth,
                                       Ncv32u rectHeight,
                                       Ncv32f curScale)
{
    ncvAssertReturn(pixelMask.ptr() != NULL && hypotheses.ptr() != NULL, NCV_NULL_PTR);
    ncvAssertReturn(pixelMask.memType() == hypotheses.memType() &&
                    pixelMask.memType() != NCVMemoryTypeDevice, NCV_MEM_RESIDENCE_ERROR);
    ncvAssertReturn(rectWidth > 0 && rectHeight > 0 && curScale > 0, NCV_INVALID_ROI);
    ncvAssertReturn(curScale > 0, NCV_INVALID_SCALE);
    ncvAssertReturn(totalMaxDetections <= hypotheses.length() &&
                    numPixelMaskDetections <= pixelMask.length() &&
                    totalDetections <= totalMaxDetections, NCV_INCONSISTENT_INPUT);

    NCVStatus ncvStat = NCV_SUCCESS;
    Ncv32u numDetsToCopy = numPixelMaskDetections;

    if (numDetsToCopy == 0)
    {
        return ncvStat;
    }

    if (totalDetections + numPixelMaskDetections > totalMaxDetections)
    {
        ncvStat = NCV_WARNING_HAAR_DETECTIONS_VECTOR_OVERFLOW;
        numDetsToCopy = totalMaxDetections - totalDetections;
    }

    for (Ncv32u i=0; i<numDetsToCopy; i++)
    {
        hypotheses.ptr()[totalDetections + i] = pixelToRect(pixelMask.ptr()[i], rectWidth, rectHeight, curScale);
    }

    totalDetections += numDetsToCopy;
    return ncvStat;
}

#define RECT_X_IDX              0
#define RECT_Y_IDX              1
#define RECT_W_IDX              2
#define RECT_H_IDX              3
#define RECT_WEIGHT_IDX         4

#define CUDA_CC_SIZE_W          0
#define CUDA_CC_SIZE_H          1

static NCVStatus loadFromXML(const cv::String &filename,
                      HaarClassifierCascadeDescriptor &haar,
                      std::vector<HaarStage64> &haarStages,
                      std::vector<HaarClassifierNode128> &haarClassifierNodes,
                      std::vector<HaarFeature64> &haarFeatures)
{
    const char *CUDA_CC_SIZE = "size";
    const char *CUDA_CC_STAGES = "stages";
    const char *CUDA_CC_STAGE_THRESHOLD = "stage_threshold";
    const char *CUDA_CC_TREES = "trees";
    const char *CUDA_CC_FEATURE = "feature";
    const char *CUDA_CC_RECT = "rects";
    const char *CUDA_CC_TILTED = "tilted";
    const char *CUDA_CC_THRESHOLD = "threshold";
    const char *CUDA_CC_LEFT_VAL = "left_val";
    const char *CUDA_CC_RIGHT_VAL = "right_val";
    const char *CUDA_CC_LEFT_NODE = "left_node";
    const char *CUDA_CC_RIGHT_NODE = "right_node";

    NCVStatus ncvStat;

    haar.NumStages = 0;
    haar.NumClassifierRootNodes = 0;
    haar.NumClassifierTotalNodes = 0;
    haar.NumFeatures = 0;
    haar.ClassifierSize.width = 0;
    haar.ClassifierSize.height = 0;
    haar.bHasStumpsOnly = true;
    haar.bNeedsTiltedII = false;
    Ncv32u curMaxTreeDepth = 0;

    std::vector<HaarClassifierNode128> h_TmpClassifierNotRootNodes;
    haarStages.resize(0);
    haarClassifierNodes.resize(0);
    haarFeatures.resize(0);

    cv::FileStorage fs(filename, cv::FileStorage::READ | cv::FileStorage::FORMAT_XML);

    if (!fs.isOpened())
        return NCV_FILE_ERROR;

    const cv::FileNode &root = fs.getFirstTopLevelNode();
    const cv::FileNode &fnSize = root[CUDA_CC_SIZE];

    // collect the cascade classifier window size
    haar.ClassifierSize.width = (int)fnSize[CUDA_CC_SIZE_W];
    haar.ClassifierSize.height = (int)fnSize[CUDA_CC_SIZE_H];
    CV_Assert(haar.ClassifierSize.height > 0 && haar.ClassifierSize.width > 0);

    const cv::FileNode &fnStages = root[CUDA_CC_STAGES];
    cv::FileNodeIterator it = fnStages.begin(), it_end = fnStages.end();

    for (; it != it_end; ++it) // by stages
    {
        cv::FileNode fnStage = *it;
        HaarStage64 curStage;

        curStage.setStartClassifierRootNodeOffset(static_cast<Ncv32u>(haarClassifierNodes.size()));
        curStage.setStageThreshold((float)fnStage[CUDA_CC_STAGE_THRESHOLD]);

        // iterate over the trees
        const cv::FileNode &fnTrees = fnStage[CUDA_CC_TREES];
        cv::FileNodeIterator it1 = fnTrees.begin(), it1_end = fnTrees.end();

        for (; it1 != it1_end; ++it1) // by trees
        {
            cv::FileNode tree = *it1;
            Ncv32u nodeId = (size_t)0;
            HaarClassifierNode128 curNode;

            curNode.setThreshold((float)tree[0][CUDA_CC_THRESHOLD]);

            NcvBool bIsLeftNodeLeaf = false;
            NcvBool bIsRightNodeLeaf = false;

            HaarClassifierNodeDescriptor32 nodeLeft;

            cv::FileNode leftNode = tree[0][CUDA_CC_LEFT_NODE];

            if (leftNode.fs == NULL)
            {
                Ncv32f leftVal = tree[0][CUDA_CC_LEFT_VAL];
                ncvStat = nodeLeft.create(leftVal);
                ncvAssertReturn(ncvStat == NCV_SUCCESS, ncvStat);
                bIsLeftNodeLeaf = true;
            }
            else
            {
                Ncv32u leftNodeOffset = (int)tree[0][CUDA_CC_LEFT_NODE];
                nodeLeft.create((Ncv32u)(h_TmpClassifierNotRootNodes.size() + leftNodeOffset - 1));
                haar.bHasStumpsOnly = false;
            }

            curNode.setLeftNodeDesc(nodeLeft);

            HaarClassifierNodeDescriptor32 nodeRight;
            cv::FileNode rightNode = tree[0][CUDA_CC_RIGHT_NODE];

            if (rightNode.fs == NULL)
            {
                Ncv32f rightVal = tree[0][CUDA_CC_RIGHT_VAL];
                ncvStat = nodeRight.create(rightVal);
                ncvAssertReturn(ncvStat == NCV_SUCCESS, ncvStat);
                bIsRightNodeLeaf = true;
            }
            else
            {
                Ncv32u rightNodeOffset = (int)tree[0][CUDA_CC_RIGHT_NODE];
                nodeRight.create((Ncv32u)(h_TmpClassifierNotRootNodes.size() + rightNodeOffset - 1));
                haar.bHasStumpsOnly = false;
            }

            curNode.setRightNodeDesc(nodeRight);

            cv::FileNode fnFeature = tree[0][CUDA_CC_FEATURE];
            Ncv32u tiltedVal = (int)fnFeature[CUDA_CC_TILTED];
            haar.bNeedsTiltedII = (tiltedVal != 0);

            cv::FileNodeIterator it2 = fnFeature[CUDA_CC_RECT].begin(), it2_end = fnFeature[CUDA_CC_RECT].end();

            Ncv32u featureId = 0;
            for (; it2 != it2_end; ++it2) // by feature
            {
                cv::FileNode rect = *it2;

                Ncv32u rectX = (int)rect[RECT_X_IDX];
                Ncv32u rectY = (int)rect[RECT_Y_IDX];
                Ncv32u rectWidth = (int)rect[RECT_W_IDX];
                Ncv32u rectHeight = (int)rect[RECT_H_IDX];

                Ncv32f rectWeight = (float)rect[RECT_WEIGHT_IDX];

                if (rectWeight == 0)
                    break;

                HaarFeature64 curFeature;
                ncvStat = curFeature.setRect(rectX, rectY, rectWidth, rectHeight, haar.ClassifierSize.width, haar.ClassifierSize.height);
                curFeature.setWeight(rectWeight);
                ncvAssertReturn(NCV_SUCCESS == ncvStat, ncvStat);

                haarFeatures.push_back(curFeature);
                featureId++;
            }

            HaarFeatureDescriptor32 tmpFeatureDesc;
            ncvStat = tmpFeatureDesc.create(haar.bNeedsTiltedII, bIsLeftNodeLeaf, bIsRightNodeLeaf,
                featureId, static_cast<Ncv32u>(haarFeatures.size()) - featureId);
            ncvAssertReturn(NCV_SUCCESS == ncvStat, ncvStat);
            curNode.setFeatureDesc(tmpFeatureDesc);

            if (!nodeId)
            {
                //root node
                haarClassifierNodes.push_back(curNode);
                curMaxTreeDepth = 1;
            }
            else
            {
                //other node
                h_TmpClassifierNotRootNodes.push_back(curNode);
                curMaxTreeDepth++;
            }

            nodeId++;
        }

        curStage.setNumClassifierRootNodes((Ncv32u)fnTrees.size());
        haarStages.push_back(curStage);
    }

    //fill in cascade stats
    haar.NumStages = static_cast<Ncv32u>(haarStages.size());
    haar.NumClassifierRootNodes = static_cast<Ncv32u>(haarClassifierNodes.size());
    haar.NumClassifierTotalNodes = static_cast<Ncv32u>(haar.NumClassifierRootNodes + h_TmpClassifierNotRootNodes.size());
    haar.NumFeatures = static_cast<Ncv32u>(haarFeatures.size());

    //merge root and leaf nodes in one classifiers array
    Ncv32u offsetRoot = static_cast<Ncv32u>(haarClassifierNodes.size());
    for (Ncv32u i=0; i<haarClassifierNodes.size(); i++)
    {
        HaarFeatureDescriptor32 featureDesc = haarClassifierNodes[i].getFeatureDesc();

        HaarClassifierNodeDescriptor32 nodeLeft = haarClassifierNodes[i].getLeftNodeDesc();
        if (!featureDesc.isLeftNodeLeaf())
        {
            Ncv32u newOffset = nodeLeft.getNextNodeOffset() + offsetRoot;
            nodeLeft.create(newOffset);
        }
        haarClassifierNodes[i].setLeftNodeDesc(nodeLeft);

        HaarClassifierNodeDescriptor32 nodeRight = haarClassifierNodes[i].getRightNodeDesc();
        if (!featureDesc.isRightNodeLeaf())
        {
            Ncv32u newOffset = nodeRight.getNextNodeOffset() + offsetRoot;
            nodeRight.create(newOffset);
        }
        haarClassifierNodes[i].setRightNodeDesc(nodeRight);
    }

    for (Ncv32u i=0; i<h_TmpClassifierNotRootNodes.size(); i++)
    {
        HaarFeatureDescriptor32 featureDesc = h_TmpClassifierNotRootNodes[i].getFeatureDesc();

        HaarClassifierNodeDescriptor32 nodeLeft = h_TmpClassifierNotRootNodes[i].getLeftNodeDesc();
        if (!featureDesc.isLeftNodeLeaf())
        {
            Ncv32u newOffset = nodeLeft.getNextNodeOffset() + offsetRoot;
            nodeLeft.create(newOffset);
        }
        h_TmpClassifierNotRootNodes[i].setLeftNodeDesc(nodeLeft);

        HaarClassifierNodeDescriptor32 nodeRight = h_TmpClassifierNotRootNodes[i].getRightNodeDesc();
        if (!featureDesc.isRightNodeLeaf())
        {
            Ncv32u newOffset = nodeRight.getNextNodeOffset() + offsetRoot;
            nodeRight.create(newOffset);
        }
        h_TmpClassifierNotRootNodes[i].setRightNodeDesc(nodeRight);

        haarClassifierNodes.push_back(h_TmpClassifierNotRootNodes[i]);
    }

    return NCV_SUCCESS;
}


#define NVBIN_HAAR_SIZERESERVED     16
#define NVBIN_HAAR_VERSION          0x1


static NCVStatus loadFromNVBIN(const cv::String &filename,
                               HaarClassifierCascadeDescriptor &haar,
                               std::vector<HaarStage64> &haarStages,
                               std::vector<HaarClassifierNode128> &haarClassifierNodes,
                               std::vector<HaarFeature64> &haarFeatures)
{
    size_t readCount;
    FILE *fp = fopen(filename.c_str(), "rb");
    ncvAssertReturn(fp != NULL, NCV_FILE_ERROR);
    Ncv32u fileVersion;
    readCount = fread(&fileVersion, sizeof(Ncv32u), 1, fp);
    ncvAssertReturn(1 == readCount, NCV_FILE_ERROR);
    ncvAssertReturn(fileVersion == NVBIN_HAAR_VERSION, NCV_FILE_ERROR);
    Ncv32u fsize;
    readCount = fread(&fsize, sizeof(Ncv32u), 1, fp);
    ncvAssertReturn(1 == readCount, NCV_FILE_ERROR);
    fseek(fp, 0, SEEK_END);
    Ncv32u fsizeActual = ftell(fp);
    ncvAssertReturn(fsize == fsizeActual, NCV_FILE_ERROR);

    std::vector<unsigned char> fdata;
    fdata.resize(fsize);
    Ncv32u dataOffset = 0;
    fseek(fp, 0, SEEK_SET);
    readCount = fread(&fdata[0], fsize, 1, fp);
    ncvAssertReturn(1 == readCount, NCV_FILE_ERROR);
    fclose(fp);

    //data
    dataOffset = NVBIN_HAAR_SIZERESERVED;
    haar.NumStages = *(Ncv32u *)(&fdata[0]+dataOffset);
    dataOffset += sizeof(Ncv32u);
    haar.NumClassifierRootNodes = *(Ncv32u *)(&fdata[0]+dataOffset);
    dataOffset += sizeof(Ncv32u);
    haar.NumClassifierTotalNodes = *(Ncv32u *)(&fdata[0]+dataOffset);
    dataOffset += sizeof(Ncv32u);
    haar.NumFeatures = *(Ncv32u *)(&fdata[0]+dataOffset);
    dataOffset += sizeof(Ncv32u);
    haar.ClassifierSize = *(NcvSize32u *)(&fdata[0]+dataOffset);
    dataOffset += sizeof(NcvSize32u);
    haar.bNeedsTiltedII = *(NcvBool *)(&fdata[0]+dataOffset);
    dataOffset += sizeof(NcvBool);
    haar.bHasStumpsOnly = *(NcvBool *)(&fdata[0]+dataOffset);
    dataOffset += sizeof(NcvBool);

    haarStages.resize(haar.NumStages);
    haarClassifierNodes.resize(haar.NumClassifierTotalNodes);
    haarFeatures.resize(haar.NumFeatures);

    Ncv32u szStages = haar.NumStages * sizeof(HaarStage64);
    Ncv32u szClassifiers = haar.NumClassifierTotalNodes * sizeof(HaarClassifierNode128);
    Ncv32u szFeatures = haar.NumFeatures * sizeof(HaarFeature64);

    memcpy(&haarStages[0], &fdata[0]+dataOffset, szStages);
    dataOffset += szStages;
    memcpy(&haarClassifierNodes[0], &fdata[0]+dataOffset, szClassifiers);
    dataOffset += szClassifiers;
    memcpy(&haarFeatures[0], &fdata[0]+dataOffset, szFeatures);
    dataOffset += szFeatures;

    return NCV_SUCCESS;
}


NCVStatus ncvHaarGetClassifierSize(const cv::String &filename, Ncv32u &numStages,
                                   Ncv32u &numNodes, Ncv32u &numFeatures)
{
    size_t readCount;
    NCVStatus ncvStat;

    cv::String fext = filename.substr(filename.find_last_of(".") + 1);
    std::transform(fext.begin(), fext.end(), fext.begin(), ::tolower);

    if (fext == "nvbin")
    {
        FILE *fp = fopen(filename.c_str(), "rb");
        ncvAssertReturn(fp != NULL, NCV_FILE_ERROR);
        Ncv32u fileVersion;
        readCount = fread(&fileVersion, sizeof(Ncv32u), 1, fp);
        ncvAssertReturn(1 == readCount, NCV_FILE_ERROR);
        ncvAssertReturn(fileVersion == NVBIN_HAAR_VERSION, NCV_FILE_ERROR);
        fseek(fp, NVBIN_HAAR_SIZERESERVED, SEEK_SET);
        Ncv32u tmp;
        readCount = fread(&numStages,   sizeof(Ncv32u), 1, fp);
        ncvAssertReturn(1 == readCount, NCV_FILE_ERROR);
        readCount = fread(&tmp,         sizeof(Ncv32u), 1, fp);
        ncvAssertReturn(1 == readCount, NCV_FILE_ERROR);
        readCount = fread(&numNodes,    sizeof(Ncv32u), 1, fp);
        ncvAssertReturn(1 == readCount, NCV_FILE_ERROR);
        readCount = fread(&numFeatures, sizeof(Ncv32u), 1, fp);
        ncvAssertReturn(1 == readCount, NCV_FILE_ERROR);
        fclose(fp);
    }
    else if (fext == "xml")
    {
        HaarClassifierCascadeDescriptor haar;
        std::vector<HaarStage64> haarStages;
        std::vector<HaarClassifierNode128> haarNodes;
        std::vector<HaarFeature64> haarFeatures;

        ncvStat = loadFromXML(filename, haar, haarStages, haarNodes, haarFeatures);
        ncvAssertReturnNcvStat(ncvStat);

        numStages = haar.NumStages;
        numNodes = haar.NumClassifierTotalNodes;
        numFeatures = haar.NumFeatures;
    }
    else
    {
        return NCV_HAAR_XML_LOADING_EXCEPTION;
    }

    return NCV_SUCCESS;
}


NCVStatus ncvHaarLoadFromFile_host(const cv::String &filename,
                                   HaarClassifierCascadeDescriptor &haar,
                                   NCVVector<HaarStage64> &h_HaarStages,
                                   NCVVector<HaarClassifierNode128> &h_HaarNodes,
                                   NCVVector<HaarFeature64> &h_HaarFeatures)
{
    ncvAssertReturn(h_HaarStages.memType() == NCVMemoryTypeHostPinned &&
                    h_HaarNodes.memType() == NCVMemoryTypeHostPinned &&
                    h_HaarFeatures.memType() == NCVMemoryTypeHostPinned, NCV_MEM_RESIDENCE_ERROR);

    NCVStatus ncvStat;

    cv::String fext = filename.substr(filename.find_last_of(".") + 1);
    std::transform(fext.begin(), fext.end(), fext.begin(), ::tolower);

    std::vector<HaarStage64> haarStages;
    std::vector<HaarClassifierNode128> haarNodes;
    std::vector<HaarFeature64> haarFeatures;

    if (fext == "nvbin")
    {
        ncvStat = loadFromNVBIN(filename, haar, haarStages, haarNodes, haarFeatures);
        ncvAssertReturnNcvStat(ncvStat);
    }
    else if (fext == "xml")
    {
        ncvStat = loadFromXML(filename, haar, haarStages, haarNodes, haarFeatures);
        ncvAssertReturnNcvStat(ncvStat);
    }
    else
    {
        return NCV_HAAR_XML_LOADING_EXCEPTION;
    }

    ncvAssertReturn(h_HaarStages.length() >= haarStages.size(), NCV_MEM_INSUFFICIENT_CAPACITY);
    ncvAssertReturn(h_HaarNodes.length() >= haarNodes.size(), NCV_MEM_INSUFFICIENT_CAPACITY);
    ncvAssertReturn(h_HaarFeatures.length() >= haarFeatures.size(), NCV_MEM_INSUFFICIENT_CAPACITY);

    memcpy(h_HaarStages.ptr(), &haarStages[0], haarStages.size()*sizeof(HaarStage64));
    memcpy(h_HaarNodes.ptr(), &haarNodes[0], haarNodes.size()*sizeof(HaarClassifierNode128));
    memcpy(h_HaarFeatures.ptr(), &haarFeatures[0], haarFeatures.size()*sizeof(HaarFeature64));

    return NCV_SUCCESS;
}


NCVStatus ncvHaarStoreNVBIN_host(const cv::String &filename,
                                 HaarClassifierCascadeDescriptor haar,
                                 NCVVector<HaarStage64> &h_HaarStages,
                                 NCVVector<HaarClassifierNode128> &h_HaarNodes,
                                 NCVVector<HaarFeature64> &h_HaarFeatures)
{
    ncvAssertReturn(h_HaarStages.length() >= haar.NumStages, NCV_INCONSISTENT_INPUT);
    ncvAssertReturn(h_HaarNodes.length() >= haar.NumClassifierTotalNodes, NCV_INCONSISTENT_INPUT);
    ncvAssertReturn(h_HaarFeatures.length() >= haar.NumFeatures, NCV_INCONSISTENT_INPUT);
    ncvAssertReturn(h_HaarStages.memType() == NCVMemoryTypeHostPinned &&
                    h_HaarNodes.memType() == NCVMemoryTypeHostPinned &&
                    h_HaarFeatures.memType() == NCVMemoryTypeHostPinned, NCV_MEM_RESIDENCE_ERROR);

    Ncv32u szStages = haar.NumStages * sizeof(HaarStage64);
    Ncv32u szClassifiers = haar.NumClassifierTotalNodes * sizeof(HaarClassifierNode128);
    Ncv32u szFeatures = haar.NumFeatures * sizeof(HaarFeature64);

    Ncv32u dataOffset = 0;
    std::vector<unsigned char> fdata;
    fdata.resize(szStages+szClassifiers+szFeatures+1024, 0);

    //header
    *(Ncv32u *)(&fdata[0]+dataOffset) = NVBIN_HAAR_VERSION;

    //data
    dataOffset = NVBIN_HAAR_SIZERESERVED;
    *(Ncv32u *)(&fdata[0]+dataOffset) = haar.NumStages;
    dataOffset += sizeof(Ncv32u);
    *(Ncv32u *)(&fdata[0]+dataOffset) = haar.NumClassifierRootNodes;
    dataOffset += sizeof(Ncv32u);
    *(Ncv32u *)(&fdata[0]+dataOffset) = haar.NumClassifierTotalNodes;
    dataOffset += sizeof(Ncv32u);
    *(Ncv32u *)(&fdata[0]+dataOffset) = haar.NumFeatures;
    dataOffset += sizeof(Ncv32u);
    *(NcvSize32u *)(&fdata[0]+dataOffset) = haar.ClassifierSize;
    dataOffset += sizeof(NcvSize32u);
    *(NcvBool *)(&fdata[0]+dataOffset) = haar.bNeedsTiltedII;
    dataOffset += sizeof(NcvBool);
    *(NcvBool *)(&fdata[0]+dataOffset) = haar.bHasStumpsOnly;
    dataOffset += sizeof(NcvBool);

    memcpy(&fdata[0]+dataOffset, h_HaarStages.ptr(), szStages);
    dataOffset += szStages;
    memcpy(&fdata[0]+dataOffset, h_HaarNodes.ptr(), szClassifiers);
    dataOffset += szClassifiers;
    memcpy(&fdata[0]+dataOffset, h_HaarFeatures.ptr(), szFeatures);
    dataOffset += szFeatures;
    Ncv32u fsize = dataOffset;

    //TODO: CRC32 here

    //update header
    dataOffset = sizeof(Ncv32u);
    *(Ncv32u *)(&fdata[0]+dataOffset) = fsize;

    FILE *fp = fopen(filename.c_str(), "wb");
    ncvAssertReturn(fp != NULL, NCV_FILE_ERROR);
    fwrite(&fdata[0], fsize, 1, fp);
    fclose(fp);
    return NCV_SUCCESS;
}
