#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "opencv2/opencv_modules.hpp"

#ifndef HAVE_OPENCV_CUDEV

#error "opencv_cudev is required"

#else

#include "opencv2/cudaarithm.hpp"
#include "opencv2/cudev.hpp"
#include "opencv2/core/private.cuda.hpp"

using namespace cv;
using namespace cv::cuda;
using namespace cv::cudev;

namespace
{
    void normDiffInf(const GpuMat& _src1, const GpuMat& _src2, GpuMat& _dst, Stream& stream)
    {
        const GpuMat_<uchar>& src1 = (const GpuMat_<uchar>&) _src1;
        const GpuMat_<uchar>& src2 = (const GpuMat_<uchar>&) _src2;
        GpuMat_<int>& dst = (GpuMat_<int>&) _dst;

        gridFindMaxVal(abs_(cvt_<int>(src1) - cvt_<int>(src2)), dst, stream);
    }

    void normDiffL1(const GpuMat& _src1, const GpuMat& _src2, GpuMat& _dst, Stream& stream)
    {
        const GpuMat_<uchar>& src1 = (const GpuMat_<uchar>&) _src1;
        const GpuMat_<uchar>& src2 = (const GpuMat_<uchar>&) _src2;
        GpuMat_<int>& dst = (GpuMat_<int>&) _dst;

        gridCalcSum(abs_(cvt_<int>(src1) - cvt_<int>(src2)), dst, stream);
    }

    void normDiffL2(const GpuMat& _src1, const GpuMat& _src2, GpuMat& _dst, Stream& stream)
    {
        const GpuMat_<uchar>& src1 = (const GpuMat_<uchar>&) _src1;
        const GpuMat_<uchar>& src2 = (const GpuMat_<uchar>&) _src2;
        GpuMat_<double>& dst = (GpuMat_<double>&) _dst;

        BufferPool pool(stream);
        GpuMat_<double> buf(1, 1, pool.getAllocator());

        gridCalcSum(sqr_(cvt_<double>(src1) - cvt_<double>(src2)), buf, stream);
        gridTransformUnary(buf, dst, sqrt_func<double>(), stream);
    }
}

void cv::cuda::calcNormDiff(InputArray _src1, InputArray _src2, OutputArray _dst, int normType, Stream& stream)
{
    typedef void (*func_t)(const GpuMat& _src1, const GpuMat& _src2, GpuMat& _dst, Stream& stream);
    static const func_t funcs[] =
    {
        0, normDiffInf, normDiffL1, 0, normDiffL2
    };

    GpuMat src1 = getInputMat(_src1, stream);
    GpuMat src2 = getInputMat(_src2, stream);

    CV_Assert( src1.type() == CV_8UC1 );
    CV_Assert( src1.size() == src2.size() && src1.type() == src2.type() );
    CV_Assert( normType == NORM_INF || normType == NORM_L1 || normType == NORM_L2 );

    GpuMat dst = getOutputMat(_dst, 1, 1, normType == NORM_L2 ? CV_64FC1 : CV_32SC1, stream);

    const func_t func = funcs[normType];
    func(src1, src2, dst, stream);

    syncOutput(dst, _dst, stream);
}

double cv::cuda::norm(InputArray _src1, InputArray _src2, int normType)
{
    Stream& stream = Stream::Null();

    HostMem dst;
    calcNormDiff(_src1, _src2, dst, normType, stream);

    stream.waitForCompletion();

    double val;
    dst.createMatHeader().convertTo(Mat(1, 1, CV_64FC1, &val), CV_64F);

    return val;
}

namespace cv { namespace cuda { namespace device {

void normL2(cv::InputArray _src, cv::OutputArray _dst, cv::InputArray _mask, Stream& stream);

}}}

namespace
{
    template <typename T, typename R>
    void normL2Impl(const GpuMat& _src, const GpuMat& mask, GpuMat& _dst, Stream& stream)
    {
        const GpuMat_<T>& src = (const GpuMat_<T>&) _src;
        GpuMat_<R>& dst = (GpuMat_<R>&) _dst;

        BufferPool pool(stream);
        GpuMat_<double> buf(1, 1, pool.getAllocator());

        if (mask.empty())
        {
            gridCalcSum(sqr_(cvt_<double>(src)), buf, stream);
        }
        else
        {
            gridCalcSum(sqr_(cvt_<double>(src)), buf, globPtr<uchar>(mask), stream);
        }

        gridTransformUnary(buf, dst, sqrt_func<double>(), stream);
    }
}

void cv::cuda::device::normL2(InputArray _src, OutputArray _dst, InputArray _mask, Stream& stream)
{
    typedef void (*func_t)(const GpuMat& _src, const GpuMat& mask, GpuMat& _dst, Stream& stream);
    static const func_t funcs[CV_DEPTH_MAX] =
    {
        normL2Impl<uchar, double>,
        normL2Impl<schar, double>,
        normL2Impl<ushort, double>,
        normL2Impl<short, double>,
        normL2Impl<int, double>,
        normL2Impl<float, double>,
        normL2Impl<double, double>
    };

    const GpuMat src = getInputMat(_src, stream);
    const GpuMat mask = getInputMat(_mask, stream);

    CV_Assert( src.channels() == 1 );
    CV_Assert( mask.empty() || (mask.size() == src.size() && mask.type() == CV_8U) );

    GpuMat dst = getOutputMat(_dst, 1, 1, CV_64FC1, stream);

    const func_t func = funcs[src.depth()];
    CV_Assert(func);
    func(src, mask, dst, stream);

    syncOutput(dst, _dst, stream);
}

#endif
