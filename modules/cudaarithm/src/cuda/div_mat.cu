#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "opencv2/opencv_modules.hpp"

#ifndef HAVE_OPENCV_CUDEV

#error "opencv_cudev is required"

#else

#include "opencv2/cudev.hpp"

using namespace cv::cudev;

void divMat(const GpuMat& src1, const GpuMat& src2, GpuMat& dst, const GpuMat&, double scale, Stream& stream, int);
void divMat_8uc4_32f(const GpuMat& src1, const GpuMat& src2, GpuMat& dst, Stream& stream);
void divMat_16sc4_32f(const GpuMat& src1, const GpuMat& src2, GpuMat& dst, Stream& stream);

namespace
{
    template <typename T, typename D> struct DivOp : binary_function<T, T, D>
    {
        __device__ __forceinline__ D operator ()(T a, T b) const
        {
            return b != 0 ? saturate_cast<D>(a / b) : 0;
        }
    };
    template <typename T> struct DivOp<T, float> : binary_function<T, T, float>
    {
        __device__ __forceinline__ float operator ()(T a, T b) const
        {
            return b != 0 ? static_cast<float>(a) / b : 0.0f;
        }
    };
    template <typename T> struct DivOp<T, double> : binary_function<T, T, double>
    {
        __device__ __forceinline__ double operator ()(T a, T b) const
        {
            return b != 0 ? static_cast<double>(a) / b : 0.0;
        }
    };

    template <typename T, typename S, typename D> struct DivScaleOp : binary_function<T, T, D>
    {
        S scale;

        __device__ __forceinline__ D operator ()(T a, T b) const
        {
            return b != 0 ? saturate_cast<D>(scale * a / b) : 0;
        }
    };

    template <typename ScalarDepth> struct TransformPolicy : DefaultTransformPolicy
    {
    };
    template <> struct TransformPolicy<double> : DefaultTransformPolicy
    {
        enum {
            shift = 1
        };
    };

    template <typename T, typename S, typename D>
    void divMatImpl(const GpuMat& src1, const GpuMat& src2, const GpuMat& dst, double scale, Stream& stream)
    {
        if (scale == 1)
        {
            DivOp<T, D> op;
            gridTransformBinary_< TransformPolicy<S> >(globPtr<T>(src1), globPtr<T>(src2), globPtr<D>(dst), op, stream);
        }
        else
        {
            DivScaleOp<T, S, D> op;
            op.scale = static_cast<S>(scale);
            gridTransformBinary_< TransformPolicy<S> >(globPtr<T>(src1), globPtr<T>(src2), globPtr<D>(dst), op, stream);
        }
    }
}

void divMat(const GpuMat& src1, const GpuMat& src2, GpuMat& dst, const GpuMat&, double scale, Stream& stream, int)
{
    typedef void (*func_t)(const GpuMat& src1, const GpuMat& src2, const GpuMat& dst, double scale, Stream& stream);
    static const func_t funcs[CV_DEPTH_MAX][CV_DEPTH_MAX] =
    {
        {
            divMatImpl<uchar, float, uchar>,
            divMatImpl<uchar, float, schar>,
            divMatImpl<uchar, float, ushort>,
            divMatImpl<uchar, float, short>,
            divMatImpl<uchar, float, int>,
            divMatImpl<uchar, float, float>,
            divMatImpl<uchar, double, double>
        },
        {
            divMatImpl<schar, float, uchar>,
            divMatImpl<schar, float, schar>,
            divMatImpl<schar, float, ushort>,
            divMatImpl<schar, float, short>,
            divMatImpl<schar, float, int>,
            divMatImpl<schar, float, float>,
            divMatImpl<schar, double, double>
        },
        {
            0 /*divMatImpl<ushort, float, uchar>*/,
            0 /*divMatImpl<ushort, float, schar>*/,
            divMatImpl<ushort, float, ushort>,
            divMatImpl<ushort, float, short>,
            divMatImpl<ushort, float, int>,
            divMatImpl<ushort, float, float>,
            divMatImpl<ushort, double, double>
        },
        {
            0 /*divMatImpl<short, float, uchar>*/,
            0 /*divMatImpl<short, float, schar>*/,
            divMatImpl<short, float, ushort>,
            divMatImpl<short, float, short>,
            divMatImpl<short, float, int>,
            divMatImpl<short, float, float>,
            divMatImpl<short, double, double>
        },
        {
            0 /*divMatImpl<int, float, uchar>*/,
            0 /*divMatImpl<int, float, schar>*/,
            0 /*divMatImpl<int, float, ushort>*/,
            0 /*divMatImpl<int, float, short>*/,
            divMatImpl<int, float, int>,
            divMatImpl<int, float, float>,
            divMatImpl<int, double, double>
        },
        {
            0 /*divMatImpl<float, float, uchar>*/,
            0 /*divMatImpl<float, float, schar>*/,
            0 /*divMatImpl<float, float, ushort>*/,
            0 /*divMatImpl<float, float, short>*/,
            0 /*divMatImpl<float, float, int>*/,
            divMatImpl<float, float, float>,
            divMatImpl<float, double, double>
        },
        {
            0 /*divMatImpl<double, double, uchar>*/,
            0 /*divMatImpl<double, double, schar>*/,
            0 /*divMatImpl<double, double, ushort>*/,
            0 /*divMatImpl<double, double, short>*/,
            0 /*divMatImpl<double, double, int>*/,
            0 /*divMatImpl<double, double, float>*/,
            divMatImpl<double, double, double>
        }
    };

    const int sdepth = src1.depth();
    const int ddepth = dst.depth();

    GpuMat src1_ = src1.reshape(1);
    GpuMat src2_ = src2.reshape(1);
    GpuMat dst_ = dst.reshape(1);

    const func_t func = funcs[sdepth][ddepth];

    if (!func)
        CV_Error(cv::Error::StsUnsupportedFormat, "Unsupported combination of source and destination types");

    func(src1_, src2_, dst_, scale, stream);
}

namespace
{
    template <typename T>
    struct DivOpSpecial : binary_function<T, float, T>
    {
        __device__ __forceinline__ T operator ()(const T& a, float b) const
        {
            typedef typename VecTraits<T>::elem_type elem_type;

            T res = VecTraits<T>::all(0);

            if (b != 0)
            {
                b = 1.0f / b;
                res.x = saturate_cast<elem_type>(a.x * b);
                res.y = saturate_cast<elem_type>(a.y * b);
                res.z = saturate_cast<elem_type>(a.z * b);
                res.w = saturate_cast<elem_type>(a.w * b);
            }

            return res;
        }
    };
}

void divMat_8uc4_32f(const GpuMat& src1, const GpuMat& src2, GpuMat& dst, Stream& stream)
{
    gridTransformBinary(globPtr<uchar4>(src1), globPtr<float>(src2), globPtr<uchar4>(dst), DivOpSpecial<uchar4>(), stream);
}

void divMat_16sc4_32f(const GpuMat& src1, const GpuMat& src2, GpuMat& dst, Stream& stream)
{
    gridTransformBinary(globPtr<short4>(src1), globPtr<float>(src2), globPtr<short4>(dst), DivOpSpecial<short4>(), stream);
}

#endif
