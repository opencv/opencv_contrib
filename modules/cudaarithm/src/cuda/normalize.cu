#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "opencv2/opencv_modules.hpp"

#ifndef HAVE_OPENCV_CUDEV

#error "opencv_cudev is required"

#else

#include "opencv2/cudaarithm.hpp"
#include "opencv2/cudev.hpp"
#include "opencv2/core/private.cuda.hpp"

using namespace cv;
using namespace cv::cuda;
using namespace cv::cudev;

namespace {

template <typename T, typename R, typename I>
struct ConvertorMinMax : unary_function<T, R>
{
    typedef typename LargerType<T, R>::type larger_type1;
    typedef typename LargerType<larger_type1, I>::type larger_type2;
    typedef typename LargerType<larger_type2, float>::type scalar_type;

    scalar_type dmin, dmax;
    const I* minMaxVals;

    __device__ R operator ()(typename TypeTraits<T>::parameter_type src) const
    {
        const scalar_type smin = minMaxVals[0];
        const scalar_type smax = minMaxVals[1];

        const scalar_type scale = (dmax - dmin) * (smax - smin > numeric_limits<scalar_type>::epsilon() ? 1.0 / (smax - smin) : 0.0);
        const scalar_type shift = dmin - smin * scale;

        return cudev::saturate_cast<R>(scale * src + shift);
    }
};

template <typename T, typename R, typename I>
void normalizeMinMax(const GpuMat& _src, GpuMat& _dst, double a, double b, const GpuMat& mask, Stream& stream)
{
    const GpuMat_<T>& src = (const GpuMat_<T>&)_src;
    GpuMat_<R>& dst = (GpuMat_<R>&)_dst;

    BufferPool pool(stream);
    GpuMat_<I> minMaxVals(1, 2, pool.getAllocator());

    if (mask.empty())
    {
        gridFindMinMaxVal(src, minMaxVals, stream);
    }
    else
    {
        gridFindMinMaxVal(src, minMaxVals, globPtr<uchar>(mask), stream);
    }

    ConvertorMinMax<T, R, I> cvt;
    cvt.dmin = std::min(a, b);
    cvt.dmax = std::max(a, b);
    cvt.minMaxVals = minMaxVals[0];

    if (mask.empty())
    {
        gridTransformUnary(src, dst, cvt, stream);
    }
    else
    {
        dst.setTo(Scalar::all(0), stream);
        gridTransformUnary(src, dst, cvt, globPtr<uchar>(mask), stream);
    }
}

template <typename T, typename R, typename I, bool normL2>
struct ConvertorNorm : unary_function<T, R>
{
    typedef typename LargerType<T, R>::type larger_type1;
    typedef typename LargerType<larger_type1, I>::type larger_type2;
    typedef typename LargerType<larger_type2, float>::type scalar_type;

    scalar_type a;
    const I* normVal;

    __device__ R operator ()(typename TypeTraits<T>::parameter_type src) const
    {
        sqrt_func<scalar_type> sqrt;

        scalar_type scale = normL2 ? sqrt(*normVal) : *normVal;
        scale = scale > numeric_limits<scalar_type>::epsilon() ? a / scale : 0.0;

        return cudev::saturate_cast<R>(scale * src);
    }
};

template <typename T, typename R, typename I>
void normalizeNorm(const GpuMat& _src, GpuMat& _dst, double a, int normType, const GpuMat& mask, Stream& stream)
{
    const GpuMat_<T>& src = (const GpuMat_<T>&)_src;
    GpuMat_<R>& dst = (GpuMat_<R>&)_dst;

    BufferPool pool(stream);
    GpuMat_<I> normVal(1, 1, pool.getAllocator());

    if (normType == NORM_L1)
    {
        if (mask.empty())
        {
            gridCalcSum(abs_(cvt_<I>(src)), normVal, stream);
        }
        else
        {
            gridCalcSum(abs_(cvt_<I>(src)), normVal, globPtr<uchar>(mask), stream);
        }
    }
    else if (normType == NORM_L2)
    {
        if (mask.empty())
        {
            gridCalcSum(sqr_(cvt_<I>(src)), normVal, stream);
        }
        else
        {
            gridCalcSum(sqr_(cvt_<I>(src)), normVal, globPtr<uchar>(mask), stream);
        }
    }
    else // NORM_INF
    {
        if (mask.empty())
        {
            gridFindMaxVal(abs_(cvt_<I>(src)), normVal, stream);
        }
        else
        {
            gridFindMaxVal(abs_(cvt_<I>(src)), normVal, globPtr<uchar>(mask), stream);
        }
    }

    if (normType == NORM_L2)
    {
        ConvertorNorm<T, R, I, true> cvt;
        cvt.a = a;
        cvt.normVal = normVal[0];

        if (mask.empty())
        {
            gridTransformUnary(src, dst, cvt, stream);
        }
        else
        {
            dst.setTo(Scalar::all(0), stream);
            gridTransformUnary(src, dst, cvt, globPtr<uchar>(mask), stream);
        }
    }
    else
    {
        ConvertorNorm<T, R, I, false> cvt;
        cvt.a = a;
        cvt.normVal = normVal[0];

        if (mask.empty())
        {
            gridTransformUnary(src, dst, cvt, stream);
        }
        else
        {
            dst.setTo(Scalar::all(0), stream);
            gridTransformUnary(src, dst, cvt, globPtr<uchar>(mask), stream);
        }
    }
}

} // namespace

void cv::cuda::normalize(InputArray _src, OutputArray _dst, double a, double b, int normType, int dtype, InputArray _mask, Stream& stream)
{
    typedef void (*func_minmax_t)(const GpuMat& _src, GpuMat& _dst, double a, double b, const GpuMat& mask, Stream& stream);
    typedef void (*func_norm_t)(const GpuMat& _src, GpuMat& _dst, double a, int normType, const GpuMat& mask, Stream& stream);

    static const func_minmax_t funcs_minmax[CV_DEPTH_MAX] =
    {
        normalizeMinMax<uchar, float, float>,
        normalizeMinMax<schar, float, float>,
        normalizeMinMax<ushort, float, float>,
        normalizeMinMax<short, float, float>,
        normalizeMinMax<int, float, float>,
        normalizeMinMax<float, float, float>,
        normalizeMinMax<double, double, double>
    };

    static const func_norm_t funcs_norm[CV_DEPTH_MAX] =
    {
        normalizeNorm<uchar, float, float>,
        normalizeNorm<schar, float, float>,
        normalizeNorm<ushort, float, float>,
        normalizeNorm<short, float, float>,
        normalizeNorm<int, float, float>,
        normalizeNorm<float, float, float>,
        normalizeNorm<double, double, double>
    };

    CV_Assert( normType == NORM_INF || normType == NORM_L1 || normType == NORM_L2 || normType == NORM_MINMAX );

    const GpuMat src = getInputMat(_src, stream);
    const GpuMat mask = getInputMat(_mask, stream);

    CV_Assert( src.channels() == 1 );
    CV_Assert( mask.empty() || (mask.size() == src.size() && mask.type() == CV_8U) );

    if (dtype < 0)
    {
        dtype = _dst.fixedType() ? _dst.type() : src.type();
    }
    dtype = CV_MAT_DEPTH(dtype);

    const int src_depth = src.depth();
    const int tmp_depth = src_depth <= CV_32F ? CV_32F : src_depth;

    GpuMat dst;
    if (dtype == tmp_depth)
    {
        _dst.create(src.size(), tmp_depth);
        dst = getOutputMat(_dst, src.size(), tmp_depth, stream);
    }
    else
    {
        BufferPool pool(stream);
        dst = pool.getBuffer(src.size(), tmp_depth);
    }

    if (normType == NORM_MINMAX)
    {
        const func_minmax_t func = funcs_minmax[src_depth];
        CV_Assert(func);
        func(src, dst, a, b, mask, stream);
    }
    else
    {
        const func_norm_t func = funcs_norm[src_depth];
        CV_Assert(func);
        func(src, dst, a, normType, mask, stream);
    }

    if (dtype == tmp_depth)
    {
        syncOutput(dst, _dst, stream);
    }
    else
    {
        dst.convertTo(_dst, dtype, stream);
    }
}

#endif
