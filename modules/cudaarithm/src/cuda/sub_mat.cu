/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "opencv2/opencv_modules.hpp"

#ifndef HAVE_OPENCV_CUDEV

#error "opencv_cudev is required"

#else

#include "opencv2/cudev.hpp"

using namespace cv::cudev;

void subMat(const GpuMat& src1, const GpuMat& src2, GpuMat& dst, const GpuMat& mask, double, Stream& _stream, int);

namespace
{
    template <typename T, typename D> struct SubOp1 : binary_function<T, T, D>
    {
        __device__ __forceinline__ D operator ()(T a, T b) const
        {
            return saturate_cast<D>(a - b);
        }
    };

    template <typename T, typename D>
    void subMat_v1(const GpuMat& src1, const GpuMat& src2, GpuMat& dst, const GpuMat& mask, Stream& stream)
    {
        if (mask.data)
            gridTransformBinary(globPtr<T>(src1), globPtr<T>(src2), globPtr<D>(dst), SubOp1<T, D>(), globPtr<uchar>(mask), stream);
        else
            gridTransformBinary(globPtr<T>(src1), globPtr<T>(src2), globPtr<D>(dst), SubOp1<T, D>(), stream);
    }

    struct SubOp2 : binary_function<uint, uint, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, uint b) const
        {
            return vsub2(a, b);
        }
    };

    void subMat_v2(const GpuMat& src1, const GpuMat& src2, GpuMat& dst, Stream& stream)
    {
        const int vcols = src1.cols >> 1;

        GlobPtrSz<uint> src1_ = globPtr((uint*) src1.data, src1.step, src1.rows, vcols);
        GlobPtrSz<uint> src2_ = globPtr((uint*) src2.data, src2.step, src1.rows, vcols);
        GlobPtrSz<uint> dst_ = globPtr((uint*) dst.data, dst.step, src1.rows, vcols);

        gridTransformBinary(src1_, src2_, dst_, SubOp2(), stream);
    }

    struct SubOp4 : binary_function<uint, uint, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, uint b) const
        {
            return vsub4(a, b);
        }
    };

    void subMat_v4(const GpuMat& src1, const GpuMat& src2, GpuMat& dst, Stream& stream)
    {
        const int vcols = src1.cols >> 2;

        GlobPtrSz<uint> src1_ = globPtr((uint*) src1.data, src1.step, src1.rows, vcols);
        GlobPtrSz<uint> src2_ = globPtr((uint*) src2.data, src2.step, src1.rows, vcols);
        GlobPtrSz<uint> dst_ = globPtr((uint*) dst.data, dst.step, src1.rows, vcols);

        gridTransformBinary(src1_, src2_, dst_, SubOp4(), stream);
    }
}

void subMat(const GpuMat& src1, const GpuMat& src2, GpuMat& dst, const GpuMat& mask, double, Stream& stream, int)
{
    typedef void (*func_t)(const GpuMat& src1, const GpuMat& src2, GpuMat& dst, const GpuMat& mask, Stream& stream);
    static const func_t funcs[7][7] =
    {
        {
            subMat_v1<uchar, uchar>,
            subMat_v1<uchar, schar>,
            subMat_v1<uchar, ushort>,
            subMat_v1<uchar, short>,
            subMat_v1<uchar, int>,
            subMat_v1<uchar, float>,
            subMat_v1<uchar, double>
        },
        {
            subMat_v1<schar, uchar>,
            subMat_v1<schar, schar>,
            subMat_v1<schar, ushort>,
            subMat_v1<schar, short>,
            subMat_v1<schar, int>,
            subMat_v1<schar, float>,
            subMat_v1<schar, double>
        },
        {
            0 /*subMat_v1<ushort, uchar>*/,
            0 /*subMat_v1<ushort, schar>*/,
            subMat_v1<ushort, ushort>,
            subMat_v1<ushort, short>,
            subMat_v1<ushort, int>,
            subMat_v1<ushort, float>,
            subMat_v1<ushort, double>
        },
        {
            0 /*subMat_v1<short, uchar>*/,
            0 /*subMat_v1<short, schar>*/,
            subMat_v1<short, ushort>,
            subMat_v1<short, short>,
            subMat_v1<short, int>,
            subMat_v1<short, float>,
            subMat_v1<short, double>
        },
        {
            0 /*subMat_v1<int, uchar>*/,
            0 /*subMat_v1<int, schar>*/,
            0 /*subMat_v1<int, ushort>*/,
            0 /*subMat_v1<int, short>*/,
            subMat_v1<int, int>,
            subMat_v1<int, float>,
            subMat_v1<int, double>
        },
        {
            0 /*subMat_v1<float, uchar>*/,
            0 /*subMat_v1<float, schar>*/,
            0 /*subMat_v1<float, ushort>*/,
            0 /*subMat_v1<float, short>*/,
            0 /*subMat_v1<float, int>*/,
            subMat_v1<float, float>,
            subMat_v1<float, double>
        },
        {
            0 /*subMat_v1<double, uchar>*/,
            0 /*subMat_v1<double, schar>*/,
            0 /*subMat_v1<double, ushort>*/,
            0 /*subMat_v1<double, short>*/,
            0 /*subMat_v1<double, int>*/,
            0 /*subMat_v1<double, float>*/,
            subMat_v1<double, double>
        }
    };

    const int sdepth = src1.depth();
    const int ddepth = dst.depth();

    CV_Assert( sdepth <= CV_64F && ddepth <= CV_64F );

    GpuMat src1_ = src1.reshape(1);
    GpuMat src2_ = src2.reshape(1);
    GpuMat dst_ = dst.reshape(1);

    if (mask.empty() && (sdepth == CV_8U || sdepth == CV_16U) && ddepth == sdepth)
    {
        const intptr_t src1ptr = reinterpret_cast<intptr_t>(src1_.data);
        const intptr_t src2ptr = reinterpret_cast<intptr_t>(src2_.data);
        const intptr_t dstptr = reinterpret_cast<intptr_t>(dst_.data);

        const bool isAllAligned = (src1ptr & 31) == 0 && (src2ptr & 31) == 0 && (dstptr & 31) == 0;

        if (isAllAligned)
        {
            if (sdepth == CV_8U && (src1_.cols & 3) == 0)
            {
                subMat_v4(src1_, src2_, dst_, stream);
                return;
            }
            else if (sdepth == CV_16U && (src1_.cols & 1) == 0)
            {
                subMat_v2(src1_, src2_, dst_, stream);
                return;
            }
        }
    }

    const func_t func = funcs[sdepth][ddepth];

    if (!func)
        CV_Error(cv::Error::StsUnsupportedFormat, "Unsupported combination of source and destination types");

    func(src1_, src2_, dst_, mask, stream);
}

#endif
