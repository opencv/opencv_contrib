#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "opencv2/opencv_modules.hpp"

#ifndef HAVE_OPENCV_CUDEV

#error "opencv_cudev is required"

#else

#include "opencv2/cudev.hpp"

using namespace cv::cudev;

void absDiffMat(const GpuMat& src1, const GpuMat& src2, GpuMat& dst, const GpuMat&, double, Stream& stream, int);

namespace
{
    __device__ __forceinline__ int _abs(int a)
    {
        return ::abs(a);
    }
    __device__ __forceinline__ float _abs(float a)
    {
        return ::fabsf(a);
    }
    __device__ __forceinline__ double _abs(double a)
    {
        return ::fabs(a);
    }

    template <typename T> struct AbsDiffOp1 : binary_function<T, T, T>
    {
        __device__ __forceinline__ T operator ()(T a, T b) const
        {
            return saturate_cast<T>(_abs(a - b));
        }
    };

    template <typename ScalarDepth> struct TransformPolicy : DefaultTransformPolicy
    {
    };
    template <> struct TransformPolicy<double> : DefaultTransformPolicy
    {
        enum {
            shift = 1
        };
    };

    template <typename T>
    void absDiffMat_v1(const GpuMat& src1, const GpuMat& src2, GpuMat& dst, Stream& stream)
    {
        gridTransformBinary_< TransformPolicy<T> >(globPtr<T>(src1), globPtr<T>(src2), globPtr<T>(dst), AbsDiffOp1<T>(), stream);
    }

    struct AbsDiffOp2 : binary_function<uint, uint, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, uint b) const
        {
            return vabsdiff2(a, b);
        }
    };

    void absDiffMat_v2(const GpuMat& src1, const GpuMat& src2, GpuMat& dst, Stream& stream)
    {
        const int vcols = src1.cols >> 1;

        GlobPtrSz<uint> src1_ = globPtr((uint*) src1.data, src1.step, src1.rows, vcols);
        GlobPtrSz<uint> src2_ = globPtr((uint*) src2.data, src2.step, src1.rows, vcols);
        GlobPtrSz<uint> dst_ = globPtr((uint*) dst.data, dst.step, src1.rows, vcols);

        gridTransformBinary(src1_, src2_, dst_, AbsDiffOp2(), stream);
    }

    struct AbsDiffOp4 : binary_function<uint, uint, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, uint b) const
        {
            return vabsdiff4(a, b);
        }
    };

    void absDiffMat_v4(const GpuMat& src1, const GpuMat& src2, GpuMat& dst, Stream& stream)
    {
        const int vcols = src1.cols >> 2;

        GlobPtrSz<uint> src1_ = globPtr((uint*) src1.data, src1.step, src1.rows, vcols);
        GlobPtrSz<uint> src2_ = globPtr((uint*) src2.data, src2.step, src1.rows, vcols);
        GlobPtrSz<uint> dst_ = globPtr((uint*) dst.data, dst.step, src1.rows, vcols);

        gridTransformBinary(src1_, src2_, dst_, AbsDiffOp4(), stream);
    }
}

void absDiffMat(const GpuMat& src1, const GpuMat& src2, GpuMat& dst, const GpuMat&, double, Stream& stream, int)
{
    typedef void (*func_t)(const GpuMat& src1, const GpuMat& src2, GpuMat& dst, Stream& stream);
    static const func_t funcs[CV_DEPTH_MAX] =
    {
        absDiffMat_v1<uchar>,
        absDiffMat_v1<schar>,
        absDiffMat_v1<ushort>,
        absDiffMat_v1<short>,
        absDiffMat_v1<int>,
        absDiffMat_v1<float>,
        absDiffMat_v1<double>
    };

    const int depth = src1.depth();

    CV_DbgAssert( depth <= CV_64F );

    GpuMat src1_ = src1.reshape(1);
    GpuMat src2_ = src2.reshape(1);
    GpuMat dst_ = dst.reshape(1);

    if (depth == CV_8U || depth == CV_16U)
    {
        const intptr_t src1ptr = reinterpret_cast<intptr_t>(src1_.data);
        const intptr_t src2ptr = reinterpret_cast<intptr_t>(src2_.data);
        const intptr_t dstptr = reinterpret_cast<intptr_t>(dst_.data);

        const bool isAllAligned = (src1ptr & 31) == 0 && (src2ptr & 31) == 0 && (dstptr & 31) == 0;

        if (isAllAligned)
        {
            if (depth == CV_8U && (src1_.cols & 3) == 0)
            {
                absDiffMat_v4(src1_, src2_, dst_, stream);
                return;
            }
            else if (depth == CV_16U && (src1_.cols & 1) == 0)
            {
                absDiffMat_v2(src1_, src2_, dst_, stream);
                return;
            }
        }
    }

    const func_t func = funcs[depth];

    if (!func)
        CV_Error(cv::Error::StsUnsupportedFormat, "Unsupported combination of source and destination types");

    func(src1_, src2_, dst_, stream);
}

#endif
