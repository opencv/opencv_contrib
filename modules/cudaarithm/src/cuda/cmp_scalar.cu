/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "opencv2/opencv_modules.hpp"

#ifndef HAVE_OPENCV_CUDEV

#error "opencv_cudev is required"

#else

#include "opencv2/cudev.hpp"

using namespace cv::cudev;

void cmpScalar(const GpuMat& src, cv::Scalar val, bool inv, GpuMat& dst, const GpuMat&, double, Stream& stream, int cmpop);

namespace
{
    template <class Op, typename T> struct CmpOp : binary_function<T, T, uchar>
    {
        __device__ __forceinline__ uchar operator()(T a, T b) const
        {
            Op op;
            return -op(a, b);
        }
    };

#define MAKE_VEC(_type, _cn) typename MakeVec<_type, _cn>::type

    template <class Op, typename T, int cn> struct CmpScalarOp;

    template <class Op, typename T>
    struct CmpScalarOp<Op, T, 1> : unary_function<T, uchar>
    {
        T val;

        __device__ __forceinline__ uchar operator()(T src) const
        {
            CmpOp<Op, T> op;
            return op(src, val);
        }
    };

    template <class Op, typename T>
    struct CmpScalarOp<Op, T, 2> : unary_function<MAKE_VEC(T, 2), MAKE_VEC(uchar, 2)>
    {
        MAKE_VEC(T, 2) val;

        __device__ __forceinline__ MAKE_VEC(uchar, 2) operator()(const MAKE_VEC(T, 2) & src) const
        {
            CmpOp<Op, T> op;
            return VecTraits<MAKE_VEC(uchar, 2)>::make(op(src.x, val.x), op(src.y, val.y));
        }
    };

    template <class Op, typename T>
    struct CmpScalarOp<Op, T, 3> : unary_function<MAKE_VEC(T, 3), MAKE_VEC(uchar, 3)>
    {
        MAKE_VEC(T, 3) val;

        __device__ __forceinline__ MAKE_VEC(uchar, 3) operator()(const MAKE_VEC(T, 3) & src) const
        {
            CmpOp<Op, T> op;
            return VecTraits<MAKE_VEC(uchar, 3)>::make(op(src.x, val.x), op(src.y, val.y), op(src.z, val.z));
        }
    };

    template <class Op, typename T>
    struct CmpScalarOp<Op, T, 4> : unary_function<MAKE_VEC(T, 4), MAKE_VEC(uchar, 4)>
    {
        MAKE_VEC(T, 4) val;

        __device__ __forceinline__ MAKE_VEC(uchar, 4) operator()(const MAKE_VEC(T, 4) & src) const
        {
            CmpOp<Op, T> op;
            return VecTraits<MAKE_VEC(uchar, 4)>::make(op(src.x, val.x), op(src.y, val.y), op(src.z, val.z), op(src.w, val.w));
        }
    };

#undef TYPE_VEC

    template <typename ScalarDepth> struct TransformPolicy : DefaultTransformPolicy
    {
    };
    template <> struct TransformPolicy<double> : DefaultTransformPolicy
    {
        enum {
            shift = 1
        };
    };

    template <template <typename> class Op, typename T, int cn>
    void cmpScalarImpl(const GpuMat& src, cv::Scalar value, GpuMat& dst, Stream& stream)
    {
        typedef typename MakeVec<T, cn>::type src_type;
        typedef typename MakeVec<uchar, cn>::type dst_type;

        cv::Scalar_<T> value_ = value;

        CmpScalarOp<Op<T>, T, cn> op;
        op.val = VecTraits<src_type>::make(value_.val);

        gridTransformUnary_< TransformPolicy<T> >(globPtr<src_type>(src), globPtr<dst_type>(dst), op, stream);
    }
}

void cmpScalar(const GpuMat& src, cv::Scalar val, bool inv, GpuMat& dst, const GpuMat&, double, Stream& stream, int cmpop)
{
    typedef void (*func_t)(const GpuMat& src, cv::Scalar value, GpuMat& dst, Stream& stream);
    static const func_t funcs[7][6][4] =
    {
        {
            {cmpScalarImpl<equal_to,      uchar, 1>, cmpScalarImpl<equal_to,      uchar, 2>, cmpScalarImpl<equal_to,      uchar, 3>, cmpScalarImpl<equal_to,      uchar, 4>},
            {cmpScalarImpl<greater,       uchar, 1>, cmpScalarImpl<greater,       uchar, 2>, cmpScalarImpl<greater,       uchar, 3>, cmpScalarImpl<greater,       uchar, 4>},
            {cmpScalarImpl<greater_equal, uchar, 1>, cmpScalarImpl<greater_equal, uchar, 2>, cmpScalarImpl<greater_equal, uchar, 3>, cmpScalarImpl<greater_equal, uchar, 4>},
            {cmpScalarImpl<less,          uchar, 1>, cmpScalarImpl<less,          uchar, 2>, cmpScalarImpl<less,          uchar, 3>, cmpScalarImpl<less,          uchar, 4>},
            {cmpScalarImpl<less_equal,    uchar, 1>, cmpScalarImpl<less_equal,    uchar, 2>, cmpScalarImpl<less_equal,    uchar, 3>, cmpScalarImpl<less_equal,    uchar, 4>},
            {cmpScalarImpl<not_equal_to,  uchar, 1>, cmpScalarImpl<not_equal_to,  uchar, 2>, cmpScalarImpl<not_equal_to,  uchar, 3>, cmpScalarImpl<not_equal_to,  uchar, 4>}
        },
        {
            {cmpScalarImpl<equal_to,      schar, 1>, cmpScalarImpl<equal_to,      schar, 2>, cmpScalarImpl<equal_to,      schar, 3>, cmpScalarImpl<equal_to,      schar, 4>},
            {cmpScalarImpl<greater,       schar, 1>, cmpScalarImpl<greater,       schar, 2>, cmpScalarImpl<greater,       schar, 3>, cmpScalarImpl<greater,       schar, 4>},
            {cmpScalarImpl<greater_equal, schar, 1>, cmpScalarImpl<greater_equal, schar, 2>, cmpScalarImpl<greater_equal, schar, 3>, cmpScalarImpl<greater_equal, schar, 4>},
            {cmpScalarImpl<less,          schar, 1>, cmpScalarImpl<less,          schar, 2>, cmpScalarImpl<less,          schar, 3>, cmpScalarImpl<less,          schar, 4>},
            {cmpScalarImpl<less_equal,    schar, 1>, cmpScalarImpl<less_equal,    schar, 2>, cmpScalarImpl<less_equal,    schar, 3>, cmpScalarImpl<less_equal,    schar, 4>},
            {cmpScalarImpl<not_equal_to,  schar, 1>, cmpScalarImpl<not_equal_to,  schar, 2>, cmpScalarImpl<not_equal_to,  schar, 3>, cmpScalarImpl<not_equal_to,  schar, 4>}
        },
        {
            {cmpScalarImpl<equal_to,      ushort, 1>, cmpScalarImpl<equal_to,      ushort, 2>, cmpScalarImpl<equal_to,      ushort, 3>, cmpScalarImpl<equal_to,      ushort, 4>},
            {cmpScalarImpl<greater,       ushort, 1>, cmpScalarImpl<greater,       ushort, 2>, cmpScalarImpl<greater,       ushort, 3>, cmpScalarImpl<greater,       ushort, 4>},
            {cmpScalarImpl<greater_equal, ushort, 1>, cmpScalarImpl<greater_equal, ushort, 2>, cmpScalarImpl<greater_equal, ushort, 3>, cmpScalarImpl<greater_equal, ushort, 4>},
            {cmpScalarImpl<less,          ushort, 1>, cmpScalarImpl<less,          ushort, 2>, cmpScalarImpl<less,          ushort, 3>, cmpScalarImpl<less,          ushort, 4>},
            {cmpScalarImpl<less_equal,    ushort, 1>, cmpScalarImpl<less_equal,    ushort, 2>, cmpScalarImpl<less_equal,    ushort, 3>, cmpScalarImpl<less_equal,    ushort, 4>},
            {cmpScalarImpl<not_equal_to,  ushort, 1>, cmpScalarImpl<not_equal_to,  ushort, 2>, cmpScalarImpl<not_equal_to,  ushort, 3>, cmpScalarImpl<not_equal_to,  ushort, 4>}
        },
        {
            {cmpScalarImpl<equal_to,      short, 1>, cmpScalarImpl<equal_to,      short, 2>, cmpScalarImpl<equal_to,      short, 3>, cmpScalarImpl<equal_to,      short, 4>},
            {cmpScalarImpl<greater,       short, 1>, cmpScalarImpl<greater,       short, 2>, cmpScalarImpl<greater,       short, 3>, cmpScalarImpl<greater,       short, 4>},
            {cmpScalarImpl<greater_equal, short, 1>, cmpScalarImpl<greater_equal, short, 2>, cmpScalarImpl<greater_equal, short, 3>, cmpScalarImpl<greater_equal, short, 4>},
            {cmpScalarImpl<less,          short, 1>, cmpScalarImpl<less,          short, 2>, cmpScalarImpl<less,          short, 3>, cmpScalarImpl<less,          short, 4>},
            {cmpScalarImpl<less_equal,    short, 1>, cmpScalarImpl<less_equal,    short, 2>, cmpScalarImpl<less_equal,    short, 3>, cmpScalarImpl<less_equal,    short, 4>},
            {cmpScalarImpl<not_equal_to,  short, 1>, cmpScalarImpl<not_equal_to,  short, 2>, cmpScalarImpl<not_equal_to,  short, 3>, cmpScalarImpl<not_equal_to,  short, 4>}
        },
        {
            {cmpScalarImpl<equal_to,      int, 1>, cmpScalarImpl<equal_to,      int, 2>, cmpScalarImpl<equal_to,      int, 3>, cmpScalarImpl<equal_to,      int, 4>},
            {cmpScalarImpl<greater,       int, 1>, cmpScalarImpl<greater,       int, 2>, cmpScalarImpl<greater,       int, 3>, cmpScalarImpl<greater,       int, 4>},
            {cmpScalarImpl<greater_equal, int, 1>, cmpScalarImpl<greater_equal, int, 2>, cmpScalarImpl<greater_equal, int, 3>, cmpScalarImpl<greater_equal, int, 4>},
            {cmpScalarImpl<less,          int, 1>, cmpScalarImpl<less,          int, 2>, cmpScalarImpl<less,          int, 3>, cmpScalarImpl<less,          int, 4>},
            {cmpScalarImpl<less_equal,    int, 1>, cmpScalarImpl<less_equal,    int, 2>, cmpScalarImpl<less_equal,    int, 3>, cmpScalarImpl<less_equal,    int, 4>},
            {cmpScalarImpl<not_equal_to,  int, 1>, cmpScalarImpl<not_equal_to,  int, 2>, cmpScalarImpl<not_equal_to,  int, 3>, cmpScalarImpl<not_equal_to,  int, 4>}
        },
        {
            {cmpScalarImpl<equal_to,      float, 1>, cmpScalarImpl<equal_to,      float, 2>, cmpScalarImpl<equal_to,      float, 3>, cmpScalarImpl<equal_to,      float, 4>},
            {cmpScalarImpl<greater,       float, 1>, cmpScalarImpl<greater,       float, 2>, cmpScalarImpl<greater,       float, 3>, cmpScalarImpl<greater,       float, 4>},
            {cmpScalarImpl<greater_equal, float, 1>, cmpScalarImpl<greater_equal, float, 2>, cmpScalarImpl<greater_equal, float, 3>, cmpScalarImpl<greater_equal, float, 4>},
            {cmpScalarImpl<less,          float, 1>, cmpScalarImpl<less,          float, 2>, cmpScalarImpl<less,          float, 3>, cmpScalarImpl<less,          float, 4>},
            {cmpScalarImpl<less_equal,    float, 1>, cmpScalarImpl<less_equal,    float, 2>, cmpScalarImpl<less_equal,    float, 3>, cmpScalarImpl<less_equal,    float, 4>},
            {cmpScalarImpl<not_equal_to,  float, 1>, cmpScalarImpl<not_equal_to,  float, 2>, cmpScalarImpl<not_equal_to,  float, 3>, cmpScalarImpl<not_equal_to,  float, 4>}
        },
        {
            {cmpScalarImpl<equal_to,      double, 1>, cmpScalarImpl<equal_to,      double, 2>, cmpScalarImpl<equal_to,      double, 3>, cmpScalarImpl<equal_to,      double, 4>},
            {cmpScalarImpl<greater,       double, 1>, cmpScalarImpl<greater,       double, 2>, cmpScalarImpl<greater,       double, 3>, cmpScalarImpl<greater,       double, 4>},
            {cmpScalarImpl<greater_equal, double, 1>, cmpScalarImpl<greater_equal, double, 2>, cmpScalarImpl<greater_equal, double, 3>, cmpScalarImpl<greater_equal, double, 4>},
            {cmpScalarImpl<less,          double, 1>, cmpScalarImpl<less,          double, 2>, cmpScalarImpl<less,          double, 3>, cmpScalarImpl<less,          double, 4>},
            {cmpScalarImpl<less_equal,    double, 1>, cmpScalarImpl<less_equal,    double, 2>, cmpScalarImpl<less_equal,    double, 3>, cmpScalarImpl<less_equal,    double, 4>},
            {cmpScalarImpl<not_equal_to,  double, 1>, cmpScalarImpl<not_equal_to,  double, 2>, cmpScalarImpl<not_equal_to,  double, 3>, cmpScalarImpl<not_equal_to,  double, 4>}
        }
    };

    if (inv)
    {
        // src1 is a scalar; swap it with src2
        cmpop = cmpop == cv::CMP_LT ? cv::CMP_GT : cmpop == cv::CMP_LE ? cv::CMP_GE :
            cmpop == cv::CMP_GE ? cv::CMP_LE : cmpop == cv::CMP_GT ? cv::CMP_LT : cmpop;
    }

    const int depth = src.depth();
    const int cn = src.channels();

    CV_Assert( depth <= CV_64F && cn <= 4 );

    funcs[depth][cmpop][cn - 1](src, val, dst, stream);
}

#endif
