/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "opencv2/opencv_modules.hpp"

#ifndef HAVE_OPENCV_CUDEV

#error "opencv_cudev is required"

#else

#include "opencv2/cudev.hpp"
#include "opencv2/core/private.cuda.hpp"

using namespace cv::cudev;

void bitScalar(const GpuMat& src, cv::Scalar value, bool, GpuMat& dst, const GpuMat& mask, double, Stream& stream, int op);

namespace
{
    template <template <typename> class Op, typename T>
    void bitScalarOp(const GpuMat& src, uint value, GpuMat& dst, Stream& stream)
    {
        gridTransformUnary(globPtr<T>(src), globPtr<T>(dst), bind2nd(Op<T>(), value), stream);
    }

    typedef void (*bit_scalar_func_t)(const GpuMat& src, uint value, GpuMat& dst, Stream& stream);

    template <typename T, bit_scalar_func_t func> struct BitScalar
    {
        static void call(const GpuMat& src, cv::Scalar value, GpuMat& dst, Stream& stream)
        {
            func(src, cv::saturate_cast<T>(value[0]), dst, stream);
        }
    };

    template <bit_scalar_func_t func> struct BitScalar4
    {
        static void call(const GpuMat& src, cv::Scalar value, GpuMat& dst, Stream& stream)
        {
            uint packedVal = 0;

            packedVal |= cv::saturate_cast<uchar>(value[0]);
            packedVal |= cv::saturate_cast<uchar>(value[1]) << 8;
            packedVal |= cv::saturate_cast<uchar>(value[2]) << 16;
            packedVal |= cv::saturate_cast<uchar>(value[3]) << 24;

            func(src, packedVal, dst, stream);
        }
    };

    template <int DEPTH, int cn> struct NppBitwiseCFunc
    {
        typedef typename NPPTypeTraits<DEPTH>::npp_type npp_type;

#if USE_NPP_STREAM_CTX
        typedef NppStatus(*func_t)(const npp_type* pSrc1, int nSrc1Step, const npp_type* pConstants, npp_type* pDst, int nDstStep, NppiSize oSizeROI, NppStreamContext ctx);
#else
        typedef NppStatus (*func_t)(const npp_type* pSrc1, int nSrc1Step, const npp_type* pConstants, npp_type* pDst, int nDstStep, NppiSize oSizeROI);
#endif
    };

    template <int DEPTH, int cn, typename NppBitwiseCFunc<DEPTH, cn>::func_t func> struct NppBitwiseC
    {
        typedef typename NppBitwiseCFunc<DEPTH, cn>::npp_type npp_type;

        static void call(const GpuMat& src, cv::Scalar value, GpuMat& dst, Stream& _stream)
        {
            hipStream_t stream = StreamAccessor::getStream(_stream);
            NppStreamHandler h(stream);

            NppiSize oSizeROI;
            oSizeROI.width = src.cols;
            oSizeROI.height = src.rows;

            const npp_type pConstants[] =
            {
                cv::saturate_cast<npp_type>(value[0]),
                cv::saturate_cast<npp_type>(value[1]),
                cv::saturate_cast<npp_type>(value[2]),
                cv::saturate_cast<npp_type>(value[3])
            };

#if USE_NPP_STREAM_CTX
            nppSafeCall(func(src.ptr<npp_type>(), static_cast<int>(src.step), pConstants, dst.ptr<npp_type>(), static_cast<int>(dst.step), oSizeROI, h));
#else
            nppSafeCall( func(src.ptr<npp_type>(), static_cast<int>(src.step), pConstants, dst.ptr<npp_type>(), static_cast<int>(dst.step), oSizeROI) );
#endif

            if (stream == 0)
                CV_CUDEV_SAFE_CALL( hipDeviceSynchronize() );
        }
    };
}

void bitScalar(const GpuMat& src, cv::Scalar value, bool, GpuMat& dst, const GpuMat& mask, double, Stream& stream, int op)
{
    CV_UNUSED(mask);

    typedef void (*func_t)(const GpuMat& src, cv::Scalar value, GpuMat& dst, Stream& stream);
    static const func_t funcs[3][6][4] =
    {
#if USE_NPP_STREAM_CTX
        {
            {BitScalar<uchar, bitScalarOp<bit_and, uchar> >::call  , 0, NppBitwiseC<CV_8U , 3, nppiAndC_8u_C3R_Ctx >::call, BitScalar4< bitScalarOp<bit_and, uint> >::call},
            {BitScalar<uchar, bitScalarOp<bit_and, uchar> >::call  , 0, NppBitwiseC<CV_8U , 3, nppiAndC_8u_C3R_Ctx >::call, BitScalar4< bitScalarOp<bit_and, uint> >::call},
            {BitScalar<ushort, bitScalarOp<bit_and, ushort> >::call, 0, NppBitwiseC<CV_16U, 3, nppiAndC_16u_C3R_Ctx>::call, NppBitwiseC<CV_16U, 4, nppiAndC_16u_C4R_Ctx>::call},
            {BitScalar<ushort, bitScalarOp<bit_and, ushort> >::call, 0, NppBitwiseC<CV_16U, 3, nppiAndC_16u_C3R_Ctx>::call, NppBitwiseC<CV_16U, 4, nppiAndC_16u_C4R_Ctx>::call},
            {BitScalar<uint, bitScalarOp<bit_and, uint> >::call    , 0, NppBitwiseC<CV_32S, 3, nppiAndC_32s_C3R_Ctx>::call, NppBitwiseC<CV_32S, 4, nppiAndC_32s_C4R_Ctx>::call},
            {BitScalar<uint, bitScalarOp<bit_and, uint> >::call    , 0, NppBitwiseC<CV_32S, 3, nppiAndC_32s_C3R_Ctx>::call, NppBitwiseC<CV_32S, 4, nppiAndC_32s_C4R_Ctx>::call}
        },
        {
            {BitScalar<uchar, bitScalarOp<bit_or, uchar> >::call  , 0, NppBitwiseC<CV_8U , 3, nppiOrC_8u_C3R_Ctx >::call, BitScalar4< bitScalarOp<bit_or, uint> >::call},
            {BitScalar<uchar, bitScalarOp<bit_or, uchar> >::call  , 0, NppBitwiseC<CV_8U , 3, nppiOrC_8u_C3R_Ctx >::call, BitScalar4< bitScalarOp<bit_or, uint> >::call},
            {BitScalar<ushort, bitScalarOp<bit_or, ushort> >::call, 0, NppBitwiseC<CV_16U, 3, nppiOrC_16u_C3R_Ctx>::call, NppBitwiseC<CV_16U, 4, nppiOrC_16u_C4R_Ctx>::call},
            {BitScalar<ushort, bitScalarOp<bit_or, ushort> >::call, 0, NppBitwiseC<CV_16U, 3, nppiOrC_16u_C3R_Ctx>::call, NppBitwiseC<CV_16U, 4, nppiOrC_16u_C4R_Ctx>::call},
            {BitScalar<uint, bitScalarOp<bit_or, uint> >::call    , 0, NppBitwiseC<CV_32S, 3, nppiOrC_32s_C3R_Ctx>::call, NppBitwiseC<CV_32S, 4, nppiOrC_32s_C4R_Ctx>::call},
            {BitScalar<uint, bitScalarOp<bit_or, uint> >::call    , 0, NppBitwiseC<CV_32S, 3, nppiOrC_32s_C3R_Ctx>::call, NppBitwiseC<CV_32S, 4, nppiOrC_32s_C4R_Ctx>::call}
        },
        {
            {BitScalar<uchar, bitScalarOp<bit_xor, uchar> >::call  , 0, NppBitwiseC<CV_8U , 3, nppiXorC_8u_C3R_Ctx >::call, BitScalar4< bitScalarOp<bit_xor, uint> >::call},
            {BitScalar<uchar, bitScalarOp<bit_xor, uchar> >::call  , 0, NppBitwiseC<CV_8U , 3, nppiXorC_8u_C3R_Ctx >::call, BitScalar4< bitScalarOp<bit_xor, uint> >::call},
            {BitScalar<ushort, bitScalarOp<bit_xor, ushort> >::call, 0, NppBitwiseC<CV_16U, 3, nppiXorC_16u_C3R_Ctx>::call, NppBitwiseC<CV_16U, 4, nppiXorC_16u_C4R_Ctx>::call},
            {BitScalar<ushort, bitScalarOp<bit_xor, ushort> >::call, 0, NppBitwiseC<CV_16U, 3, nppiXorC_16u_C3R_Ctx>::call, NppBitwiseC<CV_16U, 4, nppiXorC_16u_C4R_Ctx>::call},
            {BitScalar<uint, bitScalarOp<bit_xor, uint> >::call    , 0, NppBitwiseC<CV_32S, 3, nppiXorC_32s_C3R_Ctx>::call, NppBitwiseC<CV_32S, 4, nppiXorC_32s_C4R_Ctx>::call},
            {BitScalar<uint, bitScalarOp<bit_xor, uint> >::call    , 0, NppBitwiseC<CV_32S, 3, nppiXorC_32s_C3R_Ctx>::call, NppBitwiseC<CV_32S, 4, nppiXorC_32s_C4R_Ctx>::call}
        }
#else
        {
            { BitScalar<uchar, bitScalarOp<bit_and, uchar> >::call, 0, NppBitwiseC<CV_8U, 3, nppiAndC_8u_C3R >::call, BitScalar4< bitScalarOp<bit_and, uint> >::call },
            { BitScalar<uchar, bitScalarOp<bit_and, uchar> >::call  , 0, NppBitwiseC<CV_8U , 3, nppiAndC_8u_C3R >::call, BitScalar4< bitScalarOp<bit_and, uint> >::call },
            { BitScalar<ushort, bitScalarOp<bit_and, ushort> >::call, 0, NppBitwiseC<CV_16U, 3, nppiAndC_16u_C3R>::call, NppBitwiseC<CV_16U, 4, nppiAndC_16u_C4R>::call },
            { BitScalar<ushort, bitScalarOp<bit_and, ushort> >::call, 0, NppBitwiseC<CV_16U, 3, nppiAndC_16u_C3R>::call, NppBitwiseC<CV_16U, 4, nppiAndC_16u_C4R>::call },
            { BitScalar<uint, bitScalarOp<bit_and, uint> >::call    , 0, NppBitwiseC<CV_32S, 3, nppiAndC_32s_C3R>::call, NppBitwiseC<CV_32S, 4, nppiAndC_32s_C4R>::call },
            { BitScalar<uint, bitScalarOp<bit_and, uint> >::call    , 0, NppBitwiseC<CV_32S, 3, nppiAndC_32s_C3R>::call, NppBitwiseC<CV_32S, 4, nppiAndC_32s_C4R>::call }
        },
        {
            {BitScalar<uchar, bitScalarOp<bit_or, uchar> >::call  , 0, NppBitwiseC<CV_8U , 3, nppiOrC_8u_C3R >::call, BitScalar4< bitScalarOp<bit_or, uint> >::call},
            {BitScalar<uchar, bitScalarOp<bit_or, uchar> >::call  , 0, NppBitwiseC<CV_8U , 3, nppiOrC_8u_C3R >::call, BitScalar4< bitScalarOp<bit_or, uint> >::call},
            {BitScalar<ushort, bitScalarOp<bit_or, ushort> >::call, 0, NppBitwiseC<CV_16U, 3, nppiOrC_16u_C3R>::call, NppBitwiseC<CV_16U, 4, nppiOrC_16u_C4R>::call},
            {BitScalar<ushort, bitScalarOp<bit_or, ushort> >::call, 0, NppBitwiseC<CV_16U, 3, nppiOrC_16u_C3R>::call, NppBitwiseC<CV_16U, 4, nppiOrC_16u_C4R>::call},
            {BitScalar<uint, bitScalarOp<bit_or, uint> >::call    , 0, NppBitwiseC<CV_32S, 3, nppiOrC_32s_C3R>::call, NppBitwiseC<CV_32S, 4, nppiOrC_32s_C4R>::call},
            {BitScalar<uint, bitScalarOp<bit_or, uint> >::call    , 0, NppBitwiseC<CV_32S, 3, nppiOrC_32s_C3R>::call, NppBitwiseC<CV_32S, 4, nppiOrC_32s_C4R>::call}
        },
        {
            {BitScalar<uchar, bitScalarOp<bit_xor, uchar> >::call  , 0, NppBitwiseC<CV_8U , 3, nppiXorC_8u_C3R >::call, BitScalar4< bitScalarOp<bit_xor, uint> >::call},
            {BitScalar<uchar, bitScalarOp<bit_xor, uchar> >::call  , 0, NppBitwiseC<CV_8U , 3, nppiXorC_8u_C3R >::call, BitScalar4< bitScalarOp<bit_xor, uint> >::call},
            {BitScalar<ushort, bitScalarOp<bit_xor, ushort> >::call, 0, NppBitwiseC<CV_16U, 3, nppiXorC_16u_C3R>::call, NppBitwiseC<CV_16U, 4, nppiXorC_16u_C4R>::call},
            {BitScalar<ushort, bitScalarOp<bit_xor, ushort> >::call, 0, NppBitwiseC<CV_16U, 3, nppiXorC_16u_C3R>::call, NppBitwiseC<CV_16U, 4, nppiXorC_16u_C4R>::call},
            {BitScalar<uint, bitScalarOp<bit_xor, uint> >::call    , 0, NppBitwiseC<CV_32S, 3, nppiXorC_32s_C3R>::call, NppBitwiseC<CV_32S, 4, nppiXorC_32s_C4R>::call},
            {BitScalar<uint, bitScalarOp<bit_xor, uint> >::call    , 0, NppBitwiseC<CV_32S, 3, nppiXorC_32s_C3R>::call, NppBitwiseC<CV_32S, 4, nppiXorC_32s_C4R>::call}
        }
#endif
    };

    const int depth = src.depth();
    const int cn = src.channels();

    CV_DbgAssert( depth <= CV_32F );
    CV_DbgAssert( cn == 1 || cn == 3 || cn == 4 );
    CV_DbgAssert( mask.empty() );
    CV_DbgAssert( op >= 0 && op < 3 );

    funcs[op][depth][cn - 1](src, value, dst, stream);
}

#endif
