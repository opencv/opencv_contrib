/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "opencv2/opencv_modules.hpp"

#ifndef HAVE_OPENCV_CUDEV

#error "opencv_cudev is required"

#else

#include "opencv2/cudaarithm.hpp"
#include "opencv2/cudev.hpp"
#include "opencv2/core/private.cuda.hpp"

using namespace cv;
using namespace cv::cuda;
using namespace cv::cudev;

void cv::cuda::transpose(InputArray _src, OutputArray _dst, Stream& stream)
{
    GpuMat src = getInputMat(_src, stream);

    const size_t elemSize = src.elemSize();

    CV_Assert( elemSize == 1 || elemSize == 4 || elemSize == 8 );

    GpuMat dst = getOutputMat(_dst, src.cols, src.rows, src.type(), stream);

    if (elemSize == 1)
    {
        NppStreamHandler h(StreamAccessor::getStream(stream));

        NppiSize sz;
        sz.width  = src.cols;
        sz.height = src.rows;

#if USE_NPP_STREAM_CTX
        nppSafeCall(nppiTranspose_8u_C1R_Ctx(src.ptr<Npp8u>(), static_cast<int>(src.step),
            dst.ptr<Npp8u>(), static_cast<int>(dst.step), sz, h));
#else
        nppSafeCall( nppiTranspose_8u_C1R(src.ptr<Npp8u>(), static_cast<int>(src.step),
            dst.ptr<Npp8u>(), static_cast<int>(dst.step), sz) );
#endif

        if (!stream)
            CV_CUDEV_SAFE_CALL( hipDeviceSynchronize() );
    }
    else if (elemSize == 4)
    {
        gridTranspose(globPtr<int>(src), globPtr<int>(dst), stream);
    }
    else // if (elemSize == 8)
    {
        gridTranspose(globPtr<double>(src), globPtr<double>(dst), stream);
    }

    syncOutput(dst, _dst, stream);
}

#endif
