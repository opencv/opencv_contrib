#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "opencv2/opencv_modules.hpp"

#ifndef HAVE_OPENCV_CUDEV

#error "opencv_cudev is required"

#else

#include "opencv2/cudev.hpp"
#include "opencv2/core/cuda/cuda_compat.hpp"

using namespace cv::cudev;

void absDiffScalar(const GpuMat& src, cv::Scalar val, bool, GpuMat& dst, const GpuMat&, double, Stream& stream, int);

namespace
{
    using cv::cuda::device::compat::double4Compat;
    template <typename SrcType, typename ScalarType, typename DstType> struct AbsDiffScalarOp : unary_function<SrcType, DstType>
    {
        ScalarType val;

        __device__ __forceinline__ DstType operator ()(SrcType a) const
        {
            abs_func<ScalarType> f;
            return saturate_cast<DstType>(f(saturate_cast<ScalarType>(a) - val));
        }
    };

    template <typename ScalarDepth> struct TransformPolicy : DefaultTransformPolicy
    {
    };
    template <> struct TransformPolicy<double> : DefaultTransformPolicy
    {
        enum {
            shift = 1
        };
    };

    template <typename SrcType, typename ScalarDepth>
    void absDiffScalarImpl(const GpuMat& src, cv::Scalar value, GpuMat& dst, Stream& stream)
    {
        typedef typename MakeVec<ScalarDepth, VecTraits<SrcType>::cn>::type ScalarType;

        cv::Scalar_<ScalarDepth> value_ = value;

        AbsDiffScalarOp<SrcType, ScalarType, SrcType> op;
        op.val = VecTraits<ScalarType>::make(value_.val);
        gridTransformUnary_< TransformPolicy<ScalarDepth> >(globPtr<SrcType>(src), globPtr<SrcType>(dst), op, stream);
    }
}

void absDiffScalar(const GpuMat& src, cv::Scalar val, bool, GpuMat& dst, const GpuMat&, double, Stream& stream, int)
{
    typedef void (*func_t)(const GpuMat& src, cv::Scalar val, GpuMat& dst, Stream& stream);
    static const func_t funcs[CV_DEPTH_MAX][4] =
    {
        {
            absDiffScalarImpl<uchar, float>, absDiffScalarImpl<uchar2, float>, absDiffScalarImpl<uchar3, float>, absDiffScalarImpl<uchar4, float>
        },
        {
            absDiffScalarImpl<schar, float>, absDiffScalarImpl<char2, float>, absDiffScalarImpl<char3, float>, absDiffScalarImpl<char4, float>
        },
        {
            absDiffScalarImpl<ushort, float>, absDiffScalarImpl<ushort2, float>, absDiffScalarImpl<ushort3, float>, absDiffScalarImpl<ushort4, float>
        },
        {
            absDiffScalarImpl<short, float>, absDiffScalarImpl<short2, float>, absDiffScalarImpl<short3, float>, absDiffScalarImpl<short4, float>
        },
        {
            absDiffScalarImpl<int, float>, absDiffScalarImpl<int2, float>, absDiffScalarImpl<int3, float>, absDiffScalarImpl<int4, float>
        },
        {
          absDiffScalarImpl<float, float>, absDiffScalarImpl<float2, float>, absDiffScalarImpl<float3, float>, absDiffScalarImpl<float4, float>
        },
        {
          absDiffScalarImpl<double, double>, absDiffScalarImpl<double2, double>, absDiffScalarImpl<double3, double>, absDiffScalarImpl<double4Compat, double>
        }
    };

    const int sdepth = src.depth();
    const int cn = src.channels();

    CV_DbgAssert( sdepth <= CV_64F && cn <= 4 && src.type() == dst.type());

    const func_t func = funcs[sdepth][cn - 1];
    if (!func)
        CV_Error(cv::Error::StsUnsupportedFormat, "Unsupported combination of source and destination types");

    func(src, val, dst, stream);
}

#endif
