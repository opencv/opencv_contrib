/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "opencv2/opencv_modules.hpp"

#ifndef HAVE_OPENCV_CUDEV

#error "opencv_cudev is required"

#else

#include "opencv2/cudaarithm.hpp"
#include "opencv2/cudev.hpp"
#include "opencv2/core/private.cuda.hpp"

using namespace cv;
using namespace cv::cuda;
using namespace cv::cudev;

namespace
{
    template <typename ScalarDepth> struct TransformPolicy : DefaultTransformPolicy
    {
    };
    template <> struct TransformPolicy<double> : DefaultTransformPolicy
    {
        enum {
            shift = 1
        };
    };

    template <typename T>
    void thresholdImpl(const GpuMat& src, GpuMat& dst, double thresh, double maxVal, int type, Stream& stream)
    {
        const T thresh_ = static_cast<T>(thresh);
        const T maxVal_ = static_cast<T>(maxVal);

        switch (type)
        {
        case 0:
            gridTransformUnary_< TransformPolicy<T> >(globPtr<T>(src), globPtr<T>(dst), thresh_binary_func(thresh_, maxVal_), stream);
            break;
        case 1:
            gridTransformUnary_< TransformPolicy<T> >(globPtr<T>(src), globPtr<T>(dst), thresh_binary_inv_func(thresh_, maxVal_), stream);
            break;
        case 2:
            gridTransformUnary_< TransformPolicy<T> >(globPtr<T>(src), globPtr<T>(dst), thresh_trunc_func(thresh_), stream);
            break;
        case 3:
            gridTransformUnary_< TransformPolicy<T> >(globPtr<T>(src), globPtr<T>(dst), thresh_to_zero_func(thresh_), stream);
            break;
        case 4:
            gridTransformUnary_< TransformPolicy<T> >(globPtr<T>(src), globPtr<T>(dst), thresh_to_zero_inv_func(thresh_), stream);
            break;
        };
    }
}

double cv::cuda::threshold(InputArray _src, OutputArray _dst, double thresh, double maxVal, int type, Stream& stream)
{
    GpuMat src = getInputMat(_src, stream);

    const int depth = src.depth();

    CV_Assert( depth <= CV_64F );
    CV_Assert( type <= 4 /*THRESH_TOZERO_INV*/ );

    GpuMat dst = getOutputMat(_dst, src.size(), src.type(), stream);
    src = src.reshape(1);
    dst = dst.reshape(1);

    if (depth == CV_32F && type == 2 /*THRESH_TRUNC*/)
    {
        NppStreamHandler h(StreamAccessor::getStream(stream));

        NppiSize sz;
        sz.width  = src.cols;
        sz.height = src.rows;

#if USE_NPP_STREAM_CTX
        nppSafeCall(nppiThreshold_32f_C1R_Ctx(src.ptr<Npp32f>(), static_cast<int>(src.step),
            dst.ptr<Npp32f>(), static_cast<int>(dst.step), sz, static_cast<Npp32f>(thresh), NPP_CMP_GREATER, h));
#else
        nppSafeCall( nppiThreshold_32f_C1R(src.ptr<Npp32f>(), static_cast<int>(src.step),
            dst.ptr<Npp32f>(), static_cast<int>(dst.step), sz, static_cast<Npp32f>(thresh), NPP_CMP_GREATER) );
#endif

        if (!stream)
            CV_CUDEV_SAFE_CALL( hipDeviceSynchronize() );
    }
    else
    {
        typedef void (*func_t)(const GpuMat& src, GpuMat& dst, double thresh, double maxVal, int type, Stream& stream);
        static const func_t funcs[] =
        {
            thresholdImpl<uchar>,
            thresholdImpl<schar>,
            thresholdImpl<ushort>,
            thresholdImpl<short>,
            thresholdImpl<int>,
            thresholdImpl<float>,
            thresholdImpl<double>
        };

        if (depth != CV_32F && depth != CV_64F)
        {
            thresh = cvFloor(thresh);
            maxVal = cvRound(maxVal);
        }

        funcs[depth](src, dst, thresh, maxVal, type, stream);
    }

    syncOutput(dst, _dst, stream);

    return thresh;
}

#endif
