/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "opencv2/opencv_modules.hpp"

#ifndef HAVE_OPENCV_CUDEV

#error "opencv_cudev is required"

#else

#include "opencv2/cudev.hpp"

using namespace cv::cudev;

void minMaxMat(const GpuMat& src1, const GpuMat& src2, GpuMat& dst, const GpuMat&, double, Stream& stream, int op);

void minMaxScalar(const GpuMat& src, cv::Scalar value, bool, GpuMat& dst, const GpuMat&, double, Stream& stream, int op);

///////////////////////////////////////////////////////////////////////
/// minMaxMat

namespace
{
    template <template <typename> class Op, typename T>
    void minMaxMat_v1(const GpuMat& src1, const GpuMat& src2, GpuMat& dst, Stream& stream)
    {
        gridTransformBinary(globPtr<T>(src1), globPtr<T>(src2), globPtr<T>(dst), Op<T>(), stream);
    }

    struct MinOp2 : binary_function<uint, uint, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, uint b) const
        {
            return vmin2(a, b);
        }
    };

    struct MaxOp2 : binary_function<uint, uint, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, uint b) const
        {
            return vmax2(a, b);
        }
    };

    template <class Op2>
    void minMaxMat_v2(const GpuMat& src1, const GpuMat& src2, GpuMat& dst, Stream& stream)
    {
        const int vcols = src1.cols >> 1;

        GlobPtrSz<uint> src1_ = globPtr((uint*) src1.data, src1.step, src1.rows, vcols);
        GlobPtrSz<uint> src2_ = globPtr((uint*) src2.data, src2.step, src1.rows, vcols);
        GlobPtrSz<uint> dst_ = globPtr((uint*) dst.data, dst.step, src1.rows, vcols);

        gridTransformBinary(src1_, src2_, dst_, Op2(), stream);
    }

    struct MinOp4 : binary_function<uint, uint, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, uint b) const
        {
            return vmin4(a, b);
        }
    };

    struct MaxOp4 : binary_function<uint, uint, uint>
    {
        __device__ __forceinline__ uint operator ()(uint a, uint b) const
        {
            return vmax4(a, b);
        }
    };

    template <class Op4>
    void minMaxMat_v4(const GpuMat& src1, const GpuMat& src2, GpuMat& dst, Stream& stream)
    {
        const int vcols = src1.cols >> 2;

        GlobPtrSz<uint> src1_ = globPtr((uint*) src1.data, src1.step, src1.rows, vcols);
        GlobPtrSz<uint> src2_ = globPtr((uint*) src2.data, src2.step, src1.rows, vcols);
        GlobPtrSz<uint> dst_ = globPtr((uint*) dst.data, dst.step, src1.rows, vcols);

        gridTransformBinary(src1_, src2_, dst_, Op4(), stream);
    }
}

void minMaxMat(const GpuMat& src1, const GpuMat& src2, GpuMat& dst, const GpuMat&, double, Stream& stream, int op)
{
    typedef void (*func_t)(const GpuMat& src1, const GpuMat& src2, GpuMat& dst, Stream& stream);
    static const func_t funcs_v1[2][CV_DEPTH_MAX] =
    {
        {
            minMaxMat_v1<minimum, uchar>,
            minMaxMat_v1<minimum, schar>,
            minMaxMat_v1<minimum, ushort>,
            minMaxMat_v1<minimum, short>,
            minMaxMat_v1<minimum, int>,
            minMaxMat_v1<minimum, float>,
            minMaxMat_v1<minimum, double>
        },
        {
            minMaxMat_v1<maximum, uchar>,
            minMaxMat_v1<maximum, schar>,
            minMaxMat_v1<maximum, ushort>,
            minMaxMat_v1<maximum, short>,
            minMaxMat_v1<maximum, int>,
            minMaxMat_v1<maximum, float>,
            minMaxMat_v1<maximum, double>
        }
    };

    static const func_t funcs_v2[2] =
    {
        minMaxMat_v2<MinOp2>, minMaxMat_v2<MaxOp2>
    };

    static const func_t funcs_v4[2] =
    {
        minMaxMat_v4<MinOp4>, minMaxMat_v4<MaxOp4>
    };

    const int depth = src1.depth();

    GpuMat src1_ = src1.reshape(1);
    GpuMat src2_ = src2.reshape(1);
    GpuMat dst_ = dst.reshape(1);

    if (depth == CV_8U || depth == CV_16U)
    {
        const intptr_t src1ptr = reinterpret_cast<intptr_t>(src1_.data);
        const intptr_t src2ptr = reinterpret_cast<intptr_t>(src2_.data);
        const intptr_t dstptr = reinterpret_cast<intptr_t>(dst_.data);

        const bool isAllAligned = (src1ptr & 31) == 0 && (src2ptr & 31) == 0 && (dstptr & 31) == 0;

        if (isAllAligned)
        {
            if (depth == CV_8U && (src1_.cols & 3) == 0)
            {
                funcs_v4[op](src1_, src2_, dst_, stream);
                return;
            }
            else if (depth == CV_16U && (src1_.cols & 1) == 0)
            {
                funcs_v2[op](src1_, src2_, dst_, stream);
                return;
            }
        }
    }

    const func_t func = funcs_v1[op][depth];
    CV_Assert(func);

    func(src1_, src2_, dst_, stream);
}

///////////////////////////////////////////////////////////////////////
/// minMaxScalar

namespace
{
    template <template <typename> class Op, typename T>
    void minMaxScalar(const GpuMat& src, double value, GpuMat& dst, Stream& stream)
    {
        gridTransformUnary(globPtr<T>(src), globPtr<T>(dst), bind2nd(Op<T>(), cv::saturate_cast<T>(value)), stream);
    }
}

void minMaxScalar(const GpuMat& src, cv::Scalar value, bool, GpuMat& dst, const GpuMat&, double, Stream& stream, int op)
{
    CV_DbgAssert( src.channels() == 1 );

    typedef void (*func_t)(const GpuMat& src, double value, GpuMat& dst, Stream& stream);
    static const func_t funcs[2][CV_DEPTH_MAX] =
    {
        {
            minMaxScalar<minimum, uchar>,
            minMaxScalar<minimum, schar>,
            minMaxScalar<minimum, ushort>,
            minMaxScalar<minimum, short>,
            minMaxScalar<minimum, int>,
            minMaxScalar<minimum, float>,
            minMaxScalar<minimum, double>
        },
        {
            minMaxScalar<maximum, uchar>,
            minMaxScalar<maximum, schar>,
            minMaxScalar<maximum, ushort>,
            minMaxScalar<maximum, short>,
            minMaxScalar<maximum, int>,
            minMaxScalar<maximum, float>,
            minMaxScalar<maximum, double>
        }
    };

    auto f = funcs[op][src.depth()];
    CV_Assert(f);

    f(src, value[0], dst, stream);
}

#endif
