/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "opencv2/opencv_modules.hpp"

#ifndef HAVE_OPENCV_CUDEV

#error "opencv_cudev is required"

#else

#include "opencv2/cudaarithm.hpp"
#include "opencv2/cudev.hpp"
#include "opencv2/core/private.cuda.hpp"

using namespace cv;
using namespace cv::cuda;
using namespace cv::cudev;

namespace
{
    template <typename T, typename R>
    void minMaxImpl(const GpuMat& _src, const GpuMat& mask, GpuMat& _dst, Stream& stream)
    {
        const GpuMat_<T>& src = (const GpuMat_<T>&) _src;
        GpuMat_<R>& dst = (GpuMat_<R>&) _dst;

        if (mask.empty())
            gridFindMinMaxVal(src, dst, stream);
        else
            gridFindMinMaxVal(src, dst, globPtr<uchar>(mask), stream);
    }

    template <typename T, typename R>
    void minMaxImpl(const GpuMat& src, const GpuMat& mask, double* minVal, double* maxVal)
    {
        BufferPool pool(Stream::Null());
        GpuMat buf(pool.getBuffer(1, 2, DataType<R>::type));

        minMaxImpl<T, R>(src, mask, buf, Stream::Null());

        R data[2];
        buf.download(Mat(1, 2, buf.type(), data));

    }
}

void cv::cuda::findMinMax(InputArray _src, OutputArray _dst, InputArray _mask, Stream& stream)
{
    typedef void (*func_t)(const GpuMat& _src, const GpuMat& mask, GpuMat& _dst, Stream& stream);
    static const func_t funcs[CV_DEPTH_MAX] =
    {
        minMaxImpl<uchar, int>,
        minMaxImpl<schar, int>,
        minMaxImpl<ushort, int>,
        minMaxImpl<short, int>,
        minMaxImpl<int, int>,
        minMaxImpl<float, float>,
        minMaxImpl<double, double>
    };

    const GpuMat src = getInputMat(_src, stream);
    const GpuMat mask = getInputMat(_mask, stream);

    CV_Assert( src.channels() == 1 );
    CV_Assert( mask.empty() || (mask.size() == src.size() && mask.type() == CV_8U) );

    const int src_depth = src.depth();
    const int dst_depth = src_depth < CV_32F ? CV_32S : src_depth;

    GpuMat dst = getOutputMat(_dst, 1, 2, dst_depth, stream);

    const func_t func = funcs[src.depth()];
    CV_Assert(func);

    func(src, mask, dst, stream);

    syncOutput(dst, _dst, stream);
}

void cv::cuda::minMax(InputArray _src, double* minVal, double* maxVal, InputArray _mask)
{
    Stream& stream = Stream::Null();

    HostMem dst;
    findMinMax(_src, dst, _mask, stream);

    stream.waitForCompletion();

    double vals[2];
    dst.createMatHeader().convertTo(Mat(1, 2, CV_64FC1, &vals[0]), CV_64F);

    if (minVal)
        *minVal = vals[0];

    if (maxVal)
        *maxVal = vals[1];
}

namespace cv { namespace cuda { namespace device {

void findMaxAbs(InputArray _src, OutputArray _dst, InputArray _mask, Stream& stream);

}}}

namespace
{
    template <typename T, typename R>
    void findMaxAbsImpl(const GpuMat& _src, const GpuMat& mask, GpuMat& _dst, Stream& stream)
    {
        const GpuMat_<T>& src = (const GpuMat_<T>&) _src;
        GpuMat_<R>& dst = (GpuMat_<R>&) _dst;

        if (mask.empty())
            gridFindMaxVal(abs_(src), dst, stream);
        else
            gridFindMaxVal(abs_(src), dst, globPtr<uchar>(mask), stream);
    }
}

void cv::cuda::device::findMaxAbs(InputArray _src, OutputArray _dst, InputArray _mask, Stream& stream)
{
    typedef void (*func_t)(const GpuMat& _src, const GpuMat& mask, GpuMat& _dst, Stream& stream);
    static const func_t funcs[CV_DEPTH_MAX] =
    {
        findMaxAbsImpl<uchar, int>,
        findMaxAbsImpl<schar, int>,
        findMaxAbsImpl<ushort, int>,
        findMaxAbsImpl<short, int>,
        findMaxAbsImpl<int, int>,
        findMaxAbsImpl<float, float>,
        findMaxAbsImpl<double, double>
    };

    const GpuMat src = getInputMat(_src, stream);
    const GpuMat mask = getInputMat(_mask, stream);

    CV_Assert( src.channels() == 1 );
    CV_Assert( mask.empty() || (mask.size() == src.size() && mask.type() == CV_8U) );

    const int src_depth = src.depth();
    const int dst_depth = src_depth < CV_32F ? CV_32S : src_depth;

    GpuMat dst = getOutputMat(_dst, 1, 1, dst_depth, stream);

    const func_t func = funcs[src.depth()];
    CV_Assert(func);

    func(src, mask, dst, stream);

    syncOutput(dst, _dst, stream);
}

#endif
