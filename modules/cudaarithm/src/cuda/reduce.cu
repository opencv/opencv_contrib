/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "opencv2/opencv_modules.hpp"

#ifndef HAVE_OPENCV_CUDEV

#error "opencv_cudev is required"

#else

#include "opencv2/cudaarithm.hpp"
#include "opencv2/cudev.hpp"
#include "opencv2/core/private.cuda.hpp"

using namespace cv;
using namespace cv::cuda;
using namespace cv::cudev;

namespace
{
    template <typename T, typename S, typename D>
    void reduceToRowImpl(const GpuMat& _src, GpuMat& _dst, int reduceOp, Stream& stream)
    {
        const GpuMat_<T>& src = (const GpuMat_<T>&) _src;
        GpuMat_<D>& dst = (GpuMat_<D>&) _dst;

        switch (reduceOp)
        {
        case cv::REDUCE_SUM:
            gridReduceToRow< Sum<S> >(src, dst, stream);
            break;

        case cv::REDUCE_AVG:
            gridReduceToRow< Avg<S> >(src, dst, stream);
            break;

        case cv::REDUCE_MIN:
            gridReduceToRow< Min<S> >(src, dst, stream);
            break;

        case cv::REDUCE_MAX:
            gridReduceToRow< Max<S> >(src, dst, stream);
            break;
        };
    }

    template <typename T, typename S, typename D>
    void reduceToColumnImpl_(const GpuMat& _src, GpuMat& _dst, int reduceOp, Stream& stream)
    {
        const GpuMat_<T>& src = (const GpuMat_<T>&) _src;
        GpuMat_<D>& dst = (GpuMat_<D>&) _dst;

        switch (reduceOp)
        {
        case cv::REDUCE_SUM:
            gridReduceToColumn< Sum<S> >(src, dst, stream);
            break;

        case cv::REDUCE_AVG:
            gridReduceToColumn< Avg<S> >(src, dst, stream);
            break;

        case cv::REDUCE_MIN:
            gridReduceToColumn< Min<S> >(src, dst, stream);
            break;

        case cv::REDUCE_MAX:
            gridReduceToColumn< Max<S> >(src, dst, stream);
            break;
        };
    }

    template <typename T, typename S, typename D>
    void reduceToColumnImpl(const GpuMat& src, GpuMat& dst, int reduceOp, Stream& stream)
    {
        typedef void (*func_t)(const GpuMat& src, GpuMat& dst, int reduceOp, Stream& stream);
        static const func_t funcs[4] =
        {
            reduceToColumnImpl_<T, S, D>,
            reduceToColumnImpl_<typename MakeVec<T, 2>::type, typename MakeVec<S, 2>::type, typename MakeVec<D, 2>::type>,
            reduceToColumnImpl_<typename MakeVec<T, 3>::type, typename MakeVec<S, 3>::type, typename MakeVec<D, 3>::type>,
            reduceToColumnImpl_<typename MakeVec<T, 4>::type, typename MakeVec<S, 4>::type, typename MakeVec<D, 4>::type>
        };

        funcs[src.channels() - 1](src, dst, reduceOp, stream);
    }
}

void cv::cuda::reduce(InputArray _src, OutputArray _dst, int dim, int reduceOp, int dtype, Stream& stream)
{
    GpuMat src = getInputMat(_src, stream);

    CV_Assert( src.channels() <= 4 );
    CV_Assert( dim == 0 || dim == 1 );
    CV_Assert( reduceOp == REDUCE_SUM || reduceOp == REDUCE_AVG || reduceOp == REDUCE_MAX || reduceOp == REDUCE_MIN );

    if (dtype < 0)
        dtype = src.depth();

    GpuMat dst = getOutputMat(_dst, dim == 0 ? 1 : src.rows, dim == 0 ? src.cols : 1, CV_MAKE_TYPE(CV_MAT_DEPTH(dtype), src.channels()), stream);

    if (dim == 0)
    {
        typedef void (*func_t)(const GpuMat& _src, GpuMat& _dst, int reduceOp, Stream& stream);
        static const func_t funcs[CV_DEPTH_MAX][CV_DEPTH_MAX] =
        {
            {
                reduceToRowImpl<uchar, int, uchar>,
                0 /*reduceToRowImpl<uchar, int, schar>*/,
                0 /*reduceToRowImpl<uchar, int, ushort>*/,
                0 /*reduceToRowImpl<uchar, int, short>*/,
                reduceToRowImpl<uchar, int, int>,
                reduceToRowImpl<uchar, float, float>,
                reduceToRowImpl<uchar, double, double>
            },
            {
                0 /*reduceToRowImpl<schar, int, uchar>*/,
                0 /*reduceToRowImpl<schar, int, schar>*/,
                0 /*reduceToRowImpl<schar, int, ushort>*/,
                0 /*reduceToRowImpl<schar, int, short>*/,
                0 /*reduceToRowImpl<schar, int, int>*/,
                0 /*reduceToRowImpl<schar, float, float>*/,
                0 /*reduceToRowImpl<schar, double, double>*/
            },
            {
                0 /*reduceToRowImpl<ushort, int, uchar>*/,
                0 /*reduceToRowImpl<ushort, int, schar>*/,
                reduceToRowImpl<ushort, int, ushort>,
                0 /*reduceToRowImpl<ushort, int, short>*/,
                reduceToRowImpl<ushort, int, int>,
                reduceToRowImpl<ushort, float, float>,
                reduceToRowImpl<ushort, double, double>
            },
            {
                0 /*reduceToRowImpl<short, int, uchar>*/,
                0 /*reduceToRowImpl<short, int, schar>*/,
                0 /*reduceToRowImpl<short, int, ushort>*/,
                reduceToRowImpl<short, int, short>,
                reduceToRowImpl<short, int, int>,
                reduceToRowImpl<short, float, float>,
                reduceToRowImpl<short, double, double>
            },
            {
                0 /*reduceToRowImpl<int, int, uchar>*/,
                0 /*reduceToRowImpl<int, int, schar>*/,
                0 /*reduceToRowImpl<int, int, ushort>*/,
                0 /*reduceToRowImpl<int, int, short>*/,
                reduceToRowImpl<int, int, int>,
                reduceToRowImpl<int, float, float>,
                reduceToRowImpl<int, double, double>
            },
            {
                0 /*reduceToRowImpl<float, float, uchar>*/,
                0 /*reduceToRowImpl<float, float, schar>*/,
                0 /*reduceToRowImpl<float, float, ushort>*/,
                0 /*reduceToRowImpl<float, float, short>*/,
                0 /*reduceToRowImpl<float, float, int>*/,
                reduceToRowImpl<float, float, float>,
                reduceToRowImpl<float, double, double>
            },
            {
                0 /*reduceToRowImpl<double, double, uchar>*/,
                0 /*reduceToRowImpl<double, double, schar>*/,
                0 /*reduceToRowImpl<double, double, ushort>*/,
                0 /*reduceToRowImpl<double, double, short>*/,
                0 /*reduceToRowImpl<double, double, int>*/,
                0 /*reduceToRowImpl<double, double, float>*/,
                reduceToRowImpl<double, double, double>
            }
        };

        const func_t func = funcs[src.depth()][dst.depth()];

        if (!func)
            CV_Error(cv::Error::StsUnsupportedFormat, "Unsupported combination of input and output array formats");

        GpuMat dst_cont = dst.reshape(1);
        func(src.reshape(1), dst_cont, reduceOp, stream);
    }
    else
    {
        typedef void (*func_t)(const GpuMat& _src, GpuMat& _dst, int reduceOp, Stream& stream);
        static const func_t funcs[CV_DEPTH_MAX][CV_DEPTH_MAX] =
        {
            {
                reduceToColumnImpl<uchar, int, uchar>,
                0 /*reduceToColumnImpl<uchar, int, schar>*/,
                0 /*reduceToColumnImpl<uchar, int, ushort>*/,
                0 /*reduceToColumnImpl<uchar, int, short>*/,
                reduceToColumnImpl<uchar, int, int>,
                reduceToColumnImpl<uchar, float, float>,
                reduceToColumnImpl<uchar, double, double>
            },
            {
                0 /*reduceToColumnImpl<schar, int, uchar>*/,
                0 /*reduceToColumnImpl<schar, int, schar>*/,
                0 /*reduceToColumnImpl<schar, int, ushort>*/,
                0 /*reduceToColumnImpl<schar, int, short>*/,
                0 /*reduceToColumnImpl<schar, int, int>*/,
                0 /*reduceToColumnImpl<schar, float, float>*/,
                0 /*reduceToColumnImpl<schar, double, double>*/
            },
            {
                0 /*reduceToColumnImpl<ushort, int, uchar>*/,
                0 /*reduceToColumnImpl<ushort, int, schar>*/,
                reduceToColumnImpl<ushort, int, ushort>,
                0 /*reduceToColumnImpl<ushort, int, short>*/,
                reduceToColumnImpl<ushort, int, int>,
                reduceToColumnImpl<ushort, float, float>,
                reduceToColumnImpl<ushort, double, double>
            },
            {
                0 /*reduceToColumnImpl<short, int, uchar>*/,
                0 /*reduceToColumnImpl<short, int, schar>*/,
                0 /*reduceToColumnImpl<short, int, ushort>*/,
                reduceToColumnImpl<short, int, short>,
                reduceToColumnImpl<short, int, int>,
                reduceToColumnImpl<short, float, float>,
                reduceToColumnImpl<short, double, double>
            },
            {
                0 /*reduceToColumnImpl<int, int, uchar>*/,
                0 /*reduceToColumnImpl<int, int, schar>*/,
                0 /*reduceToColumnImpl<int, int, ushort>*/,
                0 /*reduceToColumnImpl<int, int, short>*/,
                reduceToColumnImpl<int, int, int>,
                reduceToColumnImpl<int, float, float>,
                reduceToColumnImpl<int, double, double>
            },
            {
                0 /*reduceToColumnImpl<float, float, uchar>*/,
                0 /*reduceToColumnImpl<float, float, schar>*/,
                0 /*reduceToColumnImpl<float, float, ushort>*/,
                0 /*reduceToColumnImpl<float, float, short>*/,
                0 /*reduceToColumnImpl<float, float, int>*/,
                reduceToColumnImpl<float, float, float>,
                reduceToColumnImpl<float, double, double>
            },
            {
                0 /*reduceToColumnImpl<double, double, uchar>*/,
                0 /*reduceToColumnImpl<double, double, schar>*/,
                0 /*reduceToColumnImpl<double, double, ushort>*/,
                0 /*reduceToColumnImpl<double, double, short>*/,
                0 /*reduceToColumnImpl<double, double, int>*/,
                0 /*reduceToColumnImpl<double, double, float>*/,
                reduceToColumnImpl<double, double, double>
            }
        };

        const func_t func = funcs[src.depth()][dst.depth()];

        if (!func)
            CV_Error(cv::Error::StsUnsupportedFormat, "Unsupported combination of input and output array formats");

        func(src, dst, reduceOp, stream);
    }

    syncOutput(dst, _dst, stream);
}

#endif
