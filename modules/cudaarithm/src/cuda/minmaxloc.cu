/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "opencv2/opencv_modules.hpp"

#ifndef HAVE_OPENCV_CUDEV

#error "opencv_cudev is required"

#else

#include "opencv2/cudaarithm.hpp"
#include "opencv2/cudev.hpp"
#include "opencv2/core/private.cuda.hpp"

using namespace cv;
using namespace cv::cuda;
using namespace cv::cudev;

namespace
{
    template <typename T, typename R>
    void minMaxLocImpl(const GpuMat& _src, const GpuMat& mask, GpuMat& _valBuf, GpuMat& _locBuf, Stream& stream)
    {
        const GpuMat_<T>& src = (const GpuMat_<T>&) _src;
        GpuMat_<R>& valBuf = (GpuMat_<R>&) _valBuf;
        GpuMat_<int>& locBuf = (GpuMat_<int>&) _locBuf;

        if (mask.empty())
            gridMinMaxLoc(src, valBuf, locBuf, stream);
        else
            gridMinMaxLoc(src, valBuf, locBuf, globPtr<uchar>(mask), stream);
    }
}

void cv::cuda::findMinMaxLoc(InputArray _src, OutputArray _minMaxVals, OutputArray _loc, InputArray _mask, Stream& stream)
{
    typedef void (*func_t)(const GpuMat& _src, const GpuMat& mask, GpuMat& _valBuf, GpuMat& _locBuf, Stream& stream);
    static const func_t funcs[CV_DEPTH_MAX] =
    {
        minMaxLocImpl<uchar, int>,
        minMaxLocImpl<schar, int>,
        minMaxLocImpl<ushort, int>,
        minMaxLocImpl<short, int>,
        minMaxLocImpl<int, int>,
        minMaxLocImpl<float, float>,
        minMaxLocImpl<double, double>
    };

    const GpuMat src = getInputMat(_src, stream);
    const GpuMat mask = getInputMat(_mask, stream);

    CV_Assert( src.channels() == 1 );
    CV_Assert( mask.empty() || (mask.size() == src.size() && mask.type() == CV_8U) );

    const int src_depth = src.depth();

    BufferPool pool(stream);
    GpuMat valBuf(pool.getAllocator());
    GpuMat locBuf(pool.getAllocator());

    const func_t func = funcs[src_depth];
    CV_Assert(func);
    func(src, mask, valBuf, locBuf, stream);

    GpuMat minMaxVals = valBuf.colRange(0, 1);
    GpuMat loc = locBuf.colRange(0, 1);

    if (_minMaxVals.kind() == _InputArray::CUDA_GPU_MAT)
    {
        minMaxVals.copyTo(_minMaxVals, stream);
    }
    else
    {
        minMaxVals.download(_minMaxVals, stream);
    }

    if (_loc.kind() == _InputArray::CUDA_GPU_MAT)
    {
        loc.copyTo(_loc, stream);
    }
    else
    {
        loc.download(_loc, stream);
    }
}

void cv::cuda::minMaxLoc(InputArray _src, double* minVal, double* maxVal, Point* minLoc, Point* maxLoc, InputArray _mask)
{
    Stream& stream = Stream::Null();

    HostMem minMaxVals, locVals;
    findMinMaxLoc(_src, minMaxVals, locVals, _mask, stream);

    stream.waitForCompletion();

    double vals[2];
    minMaxVals.createMatHeader().convertTo(Mat(minMaxVals.size(), CV_64FC1, &vals[0]), CV_64F);

    int locs[2];
    locVals.createMatHeader().copyTo(Mat(locVals.size(), CV_32SC1, &locs[0]));
    Size size = _src.size();
    cv::Point locs2D[] = {
        cv::Point(locs[0] % size.width, locs[0] / size.width),
        cv::Point(locs[1] % size.width, locs[1] / size.width),
    };

    if (minVal)
        *minVal = vals[0];

    if (maxVal)
        *maxVal = vals[1];

    if (minLoc)
        *minLoc = locs2D[0];

    if (maxLoc)
        *maxLoc = locs2D[1];
}

#endif
