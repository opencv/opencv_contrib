#include "hip/hip_runtime.h"
#include "opencv2/cudev/util/vec_traits.hpp"
#include "opencv2/core/cuda_types.hpp"

namespace
{
template<typename T, int cn>
constexpr T __CV_CUDA_HOST_DEVICE__ get_channel(const int index, const typename cv::cudev::MakeVec<T, cn>::type& value) noexcept
{
  return reinterpret_cast<const T*>(&value)[index];
}

template<typename T, int cn>
__global__ void extract_channel_kernel(cv::cuda::PtrStepSz<typename cv::cudev::MakeVec<T, cn>::type> many_channel,
                                       cv::cuda::PtrStepSz<T> single_channel, const int channel_index)
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= many_channel.cols || y >= many_channel.rows) {
    return;
  }

  single_channel(y, x) = ::get_channel<T, cn>(channel_index, many_channel(y, x));
}

template<typename T, int cn>
void extract_channel_impl(cv::cuda::PtrStepSz<typename cv::cudev::MakeVec<T, cn>::type> many_channel,
                          cv::cuda::PtrStepSz<T> single_channel, const int channel_index, cv::cuda::Stream& stream)
{

  static constexpr dim3 block(32, 8);
  const dim3 grid(cv::cudev::divUp(many_channel.cols, block.x), cv::cudev::divUp(many_channel.rows, block.y));
  ::extract_channel_kernel<T, cn><<<grid, block, 0, cv::cuda::StreamAccessor::getStream(stream)>>>(many_channel, single_channel, channel_index);
}

template<typename T, int depth>
void extract_channel_t(const cv::cuda::GpuMat input, cv::cuda::GpuMat& output, const int channel_index, cv::cuda::Stream& stream)
{
  static constexpr auto output_type = CV_MAKE_TYPE(depth, 1);
  if (output.size() != input.size() || output.type() != output_type) {
    output = cv::cuda::GpuMat(input.size(), output_type);
  }

  switch (input.channels()) {
  case 1:
    ::extract_channel_impl<T, 1>(input, output, channel_index, stream);
    break;
  case 2:
    ::extract_channel_impl<T, 2>(input, output, channel_index, stream);
    break;
  case 3:
    ::extract_channel_impl<T, 3>(input, output, channel_index, stream);
    break;
  case 4:
    ::extract_channel_impl<T, 4>(input, output, channel_index, stream);
    break;
    default:
    CV_Error(cv::Error::StsUnsupportedFormat, "Unsupported channel count");
  }
}
}  // namespace

namespace cv {
namespace cuda {
void extractChannel(const GpuMat input, GpuMat& output, const int channel_index, Stream& stream)
{
  switch (CV_MAT_DEPTH(input.type())) {
  case CV_8U:
    ::extract_channel_t<std::uint8_t, CV_8U>(input, output, channel_index, stream);
    break;
  case CV_8S:
    ::extract_channel_t<std::int8_t, CV_8S>(input, output, channel_index, stream);
    break;
  case CV_16S:
    ::extract_channel_t<std::int16_t, CV_16S>(input, output, channel_index, stream);
    break;
  case CV_16U:
    ::extract_channel_t<std::uint16_t, CV_16U>(input, output, channel_index, stream);
    break;
  case CV_32F:
    ::extract_channel_t<std::float_t, CV_32F>(input, output, channel_index, stream);
    break;
  case CV_32S:
    ::extract_channel_t<std::int32_t, CV_32S>(input, output, channel_index, stream);
    break;
  case CV_64F:
    ::extract_channel_t<std::double_t, CV_64F>(input, output, channel_index, stream);
    break;
  case CV_16F:
    [[fallthrough]];
  default:
    CV_Error(Error::StsUnsupportedFormat, "Unsupported data type");
  }
}
}  // namespace cuda
}  // namespace cv
