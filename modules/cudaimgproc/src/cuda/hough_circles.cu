#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/emulation.hpp"
#include "opencv2/core/cuda/dynamic_smem.hpp"

#include "opencv2/opencv_modules.hpp"

#ifdef HAVE_OPENCV_CUDAFILTERS

namespace cv { namespace cuda { namespace device
{
    namespace hough_circles
    {
        ////////////////////////////////////////////////////////////////////////
        // circlesAccumCenters

        __global__ void circlesAccumCenters(const unsigned int* list, const int count, const PtrStepi dx, const PtrStepi dy,
                                            PtrStepi accum, const int width, const int height, const int minRadius, const int maxRadius, const float idp)
        {
            const int SHIFT = 10;
            const int ONE = 1 << SHIFT;

            const int tid = blockIdx.x * blockDim.x + threadIdx.x;

            if (tid >= count)
                return;

            const unsigned int val = list[tid];

            const int x = (val & 0xFFFF);
            const int y = (val >> 16) & 0xFFFF;

            const int vx = dx(y, x);
            const int vy = dy(y, x);

            if (vx == 0 && vy == 0)
                return;

            const float mag = ::sqrtf(vx * vx + vy * vy);

            const int x0 = __float2int_rn((x * idp) * ONE);
            const int y0 = __float2int_rn((y * idp) * ONE);

            int sx = __float2int_rn((vx * idp) * ONE / mag);
            int sy = __float2int_rn((vy * idp) * ONE / mag);

            // Step from minRadius to maxRadius in both directions of the gradient
            for (int k1 = 0; k1 < 2; ++k1)
            {
                int x1 = x0 + minRadius * sx;
                int y1 = y0 + minRadius * sy;

                for (int r = minRadius; r <= maxRadius; x1 += sx, y1 += sy, ++r)
                {
                    const int x2 = x1 >> SHIFT;
                    const int y2 = y1 >> SHIFT;

                    if (x2 < 0 || x2 >= width || y2 < 0 || y2 >= height)
                        break;

                    ::atomicAdd(accum.ptr(y2 + 1) + x2 + 1, 1);
                }

                sx = -sx;
                sy = -sy;
            }
        }

        void circlesAccumCenters_gpu(const unsigned int* list, int count, PtrStepi dx, PtrStepi dy, PtrStepSzi accum, int minRadius, int maxRadius, float idp, hipStream_t stream)
        {
            const dim3 block(256);
            const dim3 grid(divUp(count, block.x));

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(circlesAccumCenters), hipFuncCachePreferL1) );

            circlesAccumCenters<<<grid, block, 0, stream>>>(list, count, dx, dy, accum, accum.cols - 2, accum.rows - 2, minRadius, maxRadius, idp);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipStreamSynchronize(stream) );
        }

        ////////////////////////////////////////////////////////////////////////
        // buildCentersList
        __global__ void buildCentersList(const PtrStepSzi accum, unsigned int* centers, const int threshold, int* counterPtr)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x < accum.cols - 2 && y < accum.rows - 2)
            {
                const int top = accum(y, x + 1);

                const int left = accum(y + 1, x);
                const int cur = accum(y + 1, x + 1);
                const int right = accum(y + 1, x + 2);

                const int bottom = accum(y + 2, x + 1);

                if (cur > threshold && cur > top && cur >= bottom && cur >  left && cur >= right)
                {
                    const unsigned int val = (y << 16) | x;
                    const int idx = ::atomicAdd(counterPtr, 1);
                    centers[idx] = val;
                }
            }
        }

        int buildCentersList_gpu(PtrStepSzi accum, unsigned int* centers, int threshold, int* counterPtr, hipStream_t stream)
        {
            cudaSafeCall( hipMemsetAsync(counterPtr, 0, sizeof(int), stream) );

            const dim3 block(32, 8);
            const dim3 grid(divUp(accum.cols - 2, block.x), divUp(accum.rows - 2, block.y));

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(buildCentersList), hipFuncCachePreferL1) );

            buildCentersList<<<grid, block, 0, stream>>>(accum, centers, threshold, counterPtr);
            cudaSafeCall( hipGetLastError() );

            int totalCount;
            cudaSafeCall( hipMemcpyAsync(&totalCount, counterPtr, sizeof(int), hipMemcpyDeviceToHost, stream) );
            cudaSafeCall( hipStreamSynchronize(stream) );

            return totalCount;
        }

        ////////////////////////////////////////////////////////////////////////
        // circlesAccumRadius

        __global__ void circlesAccumRadius(const unsigned int* centers, const unsigned int* list, const int count,
                                           float3* circles, const int maxCircles, const float dp,
                                           const int minRadius, const int maxRadius, const int histSize, const int threshold,
                                           int* counterPtr)
        {
            int* smem = DynamicSharedMem<int>();

            for (int i = threadIdx.x; i < histSize + 2; i += blockDim.x)
                smem[i] = 0;
            __syncthreads();

            unsigned int val = centers[blockIdx.x];

            float cx = (val & 0xFFFF);
            float cy = (val >> 16) & 0xFFFF;

            cx = (cx + 0.5f) * dp;
            cy = (cy + 0.5f) * dp;

            for (int i = threadIdx.x; i < count; i += blockDim.x)
            {
                val = list[i];

                const int x = (val & 0xFFFF);
                const int y = (val >> 16) & 0xFFFF;

                const float rad = ::sqrtf((cx - x) * (cx - x) + (cy - y) * (cy - y));
                if (rad >= minRadius && rad <= maxRadius)
                {
                    const int r = __float2int_rn(rad - minRadius);

                    Emulation::smem::atomicAdd(&smem[r + 1], 1);
                }
            }

            __syncthreads();

            for (int i = threadIdx.x; i < histSize; i += blockDim.x)
            {
                const int curVotes = smem[i + 1];

                if (curVotes >= threshold && curVotes > smem[i] && curVotes >= smem[i + 2])
                {
                    const int ind = ::atomicAdd(counterPtr, 1);
                    if (ind < maxCircles)
                        circles[ind] = make_float3(cx, cy, i + minRadius);
                }
            }
        }

        int circlesAccumRadius_gpu(const unsigned int* centers, int centersCount, const unsigned int* list, int count,
                                   float3* circles, int maxCircles, float dp, int minRadius, int maxRadius, int threshold, bool has20, int* counterPtr, hipStream_t stream)
        {
            cudaSafeCall( hipMemsetAsync(counterPtr, 0, sizeof(int), stream) );

            const dim3 block(has20 ? 1024 : 512);
            const dim3 grid(centersCount);

            const int histSize = maxRadius - minRadius + 1;
            size_t smemSize = (histSize + 2) * sizeof(int);

            circlesAccumRadius<<<grid, block, smemSize, stream>>>(centers, list, count, circles, maxCircles, dp, minRadius, maxRadius, histSize, threshold, counterPtr);
            cudaSafeCall( hipGetLastError() );

            int totalCount;
            cudaSafeCall( hipMemcpyAsync(&totalCount, counterPtr, sizeof(int), hipMemcpyDeviceToHost, stream) );
            cudaSafeCall( hipStreamSynchronize(stream) );

            totalCount = std::min(totalCount, maxCircles);

            return totalCount;
        }
    }
}}}

#endif // HAVE_OPENCV_CUDAFILTERS

#endif /* CUDA_DISABLER */
