#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include <thrust/device_ptr.h>
#include <thrust/transform.h>

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/emulation.hpp"
#include "opencv2/core/cuda/vec_math.hpp"
#include "opencv2/core/cuda/functional.hpp"

#include "opencv2/opencv_modules.hpp"

#ifdef HAVE_OPENCV_CUDAARITHM

namespace cv { namespace cuda { namespace device
{
    namespace ght
    {
        __device__ int g_counter;

        template <typename T, int PIXELS_PER_THREAD>
        __global__ void buildEdgePointList(const PtrStepSzb edges, const PtrStep<T> dx, const PtrStep<T> dy,
                                           unsigned int* coordList, float* thetaList)
        {
            __shared__ unsigned int s_coordLists[4][32 * PIXELS_PER_THREAD];
            __shared__ float s_thetaLists[4][32 * PIXELS_PER_THREAD];
            __shared__ int s_sizes[4];
            __shared__ int s_globStart[4];

            const int x = blockIdx.x * blockDim.x * PIXELS_PER_THREAD + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (threadIdx.x == 0)
                s_sizes[threadIdx.y] = 0;
            __syncthreads();

            if (y < edges.rows)
            {
                // fill the queue
                const uchar* edgesRow = edges.ptr(y);
                const T* dxRow = dx.ptr(y);
                const T* dyRow = dy.ptr(y);

                for (int i = 0, xx = x; i < PIXELS_PER_THREAD && xx < edges.cols; ++i, xx += blockDim.x)
                {
                    const T dxVal = dxRow[xx];
                    const T dyVal = dyRow[xx];

                    if (edgesRow[xx] && (dxVal != 0 || dyVal != 0))
                    {
                        const unsigned int coord = (y << 16) | xx;

                        float theta = ::atan2f(dyVal, dxVal);
                        if (theta < 0)
                            theta += 2.0f * CV_PI_F;

                        const int qidx = Emulation::smem::atomicAdd(&s_sizes[threadIdx.y], 1);

                        s_coordLists[threadIdx.y][qidx] = coord;
                        s_thetaLists[threadIdx.y][qidx] = theta;
                    }
                }
            }

            __syncthreads();

            // let one thread reserve the space required in the global list
            if (threadIdx.x == 0 && threadIdx.y == 0)
            {
                // find how many items are stored in each list
                int totalSize = 0;
                for (int i = 0; i < blockDim.y; ++i)
                {
                    s_globStart[i] = totalSize;
                    totalSize += s_sizes[i];
                }

                // calculate the offset in the global list
                const int globalOffset = atomicAdd(&g_counter, totalSize);
                for (int i = 0; i < blockDim.y; ++i)
                    s_globStart[i] += globalOffset;
            }

            __syncthreads();

            // copy local queues to global queue
            const int qsize = s_sizes[threadIdx.y];
            int gidx = s_globStart[threadIdx.y] + threadIdx.x;
            for(int i = threadIdx.x; i < qsize; i += blockDim.x, gidx += blockDim.x)
            {
                coordList[gidx] = s_coordLists[threadIdx.y][i];
                thetaList[gidx] = s_thetaLists[threadIdx.y][i];
            }
        }

        template <typename T>
        int buildEdgePointList_gpu(PtrStepSzb edges, PtrStepSzb dx, PtrStepSzb dy, unsigned int* coordList, float* thetaList)
        {
            const int PIXELS_PER_THREAD = 8;

            void* counterPtr;
            cudaSafeCall( hipGetSymbolAddress(&counterPtr, HIP_SYMBOL(g_counter)) );

            cudaSafeCall( hipMemset(counterPtr, 0, sizeof(int)) );

            const dim3 block(32, 4);
            const dim3 grid(divUp(edges.cols, block.x * PIXELS_PER_THREAD), divUp(edges.rows, block.y));

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(buildEdgePointList<T), PIXELS_PER_THREAD>, hipFuncCachePreferShared) );

            buildEdgePointList<T, PIXELS_PER_THREAD><<<grid, block>>>(edges, (PtrStepSz<T>) dx, (PtrStepSz<T>) dy, coordList, thetaList);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );

            int totalCount;
            cudaSafeCall( hipMemcpy(&totalCount, counterPtr, sizeof(int), hipMemcpyDeviceToHost) );

            return totalCount;
        }

        template int buildEdgePointList_gpu<short>(PtrStepSzb edges, PtrStepSzb dx, PtrStepSzb dy, unsigned int* coordList, float* thetaList);
        template int buildEdgePointList_gpu<int>(PtrStepSzb edges, PtrStepSzb dx, PtrStepSzb dy, unsigned int* coordList, float* thetaList);
        template int buildEdgePointList_gpu<float>(PtrStepSzb edges, PtrStepSzb dx, PtrStepSzb dy, unsigned int* coordList, float* thetaList);

        __global__ void buildRTable(const unsigned int* coordList, const float* thetaList, const int pointsCount,
                                    PtrStep<short2> r_table, int* r_sizes, int maxSize,
                                    const short2 templCenter, const float thetaScale)
        {
            const int tid = blockIdx.x * blockDim.x + threadIdx.x;

            if (tid >= pointsCount)
                return;

            const unsigned int coord = coordList[tid];
            short2 p;
            p.x = (coord & 0xFFFF);
            p.y = (coord >> 16) & 0xFFFF;

            const float theta = thetaList[tid];
            const int n = __float2int_rn(theta * thetaScale);

            const int ind = ::atomicAdd(r_sizes + n, 1);
            if (ind < maxSize)
                r_table(n, ind) = saturate_cast<short2>(p - templCenter);
        }

        void buildRTable_gpu(const unsigned int* coordList, const float* thetaList, int pointsCount,
                             PtrStepSz<short2> r_table, int* r_sizes,
                             short2 templCenter, int levels)
        {
            const dim3 block(256);
            const dim3 grid(divUp(pointsCount, block.x));

            const float thetaScale = levels / (2.0f * CV_PI_F);

            buildRTable<<<grid, block>>>(coordList, thetaList, pointsCount, r_table, r_sizes, r_table.cols, templCenter, thetaScale);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }

        ////////////////////////////////////////////////////////////////////////
        // Ballard_Pos

        __global__ void Ballard_Pos_calcHist(const unsigned int* coordList, const float* thetaList, const int pointsCount,
                                             const PtrStep<short2> r_table, const int* r_sizes,
                                             PtrStepSzi hist,
                                             const float idp, const float thetaScale)
        {
            const int tid = blockIdx.x * blockDim.x + threadIdx.x;

            if (tid >= pointsCount)
                return;

            const unsigned int coord = coordList[tid];
            short2 p;
            p.x = (coord & 0xFFFF);
            p.y = (coord >> 16) & 0xFFFF;

            const float theta = thetaList[tid];
            const int n = __float2int_rn(theta * thetaScale);

            const short2* r_row = r_table.ptr(n);
            const int r_row_size = r_sizes[n];

            for (int j = 0; j < r_row_size; ++j)
            {
                short2 c = saturate_cast<short2>(p - r_row[j]);

                c.x = __float2int_rn(c.x * idp);
                c.y = __float2int_rn(c.y * idp);

                if (c.x >= 0 && c.x < hist.cols - 2 && c.y >= 0 && c.y < hist.rows - 2)
                    ::atomicAdd(hist.ptr(c.y + 1) + c.x + 1, 1);
            }
        }

        void Ballard_Pos_calcHist_gpu(const unsigned int* coordList, const float* thetaList, int pointsCount,
                                      PtrStepSz<short2> r_table, const int* r_sizes,
                                      PtrStepSzi hist,
                                      float dp, int levels)
        {
            const dim3 block(256);
            const dim3 grid(divUp(pointsCount, block.x));

            const float idp = 1.0f / dp;
            const float thetaScale = levels / (2.0f * CV_PI_F);

            Ballard_Pos_calcHist<<<grid, block>>>(coordList, thetaList, pointsCount, r_table, r_sizes, hist, idp, thetaScale);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }

        __global__ void Ballard_Pos_findPosInHist(const PtrStepSzi hist, float4* out, int3* votes,
                                                  const int maxSize, const float dp, const int threshold)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x >= hist.cols - 2 || y >= hist.rows - 2)
                return;

            const int curVotes = hist(y + 1, x + 1);

            if (curVotes > threshold &&
                curVotes >  hist(y + 1, x) &&
                curVotes >= hist(y + 1, x + 2) &&
                curVotes >  hist(y, x + 1) &&
                curVotes >= hist(y + 2, x + 1))
            {
                const int ind = ::atomicAdd(&g_counter, 1);

                if (ind < maxSize)
                {
                    out[ind] = make_float4(x * dp, y * dp, 1.0f, 0.0f);
                    votes[ind] = make_int3(curVotes, 0, 0);
                }
            }
        }

        int Ballard_Pos_findPosInHist_gpu(PtrStepSzi hist, float4* out, int3* votes, int maxSize, float dp, int threshold)
        {
            void* counterPtr;
            cudaSafeCall( hipGetSymbolAddress(&counterPtr, HIP_SYMBOL(g_counter)) );

            cudaSafeCall( hipMemset(counterPtr, 0, sizeof(int)) );

            const dim3 block(32, 8);
            const dim3 grid(divUp(hist.cols - 2, block.x), divUp(hist.rows - 2, block.y));

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(Ballard_Pos_findPosInHist), hipFuncCachePreferL1) );

            Ballard_Pos_findPosInHist<<<grid, block>>>(hist, out, votes, maxSize, dp, threshold);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );

            int totalCount;
            cudaSafeCall( hipMemcpy(&totalCount, counterPtr, sizeof(int), hipMemcpyDeviceToHost) );

            totalCount = std::min(totalCount, maxSize);

            return totalCount;
        }

        ////////////////////////////////////////////////////////////////////////
        // Guil_Full

        struct FeatureTable
        {
            uchar* p1_pos_data;
            size_t p1_pos_step;

            uchar* p1_theta_data;
            size_t p1_theta_step;

            uchar* p2_pos_data;
            size_t p2_pos_step;

            uchar* d12_data;
            size_t d12_step;

            uchar* r1_data;
            size_t r1_step;

            uchar* r2_data;
            size_t r2_step;
        };

        __constant__ FeatureTable c_templFeatures;
        __constant__ FeatureTable c_imageFeatures;

        void Guil_Full_setTemplFeatures(PtrStepb p1_pos, PtrStepb p1_theta, PtrStepb p2_pos, PtrStepb d12, PtrStepb r1, PtrStepb r2)
        {
            FeatureTable tbl;

            tbl.p1_pos_data = p1_pos.data;
            tbl.p1_pos_step = p1_pos.step;

            tbl.p1_theta_data = p1_theta.data;
            tbl.p1_theta_step = p1_theta.step;

            tbl.p2_pos_data = p2_pos.data;
            tbl.p2_pos_step = p2_pos.step;

            tbl.d12_data = d12.data;
            tbl.d12_step = d12.step;

            tbl.r1_data = r1.data;
            tbl.r1_step = r1.step;

            tbl.r2_data = r2.data;
            tbl.r2_step = r2.step;

            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_templFeatures), &tbl, sizeof(FeatureTable)) );
        }
        void Guil_Full_setImageFeatures(PtrStepb p1_pos, PtrStepb p1_theta, PtrStepb p2_pos, PtrStepb d12, PtrStepb r1, PtrStepb r2)
        {
            FeatureTable tbl;

            tbl.p1_pos_data = p1_pos.data;
            tbl.p1_pos_step = p1_pos.step;

            tbl.p1_theta_data = p1_theta.data;
            tbl.p1_theta_step = p1_theta.step;

            tbl.p2_pos_data = p2_pos.data;
            tbl.p2_pos_step = p2_pos.step;

            tbl.d12_data = d12.data;
            tbl.d12_step = d12.step;

            tbl.r1_data = r1.data;
            tbl.r1_step = r1.step;

            tbl.r2_data = r2.data;
            tbl.r2_step = r2.step;

            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_imageFeatures), &tbl, sizeof(FeatureTable)) );
        }

        struct TemplFeatureTable
        {
            static __device__ float2* p1_pos(int n)
            {
                return (float2*)(c_templFeatures.p1_pos_data + n * c_templFeatures.p1_pos_step);
            }
            static __device__ float* p1_theta(int n)
            {
                return (float*)(c_templFeatures.p1_theta_data + n * c_templFeatures.p1_theta_step);
            }
            static __device__ float2* p2_pos(int n)
            {
                return (float2*)(c_templFeatures.p2_pos_data + n * c_templFeatures.p2_pos_step);
            }

            static __device__ float* d12(int n)
            {
                return (float*)(c_templFeatures.d12_data + n * c_templFeatures.d12_step);
            }

            static __device__ float2* r1(int n)
            {
                return (float2*)(c_templFeatures.r1_data + n * c_templFeatures.r1_step);
            }
            static __device__ float2* r2(int n)
            {
                return (float2*)(c_templFeatures.r2_data + n * c_templFeatures.r2_step);
            }
        };
        struct ImageFeatureTable
        {
            static __device__ float2* p1_pos(int n)
            {
                return (float2*)(c_imageFeatures.p1_pos_data + n * c_imageFeatures.p1_pos_step);
            }
            static __device__ float* p1_theta(int n)
            {
                return (float*)(c_imageFeatures.p1_theta_data + n * c_imageFeatures.p1_theta_step);
            }
            static __device__ float2* p2_pos(int n)
            {
                return (float2*)(c_imageFeatures.p2_pos_data + n * c_imageFeatures.p2_pos_step);
            }

            static __device__ float* d12(int n)
            {
                return (float*)(c_imageFeatures.d12_data + n * c_imageFeatures.d12_step);
            }

            static __device__ float2* r1(int n)
            {
                return (float2*)(c_imageFeatures.r1_data + n * c_imageFeatures.r1_step);
            }
            static __device__ float2* r2(int n)
            {
                return (float2*)(c_imageFeatures.r2_data + n * c_imageFeatures.r2_step);
            }
        };

        __device__ float clampAngle(float a)
        {
            float res = a;

            while (res > 2.0f * CV_PI_F)
                res -= 2.0f * CV_PI_F;
            while (res < 0.0f)
                res += 2.0f * CV_PI_F;

            return res;
        }

        __device__ bool angleEq(float a, float b, float eps)
        {
            return (::fabs(clampAngle(a - b)) <= eps);
        }

        template <class FT, bool isTempl>
        __global__ void Guil_Full_buildFeatureList(const unsigned int* coordList, const float* thetaList, const int pointsCount,
                                                   int* sizes, const int maxSize,
                                                   const float xi, const float angleEpsilon, const float alphaScale,
                                                   const float2 center, const float maxDist)
        {
            const float p1_theta = thetaList[blockIdx.x];
            const unsigned int coord1 = coordList[blockIdx.x];
            float2 p1_pos;
            p1_pos.x = (coord1 & 0xFFFF);
            p1_pos.y = (coord1 >> 16) & 0xFFFF;

            for (int i = threadIdx.x; i < pointsCount; i += blockDim.x)
            {
                const float p2_theta = thetaList[i];
                const unsigned int coord2 = coordList[i];
                float2 p2_pos;
                p2_pos.x = (coord2 & 0xFFFF);
                p2_pos.y = (coord2 >> 16) & 0xFFFF;

                if (angleEq(p1_theta - p2_theta, xi, angleEpsilon))
                {
                    const float2 d = p1_pos - p2_pos;

                    float alpha12 = clampAngle(::atan2(d.y, d.x) - p1_theta);
                    float d12 = ::sqrtf(d.x * d.x + d.y * d.y);

                    if (d12 > maxDist)
                        continue;

                    float2 r1 = p1_pos - center;
                    float2 r2 = p2_pos - center;

                    const int n = __float2int_rn(alpha12 * alphaScale);

                    const int ind = ::atomicAdd(sizes + n, 1);

                    if (ind < maxSize)
                    {
                        if (!isTempl)
                        {
                            FT::p1_pos(n)[ind] = p1_pos;
                            FT::p2_pos(n)[ind] = p2_pos;
                        }

                        FT::p1_theta(n)[ind] = p1_theta;

                        FT::d12(n)[ind] = d12;

                        if (isTempl)
                        {
                            FT::r1(n)[ind] = r1;
                            FT::r2(n)[ind] = r2;
                        }
                    }
                }
            }
        }

        template <class FT, bool isTempl>
        void Guil_Full_buildFeatureList_caller(const unsigned int* coordList, const float* thetaList, int pointsCount,
                                               int* sizes, int maxSize,
                                               float xi, float angleEpsilon, int levels,
                                               float2 center, float maxDist)
        {
            const dim3 block(256);
            const dim3 grid(pointsCount);

            const float alphaScale = levels / (2.0f * CV_PI_F);

            Guil_Full_buildFeatureList<FT, isTempl><<<grid, block>>>(coordList, thetaList, pointsCount,
                                                                     sizes, maxSize,
                                                                     xi * (CV_PI_F / 180.0f), angleEpsilon * (CV_PI_F / 180.0f), alphaScale,
                                                                     center, maxDist);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );

            thrust::device_ptr<int> sizesPtr(sizes);
            thrust::transform(sizesPtr, sizesPtr + levels + 1, sizesPtr, device::bind2nd(device::minimum<int>(), maxSize));
        }

        void Guil_Full_buildTemplFeatureList_gpu(const unsigned int* coordList, const float* thetaList, int pointsCount,
                                                 int* sizes, int maxSize,
                                                 float xi, float angleEpsilon, int levels,
                                                 float2 center, float maxDist)
        {
            Guil_Full_buildFeatureList_caller<TemplFeatureTable, true>(coordList, thetaList, pointsCount,
                                                                       sizes, maxSize,
                                                                       xi, angleEpsilon, levels,
                                                                       center, maxDist);
        }
        void Guil_Full_buildImageFeatureList_gpu(const unsigned int* coordList, const float* thetaList, int pointsCount,
                                                 int* sizes, int maxSize,
                                                 float xi, float angleEpsilon, int levels,
                                                 float2 center, float maxDist)
        {
            Guil_Full_buildFeatureList_caller<ImageFeatureTable, false>(coordList, thetaList, pointsCount,
                                                                        sizes, maxSize,
                                                                        xi, angleEpsilon, levels,
                                                                        center, maxDist);
        }

        __global__ void Guil_Full_calcOHist(const int* templSizes, const int* imageSizes, int* OHist,
                                            const float minAngle, const float maxAngle, const float iAngleStep, const int angleRange)
        {
            extern __shared__ int s_OHist[];
            for (int i = threadIdx.x; i <= angleRange; i += blockDim.x)
                s_OHist[i] = 0;
            __syncthreads();

            const int tIdx = blockIdx.x;
            const int level = blockIdx.y;

            const int tSize = templSizes[level];

            if (tIdx < tSize)
            {
                const int imSize = imageSizes[level];

                const float t_p1_theta = TemplFeatureTable::p1_theta(level)[tIdx];

                for (int i = threadIdx.x; i < imSize; i += blockDim.x)
                {
                    const float im_p1_theta = ImageFeatureTable::p1_theta(level)[i];

                    const float angle = clampAngle(im_p1_theta - t_p1_theta);

                    if (angle >= minAngle && angle <= maxAngle)
                    {
                        const int n = __float2int_rn((angle - minAngle) * iAngleStep);
                        Emulation::smem::atomicAdd(&s_OHist[n], 1);
                    }
                }
            }
            __syncthreads();

            for (int i = threadIdx.x; i <= angleRange; i += blockDim.x)
                ::atomicAdd(OHist + i, s_OHist[i]);
        }

        void Guil_Full_calcOHist_gpu(const int* templSizes, const int* imageSizes, int* OHist,
                                     float minAngle, float maxAngle, float angleStep, int angleRange,
                                     int levels, int tMaxSize)
        {
            const dim3 block(256);
            const dim3 grid(tMaxSize, levels + 1);

            minAngle *= (CV_PI_F / 180.0f);
            maxAngle *= (CV_PI_F / 180.0f);
            angleStep *= (CV_PI_F / 180.0f);

            const size_t smemSize = (angleRange + 1) * sizeof(float);

            Guil_Full_calcOHist<<<grid, block, smemSize>>>(templSizes, imageSizes, OHist,
                                                           minAngle, maxAngle, 1.0f / angleStep, angleRange);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }

        __global__ void Guil_Full_calcSHist(const int* templSizes, const int* imageSizes, int* SHist,
                                            const float angle, const float angleEpsilon,
                                            const float minScale, const float maxScale, const float iScaleStep, const int scaleRange)
        {
            extern __shared__ int s_SHist[];
            for (int i = threadIdx.x; i <= scaleRange; i += blockDim.x)
                s_SHist[i] = 0;
            __syncthreads();

            const int tIdx = blockIdx.x;
            const int level = blockIdx.y;

            const int tSize = templSizes[level];

            if (tIdx < tSize)
            {
                const int imSize = imageSizes[level];

                const float t_p1_theta = TemplFeatureTable::p1_theta(level)[tIdx] + angle;
                const float t_d12 = TemplFeatureTable::d12(level)[tIdx] + angle;

                for (int i = threadIdx.x; i < imSize; i += blockDim.x)
                {
                    const float im_p1_theta = ImageFeatureTable::p1_theta(level)[i];
                    const float im_d12 = ImageFeatureTable::d12(level)[i];

                    if (angleEq(im_p1_theta, t_p1_theta, angleEpsilon))
                    {
                        const float scale = im_d12 / t_d12;

                        if (scale >= minScale && scale <= maxScale)
                        {
                            const int s = __float2int_rn((scale - minScale) * iScaleStep);
                            Emulation::smem::atomicAdd(&s_SHist[s], 1);
                        }
                    }
                }
            }
            __syncthreads();

            for (int i = threadIdx.x; i <= scaleRange; i += blockDim.x)
                ::atomicAdd(SHist + i, s_SHist[i]);
        }

        void Guil_Full_calcSHist_gpu(const int* templSizes, const int* imageSizes, int* SHist,
                                     float angle, float angleEpsilon,
                                     float minScale, float maxScale, float iScaleStep, int scaleRange,
                                     int levels, int tMaxSize)
        {
            const dim3 block(256);
            const dim3 grid(tMaxSize, levels + 1);

            angle *= (CV_PI_F / 180.0f);
            angleEpsilon *= (CV_PI_F / 180.0f);

            const size_t smemSize = (scaleRange + 1) * sizeof(float);

            Guil_Full_calcSHist<<<grid, block, smemSize>>>(templSizes, imageSizes, SHist,
                                                           angle, angleEpsilon,
                                                           minScale, maxScale, iScaleStep, scaleRange);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }

        __global__ void Guil_Full_calcPHist(const int* templSizes, const int* imageSizes, PtrStepSzi PHist,
                                            const float angle, const float sinVal, const float cosVal, const float angleEpsilon, const float scale,
                                            const float idp)
        {
            const int tIdx = blockIdx.x;
            const int level = blockIdx.y;

            const int tSize = templSizes[level];

            if (tIdx < tSize)
            {
                const int imSize = imageSizes[level];

                const float t_p1_theta = TemplFeatureTable::p1_theta(level)[tIdx] + angle;

                float2 r1 = TemplFeatureTable::r1(level)[tIdx];
                float2 r2 = TemplFeatureTable::r2(level)[tIdx];

                r1 = r1 * scale;
                r2 = r2 * scale;

                r1 = make_float2(cosVal * r1.x - sinVal * r1.y, sinVal * r1.x + cosVal * r1.y);
                r2 = make_float2(cosVal * r2.x - sinVal * r2.y, sinVal * r2.x + cosVal * r2.y);

                for (int i = threadIdx.x; i < imSize; i += blockDim.x)
                {
                    const float im_p1_theta = ImageFeatureTable::p1_theta(level)[i];

                    const float2 im_p1_pos = ImageFeatureTable::p1_pos(level)[i];
                    const float2 im_p2_pos = ImageFeatureTable::p2_pos(level)[i];

                    if (angleEq(im_p1_theta, t_p1_theta, angleEpsilon))
                    {
                        float2 c1, c2;

                        c1 = im_p1_pos - r1;
                        c1 = c1 * idp;

                        c2 = im_p2_pos - r2;
                        c2 = c2 * idp;

                        if (::fabs(c1.x - c2.x) > 1 || ::fabs(c1.y - c2.y) > 1)
                            continue;

                        if (c1.y >= 0 && c1.y < PHist.rows - 2 && c1.x >= 0 && c1.x < PHist.cols - 2)
                            ::atomicAdd(PHist.ptr(__float2int_rn(c1.y) + 1) + __float2int_rn(c1.x) + 1, 1);
                    }
                }
            }
        }

        void Guil_Full_calcPHist_gpu(const int* templSizes, const int* imageSizes, PtrStepSzi PHist,
                                     float angle, float angleEpsilon, float scale,
                                     float dp,
                                     int levels, int tMaxSize)
        {
            const dim3 block(256);
            const dim3 grid(tMaxSize, levels + 1);

            angle *= (CV_PI_F / 180.0f);
            angleEpsilon *= (CV_PI_F / 180.0f);

            const float sinVal = ::sinf(angle);
            const float cosVal = ::cosf(angle);

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(Guil_Full_calcPHist), hipFuncCachePreferL1) );

            Guil_Full_calcPHist<<<grid, block>>>(templSizes, imageSizes, PHist,
                                                 angle, sinVal, cosVal, angleEpsilon, scale,
                                                 1.0f / dp);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }

        __global__ void Guil_Full_findPosInHist(const PtrStepSzi hist, float4* out, int3* votes, const int maxSize,
                                                const float angle, const int angleVotes, const float scale, const int scaleVotes,
                                                const float dp, const int threshold)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x >= hist.cols - 2 || y >= hist.rows - 2)
                return;

            const int curVotes = hist(y + 1, x + 1);

            if (curVotes > threshold &&
                curVotes >  hist(y + 1, x) &&
                curVotes >= hist(y + 1, x + 2) &&
                curVotes >  hist(y, x + 1) &&
                curVotes >= hist(y + 2, x + 1))
            {
                const int ind = ::atomicAdd(&g_counter, 1);

                if (ind < maxSize)
                {
                    out[ind] = make_float4(x * dp, y * dp, scale, angle);
                    votes[ind] = make_int3(curVotes, scaleVotes, angleVotes);
                }
            }
        }

        int Guil_Full_findPosInHist_gpu(PtrStepSzi hist, float4* out, int3* votes, int curSize, int maxSize,
                                        float angle, int angleVotes, float scale, int scaleVotes,
                                        float dp, int threshold)
        {
            void* counterPtr;
            cudaSafeCall( hipGetSymbolAddress(&counterPtr, HIP_SYMBOL(g_counter)) );

            cudaSafeCall( hipMemcpy(counterPtr, &curSize, sizeof(int), hipMemcpyHostToDevice) );

            const dim3 block(32, 8);
            const dim3 grid(divUp(hist.cols - 2, block.x), divUp(hist.rows - 2, block.y));

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(Guil_Full_findPosInHist), hipFuncCachePreferL1) );

            Guil_Full_findPosInHist<<<grid, block>>>(hist, out, votes, maxSize,
                                                     angle, angleVotes, scale, scaleVotes,
                                                     dp, threshold);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );

            int totalCount;
            cudaSafeCall( hipMemcpy(&totalCount, counterPtr, sizeof(int), hipMemcpyDeviceToHost) );

            totalCount = std::min(totalCount, maxSize);

            return totalCount;
        }
    }
}}}

#endif // HAVE_OPENCV_CUDAARITHM

#endif /* CUDA_DISABLER */
