#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <thrust/system/cuda/execution_policy.h>

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/emulation.hpp"
#include "opencv2/core/cuda/dynamic_smem.hpp"

namespace cv { namespace cuda { namespace device
{
    namespace hough_lines
    {
        ////////////////////////////////////////////////////////////////////////
        // linesAccum

        __global__ void linesAccumGlobal(const unsigned int* list, const int count, PtrStepi accum, const float irho, const float theta, const int numrho)
        {
            const int n = blockIdx.x;
            const float ang = n * theta;

            float sinVal;
            float cosVal;
            sincosf(ang, &sinVal, &cosVal);
            sinVal *= irho;
            cosVal *= irho;

            const int shift = (numrho - 1) / 2;

            int* accumRow = accum.ptr(n + 1);
            for (int i = threadIdx.x; i < count; i += blockDim.x)
            {
                const unsigned int val = list[i];

                const int x = (val & 0xFFFF);
                const int y = (val >> 16) & 0xFFFF;

                int r = __float2int_rn(x * cosVal + y * sinVal);
                r += shift;

                ::atomicAdd(accumRow + r + 1, 1);
            }
        }

        __global__ void linesAccumShared(const unsigned int* list, const int count, PtrStepi accum, const float irho, const float theta, const int numrho)
        {
            int* smem = DynamicSharedMem<int>();

            for (int i = threadIdx.x; i < numrho + 1; i += blockDim.x)
                smem[i] = 0;

            __syncthreads();

            const int n = blockIdx.x;
            const float ang = n * theta;

            float sinVal;
            float cosVal;
            sincosf(ang, &sinVal, &cosVal);
            sinVal *= irho;
            cosVal *= irho;

            const int shift = (numrho - 1) / 2;

            for (int i = threadIdx.x; i < count; i += blockDim.x)
            {
                const unsigned int val = list[i];

                const int x = (val & 0xFFFF);
                const int y = (val >> 16) & 0xFFFF;

                int r = __float2int_rn(x * cosVal + y * sinVal);
                r += shift;

                Emulation::smem::atomicAdd(&smem[r + 1], 1);
            }

            __syncthreads();

            int* accumRow = accum.ptr(n + 1);
            for (int i = threadIdx.x; i < numrho + 1; i += blockDim.x)
                accumRow[i] = smem[i];
        }

        void linesAccum_gpu(const unsigned int* list, int count, PtrStepSzi accum, float rho, float theta, size_t sharedMemPerBlock, bool has20, hipStream_t stream)
        {
            const dim3 block(has20 ? 1024 : 512);
            const dim3 grid(accum.rows - 2);

            size_t smemSize = (accum.cols - 1) * sizeof(int);

            if (smemSize < sharedMemPerBlock - 1000)
                linesAccumShared<<<grid, block, smemSize, stream>>>(list, count, accum, 1.0f / rho, theta, accum.cols - 2);
            else
                linesAccumGlobal<<<grid, block, 0, stream>>>(list, count, accum, 1.0f / rho, theta, accum.cols - 2);

            cudaSafeCall( hipGetLastError() );
            cudaSafeCall( hipStreamSynchronize(stream) );
        }

        ////////////////////////////////////////////////////////////////////////
        // linesGetResult

        __global__ void linesGetResult(const PtrStepSzi accum, float2* out, int* votes, const int maxSize, const float rho, const float theta, const int threshold, const int numrho, int* counterPtr)
        {
            const int r = blockIdx.x * blockDim.x + threadIdx.x;
            const int n = blockIdx.y * blockDim.y + threadIdx.y;

            if (r >= accum.cols - 2 || n >= accum.rows - 2)
                return;

            const int curVotes = accum(n + 1, r + 1);

            if (curVotes > threshold &&
                curVotes >  accum(n + 1, r) &&
                curVotes >= accum(n + 1, r + 2) &&
                curVotes >  accum(n, r + 1) &&
                curVotes >= accum(n + 2, r + 1))
            {
                const float radius = (r - (numrho - 1) * 0.5f) * rho;
                const float angle = n * theta;

                const int ind = ::atomicAdd(counterPtr, 1);
                if (ind < maxSize)
                {
                    out[ind] = make_float2(radius, angle);
                    votes[ind] = curVotes;
                }
            }
        }

        int linesGetResult_gpu(PtrStepSzi accum, float2* out, int* votes, int maxSize, float rho, float theta, int threshold, bool doSort, int* counterPtr, hipStream_t stream)
        {
            cudaSafeCall( hipMemsetAsync(counterPtr, 0, sizeof(int), stream) );

            const dim3 block(32, 8);
            const dim3 grid(divUp(accum.cols - 2, block.x), divUp(accum.rows - 2, block.y));

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(linesGetResult), hipFuncCachePreferL1) );

            linesGetResult<<<grid, block, 0, stream>>>(accum, out, votes, maxSize, rho, theta, threshold, accum.cols - 2, counterPtr);
            cudaSafeCall( hipGetLastError() );

            int totalCount;
            cudaSafeCall( hipMemcpyAsync(&totalCount, counterPtr, sizeof(int), hipMemcpyDeviceToHost, stream) );

            cudaSafeCall( hipStreamSynchronize(stream) );

            totalCount = std::min(totalCount, maxSize);

            if (doSort && totalCount > 0)
            {
                thrust::device_ptr<float2> outPtr(out);
                thrust::device_ptr<int> votesPtr(votes);
                thrust::sort_by_key(thrust::cuda::par.on(stream), votesPtr, votesPtr + totalCount, outPtr, thrust::greater<int>());
            }

            return totalCount;
        }
    }
}}}


#endif /* CUDA_DISABLER */
