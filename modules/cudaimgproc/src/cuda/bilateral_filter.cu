#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/vec_traits.hpp"
#include "opencv2/core/cuda/vec_math.hpp"
#include "opencv2/core/cuda/border_interpolate.hpp"

using namespace cv::cuda;

typedef unsigned char uchar;
typedef unsigned short ushort;

//////////////////////////////////////////////////////////////////////////////////
/// Bilateral filtering

namespace cv { namespace cuda { namespace device
{
    namespace imgproc
    {
        __device__ __forceinline__ float norm_l1(const float& a)  { return ::fabs(a); }
        __device__ __forceinline__ float norm_l1(const float2& a) { return ::fabs(a.x) + ::fabs(a.y); }
        __device__ __forceinline__ float norm_l1(const float3& a) { return ::fabs(a.x) + ::fabs(a.y) + ::fabs(a.z); }
        __device__ __forceinline__ float norm_l1(const float4& a) { return ::fabs(a.x) + ::fabs(a.y) + ::fabs(a.z) + ::fabs(a.w); }

        __device__ __forceinline__ float sqr(const float& a)  { return a * a; }

        template<typename T, typename B>
        __global__ void bilateral_kernel(const PtrStepSz<T> src, PtrStep<T> dst, const B b, const int ksz, const float sigma_spatial2_inv_half, const float sigma_color2_inv_half)
        {
            typedef typename TypeVec<float, VecTraits<T>::cn>::vec_type value_type;

            int x = threadIdx.x + blockIdx.x * blockDim.x;
            int y = threadIdx.y + blockIdx.y * blockDim.y;

            if (x >= src.cols || y >= src.rows)
                return;

            value_type center = saturate_cast<value_type>(src(y, x));

            value_type sum1 = VecTraits<value_type>::all(0);
            float sum2 = 0;

            int r = ksz / 2;
            float r2 = (float)(r * r);

            int tx = x - r + ksz;
            int ty = y - r + ksz;

            if (x - ksz/2 >=0 && y - ksz/2 >=0 && tx < src.cols && ty < src.rows)
            {
                for (int cy = y - r; cy < ty; ++cy)
                    for (int cx = x - r; cx < tx; ++cx)
                    {
                        float space2 = (x - cx) * (x - cx) + (y - cy) * (y - cy);
                        if (space2 > r2)
                            continue;

                        value_type value = saturate_cast<value_type>(src(cy, cx));
                        if (isnan(norm_l1(value)))
                            continue;

                        float weight = ::exp(space2 * sigma_spatial2_inv_half + sqr(norm_l1(value - center)) * sigma_color2_inv_half);
                        sum1 = sum1 + weight * value;
                        sum2 = sum2 + weight;
                    }
            }
            else
            {
                for (int cy = y - r; cy < ty; ++cy)
                    for (int cx = x - r; cx < tx; ++cx)
                    {
                        float space2 = (x - cx) * (x - cx) + (y - cy) * (y - cy);
                        if (space2 > r2)
                            continue;

                        value_type value = saturate_cast<value_type>(b.at(cy, cx, src.data, src.step));
                        if (isnan(norm_l1(value)))
                            continue;

                        float weight = ::exp(space2 * sigma_spatial2_inv_half + sqr(norm_l1(value - center)) * sigma_color2_inv_half);

                        sum1 = sum1 + weight * value;
                        sum2 = sum2 + weight;
                    }
            }
            dst(y, x) = saturate_cast<T>(sum1 / sum2);
        }

        template<typename T, template <typename> class B>
        void bilateral_caller(const PtrStepSzb& src, PtrStepSzb dst, int kernel_size, float sigma_spatial, float sigma_color, hipStream_t stream)
        {
            dim3 block (32, 8);
            dim3 grid (divUp (src.cols, block.x), divUp (src.rows, block.y));

            B<T> b(src.rows, src.cols);

            float sigma_spatial2_inv_half = -0.5f/(sigma_spatial * sigma_spatial);
            float sigma_color2_inv_half = -0.5f/(sigma_color * sigma_color);

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(bilateral_kernel<T), B<T> >, hipFuncCachePreferL1) );
            bilateral_kernel<<<grid, block, 0, stream>>>((PtrStepSz<T>)src, (PtrStepSz<T>)dst, b, kernel_size, sigma_spatial2_inv_half, sigma_color2_inv_half);
            cudaSafeCall ( hipGetLastError () );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template<typename T>
        void bilateral_filter_gpu(const PtrStepSzb& src, PtrStepSzb dst, int kernel_size, float gauss_spatial_coeff, float gauss_color_coeff, int borderMode, hipStream_t stream)
        {
            typedef void (*caller_t)(const PtrStepSzb& src, PtrStepSzb dst, int kernel_size, float sigma_spatial, float sigma_color, hipStream_t stream);

            static caller_t funcs[] =
            {
                bilateral_caller<T, BrdConstant>,
                bilateral_caller<T, BrdReplicate>,
                bilateral_caller<T, BrdReflect>,
                bilateral_caller<T, BrdWrap>,
                bilateral_caller<T, BrdReflect101>
            };
            funcs[borderMode](src, dst, kernel_size, gauss_spatial_coeff, gauss_color_coeff, stream);
        }
    }
}}}


#define OCV_INSTANTIATE_BILATERAL_FILTER(T) \
    template void cv::cuda::device::imgproc::bilateral_filter_gpu<T>(const PtrStepSzb&, PtrStepSzb, int, float, float, int, hipStream_t);

OCV_INSTANTIATE_BILATERAL_FILTER(uchar)
//OCV_INSTANTIATE_BILATERAL_FILTER(uchar2)
OCV_INSTANTIATE_BILATERAL_FILTER(uchar3)
OCV_INSTANTIATE_BILATERAL_FILTER(uchar4)

//OCV_INSTANTIATE_BILATERAL_FILTER(schar)
//OCV_INSTANTIATE_BILATERAL_FILTER(schar2)
//OCV_INSTANTIATE_BILATERAL_FILTER(schar3)
//OCV_INSTANTIATE_BILATERAL_FILTER(schar4)

OCV_INSTANTIATE_BILATERAL_FILTER(short)
//OCV_INSTANTIATE_BILATERAL_FILTER(short2)
OCV_INSTANTIATE_BILATERAL_FILTER(short3)
OCV_INSTANTIATE_BILATERAL_FILTER(short4)

OCV_INSTANTIATE_BILATERAL_FILTER(ushort)
//OCV_INSTANTIATE_BILATERAL_FILTER(ushort2)
OCV_INSTANTIATE_BILATERAL_FILTER(ushort3)
OCV_INSTANTIATE_BILATERAL_FILTER(ushort4)

//OCV_INSTANTIATE_BILATERAL_FILTER(int)
//OCV_INSTANTIATE_BILATERAL_FILTER(int2)
//OCV_INSTANTIATE_BILATERAL_FILTER(int3)
//OCV_INSTANTIATE_BILATERAL_FILTER(int4)

OCV_INSTANTIATE_BILATERAL_FILTER(float)
//OCV_INSTANTIATE_BILATERAL_FILTER(float2)
OCV_INSTANTIATE_BILATERAL_FILTER(float3)
OCV_INSTANTIATE_BILATERAL_FILTER(float4)


#endif /* CUDA_DISABLER */
