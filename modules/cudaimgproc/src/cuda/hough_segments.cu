#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/vec_math.hpp"
#include "opencv2/cudev.hpp"

namespace cv { namespace cuda { namespace device
{
    namespace hough_segments
    {
        template<class Ptr2D>
        __global__ void houghLinesProbabilistic(Ptr2D src, const PtrStepSzi accum,
                                                int4* out, const int maxSize,
                                                const float rho, const float theta,
                                                const int lineGap, const int lineLength,
                                                const int threshold,
                                                const int rows, const int cols,
                                                int* counterPtr)
        {
            const int r = blockIdx.x * blockDim.x + threadIdx.x;
            const int n = blockIdx.y * blockDim.y + threadIdx.y;

            if (r >= accum.cols - 2 || n >= accum.rows - 2)
                return;

            const int curVotes = accum(n + 1, r + 1);
            const int threshold_ = (threshold > 0) ? threshold : lineLength;

            if (curVotes >= threshold_ &&
                curVotes > accum(n, r) &&
                curVotes > accum(n, r + 1) &&
                curVotes > accum(n, r + 2) &&
                curVotes > accum(n + 1, r) &&
                curVotes > accum(n + 1, r + 2) &&
                curVotes > accum(n + 2, r) &&
                curVotes > accum(n + 2, r + 1) &&
                curVotes > accum(n + 2, r + 2))
            {
                const float radius = (r - (accum.cols - 2 - 1) * 0.5f) * rho;
                const float angle = n * theta;

                float cosa;
                float sina;
                sincosf(angle, &sina, &cosa);

                float2 p0 = make_float2(cosa * radius, sina * radius);
                float2 dir = make_float2(-sina, cosa);

                float2 pb[4] = {make_float2(-1, -1), make_float2(-1, -1), make_float2(-1, -1), make_float2(-1, -1)};
                float a;

                if (dir.x != 0)
                {
                    a = -p0.x / dir.x;
                    pb[0].x = 0;
                    pb[0].y = p0.y + a * dir.y;

                    a = (cols - 1 - p0.x) / dir.x;
                    pb[1].x = cols - 1;
                    pb[1].y = p0.y + a * dir.y;
                }
                if (dir.y != 0)
                {
                    a = -p0.y / dir.y;
                    pb[2].x = p0.x + a * dir.x;
                    pb[2].y = 0;

                    a = (rows - 1 - p0.y) / dir.y;
                    pb[3].x = p0.x + a * dir.x;
                    pb[3].y = rows - 1;
                }

                if (pb[0].x == 0 && (pb[0].y >= 0 && pb[0].y < rows))
                {
                    p0 = pb[0];
                    if (dir.x < 0)
                        dir = -dir;
                }
                else if (pb[1].x == cols - 1 && (pb[1].y >= 0 && pb[1].y < rows))
                {
                    p0 = pb[1];
                    if (dir.x > 0)
                        dir = -dir;
                }
                else if (pb[2].y == 0 && (pb[2].x >= 0 && pb[2].x < cols))
                {
                    p0 = pb[2];
                    if (dir.y < 0)
                        dir = -dir;
                }
                else if (pb[3].y == rows - 1 && (pb[3].x >= 0 && pb[3].x < cols))
                {
                    p0 = pb[3];
                    if (dir.y > 0)
                        dir = -dir;
                }

                float2 d;
                if (::fabsf(dir.x) > ::fabsf(dir.y))
                {
                    d.x = dir.x > 0 ? 1 : -1;
                    d.y = dir.y / ::fabsf(dir.x);
                }
                else
                {
                    d.x = dir.x / ::fabsf(dir.y);
                    d.y = dir.y > 0 ? 1 : -1;
                }

                float2 line_end[2];
                int gap;
                bool inLine = false;

                float2 p1 = p0;
                if (p1.x < 0 || p1.x >= cols || p1.y < 0 || p1.y >= rows)
                    return;

                for (;;)
                {
                    if (src(p1.y, p1.x))
                    {
                        gap = 0;

                        if (!inLine)
                        {
                            line_end[0] = p1;
                            line_end[1] = p1;
                            inLine = true;
                        }
                        else
                        {
                            line_end[1] = p1;
                        }
                    }
                    else if (inLine)
                    {
                        if (++gap > lineGap)
                        {
                            bool good_line = ::abs(line_end[1].x - line_end[0].x) >= lineLength ||
                                             ::abs(line_end[1].y - line_end[0].y) >= lineLength;

                            if (good_line)
                            {
                                const int ind = ::atomicAdd(counterPtr, 1);
                                if (ind < maxSize)
                                    out[ind] = make_int4(line_end[0].x, line_end[0].y, line_end[1].x, line_end[1].y);
                            }

                            gap = 0;
                            inLine = false;
                        }
                    }

                    p1 = p1 + d;
                    if (p1.x < 0 || p1.x >= cols || p1.y < 0 || p1.y >= rows)
                    {
                        if (inLine)
                        {
                            bool good_line = ::abs(line_end[1].x - line_end[0].x) >= lineLength ||
                                             ::abs(line_end[1].y - line_end[0].y) >= lineLength;

                            if (good_line)
                            {
                                const int ind = ::atomicAdd(counterPtr, 1);
                                if (ind < maxSize)
                                    out[ind] = make_int4(line_end[0].x, line_end[0].y, line_end[1].x, line_end[1].y);
                            }

                        }
                        break;
                    }
                }
            }
        }

        int houghLinesProbabilistic_gpu(GpuMat &mask, PtrStepSzi accum, int4* out, int maxSize, float rho, float theta, int lineGap, int lineLength, int threshold, int* counterPtr, hipStream_t stream)
        {
            cudaSafeCall( hipMemsetAsync(counterPtr, 0, sizeof(int), stream) );

            const dim3 block(32, 8);
            const dim3 grid(divUp(accum.cols - 2, block.x), divUp(accum.rows - 2, block.y));
            
            Size wholeSize;
            Point ofs;
            mask.locateROI(wholeSize, ofs);
            if (ofs.x || ofs.y) {
                cv::cudev::TextureOff<uchar> texMask(wholeSize.height, wholeSize.width, mask.datastart, mask.step, ofs.y, ofs.x);
                houghLinesProbabilistic<cv::cudev::TextureOffPtr<uchar>><<<grid, block, 0, stream>>>(texMask, accum, out, maxSize, rho, theta, lineGap, lineLength, threshold, mask.rows, mask.cols, counterPtr);
            }
            else {
                cv::cudev::Texture<uchar> texMask(mask);
                houghLinesProbabilistic<cv::cudev::TexturePtr<uchar>><<<grid, block, 0, stream>>>(texMask, accum, out, maxSize, rho, theta, lineGap, lineLength, threshold, mask.rows, mask.cols, counterPtr);
            }

            cudaSafeCall( hipGetLastError() );

            int totalCount;
            cudaSafeCall( hipMemcpyAsync(&totalCount, counterPtr, sizeof(int), hipMemcpyDeviceToHost, stream) );

            cudaSafeCall( hipStreamSynchronize(stream) );

            totalCount = std::min(totalCount, maxSize);
            return totalCount;
        }
    }
}}}


#endif /* CUDA_DISABLER */
