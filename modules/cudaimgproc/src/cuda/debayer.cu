#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/vec_traits.hpp"
#include "opencv2/core/cuda/vec_math.hpp"
#include "opencv2/core/cuda/limits.hpp"
#include "opencv2/core/cuda/color.hpp"
#include "opencv2/core/cuda/saturate_cast.hpp"
#include "opencv2/cudev/ptr2d/texture.hpp"

namespace cv { namespace cuda { namespace device
{
    template <typename T> struct Bayer2BGR;

    template <> struct Bayer2BGR<uchar>
    {
        uchar3 res0;
        uchar3 res1;
        uchar3 res2;
        uchar3 res3;

        __device__ void apply(const PtrStepSzb& src, int s_x, int s_y, bool blue_last, bool start_with_green)
        {
            uchar4 patch[3][3];
            patch[0][1] = ((const uchar4*) src.ptr(s_y - 1))[s_x];
            patch[0][0] = ((const uchar4*) src.ptr(s_y - 1))[::max(s_x - 1, 0)];
            patch[0][2] = ((const uchar4*) src.ptr(s_y - 1))[::min(s_x + 1, ((src.cols + 3) >> 2) - 1)];

            patch[1][1] = ((const uchar4*) src.ptr(s_y))[s_x];
            patch[1][0] = ((const uchar4*) src.ptr(s_y))[::max(s_x - 1, 0)];
            patch[1][2] = ((const uchar4*) src.ptr(s_y))[::min(s_x + 1, ((src.cols + 3) >> 2) - 1)];

            patch[2][1] = ((const uchar4*) src.ptr(s_y + 1))[s_x];
            patch[2][0] = ((const uchar4*) src.ptr(s_y + 1))[::max(s_x - 1, 0)];
            patch[2][2] = ((const uchar4*) src.ptr(s_y + 1))[::min(s_x + 1, ((src.cols + 3) >> 2) - 1)];

            if ((s_y & 1) ^ start_with_green)
            {
                const int t0 = (patch[0][1].x + patch[2][1].x + 1) >> 1;
                const int t1 = (patch[1][0].w + patch[1][1].y + 1) >> 1;

                const int t2 = (patch[0][1].x + patch[0][1].z + patch[2][1].x + patch[2][1].z + 2) >> 2;
                const int t3 = (patch[0][1].y + patch[1][1].x + patch[1][1].z + patch[2][1].y + 2) >> 2;

                const int t4 = (patch[0][1].z + patch[2][1].z + 1) >> 1;
                const int t5 = (patch[1][1].y + patch[1][1].w + 1) >> 1;

                const int t6 = (patch[0][1].z + patch[0][2].x + patch[2][1].z + patch[2][2].x + 2) >> 2;
                const int t7 = (patch[0][1].w + patch[1][1].z + patch[1][2].x + patch[2][1].w + 2) >> 2;

                if ((s_y & 1) ^ blue_last)
                {
                    res0.x = t1;
                    res0.y = patch[1][1].x;
                    res0.z = t0;

                    res1.x = patch[1][1].y;
                    res1.y = t3;
                    res1.z = t2;

                    res2.x = t5;
                    res2.y = patch[1][1].z;
                    res2.z = t4;

                    res3.x = patch[1][1].w;
                    res3.y = t7;
                    res3.z = t6;
                }
                else
                {
                    res0.x = t0;
                    res0.y = patch[1][1].x;
                    res0.z = t1;

                    res1.x = t2;
                    res1.y = t3;
                    res1.z = patch[1][1].y;

                    res2.x = t4;
                    res2.y = patch[1][1].z;
                    res2.z = t5;

                    res3.x = t6;
                    res3.y = t7;
                    res3.z = patch[1][1].w;
                }
            }
            else
            {
                const int t0 = (patch[0][0].w + patch[0][1].y + patch[2][0].w + patch[2][1].y + 2) >> 2;
                const int t1 = (patch[0][1].x + patch[1][0].w + patch[1][1].y + patch[2][1].x + 2) >> 2;

                const int t2 = (patch[0][1].y + patch[2][1].y + 1) >> 1;
                const int t3 = (patch[1][1].x + patch[1][1].z + 1) >> 1;

                const int t4 = (patch[0][1].y + patch[0][1].w + patch[2][1].y + patch[2][1].w + 2) >> 2;
                const int t5 = (patch[0][1].z + patch[1][1].y + patch[1][1].w + patch[2][1].z + 2) >> 2;

                const int t6 = (patch[0][1].w + patch[2][1].w + 1) >> 1;
                const int t7 = (patch[1][1].z + patch[1][2].x + 1) >> 1;

                if ((s_y & 1) ^ blue_last)
                {
                    res0.x = patch[1][1].x;
                    res0.y = t1;
                    res0.z = t0;

                    res1.x = t3;
                    res1.y = patch[1][1].y;
                    res1.z = t2;

                    res2.x = patch[1][1].z;
                    res2.y = t5;
                    res2.z = t4;

                    res3.x = t7;
                    res3.y = patch[1][1].w;
                    res3.z = t6;
                }
                else
                {
                    res0.x = t0;
                    res0.y = t1;
                    res0.z = patch[1][1].x;

                    res1.x = t2;
                    res1.y = patch[1][1].y;
                    res1.z = t3;

                    res2.x = t4;
                    res2.y = t5;
                    res2.z = patch[1][1].z;

                    res3.x = t6;
                    res3.y = patch[1][1].w;
                    res3.z = t7;
                }
            }
        }
    };

    template <typename D> __device__ __forceinline__ D toDst(const uchar3& pix);
    template <> __device__ __forceinline__ uchar toDst<uchar>(const uchar3& pix)
    {
        typename bgr_to_gray_traits<uchar>::functor_type f = bgr_to_gray_traits<uchar>::create_functor();
        return f(pix);
    }
    template <> __device__ __forceinline__ uchar3 toDst<uchar3>(const uchar3& pix)
    {
        return pix;
    }
    template <> __device__ __forceinline__ uchar4 toDst<uchar4>(const uchar3& pix)
    {
        return make_uchar4(pix.x, pix.y, pix.z, 255);
    }

    template <typename D>
    __global__ void Bayer2BGR_8u(const PtrStepSzb src, PtrStep<D> dst, const bool blue_last, const bool start_with_green)
    {
        const int s_x = blockIdx.x * blockDim.x + threadIdx.x;
        int s_y = blockIdx.y * blockDim.y + threadIdx.y;

        if (s_y >= src.rows || (s_x << 2) >= src.cols)
            return;

        s_y = ::min(::max(s_y, 1), src.rows - 2);

        Bayer2BGR<uchar> bayer;
        bayer.apply(src, s_x, s_y, blue_last, start_with_green);

        const int d_x = (blockIdx.x * blockDim.x + threadIdx.x) << 2;
        const int d_y = blockIdx.y * blockDim.y + threadIdx.y;

        dst(d_y, d_x) = toDst<D>(bayer.res0);
        if (d_x + 1 < src.cols)
            dst(d_y, d_x + 1) = toDst<D>(bayer.res1);
        if (d_x + 2 < src.cols)
            dst(d_y, d_x + 2) = toDst<D>(bayer.res2);
        if (d_x + 3 < src.cols)
            dst(d_y, d_x + 3) = toDst<D>(bayer.res3);
    }

    template <> struct Bayer2BGR<ushort>
    {
        ushort3 res0;
        ushort3 res1;

        __device__ void apply(const PtrStepSzb& src, int s_x, int s_y, bool blue_last, bool start_with_green)
        {
            ushort2 patch[3][3];
            patch[0][1] = ((const ushort2*) src.ptr(s_y - 1))[s_x];
            patch[0][0] = ((const ushort2*) src.ptr(s_y - 1))[::max(s_x - 1, 0)];
            patch[0][2] = ((const ushort2*) src.ptr(s_y - 1))[::min(s_x + 1, ((src.cols + 1) >> 1) - 1)];

            patch[1][1] = ((const ushort2*) src.ptr(s_y))[s_x];
            patch[1][0] = ((const ushort2*) src.ptr(s_y))[::max(s_x - 1, 0)];
            patch[1][2] = ((const ushort2*) src.ptr(s_y))[::min(s_x + 1, ((src.cols + 1) >> 1) - 1)];

            patch[2][1] = ((const ushort2*) src.ptr(s_y + 1))[s_x];
            patch[2][0] = ((const ushort2*) src.ptr(s_y + 1))[::max(s_x - 1, 0)];
            patch[2][2] = ((const ushort2*) src.ptr(s_y + 1))[::min(s_x + 1, ((src.cols + 1) >> 1) - 1)];

            if ((s_y & 1) ^ start_with_green)
            {
                const int t0 = (patch[0][1].x + patch[2][1].x + 1) >> 1;
                const int t1 = (patch[1][0].y + patch[1][1].y + 1) >> 1;

                const int t2 = (patch[0][1].x + patch[0][2].x + patch[2][1].x + patch[2][2].x + 2) >> 2;
                const int t3 = (patch[0][1].y + patch[1][1].x + patch[1][2].x + patch[2][1].y + 2) >> 2;

                if ((s_y & 1) ^ blue_last)
                {
                    res0.x = t1;
                    res0.y = patch[1][1].x;
                    res0.z = t0;

                    res1.x = patch[1][1].y;
                    res1.y = t3;
                    res1.z = t2;
                }
                else
                {
                    res0.x = t0;
                    res0.y = patch[1][1].x;
                    res0.z = t1;

                    res1.x = t2;
                    res1.y = t3;
                    res1.z = patch[1][1].y;
                }
            }
            else
            {
                const int t0 = (patch[0][0].y + patch[0][1].y + patch[2][0].y + patch[2][1].y + 2) >> 2;
                const int t1 = (patch[0][1].x + patch[1][0].y + patch[1][1].y + patch[2][1].x + 2) >> 2;

                const int t2 = (patch[0][1].y + patch[2][1].y + 1) >> 1;
                const int t3 = (patch[1][1].x + patch[1][2].x + 1) >> 1;

                if ((s_y & 1) ^ blue_last)
                {
                    res0.x = patch[1][1].x;
                    res0.y = t1;
                    res0.z = t0;

                    res1.x = t3;
                    res1.y = patch[1][1].y;
                    res1.z = t2;
                }
                else
                {
                    res0.x = t0;
                    res0.y = t1;
                    res0.z = patch[1][1].x;

                    res1.x = t2;
                    res1.y = patch[1][1].y;
                    res1.z = t3;
                }
            }
        }
    };

    template <typename D> __device__ __forceinline__ D toDst(const ushort3& pix);
    template <> __device__ __forceinline__ ushort toDst<ushort>(const ushort3& pix)
    {
        typename bgr_to_gray_traits<ushort>::functor_type f = bgr_to_gray_traits<ushort>::create_functor();
        return f(pix);
    }
    template <> __device__ __forceinline__ ushort3 toDst<ushort3>(const ushort3& pix)
    {
        return pix;
    }
    template <> __device__ __forceinline__ ushort4 toDst<ushort4>(const ushort3& pix)
    {
        return make_ushort4(pix.x, pix.y, pix.z, numeric_limits<ushort>::max());
    }

    template <typename D>
    __global__ void Bayer2BGR_16u(const PtrStepSzb src, PtrStep<D> dst, const bool blue_last, const bool start_with_green)
    {
        const int s_x = blockIdx.x * blockDim.x + threadIdx.x;
        int s_y = blockIdx.y * blockDim.y + threadIdx.y;

        if (s_y >= src.rows || (s_x << 1) >= src.cols)
            return;

        s_y = ::min(::max(s_y, 1), src.rows - 2);

        Bayer2BGR<ushort> bayer;
        bayer.apply(src, s_x, s_y, blue_last, start_with_green);

        const int d_x = (blockIdx.x * blockDim.x + threadIdx.x) << 1;
        const int d_y = blockIdx.y * blockDim.y + threadIdx.y;

        dst(d_y, d_x) = toDst<D>(bayer.res0);
        if (d_x + 1 < src.cols)
            dst(d_y, d_x + 1) = toDst<D>(bayer.res1);
    }

    template <int cn>
    void Bayer2BGR_8u_gpu(PtrStepSzb src, PtrStepSzb dst, bool blue_last, bool start_with_green, hipStream_t stream)
    {
        typedef typename TypeVec<uchar, cn>::vec_type dst_t;

        const dim3 block(32, 8);
        const dim3 grid(divUp(src.cols, 4 * block.x), divUp(src.rows, block.y));

        cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(Bayer2BGR_8u<dst_t>), hipFuncCachePreferL1) );

        Bayer2BGR_8u<dst_t><<<grid, block, 0, stream>>>(src, (PtrStepSz<dst_t>)dst, blue_last, start_with_green);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    template <int cn>
    void Bayer2BGR_16u_gpu(PtrStepSzb src, PtrStepSzb dst, bool blue_last, bool start_with_green, hipStream_t stream)
    {
        typedef typename TypeVec<ushort, cn>::vec_type dst_t;

        const dim3 block(32, 8);
        const dim3 grid(divUp(src.cols, 2 * block.x), divUp(src.rows, block.y));

        cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(Bayer2BGR_16u<dst_t>), hipFuncCachePreferL1) );

        Bayer2BGR_16u<dst_t><<<grid, block, 0, stream>>>(src, (PtrStepSz<dst_t>)dst, blue_last, start_with_green);
        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    template void Bayer2BGR_8u_gpu<1>(PtrStepSzb src, PtrStepSzb dst, bool blue_last, bool start_with_green, hipStream_t stream);
    template void Bayer2BGR_8u_gpu<3>(PtrStepSzb src, PtrStepSzb dst, bool blue_last, bool start_with_green, hipStream_t stream);
    template void Bayer2BGR_8u_gpu<4>(PtrStepSzb src, PtrStepSzb dst, bool blue_last, bool start_with_green, hipStream_t stream);

    template void Bayer2BGR_16u_gpu<1>(PtrStepSzb src, PtrStepSzb dst, bool blue_last, bool start_with_green, hipStream_t stream);
    template void Bayer2BGR_16u_gpu<3>(PtrStepSzb src, PtrStepSzb dst, bool blue_last, bool start_with_green, hipStream_t stream);
    template void Bayer2BGR_16u_gpu<4>(PtrStepSzb src, PtrStepSzb dst, bool blue_last, bool start_with_green, hipStream_t stream);

    //////////////////////////////////////////////////////////////
    // Bayer Demosaicing (Malvar, He, and Cutler)
    //
    // by Morgan McGuire, Williams College
    // http://graphics.cs.williams.edu/papers/BayerJGT09/#shaders
    //
    // ported to CUDA

    template<typename Depth> __device__
    typename TypeVec<Depth, 3>::vec_type make_3(Depth x, Depth y, Depth z);

    template<> __device__ TypeVec<uchar, 3>::vec_type make_3<uchar>(uchar x, uchar y, uchar z) {
        return make_uchar3(x, y, z);
    }

    template<> __device__ TypeVec<ushort, 3>::vec_type make_3<ushort>(ushort x, ushort y, ushort z) {
        return make_ushort3(x, y, z);
    }

    template <typename DstType, class Ptr2D>
    __global__ void MHCdemosaic(PtrStepSz<DstType> dst, Ptr2D src, const int2 firstRed)
    {
        const float   kAx = -1.0f / 8.0f,     kAy = -1.5f / 8.0f,     kAz =  0.5f / 8.0f    /*kAw = -1.0f / 8.0f*/;
        const float   kBx =  2.0f / 8.0f,   /*kBy =  0.0f / 8.0f,*/ /*kBz =  0.0f / 8.0f,*/   kBw =  4.0f / 8.0f  ;
        const float   kCx =  4.0f / 8.0f,     kCy =  6.0f / 8.0f,     kCz =  5.0f / 8.0f    /*kCw =  5.0f / 8.0f*/;
        const float /*kDx =  0.0f / 8.0f,*/   kDy =  2.0f / 8.0f,     kDz = -1.0f / 8.0f    /*kDw = -1.0f / 8.0f*/;
        const float   kEx = -1.0f / 8.0f,     kEy = -1.5f / 8.0f,   /*kEz = -1.0f / 8.0f,*/   kEw =  0.5f / 8.0f  ;
        const float   kFx =  2.0f / 8.0f,   /*kFy =  0.0f / 8.0f,*/   kFz =  4.0f / 8.0f    /*kFw =  0.0f / 8.0f*/;

        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x == 0 || x >= dst.cols - 1 || y == 0 || y >= dst.rows - 1)
            return;

        int2 center;
        center.x = x;
        center.y = y;

        int4 xCoord;
        xCoord.x = center.x - 2;
        xCoord.y = center.x - 1;
        xCoord.z = center.x + 1;
        xCoord.w = center.x + 2;

        int4 yCoord;
        yCoord.x = center.y - 2;
        yCoord.y = center.y - 1;
        yCoord.z = center.y + 1;
        yCoord.w = center.y + 2;

        float C = src(center.y, center.x); // ( 0, 0)

        float4 Dvec;
        Dvec.x = src(yCoord.y, xCoord.y); // (-1,-1)
        Dvec.y = src(yCoord.z, xCoord.y); // (-1, 1)
        Dvec.z = src(yCoord.y, xCoord.z); // ( 1,-1)
        Dvec.w = src(yCoord.z, xCoord.z); // ( 1, 1)


        float4 value;
        value.x = src(yCoord.x, center.x); // ( 0,-2) A0
        value.y = src(yCoord.y, center.x); // ( 0,-1) B0
        value.z = src(center.y, xCoord.x); // (-2, 0) E0
        value.w = src(center.y, xCoord.y); // (-1, 0) F0

        // (A0 + A1), (B0 + B1), (E0 + E1), (F0 + F1)
        value.x += src(yCoord.w, center.x); // ( 0, 2) A1
        value.y += src(yCoord.z, center.x); // ( 0, 1) B1
        value.z += src(center.y, xCoord.w); // ( 2, 0) E1
        value.w += src(center.y, xCoord.z); // ( 1, 0) F1

        float4 PATTERN;
        PATTERN.x = kCx * C;
        PATTERN.y = kCy * C;
        PATTERN.z = kCz * C;
        PATTERN.w = PATTERN.z;

        float D = Dvec.x + Dvec.y + Dvec.z + Dvec.w;

        // There are five filter patterns (identity, cross, checker,
        // theta, phi). Precompute the terms from all of them and then
        // use swizzles to assign to color channels.
        //
        // Channel Matches
        // x cross (e.g., EE G)
        // y checker (e.g., EE B)
        // z theta (e.g., EO R)
        // w phi (e.g., EO B)

        #define A value.x  // A0 + A1
        #define B value.y  // B0 + B1
        #define E value.z  // E0 + E1
        #define F value.w  // F0 + F1

        float3 temp;

        // PATTERN.yzw += (kD.yz * D).xyy;
        temp.x = kDy * D;
        temp.y = kDz * D;
        PATTERN.y += temp.x;
        PATTERN.z += temp.y;
        PATTERN.w += temp.y;

        // PATTERN += (kA.xyz * A).xyzx;
        temp.x = kAx * A;
        temp.y = kAy * A;
        temp.z = kAz * A;
        PATTERN.x += temp.x;
        PATTERN.y += temp.y;
        PATTERN.z += temp.z;
        PATTERN.w += temp.x;

        // PATTERN += (kE.xyw * E).xyxz;
        temp.x = kEx * E;
        temp.y = kEy * E;
        temp.z = kEw * E;
        PATTERN.x += temp.x;
        PATTERN.y += temp.y;
        PATTERN.z += temp.x;
        PATTERN.w += temp.z;

        // PATTERN.xw += kB.xw * B;
        PATTERN.x += kBx * B;
        PATTERN.w += kBw * B;

        // PATTERN.xz += kF.xz * F;
        PATTERN.x += kFx * F;
        PATTERN.z += kFz * F;

        // Determine which of four types of pixels we are on.
        int2 alternate;
        alternate.x = (x + firstRed.x) % 2;
        alternate.y = (y + firstRed.y) % 2;

        typedef typename VecTraits<DstType>::elem_type SrcElemType;
        typedef typename TypeVec<SrcElemType, 3>::vec_type SrcType;

        SrcType pixelColor =
            (alternate.y == 0) ?
                ((alternate.x == 0) ?
                    make_3<SrcElemType>(saturate_cast<SrcElemType>(PATTERN.y), saturate_cast<SrcElemType>(PATTERN.x), saturate_cast<SrcElemType>(C)) :
                    make_3<SrcElemType>(saturate_cast<SrcElemType>(PATTERN.w), saturate_cast<SrcElemType>(C), saturate_cast<SrcElemType>(PATTERN.z))) :
                ((alternate.x == 0) ?
                    make_3<SrcElemType>(saturate_cast<SrcElemType>(PATTERN.z), saturate_cast<SrcElemType>(C), saturate_cast<SrcElemType>(PATTERN.w)) :
                    make_3<SrcElemType>(saturate_cast<SrcElemType>(C), saturate_cast<SrcElemType>(PATTERN.x), saturate_cast<SrcElemType>(PATTERN.y)));

        dst(y, x) = toDst<DstType>(pixelColor);
    }

    template <int cn, typename Depth>
    void MHCdemosaic(PtrStepSz<Depth> src, int2 sourceOffset, PtrStepSz<Depth> dst, int2 firstRed, hipStream_t stream)
    {
        typedef typename TypeVec<Depth, cn>::vec_type dst_t;

        const dim3 block(32, 8);
        const dim3 grid(divUp(src.cols, block.x), divUp(src.rows, block.y));

        if (sourceOffset.x || sourceOffset.y) {
            cv::cudev::TextureOff<Depth> texSrc(src, sourceOffset.y, sourceOffset.x);
            MHCdemosaic<dst_t, cv::cudev::TextureOffPtr<Depth>><<<grid, block, 0, stream>>>((PtrStepSz<dst_t>)dst, texSrc, firstRed);
        }
        else {
            cv::cudev::Texture<Depth> texSrc(src);
            MHCdemosaic<dst_t, cv::cudev::TexturePtr<Depth>><<<grid, block, 0, stream>>>((PtrStepSz<dst_t>)dst, texSrc, firstRed);
        }

        cudaSafeCall( hipGetLastError() );

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    template void MHCdemosaic<1, uchar>(PtrStepSzb src, int2 sourceOffset, PtrStepSzb dst, int2 firstRed, hipStream_t stream);
    template void MHCdemosaic<3, uchar>(PtrStepSzb src, int2 sourceOffset, PtrStepSzb dst, int2 firstRed, hipStream_t stream);
    template void MHCdemosaic<4, uchar>(PtrStepSzb src, int2 sourceOffset, PtrStepSzb dst, int2 firstRed, hipStream_t stream);
    template void MHCdemosaic<1, ushort>(PtrStepSz<ushort> src, int2 sourceOffset, PtrStepSz<ushort> dst, int2 firstRed, hipStream_t stream);
    template void MHCdemosaic<3, ushort>(PtrStepSz<ushort> src, int2 sourceOffset, PtrStepSz<ushort> dst, int2 firstRed, hipStream_t stream);
    template void MHCdemosaic<4, ushort>(PtrStepSz<ushort> src, int2 sourceOffset, PtrStepSz<ushort> dst, int2 firstRed, hipStream_t stream);
}}}

#endif /* CUDA_DISABLER */
