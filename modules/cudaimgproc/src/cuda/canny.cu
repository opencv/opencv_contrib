#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/emulation.hpp"
#include "opencv2/core/cuda/transform.hpp"
#include "opencv2/core/cuda/functional.hpp"
#include "opencv2/core/cuda/utility.hpp"
#include "opencv2/core/cuda.hpp"
#include <opencv2/cudev/ptr2d/texture.hpp>

using namespace cv::cuda;
using namespace cv::cuda::device;

namespace canny
{
    struct L1 : binary_function<int, int, float>
    {
        __device__ __forceinline__ float operator ()(int x, int y) const
        {
            return ::abs(x) + ::abs(y);
        }

        __host__ __device__ __forceinline__ L1() {}
        __host__ __device__ __forceinline__ L1(const L1&) {}
    };
    struct L2 : binary_function<int, int, float>
    {
        __device__ __forceinline__ float operator ()(int x, int y) const
        {
            return ::sqrtf(x * x + y * y);
        }

        __host__ __device__ __forceinline__ L2() {}
        __host__ __device__ __forceinline__ L2(const L2&) {}
    };
}

namespace cv { namespace cuda { namespace device
{
    template <> struct TransformFunctorTraits<canny::L1> : DefaultTransformFunctorTraits<canny::L1>
    {
        enum { smart_shift = 4 };
    };
    template <> struct TransformFunctorTraits<canny::L2> : DefaultTransformFunctorTraits<canny::L2>
    {
        enum { smart_shift = 4 };
    };
}}}

namespace canny
{
    template <class Norm>
    __global__ void calcMagnitudeKernel(cv::cudev::TextureOffPtr<uchar> texSrc, PtrStepi dx, PtrStepi dy, PtrStepSzf mag, const Norm norm)
    {
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (y >= mag.rows || x >= mag.cols)
            return;

        int dxVal = (texSrc(y - 1, x + 1) + 2 * texSrc(y, x + 1) + texSrc(y + 1, x + 1)) - (texSrc(y - 1, x - 1) + 2 * texSrc(y, x - 1) + texSrc(y + 1, x - 1));
        int dyVal = (texSrc(y + 1, x - 1) + 2 * texSrc(y + 1, x) + texSrc(y + 1, x + 1)) - (texSrc(y - 1, x - 1) + 2 * texSrc(y - 1, x) + texSrc(y - 1, x + 1));

        dx(y, x) = dxVal;
        dy(y, x) = dyVal;

        mag(y, x) = norm(dxVal, dyVal);
    }

    void calcMagnitude(PtrStepSzb srcWhole, int xoff, int yoff, PtrStepSzi dx, PtrStepSzi dy, PtrStepSzf mag, bool L2Grad, hipStream_t stream)
    {
        const dim3 block(16, 16);
        const dim3 grid(divUp(mag.cols, block.x), divUp(mag.rows, block.y));
        cv::cudev::TextureOff<uchar> texSrc(srcWhole, yoff, xoff);
        if (L2Grad)
        {
            L2 norm;
            calcMagnitudeKernel<<<grid, block, 0, stream>>>(texSrc, dx, dy, mag, norm);
        }
        else
        {
            L1 norm;
            calcMagnitudeKernel<<<grid, block, 0, stream>>>(texSrc, dx, dy, mag, norm);
        }

        if (stream == NULL)
            cudaSafeCall(hipDeviceSynchronize());
    }

    void calcMagnitude(PtrStepSzi dx, PtrStepSzi dy, PtrStepSzf mag, bool L2Grad, hipStream_t stream)
    {
        if (L2Grad)
        {
            L2 norm;
            transform(dx, dy, mag, norm, WithOutMask(), stream);
        }
        else
        {
            L1 norm;
            transform(dx, dy, mag, norm, WithOutMask(), stream);
        }
    }
}

//////////////////////////////////////////////////////////////////////////////////////////

namespace canny
{
    __global__ void calcMapKernel(cv::cudev::TexturePtr<float> texMag, const PtrStepSzi dx, const PtrStepi dy, PtrStepi map, const float low_thresh, const float high_thresh)
    {
        const int CANNY_SHIFT = 15;
        const int TG22 = (int)(0.4142135623730950488016887242097*(1<<CANNY_SHIFT) + 0.5);

        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x == 0 || x >= dx.cols - 1 || y == 0 || y >= dx.rows - 1)
            return;

        int dxVal = dx(y, x);
        int dyVal = dy(y, x);

        const int s = (dxVal ^ dyVal) < 0 ? -1 : 1;
        const float m = texMag(y, x);

        dxVal = ::abs(dxVal);
        dyVal = ::abs(dyVal);

        // 0 - the pixel can not belong to an edge
        // 1 - the pixel might belong to an edge
        // 2 - the pixel does belong to an edge
        int edge_type = 0;

        if (m > low_thresh)
        {
            const int tg22x = dxVal * TG22;
            const int tg67x = tg22x + ((dxVal + dxVal) << CANNY_SHIFT);

            dyVal <<= CANNY_SHIFT;

            if (dyVal < tg22x)
            {
                if (m > texMag(y, x - 1) && m >= texMag(y, x + 1))
                    edge_type = 1 + (int)(m > high_thresh);
            }
            else if(dyVal > tg67x)
            {
                if (m > texMag(y - 1, x) && m >= texMag(y + 1, x))
                    edge_type = 1 + (int)(m > high_thresh);
            }
            else
            {
                if (m > texMag(y - 1, x - s) && m >= texMag(y + 1, x + s))
                    edge_type = 1 + (int)(m > high_thresh);
            }
        }

        map(y, x) = edge_type;
    }

    void calcMap(PtrStepSzi dx, PtrStepSzi dy, PtrStepSzf mag, PtrStepSzi map, float low_thresh, float high_thresh, hipStream_t stream)
    {
        const dim3 block(16, 16);
        const dim3 grid(divUp(dx.cols, block.x), divUp(dx.rows, block.y));
        cv::cudev::Texture<float> texMag(mag);
        calcMapKernel<<<grid, block, 0, stream>>>(texMag, dx, dy, map, low_thresh, high_thresh);
        if (stream == NULL)
            cudaSafeCall( hipDeviceSynchronize() );
    }
}

//////////////////////////////////////////////////////////////////////////////////////////

namespace canny
{
    __device__ __forceinline__ bool checkIdx(int y, int x, int rows, int cols)
    {
        return (y >= 0) && (y < rows) && (x >= 0) && (x < cols);
    }

    __global__ void edgesHysteresisLocalKernel(PtrStepSzi map, short2* st, int* d_counter)
    {
        __shared__ volatile int smem[18][18];

        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        smem[threadIdx.y + 1][threadIdx.x + 1] = checkIdx(y, x, map.rows, map.cols) ? map(y, x) : 0;
        if (threadIdx.y == 0)
            smem[0][threadIdx.x + 1] = checkIdx(y - 1, x, map.rows, map.cols) ? map(y - 1, x) : 0;
        if (threadIdx.y == blockDim.y - 1)
            smem[blockDim.y + 1][threadIdx.x + 1] = checkIdx(y + 1, x, map.rows, map.cols) ? map(y + 1, x) : 0;
        if (threadIdx.x == 0)
            smem[threadIdx.y + 1][0] = checkIdx(y, x - 1, map.rows, map.cols) ? map(y, x - 1) : 0;
        if (threadIdx.x == blockDim.x - 1)
            smem[threadIdx.y + 1][blockDim.x + 1] = checkIdx(y, x + 1, map.rows, map.cols) ? map(y, x + 1) : 0;
        if (threadIdx.x == 0 && threadIdx.y == 0)
            smem[0][0] = checkIdx(y - 1, x - 1, map.rows, map.cols) ? map(y - 1, x - 1) : 0;
        if (threadIdx.x == blockDim.x - 1 && threadIdx.y == 0)
            smem[0][blockDim.x + 1] = checkIdx(y - 1, x + 1, map.rows, map.cols) ? map(y - 1, x + 1) : 0;
        if (threadIdx.x == 0 && threadIdx.y == blockDim.y - 1)
            smem[blockDim.y + 1][0] = checkIdx(y + 1, x - 1, map.rows, map.cols) ? map(y + 1, x - 1) : 0;
        if (threadIdx.x == blockDim.x - 1 && threadIdx.y == blockDim.y - 1)
            smem[blockDim.y + 1][blockDim.x + 1] = checkIdx(y + 1, x + 1, map.rows, map.cols) ? map(y + 1, x + 1) : 0;

        __syncthreads();

        if (x >= map.cols || y >= map.rows)
            return;

        int n;

        #pragma unroll
        for (int k = 0; k < 16; ++k)
        {
            n = 0;

            if (smem[threadIdx.y + 1][threadIdx.x + 1] == 1)
            {
                n += smem[threadIdx.y    ][threadIdx.x    ] == 2;
                n += smem[threadIdx.y    ][threadIdx.x + 1] == 2;
                n += smem[threadIdx.y    ][threadIdx.x + 2] == 2;

                n += smem[threadIdx.y + 1][threadIdx.x    ] == 2;
                n += smem[threadIdx.y + 1][threadIdx.x + 2] == 2;

                n += smem[threadIdx.y + 2][threadIdx.x    ] == 2;
                n += smem[threadIdx.y + 2][threadIdx.x + 1] == 2;
                n += smem[threadIdx.y + 2][threadIdx.x + 2] == 2;
            }

            __syncthreads();

            if (n > 0)
                smem[threadIdx.y + 1][threadIdx.x + 1] = 2;

            __syncthreads();
        }

        const int e = smem[threadIdx.y + 1][threadIdx.x + 1];

        map(y, x) = e;

        n = 0;

        if (e == 2)
        {
            n += smem[threadIdx.y    ][threadIdx.x    ] == 1;
            n += smem[threadIdx.y    ][threadIdx.x + 1] == 1;
            n += smem[threadIdx.y    ][threadIdx.x + 2] == 1;

            n += smem[threadIdx.y + 1][threadIdx.x    ] == 1;
            n += smem[threadIdx.y + 1][threadIdx.x + 2] == 1;

            n += smem[threadIdx.y + 2][threadIdx.x    ] == 1;
            n += smem[threadIdx.y + 2][threadIdx.x + 1] == 1;
            n += smem[threadIdx.y + 2][threadIdx.x + 2] == 1;
        }

        if (n > 0)
        {
            const int ind =  ::atomicAdd(d_counter, 1);
            st[ind] = make_short2(x, y);
        }
    }

    void edgesHysteresisLocal(PtrStepSzi map, short2* st1, int* d_counter, hipStream_t stream)
    {
        cudaSafeCall( hipMemsetAsync(d_counter, 0, sizeof(int), stream) );

        const dim3 block(16, 16);
        const dim3 grid(divUp(map.cols, block.x), divUp(map.rows, block.y));

        edgesHysteresisLocalKernel<<<grid, block, 0, stream>>>(map, st1, d_counter);
        cudaSafeCall( hipGetLastError() );

        if (stream == NULL)
            cudaSafeCall( hipDeviceSynchronize() );
    }
}

//////////////////////////////////////////////////////////////////////////////////////////

namespace canny
{
    __constant__ int c_dx[8] = {-1,  0,  1, -1, 1, -1, 0, 1};
    __constant__ int c_dy[8] = {-1, -1, -1,  0, 0,  1, 1, 1};

    __global__ void edgesHysteresisGlobalKernel(PtrStepSzi map, short2* st1, short2* st2, int* d_counter, const int count)
    {
        const int stack_size = 512;

        __shared__ int s_counter;
        __shared__ int s_ind;
        __shared__ short2 s_st[stack_size];

        if (threadIdx.x == 0)
            s_counter = 0;

        __syncthreads();

        int ind = blockIdx.y * gridDim.x + blockIdx.x;

        if (ind >= count)
            return;

        short2 pos = st1[ind];

        if (threadIdx.x < 8)
        {
            pos.x += c_dx[threadIdx.x];
            pos.y += c_dy[threadIdx.x];

            if (pos.x > 0 && pos.x < map.cols - 1 && pos.y > 0 && pos.y < map.rows - 1 && map(pos.y, pos.x) == 1)
            {
                map(pos.y, pos.x) = 2;

                ind = Emulation::smem::atomicAdd(&s_counter, 1);

                s_st[ind] = pos;
            }
        }

        __syncthreads();

        while (s_counter > 0 && s_counter <= stack_size - blockDim.x)
        {
            const int subTaskIdx = threadIdx.x >> 3;
            const int portion = ::min(s_counter, blockDim.x >> 3);

            if (subTaskIdx < portion)
                pos = s_st[s_counter - 1 - subTaskIdx];

            __syncthreads();

            if (threadIdx.x == 0)
                s_counter -= portion;

            __syncthreads();

            if (subTaskIdx < portion)
            {
                pos.x += c_dx[threadIdx.x & 7];
                pos.y += c_dy[threadIdx.x & 7];

                if (pos.x > 0 && pos.x < map.cols - 1 && pos.y > 0 && pos.y < map.rows - 1 && map(pos.y, pos.x) == 1)
                {
                    map(pos.y, pos.x) = 2;

                    ind = Emulation::smem::atomicAdd(&s_counter, 1);

                    s_st[ind] = pos;
                }
            }

            __syncthreads();
        }

        if (s_counter > 0)
        {
            if (threadIdx.x == 0)
            {
                s_ind = ::atomicAdd(d_counter, s_counter);

                if (s_ind + s_counter > map.cols * map.rows)
                    s_counter = 0;
            }

            __syncthreads();

            ind = s_ind;

            for (int i = threadIdx.x; i < s_counter; i += blockDim.x)
                st2[ind + i] = s_st[i];
        }
    }

    void edgesHysteresisGlobal(PtrStepSzi map, short2* st1, short2* st2, int* d_counter, hipStream_t stream)
    {
        int count;
        cudaSafeCall( hipMemcpyAsync(&count, d_counter, sizeof(int), hipMemcpyDeviceToHost, stream) );
        cudaSafeCall( hipStreamSynchronize(stream) );

        while (count > 0)
        {
            cudaSafeCall( hipMemsetAsync(d_counter, 0, sizeof(int), stream) );

            const dim3 block(128);
            const dim3 grid(std::min(count, 65535), divUp(count, 65535), 1);

            edgesHysteresisGlobalKernel<<<grid, block, 0, stream>>>(map, st1, st2, d_counter, count);
            cudaSafeCall( hipGetLastError() );

            if (stream == NULL)
                cudaSafeCall( hipDeviceSynchronize() );

            cudaSafeCall( hipMemcpyAsync(&count, d_counter, sizeof(int), hipMemcpyDeviceToHost, stream) );
            cudaSafeCall( hipStreamSynchronize(stream) );

            count = std::min(count, map.cols * map.rows);

            //std::swap(st1, st2);
            short2* tmp = st1;
            st1 = st2;
            st2 = tmp;
        }
    }
}

//////////////////////////////////////////////////////////////////////////////////////////

namespace canny
{
    struct GetEdges : unary_function<int, uchar>
    {
        __device__ __forceinline__ uchar operator ()(int e) const
        {
            return (uchar)(-(e >> 1));
        }

        __host__ __device__ __forceinline__ GetEdges() {}
        __host__ __device__ __forceinline__ GetEdges(const GetEdges&) {}
    };
}

namespace cv { namespace cuda { namespace device
{
    template <> struct TransformFunctorTraits<canny::GetEdges> : DefaultTransformFunctorTraits<canny::GetEdges>
    {
        enum { smart_shift = 4 };
    };
}}}

namespace canny
{
    void getEdges(PtrStepSzi map, PtrStepSzb dst, hipStream_t stream)
    {
        transform(map, dst, GetEdges(), WithOutMask(), stream);
    }
}

#endif /* CUDA_DISABLER */
