#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "opencv2/opencv_modules.hpp"

#if defined(HAVE_OPENCV_CUDAARITHM) && defined(OPENCV_ENABLE_NONFREE)

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/limits.hpp"
#include "opencv2/core/cuda/saturate_cast.hpp"
#include "opencv2/core/cuda/reduce.hpp"
#include "opencv2/core/cuda/utility.hpp"
#include "opencv2/core/cuda/functional.hpp"
#include "opencv2/core/cuda/filters.hpp"
#include <opencv2/cudev/ptr2d/texture.hpp>

namespace cv { namespace cuda { namespace device
{
    namespace surf
    {
        void loadGlobalConstants(int maxCandidates, int maxFeatures, int img_rows, int img_cols, int nOctaveLayers, float hessianThreshold);
        void loadOctaveConstants(int octave, int layer_rows, int layer_cols);

        void icvCalcLayerDetAndTrace_gpu(const PtrStepSz<unsigned int>& sum, const PtrStepf& det, const PtrStepf& trace, int img_rows, int img_cols,
            int octave, int nOctaveLayer);

        void icvFindMaximaInLayer_gpu(const PtrStepSz<unsigned int>& maskSum, const PtrStepf& det, const PtrStepf& trace, int4* maxPosBuffer, unsigned int* maxCounter,
            int img_rows, int img_cols, int octave, bool use_mask, int nLayers);

        void icvInterpolateKeypoint_gpu(const PtrStepf& det, const int4* maxPosBuffer, unsigned int maxCounter,
            float* featureX, float* featureY, int* featureLaplacian, int* featureOctave, float* featureSize, float* featureHessian,
            unsigned int* featureCounter);

        void icvCalcOrientation_gpu(const PtrStepSz<unsigned int>& sum, const float* featureX, const float* featureY, const float* featureSize, float* featureDir, int nFeatures);

        void compute_descriptors_gpu(const PtrStepSzb& img, PtrStepSz<float4> descriptors, const float* featureX, const float* featureY, const float* featureSize, const float* featureDir, int nFeatures);
    }
}}}

namespace cv { namespace cuda { namespace device
{
    namespace surf
    {
        ////////////////////////////////////////////////////////////////////////
        // Global parameters

        // The maximum number of features (before subpixel interpolation) that memory is reserved for.
        __constant__ int c_max_candidates;
        // The maximum number of features that memory is reserved for.
        __constant__ int c_max_features;
        // The image size.
        __constant__ int c_img_rows;
        __constant__ int c_img_cols;
        // The number of layers.
        __constant__ int c_nOctaveLayers;
        // The hessian threshold.
        __constant__ float c_hessianThreshold;

        // The current octave.
        __constant__ int c_octave;
        // The current layer size.
        __constant__ int c_layer_rows;
        __constant__ int c_layer_cols;

        void loadGlobalConstants(int maxCandidates, int maxFeatures, int img_rows, int img_cols, int nOctaveLayers, float hessianThreshold)
        {
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_max_candidates), &maxCandidates, sizeof(maxCandidates)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_max_features), &maxFeatures, sizeof(maxFeatures)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_img_rows), &img_rows, sizeof(img_rows)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_img_cols), &img_cols, sizeof(img_cols)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_nOctaveLayers), &nOctaveLayers, sizeof(nOctaveLayers)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_hessianThreshold), &hessianThreshold, sizeof(hessianThreshold)) );
        }

        void loadOctaveConstants(int octave, int layer_rows, int layer_cols)
        {
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_octave), &octave, sizeof(octave)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_layer_rows), &layer_rows, sizeof(layer_rows)) );
            cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_layer_cols), &layer_cols, sizeof(layer_cols)) );
        }


        template <int N> __device__ float icvCalcHaarPatternSum(cudev::TexturePtr<unsigned int> texSum, const float src[][5], int oldSize, int newSize, int y, int x)
        {
        #if defined __CUDA_ARCH__ && __CUDA_ARCH__ >= 200
            typedef double real_t;
        #else
            typedef float  real_t;
        #endif

            float ratio = (float)newSize / oldSize;

            real_t d = 0;

            #pragma unroll
            for (int k = 0; k < N; ++k)
            {
                int dx1 = __float2int_rn(ratio * src[k][0]);
                int dy1 = __float2int_rn(ratio * src[k][1]);
                int dx2 = __float2int_rn(ratio * src[k][2]);
                int dy2 = __float2int_rn(ratio * src[k][3]);

                real_t t = 0;
                t += texSum(y + dy1, x + dx1);
                t -= texSum(y + dy2, x + dx1);
                t -= texSum(y + dy1, x + dx2);
                t += texSum(y + dy2, x + dx2);

                d += t * src[k][4] / ((dx2 - dx1) * (dy2 - dy1));
            }

            return (float)d;
        }

        ////////////////////////////////////////////////////////////////////////
        // Hessian

        __constant__ float c_DX [3][5] = { {0, 2, 3, 7, 1}, {3, 2, 6, 7, -2}, {6, 2, 9, 7, 1} };
        __constant__ float c_DY [3][5] = { {2, 0, 7, 3, 1}, {2, 3, 7, 6, -2}, {2, 6, 7, 9, 1} };
        __constant__ float c_DXY[4][5] = { {1, 1, 4, 4, 1}, {5, 1, 8, 4, -1}, {1, 5, 4, 8, -1}, {5, 5, 8, 8, 1} };

        __host__ __device__ __forceinline__ int calcSize(int octave, int layer)
        {
            /* Wavelet size at first layer of first octave. */
            const int HAAR_SIZE0 = 9;

            /* Wavelet size increment between layers. This should be an even number,
             such that the wavelet sizes in an octave are either all even or all odd.
             This ensures that when looking for the neighbours of a sample, the layers
             above and below are aligned correctly. */
            const int HAAR_SIZE_INC = 6;

            return (HAAR_SIZE0 + HAAR_SIZE_INC * layer) << octave;
        }

        __global__ void icvCalcLayerDetAndTrace(cudev::TexturePtr<unsigned int> texSum, PtrStepf det, PtrStepf trace)
        {
            // Determine the indices
            const int gridDim_y = gridDim.y / (c_nOctaveLayers + 2);
            const int blockIdx_y = blockIdx.y % gridDim_y;
            const int blockIdx_z = blockIdx.y / gridDim_y;

            const int j = threadIdx.x + blockIdx.x * blockDim.x;
            const int i = threadIdx.y + blockIdx_y * blockDim.y;
            const int layer = blockIdx_z;

            const int size = calcSize(c_octave, layer);

            const int samples_i = 1 + ((c_img_rows - size) >> c_octave);
            const int samples_j = 1 + ((c_img_cols - size) >> c_octave);

            // Ignore pixels where some of the kernel is outside the image
            const int margin = (size >> 1) >> c_octave;

            if (size <= c_img_rows && size <= c_img_cols && i < samples_i && j < samples_j)
            {
                const float dx  = icvCalcHaarPatternSum<3>(texSum, c_DX , 9, size, (i << c_octave), (j << c_octave));
                const float dy  = icvCalcHaarPatternSum<3>(texSum, c_DY , 9, size, (i << c_octave), (j << c_octave));
                const float dxy = icvCalcHaarPatternSum<4>(texSum, c_DXY, 9, size, (i << c_octave), (j << c_octave));

                det.ptr(layer * c_layer_rows + i + margin)[j + margin] = dx * dy - 0.81f * dxy * dxy;
                trace.ptr(layer * c_layer_rows + i + margin)[j + margin] = dx + dy;
            }
        }

        void icvCalcLayerDetAndTrace_gpu(const PtrStepSz<unsigned int>& sum, const PtrStepf& det, const PtrStepf& trace, int img_rows, int img_cols,
            int octave, int nOctaveLayers)
        {
            const int min_size = calcSize(octave, 0);
            const int max_samples_i = 1 + ((img_rows - min_size) >> octave);
            const int max_samples_j = 1 + ((img_cols - min_size) >> octave);
            cudev::Texture<unsigned int> texSum(sum);
            dim3 threads(16, 16);

            dim3 grid;
            grid.x = divUp(max_samples_j, threads.x);
            grid.y = divUp(max_samples_i, threads.y) * (nOctaveLayers + 2);

            icvCalcLayerDetAndTrace<<<grid, threads>>>(texSum, det, trace);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }

        ////////////////////////////////////////////////////////////////////////
        // NONMAX

        __constant__ float c_DM[5] = {0, 0, 9, 9, 1};

        template<bool useMask = true>
        struct Mask
        {
            __host__ Mask(){};
            __host__ Mask(cudev::TexturePtr<unsigned int> tex_): tex(tex_) {};
            __device__ bool check(int sum_i, int sum_j, int size)
            {
                int dx1 = 0;
                int dy1 = 0;
                int dx2 = 0;
                int dy2 = 0;
                float ratio = 0;
                float d = 0;
                float t = 0;

                if (!useMask) return true;
                ratio = (float)size / 9.0f;
                dx1 = __float2int_rn(ratio * c_DM[0]);
                dy1 = __float2int_rn(ratio * c_DM[1]);
                dx2 = __float2int_rn(ratio * c_DM[2]);
                dy2 = __float2int_rn(ratio * c_DM[3]);

                t += tex(sum_i + dy1, sum_j + dx1);
                t -= tex(sum_i + dy2, sum_j + dx1);
                t -= tex(sum_i + dy1, sum_j + dx2);
                t += tex(sum_i + dy2, sum_j + dx2);

                d += t * c_DM[4] / ((dx2 - dx1) * (dy2 - dy1));

                return (d >= 0.5f);
            }
            cudev::TexturePtr<unsigned int> tex;
        };

        template<class T>
        __global__ void icvFindMaximaInLayer(T mask, const PtrStepf det, const PtrStepf trace, int4* maxPosBuffer,
            unsigned int* maxCounter)
        {
            #if defined __CUDA_ARCH__ && __CUDA_ARCH__ >= 110

            extern __shared__ float N9[];

            // The hidx variables are the indices to the hessian buffer.
            const int gridDim_y = gridDim.y / c_nOctaveLayers;
            const int blockIdx_y = blockIdx.y % gridDim_y;
            const int blockIdx_z = blockIdx.y / gridDim_y;

            const int layer = blockIdx_z + 1;

            const int size = calcSize(c_octave, layer);

            // Ignore pixels without a 3x3x3 neighbourhood in the layer above
            const int margin = ((calcSize(c_octave, layer + 1) >> 1) >> c_octave) + 1;

            const int j = threadIdx.x + blockIdx.x * (blockDim.x - 2) + margin - 1;
            const int i = threadIdx.y + blockIdx_y * (blockDim.y - 2) + margin - 1;

            // Is this thread within the hessian buffer?
            const int zoff = blockDim.x * blockDim.y;
            const int localLin = threadIdx.x + threadIdx.y * blockDim.x + zoff;
            N9[localLin - zoff] = det.ptr(c_layer_rows * (layer - 1) + ::min(::max(i, 0), c_img_rows - 1))[::min(::max(j, 0), c_img_cols - 1)];
            N9[localLin       ] = det.ptr(c_layer_rows * (layer    ) + ::min(::max(i, 0), c_img_rows - 1))[::min(::max(j, 0), c_img_cols - 1)];
            N9[localLin + zoff] = det.ptr(c_layer_rows * (layer + 1) + ::min(::max(i, 0), c_img_rows - 1))[::min(::max(j, 0), c_img_cols - 1)];
            __syncthreads();

            if (i < c_layer_rows - margin && j < c_layer_cols - margin && threadIdx.x > 0 && threadIdx.x < blockDim.x - 1 && threadIdx.y > 0 && threadIdx.y < blockDim.y - 1)
            {
                float val0 = N9[localLin];

                if (val0 > c_hessianThreshold)
                {
                    // Coordinates for the start of the wavelet in the sum image. There
                    // is some integer division involved, so don't try to simplify this
                    // (cancel out sampleStep) without checking the result is the same
                    const int sum_i = (i - ((size >> 1) >> c_octave)) << c_octave;
                    const int sum_j = (j - ((size >> 1) >> c_octave)) << c_octave;

                    if (mask.check(sum_i, sum_j, size))
                    {
                        // Check to see if we have a max (in its 26 neighbours)
                        const bool condmax = val0 > N9[localLin - 1 - blockDim.x - zoff]
                        &&                   val0 > N9[localLin     - blockDim.x - zoff]
                        &&                   val0 > N9[localLin + 1 - blockDim.x - zoff]
                        &&                   val0 > N9[localLin - 1              - zoff]
                        &&                   val0 > N9[localLin                  - zoff]
                        &&                   val0 > N9[localLin + 1              - zoff]
                        &&                   val0 > N9[localLin - 1 + blockDim.x - zoff]
                        &&                   val0 > N9[localLin     + blockDim.x - zoff]
                        &&                   val0 > N9[localLin + 1 + blockDim.x - zoff]

                        &&                   val0 > N9[localLin - 1 - blockDim.x]
                        &&                   val0 > N9[localLin     - blockDim.x]
                        &&                   val0 > N9[localLin + 1 - blockDim.x]
                        &&                   val0 > N9[localLin - 1             ]
                        &&                   val0 > N9[localLin + 1             ]
                        &&                   val0 > N9[localLin - 1 + blockDim.x]
                        &&                   val0 > N9[localLin     + blockDim.x]
                        &&                   val0 > N9[localLin + 1 + blockDim.x]

                        &&                   val0 > N9[localLin - 1 - blockDim.x + zoff]
                        &&                   val0 > N9[localLin     - blockDim.x + zoff]
                        &&                   val0 > N9[localLin + 1 - blockDim.x + zoff]
                        &&                   val0 > N9[localLin - 1              + zoff]
                        &&                   val0 > N9[localLin                  + zoff]
                        &&                   val0 > N9[localLin + 1              + zoff]
                        &&                   val0 > N9[localLin - 1 + blockDim.x + zoff]
                        &&                   val0 > N9[localLin     + blockDim.x + zoff]
                        &&                   val0 > N9[localLin + 1 + blockDim.x + zoff]
                        ;

                        if(condmax)
                        {
                            unsigned int ind = atomicInc(maxCounter,(unsigned int) -1);

                            if (ind < c_max_candidates)
                            {
                                const int laplacian = (int) copysignf(1.0f, trace.ptr(layer * c_layer_rows + i)[j]);

                                maxPosBuffer[ind] = make_int4(j, i, layer, laplacian);
                            }
                        }
                    }
                }
            }

            #endif
        }

        void icvFindMaximaInLayer_gpu(const PtrStepSz<unsigned int>& maskSum, const PtrStepf& det, const PtrStepf& trace, int4* maxPosBuffer, unsigned int* maxCounter,
            int img_rows, int img_cols, int octave, bool use_mask, int nOctaveLayers)
        {
            const int layer_rows = img_rows >> octave;
            const int layer_cols = img_cols >> octave;

            const int min_margin = ((calcSize(octave, 2) >> 1) >> octave) + 1;

            dim3 threads(16, 16);

            dim3 grid;
            grid.x = divUp(layer_cols - 2 * min_margin, threads.x - 2);
            grid.y = divUp(layer_rows - 2 * min_margin, threads.y - 2) * nOctaveLayers;

            const size_t smem_size = threads.x * threads.y * 3 * sizeof(float);

            if (use_mask) {
                cudev::Texture<unsigned int> texMaskSum(maskSum);
                Mask<true> mask(texMaskSum);
                icvFindMaximaInLayer<<<grid, threads, smem_size>>>(mask, det, trace, maxPosBuffer, maxCounter);
            }
            else {
                Mask<false> mask;
                icvFindMaximaInLayer<<<grid, threads, smem_size>>>(mask, det, trace, maxPosBuffer, maxCounter);
            }

            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }

        ////////////////////////////////////////////////////////////////////////
        // INTERPOLATION

        __global__ void icvInterpolateKeypoint(const PtrStepf det, const int4* maxPosBuffer,
            float* featureX, float* featureY, int* featureLaplacian, int* featureOctave, float* featureSize, float* featureHessian,
            unsigned int* featureCounter)
        {
            #if defined __CUDA_ARCH__ && __CUDA_ARCH__ >= 110

            const int4 maxPos = maxPosBuffer[blockIdx.x];

            const int j = maxPos.x - 1 + threadIdx.x;
            const int i = maxPos.y - 1 + threadIdx.y;
            const int layer = maxPos.z - 1 + threadIdx.z;

            __shared__ float N9[3][3][3];

            N9[threadIdx.z][threadIdx.y][threadIdx.x] = det.ptr(c_layer_rows * layer + i)[j];
            __syncthreads();

            if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0)
            {
                __shared__ float dD[3];

                //dx
                dD[0] = -0.5f * (N9[1][1][2] - N9[1][1][0]);
                //dy
                dD[1] = -0.5f * (N9[1][2][1] - N9[1][0][1]);
                //ds
                dD[2] = -0.5f * (N9[2][1][1] - N9[0][1][1]);

                __shared__ float H[3][3];

                //dxx
                H[0][0] = N9[1][1][0] - 2.0f * N9[1][1][1] + N9[1][1][2];
                //dxy
                H[0][1]= 0.25f * (N9[1][2][2] - N9[1][2][0] - N9[1][0][2] + N9[1][0][0]);
                //dxs
                H[0][2]= 0.25f * (N9[2][1][2] - N9[2][1][0] - N9[0][1][2] + N9[0][1][0]);
                //dyx = dxy
                H[1][0] = H[0][1];
                //dyy
                H[1][1] = N9[1][0][1] - 2.0f * N9[1][1][1] + N9[1][2][1];
                //dys
                H[1][2]= 0.25f * (N9[2][2][1] - N9[2][0][1] - N9[0][2][1] + N9[0][0][1]);
                //dsx = dxs
                H[2][0] = H[0][2];
                //dsy = dys
                H[2][1] = H[1][2];
                //dss
                H[2][2] = N9[0][1][1] - 2.0f * N9[1][1][1] + N9[2][1][1];

                __shared__ float x[3];

                if (solve3x3(H, dD, x))
                {
                    if (::fabs(x[0]) <= 1.f && ::fabs(x[1]) <= 1.f && ::fabs(x[2]) <= 1.f)
                    {
                        // if the step is within the interpolation region, perform it

                        const int size = calcSize(c_octave, maxPos.z);

                        const int sum_i = (maxPos.y - ((size >> 1) >> c_octave)) << c_octave;
                        const int sum_j = (maxPos.x - ((size >> 1) >> c_octave)) << c_octave;

                        const float center_i = sum_i + (float)(size - 1) / 2;
                        const float center_j = sum_j + (float)(size - 1) / 2;

                        const float px = center_j + x[0] * (1 << c_octave);
                        const float py = center_i + x[1] * (1 << c_octave);

                        const int ds = size - calcSize(c_octave, maxPos.z - 1);
                        const float psize = roundf(size + x[2] * ds);

                        /* The sampling intervals and wavelet sized for selecting an orientation
                         and building the keypoint descriptor are defined relative to 's' */
                        const float s = psize * 1.2f / 9.0f;

                        /* To find the dominant orientation, the gradients in x and y are
                         sampled in a circle of radius 6s using wavelets of size 4s.
                         We ensure the gradient wavelet size is even to ensure the
                         wavelet pattern is balanced and symmetric around its center */
                        const int grad_wav_size = 2 * __float2int_rn(2.0f * s);

                        // check when grad_wav_size is too big
                        if ((c_img_rows + 1) >= grad_wav_size && (c_img_cols + 1) >= grad_wav_size)
                        {
                            // Get a new feature index.
                            unsigned int ind = atomicInc(featureCounter, (unsigned int)-1);

                            if (ind < c_max_features)
                            {
                                featureX[ind] = px;
                                featureY[ind] = py;
                                featureLaplacian[ind] = maxPos.w;
                                featureOctave[ind] = c_octave;
                                featureSize[ind] = psize;
                                featureHessian[ind] = N9[1][1][1];
                            }
                        } // grad_wav_size check
                    } // If the subpixel interpolation worked
                }
            } // If this is thread 0.

            #endif
        }

        void icvInterpolateKeypoint_gpu(const PtrStepf& det, const int4* maxPosBuffer, unsigned int maxCounter,
            float* featureX, float* featureY, int* featureLaplacian, int* featureOctave, float* featureSize, float* featureHessian,
            unsigned int* featureCounter)
        {
            dim3 threads;
            threads.x = 3;
            threads.y = 3;
            threads.z = 3;

            dim3 grid;
            grid.x = maxCounter;

            icvInterpolateKeypoint<<<grid, threads>>>(det, maxPosBuffer, featureX, featureY, featureLaplacian, featureOctave, featureSize, featureHessian, featureCounter);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }

        ////////////////////////////////////////////////////////////////////////
        // Orientation

        #define ORI_SEARCH_INC 5
        #define ORI_WIN        60
        #define ORI_SAMPLES    113

        __constant__ float c_aptX[ORI_SAMPLES] = {-6, -5, -5, -5, -5, -5, -5, -5, -4, -4, -4, -4, -4, -4, -4, -4, -4, -3, -3, -3, -3, -3, -3, -3, -3, -3, -3, -3, -2, -2, -2, -2, -2, -2, -2, -2, -2, -2, -2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 3, 4, 4, 4, 4, 4, 4, 4, 4, 4, 5, 5, 5, 5, 5, 5, 5, 6};
        __constant__ float c_aptY[ORI_SAMPLES] = {0, -3, -2, -1, 0, 1, 2, 3, -4, -3, -2, -1, 0, 1, 2, 3, 4, -5, -4, -3, -2, -1, 0, 1, 2, 3, 4, 5, -5, -4, -3, -2, -1, 0, 1, 2, 3, 4, 5, -5, -4, -3, -2, -1, 0, 1, 2, 3, 4, 5, -6, -5, -4, -3, -2, -1, 0, 1, 2, 3, 4, 5, 6, -5, -4, -3, -2, -1, 0, 1, 2, 3, 4, 5, -5, -4, -3, -2, -1, 0, 1, 2, 3, 4, 5, -5, -4, -3, -2, -1, 0, 1, 2, 3, 4, 5, -4, -3, -2, -1, 0, 1, 2, 3, 4, -3, -2, -1, 0, 1, 2, 3, 0};
        __constant__ float c_aptW[ORI_SAMPLES] = {0.001455130288377404f, 0.001707611023448408f, 0.002547456417232752f, 0.003238451667129993f, 0.0035081731621176f, 0.003238451667129993f, 0.002547456417232752f, 0.001707611023448408f, 0.002003900473937392f, 0.0035081731621176f, 0.005233579315245152f, 0.00665318313986063f, 0.00720730796456337f, 0.00665318313986063f, 0.005233579315245152f, 0.0035081731621176f, 0.002003900473937392f, 0.001707611023448408f, 0.0035081731621176f, 0.006141661666333675f, 0.009162282571196556f, 0.01164754293859005f, 0.01261763460934162f, 0.01164754293859005f, 0.009162282571196556f, 0.006141661666333675f, 0.0035081731621176f, 0.001707611023448408f, 0.002547456417232752f, 0.005233579315245152f, 0.009162282571196556f, 0.01366852037608624f, 0.01737609319388866f, 0.0188232995569706f, 0.01737609319388866f, 0.01366852037608624f, 0.009162282571196556f, 0.005233579315245152f, 0.002547456417232752f, 0.003238451667129993f, 0.00665318313986063f, 0.01164754293859005f, 0.01737609319388866f, 0.02208934165537357f, 0.02392910048365593f, 0.02208934165537357f, 0.01737609319388866f, 0.01164754293859005f, 0.00665318313986063f, 0.003238451667129993f, 0.001455130288377404f, 0.0035081731621176f, 0.00720730796456337f, 0.01261763460934162f, 0.0188232995569706f, 0.02392910048365593f, 0.02592208795249462f, 0.02392910048365593f, 0.0188232995569706f, 0.01261763460934162f, 0.00720730796456337f, 0.0035081731621176f, 0.001455130288377404f, 0.003238451667129993f, 0.00665318313986063f, 0.01164754293859005f, 0.01737609319388866f, 0.02208934165537357f, 0.02392910048365593f, 0.02208934165537357f, 0.01737609319388866f, 0.01164754293859005f, 0.00665318313986063f, 0.003238451667129993f, 0.002547456417232752f, 0.005233579315245152f, 0.009162282571196556f, 0.01366852037608624f, 0.01737609319388866f, 0.0188232995569706f, 0.01737609319388866f, 0.01366852037608624f, 0.009162282571196556f, 0.005233579315245152f, 0.002547456417232752f, 0.001707611023448408f, 0.0035081731621176f, 0.006141661666333675f, 0.009162282571196556f, 0.01164754293859005f, 0.01261763460934162f, 0.01164754293859005f, 0.009162282571196556f, 0.006141661666333675f, 0.0035081731621176f, 0.001707611023448408f, 0.002003900473937392f, 0.0035081731621176f, 0.005233579315245152f, 0.00665318313986063f, 0.00720730796456337f, 0.00665318313986063f, 0.005233579315245152f, 0.0035081731621176f, 0.002003900473937392f, 0.001707611023448408f, 0.002547456417232752f, 0.003238451667129993f, 0.0035081731621176f, 0.003238451667129993f, 0.002547456417232752f, 0.001707611023448408f, 0.001455130288377404f};

        __constant__ float c_NX[2][5] = {{0, 0, 2, 4, -1}, {2, 0, 4, 4, 1}};
        __constant__ float c_NY[2][5] = {{0, 0, 4, 2, 1}, {0, 2, 4, 4, -1}};

        __global__ void icvCalcOrientation(cudev::TexturePtr<unsigned int> texSum, const float* featureX, const float* featureY, const float* featureSize, float* featureDir)
        {
            __shared__ float s_X[128];
            __shared__ float s_Y[128];
            __shared__ float s_angle[128];

            __shared__ float s_sumx[32 * 4];
            __shared__ float s_sumy[32 * 4];

            /* The sampling intervals and wavelet sized for selecting an orientation
             and building the keypoint descriptor are defined relative to 's' */
            const float s = featureSize[blockIdx.x] * 1.2f / 9.0f;

            /* To find the dominant orientation, the gradients in x and y are
             sampled in a circle of radius 6s using wavelets of size 4s.
             We ensure the gradient wavelet size is even to ensure the
             wavelet pattern is balanced and symmetric around its center */
            const int grad_wav_size = 2 * __float2int_rn(2.0f * s);

            // check when grad_wav_size is too big
            if ((c_img_rows + 1) < grad_wav_size || (c_img_cols + 1) < grad_wav_size)
                return;

            // Calc X, Y, angle and store it to shared memory
            const int tid = threadIdx.y * blockDim.x + threadIdx.x;

            float X = 0.0f, Y = 0.0f, angle = 0.0f;

            if (tid < ORI_SAMPLES)
            {
                const float margin = (float)(grad_wav_size - 1) / 2.0f;
                const int x = __float2int_rn(featureX[blockIdx.x] + c_aptX[tid] * s - margin);
                const int y = __float2int_rn(featureY[blockIdx.x] + c_aptY[tid] * s - margin);

                if (y >= 0 && y < (c_img_rows + 1) - grad_wav_size &&
                    x >= 0 && x < (c_img_cols + 1) - grad_wav_size)
                {
                    X = c_aptW[tid] * icvCalcHaarPatternSum<2>(texSum, c_NX, 4, grad_wav_size, y, x);
                    Y = c_aptW[tid] * icvCalcHaarPatternSum<2>(texSum, c_NY, 4, grad_wav_size, y, x);

                    angle = atan2f(Y, X);
                    if (angle < 0)
                        angle += 2.0f * CV_PI_F;
                    angle *= 180.0f / CV_PI_F;
                }
            }
            s_X[tid] = X;
            s_Y[tid] = Y;
            s_angle[tid] = angle;
            __syncthreads();

            float bestx = 0, besty = 0, best_mod = 0;

        #if defined __CUDA_ARCH__ && __CUDA_ARCH__ >= 200
            #pragma unroll
        #endif
            for (int i = 0; i < 18; ++i)
            {
                const int dir = (i * 4 + threadIdx.y) * ORI_SEARCH_INC;

                float sumx = 0.0f, sumy = 0.0f;
                int d = ::abs(__float2int_rn(s_angle[threadIdx.x]) - dir);
                if (d < ORI_WIN / 2 || d > 360 - ORI_WIN / 2)
                {
                    sumx = s_X[threadIdx.x];
                    sumy = s_Y[threadIdx.x];
                }
                d = ::abs(__float2int_rn(s_angle[threadIdx.x + 32]) - dir);
                if (d < ORI_WIN / 2 || d > 360 - ORI_WIN / 2)
                {
                    sumx += s_X[threadIdx.x + 32];
                    sumy += s_Y[threadIdx.x + 32];
                }
                d = ::abs(__float2int_rn(s_angle[threadIdx.x + 64]) - dir);
                if (d < ORI_WIN / 2 || d > 360 - ORI_WIN / 2)
                {
                    sumx += s_X[threadIdx.x + 64];
                    sumy += s_Y[threadIdx.x + 64];
                }
                d = ::abs(__float2int_rn(s_angle[threadIdx.x + 96]) - dir);
                if (d < ORI_WIN / 2 || d > 360 - ORI_WIN / 2)
                {
                    sumx += s_X[threadIdx.x + 96];
                    sumy += s_Y[threadIdx.x + 96];
                }

                plus<float> op;
                device::reduce<32>(smem_tuple(s_sumx + threadIdx.y * 32, s_sumy + threadIdx.y * 32),
                                   thrust::tie(sumx, sumy), threadIdx.x, thrust::make_tuple(op, op));

                const float temp_mod = sumx * sumx + sumy * sumy;
                if (temp_mod > best_mod)
                {
                    best_mod = temp_mod;
                    bestx = sumx;
                    besty = sumy;
                }

                __syncthreads();
            }

            if (threadIdx.x == 0)
            {
                s_X[threadIdx.y] = bestx;
                s_Y[threadIdx.y] = besty;
                s_angle[threadIdx.y] = best_mod;
            }
            __syncthreads();

            if (threadIdx.x == 0 && threadIdx.y == 0)
            {
                int bestIdx = 0;

                if (s_angle[1] > s_angle[bestIdx])
                    bestIdx = 1;
                if (s_angle[2] > s_angle[bestIdx])
                    bestIdx = 2;
                if (s_angle[3] > s_angle[bestIdx])
                    bestIdx = 3;

                float kp_dir = atan2f(s_Y[bestIdx], s_X[bestIdx]);
                if (kp_dir < 0)
                    kp_dir += 2.0f * CV_PI_F;
                kp_dir *= 180.0f / CV_PI_F;

                kp_dir = 360.0f - kp_dir;
                if (::fabsf(kp_dir - 360.f) < numeric_limits<float>::epsilon())
                    kp_dir = 0.f;

                featureDir[blockIdx.x] = kp_dir;
            }
        }

        #undef ORI_SEARCH_INC
        #undef ORI_WIN
        #undef ORI_SAMPLES

        void icvCalcOrientation_gpu(const PtrStepSz<unsigned int>& sum, const float* featureX, const float* featureY, const float* featureSize, float* featureDir, int nFeatures)
        {
            cudev::Texture<unsigned int> texSum(sum);
            dim3 threads;
            threads.x = 32;
            threads.y = 4;

            dim3 grid;
            grid.x = nFeatures;

            icvCalcOrientation<<<grid, threads>>>(texSum, featureX, featureY, featureSize, featureDir);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }

        ////////////////////////////////////////////////////////////////////////
        // Descriptors

        #define PATCH_SZ 20

        __constant__ float c_DW[PATCH_SZ * PATCH_SZ] =
        {
            3.695352233989979e-006f, 8.444558261544444e-006f, 1.760426494001877e-005f, 3.34794785885606e-005f, 5.808438800158911e-005f, 9.193058212986216e-005f, 0.0001327334757661447f, 0.0001748319627949968f, 0.0002100782439811155f, 0.0002302826324012131f, 0.0002302826324012131f, 0.0002100782439811155f, 0.0001748319627949968f, 0.0001327334757661447f, 9.193058212986216e-005f, 5.808438800158911e-005f, 3.34794785885606e-005f, 1.760426494001877e-005f, 8.444558261544444e-006f, 3.695352233989979e-006f,
            8.444558261544444e-006f, 1.929736572492402e-005f, 4.022897701361217e-005f, 7.650675252079964e-005f, 0.0001327334903180599f, 0.0002100782585330308f, 0.0003033203829545528f, 0.0003995231236331165f, 0.0004800673632416874f, 0.0005262381164357066f, 0.0005262381164357066f, 0.0004800673632416874f, 0.0003995231236331165f, 0.0003033203829545528f, 0.0002100782585330308f, 0.0001327334903180599f, 7.650675252079964e-005f, 4.022897701361217e-005f, 1.929736572492402e-005f, 8.444558261544444e-006f,
            1.760426494001877e-005f, 4.022897701361217e-005f, 8.386484114453197e-005f, 0.0001594926579855382f, 0.0002767078403849155f, 0.0004379475140012801f, 0.0006323281559161842f, 0.0008328808471560478f, 0.001000790391117334f, 0.001097041997127235f, 0.001097041997127235f, 0.001000790391117334f, 0.0008328808471560478f, 0.0006323281559161842f, 0.0004379475140012801f, 0.0002767078403849155f, 0.0001594926579855382f, 8.386484114453197e-005f, 4.022897701361217e-005f, 1.760426494001877e-005f,
            3.34794785885606e-005f, 7.650675252079964e-005f, 0.0001594926579855382f, 0.0003033203247468919f, 0.0005262380582280457f, 0.0008328807889483869f, 0.001202550483867526f, 0.001583957928232849f, 0.001903285388834775f, 0.002086334861814976f, 0.002086334861814976f, 0.001903285388834775f, 0.001583957928232849f, 0.001202550483867526f, 0.0008328807889483869f, 0.0005262380582280457f, 0.0003033203247468919f, 0.0001594926579855382f, 7.650675252079964e-005f, 3.34794785885606e-005f,
            5.808438800158911e-005f, 0.0001327334903180599f, 0.0002767078403849155f, 0.0005262380582280457f, 0.0009129836107604206f, 0.001444985857233405f, 0.002086335094645619f, 0.002748048631474376f, 0.00330205773934722f, 0.003619635012000799f, 0.003619635012000799f, 0.00330205773934722f, 0.002748048631474376f, 0.002086335094645619f, 0.001444985857233405f, 0.0009129836107604206f, 0.0005262380582280457f, 0.0002767078403849155f, 0.0001327334903180599f, 5.808438800158911e-005f,
            9.193058212986216e-005f, 0.0002100782585330308f, 0.0004379475140012801f, 0.0008328807889483869f, 0.001444985857233405f, 0.002286989474669099f, 0.00330205773934722f, 0.004349356517195702f, 0.00522619066759944f, 0.005728822201490402f, 0.005728822201490402f, 0.00522619066759944f, 0.004349356517195702f, 0.00330205773934722f, 0.002286989474669099f, 0.001444985857233405f, 0.0008328807889483869f, 0.0004379475140012801f, 0.0002100782585330308f, 9.193058212986216e-005f,
            0.0001327334757661447f, 0.0003033203829545528f, 0.0006323281559161842f, 0.001202550483867526f, 0.002086335094645619f, 0.00330205773934722f, 0.004767658654600382f, 0.006279794964939356f, 0.007545807864516974f, 0.008271530270576477f, 0.008271530270576477f, 0.007545807864516974f, 0.006279794964939356f, 0.004767658654600382f, 0.00330205773934722f, 0.002086335094645619f, 0.001202550483867526f, 0.0006323281559161842f, 0.0003033203829545528f, 0.0001327334757661447f,
            0.0001748319627949968f, 0.0003995231236331165f, 0.0008328808471560478f, 0.001583957928232849f, 0.002748048631474376f, 0.004349356517195702f, 0.006279794964939356f, 0.008271529339253902f, 0.009939077310264111f, 0.01089497376233339f, 0.01089497376233339f, 0.009939077310264111f, 0.008271529339253902f, 0.006279794964939356f, 0.004349356517195702f, 0.002748048631474376f, 0.001583957928232849f, 0.0008328808471560478f, 0.0003995231236331165f, 0.0001748319627949968f,
            0.0002100782439811155f, 0.0004800673632416874f, 0.001000790391117334f, 0.001903285388834775f, 0.00330205773934722f, 0.00522619066759944f, 0.007545807864516974f, 0.009939077310264111f, 0.01194280479103327f, 0.01309141051024199f, 0.01309141051024199f, 0.01194280479103327f, 0.009939077310264111f, 0.007545807864516974f, 0.00522619066759944f, 0.00330205773934722f, 0.001903285388834775f, 0.001000790391117334f, 0.0004800673632416874f, 0.0002100782439811155f,
            0.0002302826324012131f, 0.0005262381164357066f, 0.001097041997127235f, 0.002086334861814976f, 0.003619635012000799f, 0.005728822201490402f, 0.008271530270576477f, 0.01089497376233339f, 0.01309141051024199f, 0.01435048412531614f, 0.01435048412531614f, 0.01309141051024199f, 0.01089497376233339f, 0.008271530270576477f, 0.005728822201490402f, 0.003619635012000799f, 0.002086334861814976f, 0.001097041997127235f, 0.0005262381164357066f, 0.0002302826324012131f,
            0.0002302826324012131f, 0.0005262381164357066f, 0.001097041997127235f, 0.002086334861814976f, 0.003619635012000799f, 0.005728822201490402f, 0.008271530270576477f, 0.01089497376233339f, 0.01309141051024199f, 0.01435048412531614f, 0.01435048412531614f, 0.01309141051024199f, 0.01089497376233339f, 0.008271530270576477f, 0.005728822201490402f, 0.003619635012000799f, 0.002086334861814976f, 0.001097041997127235f, 0.0005262381164357066f, 0.0002302826324012131f,
            0.0002100782439811155f, 0.0004800673632416874f, 0.001000790391117334f, 0.001903285388834775f, 0.00330205773934722f, 0.00522619066759944f, 0.007545807864516974f, 0.009939077310264111f, 0.01194280479103327f, 0.01309141051024199f, 0.01309141051024199f, 0.01194280479103327f, 0.009939077310264111f, 0.007545807864516974f, 0.00522619066759944f, 0.00330205773934722f, 0.001903285388834775f, 0.001000790391117334f, 0.0004800673632416874f, 0.0002100782439811155f,
            0.0001748319627949968f, 0.0003995231236331165f, 0.0008328808471560478f, 0.001583957928232849f, 0.002748048631474376f, 0.004349356517195702f, 0.006279794964939356f, 0.008271529339253902f, 0.009939077310264111f, 0.01089497376233339f, 0.01089497376233339f, 0.009939077310264111f, 0.008271529339253902f, 0.006279794964939356f, 0.004349356517195702f, 0.002748048631474376f, 0.001583957928232849f, 0.0008328808471560478f, 0.0003995231236331165f, 0.0001748319627949968f,
            0.0001327334757661447f, 0.0003033203829545528f, 0.0006323281559161842f, 0.001202550483867526f, 0.002086335094645619f, 0.00330205773934722f, 0.004767658654600382f, 0.006279794964939356f, 0.007545807864516974f, 0.008271530270576477f, 0.008271530270576477f, 0.007545807864516974f, 0.006279794964939356f, 0.004767658654600382f, 0.00330205773934722f, 0.002086335094645619f, 0.001202550483867526f, 0.0006323281559161842f, 0.0003033203829545528f, 0.0001327334757661447f,
            9.193058212986216e-005f, 0.0002100782585330308f, 0.0004379475140012801f, 0.0008328807889483869f, 0.001444985857233405f, 0.002286989474669099f, 0.00330205773934722f, 0.004349356517195702f, 0.00522619066759944f, 0.005728822201490402f, 0.005728822201490402f, 0.00522619066759944f, 0.004349356517195702f, 0.00330205773934722f, 0.002286989474669099f, 0.001444985857233405f, 0.0008328807889483869f, 0.0004379475140012801f, 0.0002100782585330308f, 9.193058212986216e-005f,
            5.808438800158911e-005f, 0.0001327334903180599f, 0.0002767078403849155f, 0.0005262380582280457f, 0.0009129836107604206f, 0.001444985857233405f, 0.002086335094645619f, 0.002748048631474376f, 0.00330205773934722f, 0.003619635012000799f, 0.003619635012000799f, 0.00330205773934722f, 0.002748048631474376f, 0.002086335094645619f, 0.001444985857233405f, 0.0009129836107604206f, 0.0005262380582280457f, 0.0002767078403849155f, 0.0001327334903180599f, 5.808438800158911e-005f,
            3.34794785885606e-005f, 7.650675252079964e-005f, 0.0001594926579855382f, 0.0003033203247468919f, 0.0005262380582280457f, 0.0008328807889483869f, 0.001202550483867526f, 0.001583957928232849f, 0.001903285388834775f, 0.002086334861814976f, 0.002086334861814976f, 0.001903285388834775f, 0.001583957928232849f, 0.001202550483867526f, 0.0008328807889483869f, 0.0005262380582280457f, 0.0003033203247468919f, 0.0001594926579855382f, 7.650675252079964e-005f, 3.34794785885606e-005f,
            1.760426494001877e-005f, 4.022897701361217e-005f, 8.386484114453197e-005f, 0.0001594926579855382f, 0.0002767078403849155f, 0.0004379475140012801f, 0.0006323281559161842f, 0.0008328808471560478f, 0.001000790391117334f, 0.001097041997127235f, 0.001097041997127235f, 0.001000790391117334f, 0.0008328808471560478f, 0.0006323281559161842f, 0.0004379475140012801f, 0.0002767078403849155f, 0.0001594926579855382f, 8.386484114453197e-005f, 4.022897701361217e-005f, 1.760426494001877e-005f,
            8.444558261544444e-006f, 1.929736572492402e-005f, 4.022897701361217e-005f, 7.650675252079964e-005f, 0.0001327334903180599f, 0.0002100782585330308f, 0.0003033203829545528f, 0.0003995231236331165f, 0.0004800673632416874f, 0.0005262381164357066f, 0.0005262381164357066f, 0.0004800673632416874f, 0.0003995231236331165f, 0.0003033203829545528f, 0.0002100782585330308f, 0.0001327334903180599f, 7.650675252079964e-005f, 4.022897701361217e-005f, 1.929736572492402e-005f, 8.444558261544444e-006f,
            3.695352233989979e-006f, 8.444558261544444e-006f, 1.760426494001877e-005f, 3.34794785885606e-005f, 5.808438800158911e-005f, 9.193058212986216e-005f, 0.0001327334757661447f, 0.0001748319627949968f, 0.0002100782439811155f, 0.0002302826324012131f, 0.0002302826324012131f, 0.0002100782439811155f, 0.0001748319627949968f, 0.0001327334757661447f, 9.193058212986216e-005f, 5.808438800158911e-005f, 3.34794785885606e-005f, 1.760426494001877e-005f, 8.444558261544444e-006f, 3.695352233989979e-006f
        };

        struct WinReader
        {
            typedef uchar elem_type;

            __device__ WinReader(cudev::TexturePtr<uchar> tex_) : tex(tex_) {};

            __device__ __forceinline__ uchar operator ()(int i, int j) const
            {
                float pixel_x = centerX + (win_offset + j) * cos_dir + (win_offset + i) * sin_dir;
                float pixel_y = centerY - (win_offset + j) * sin_dir + (win_offset + i) * cos_dir;

                return tex(pixel_y, pixel_x);
            }

            float centerX;
            float centerY;
            float win_offset;
            float cos_dir;
            float sin_dir;
            int width;
            int height;
            cudev::TexturePtr<uchar> tex;
        };

        __device__ void calc_dx_dy(cudev::TexturePtr<uchar> tex, const float* featureX, const float* featureY, const float* featureSize, const float* featureDir,
                                   float& dx, float& dy)
        {
            __shared__ float s_PATCH[PATCH_SZ + 1][PATCH_SZ + 1];

            dx = dy = 0.0f;

            WinReader win(tex);

            win.centerX = featureX[blockIdx.x];
            win.centerY = featureY[blockIdx.x];

            // The sampling intervals and wavelet sized for selecting an orientation
            // and building the keypoint descriptor are defined relative to 's'
            const float s = featureSize[blockIdx.x] * 1.2f / 9.0f;

            // Extract a window of pixels around the keypoint of size 20s
            const int win_size = (int)((PATCH_SZ + 1) * s);

            win.width = win.height = win_size;

            // Nearest neighbour version (faster)
            win.win_offset = -(win_size - 1.0f) / 2.0f;

            float descriptor_dir = 360.0f - featureDir[blockIdx.x];
            if (::fabsf(descriptor_dir - 360.f) < numeric_limits<float>::epsilon())
                descriptor_dir = 0.f;
            descriptor_dir *= CV_PI_F / 180.0f;
            sincosf(descriptor_dir, &win.sin_dir, &win.cos_dir);

            const int tid = threadIdx.y * blockDim.x + threadIdx.x;

            const int xLoadInd = tid % (PATCH_SZ + 1);
            const int yLoadInd = tid / (PATCH_SZ + 1);

            if (yLoadInd < (PATCH_SZ + 1))
            {
                if (s > 1)
                {
                    AreaFilter<WinReader> filter(win, s, s);
                    s_PATCH[yLoadInd][xLoadInd] = filter(yLoadInd, xLoadInd);
                }
                else
                {
                    LinearFilter<WinReader> filter(win);
                    s_PATCH[yLoadInd][xLoadInd] = filter(yLoadInd * s, xLoadInd * s);
                }
            }

            __syncthreads();

            const int xPatchInd = threadIdx.x % 5;
            const int yPatchInd = threadIdx.x / 5;

            if (yPatchInd < 5)
            {
                const int xBlockInd = threadIdx.y % 4;
                const int yBlockInd = threadIdx.y / 4;

                const int xInd = xBlockInd * 5 + xPatchInd;
                const int yInd = yBlockInd * 5 + yPatchInd;

                const float dw = c_DW[yInd * PATCH_SZ + xInd];

                dx = (s_PATCH[yInd    ][xInd + 1] - s_PATCH[yInd][xInd] + s_PATCH[yInd + 1][xInd + 1] - s_PATCH[yInd + 1][xInd    ]) * dw;
                dy = (s_PATCH[yInd + 1][xInd    ] - s_PATCH[yInd][xInd] + s_PATCH[yInd + 1][xInd + 1] - s_PATCH[yInd    ][xInd + 1]) * dw;
            }
        }

        __global__ void compute_descriptors_64(cudev::TexturePtr<uchar> texImg, PtrStep<float4> descriptors, const float* featureX, const float* featureY, const float* featureSize, const float* featureDir)
        {
            __shared__ float smem[32 * 16];

            float* sRow = smem + threadIdx.y * 32;

            float dx, dy;
            calc_dx_dy(texImg, featureX, featureY, featureSize, featureDir, dx, dy);

            float dxabs = ::fabsf(dx);
            float dyabs = ::fabsf(dy);

            plus<float> op;

            reduce<32>(sRow, dx, threadIdx.x, op);
            reduce<32>(sRow, dy, threadIdx.x, op);
            reduce<32>(sRow, dxabs, threadIdx.x, op);
            reduce<32>(sRow, dyabs, threadIdx.x, op);

            float4* descriptors_block = descriptors.ptr(blockIdx.x) + threadIdx.y;

            // write dx, dy, |dx|, |dy|
            if (threadIdx.x == 0)
                *descriptors_block = make_float4(dx, dy, dxabs, dyabs);
        }

        __global__ void compute_descriptors_128(cudev::TexturePtr<uchar> texImg, PtrStep<float4> descriptors, const float* featureX, const float* featureY, const float* featureSize, const float* featureDir)
        {
            __shared__ float smem[32 * 16];

            float* sRow = smem + threadIdx.y * 32;

            float dx, dy;
            calc_dx_dy(texImg, featureX, featureY, featureSize, featureDir, dx, dy);

            float4* descriptors_block = descriptors.ptr(blockIdx.x) + threadIdx.y * 2;

            plus<float> op;

            float d1 = 0.0f;
            float d2 = 0.0f;
            float abs1 = 0.0f;
            float abs2 = 0.0f;

            if (dy >= 0)
            {
                d1 = dx;
                abs1 = ::fabsf(dx);
            }
            else
            {
                d2 = dx;
                abs2 = ::fabsf(dx);
            }

            reduce<32>(sRow, d1, threadIdx.x, op);
            reduce<32>(sRow, d2, threadIdx.x, op);
            reduce<32>(sRow, abs1, threadIdx.x, op);
            reduce<32>(sRow, abs2, threadIdx.x, op);

            // write dx (dy >= 0), |dx| (dy >= 0), dx (dy < 0), |dx| (dy < 0)
            if (threadIdx.x == 0)
                descriptors_block[0] = make_float4(d1, abs1, d2, abs2);

            if (dx >= 0)
            {
                d1 = dy;
                abs1 = ::fabsf(dy);
                d2 = 0.0f;
                abs2 = 0.0f;
            }
            else
            {
                d1 = 0.0f;
                abs1 = 0.0f;
                d2 = dy;
                abs2 = ::fabsf(dy);
            }

            reduce<32>(sRow, d1, threadIdx.x, op);
            reduce<32>(sRow, d2, threadIdx.x, op);
            reduce<32>(sRow, abs1, threadIdx.x, op);
            reduce<32>(sRow, abs2, threadIdx.x, op);

            // write dy (dx >= 0), |dy| (dx >= 0), dy (dx < 0), |dy| (dx < 0)
            if (threadIdx.x == 0)
                descriptors_block[1] = make_float4(d1, abs1, d2, abs2);
        }

        template <int BLOCK_DIM_X> __global__ void normalize_descriptors(PtrStepf descriptors)
        {
            __shared__ float smem[BLOCK_DIM_X];
            __shared__ float s_len;

            // no need for thread ID
            float* descriptor_base = descriptors.ptr(blockIdx.x);

            // read in the unnormalized descriptor values (squared)
            const float val = descriptor_base[threadIdx.x];

            float len = val * val;
            reduce<BLOCK_DIM_X>(smem, len, threadIdx.x, plus<float>());

            if (threadIdx.x == 0)
                s_len = ::sqrtf(len);

            __syncthreads();

            // normalize and store in output
            descriptor_base[threadIdx.x] = val / s_len;
        }

        void compute_descriptors_gpu(const PtrStepSzb& img, PtrStepSz<float4> descriptors, const float* featureX, const float* featureY, const float* featureSize, const float* featureDir, int nFeatures)
        {
            // compute unnormalized descriptors, then normalize them - odd indexing since grid must be 2D
            cudev::Texture<unsigned char> texImg(img);
            if (descriptors.cols == 64)
            {
                compute_descriptors_64<<<nFeatures, dim3(32, 16)>>>(texImg, descriptors, featureX, featureY, featureSize, featureDir);
                cudaSafeCall( hipGetLastError() );

                cudaSafeCall( hipDeviceSynchronize() );

                normalize_descriptors<64><<<nFeatures, 64>>>((PtrStepSzf) descriptors);
                cudaSafeCall( hipGetLastError() );

                cudaSafeCall( hipDeviceSynchronize() );
            }
            else
            {
                compute_descriptors_128<<<nFeatures, dim3(32, 16)>>>(texImg, descriptors, featureX, featureY, featureSize, featureDir);
                cudaSafeCall( hipGetLastError() );

                cudaSafeCall( hipDeviceSynchronize() );

                normalize_descriptors<128><<<nFeatures, 128>>>((PtrStepSzf) descriptors);
                cudaSafeCall( hipGetLastError() );

                cudaSafeCall( hipDeviceSynchronize() );
            }
        }
    } // namespace surf
}}} // namespace cv { namespace cuda { namespace cudev

#endif // HAVE_OPENCV_CUDAARITHM
